// ====------ BlasUtils_api_test32.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_trsm_batch

int main() {
  hipblasHandle_t handle;
  float * alpha;
  const float **a;
  float **b;

  hipblasStrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, 4, 4, alpha, a, 4, b, 4, 2);
  return 0;
}
