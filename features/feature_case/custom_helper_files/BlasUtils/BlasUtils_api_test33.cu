// ====------ BlasUtils_api_test33.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_trmm

int main() {
  hipblasHandle_t handle;
  float * alpha;
  float *a, *b, *c;

  hipblasStrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, 4, 4, alpha, a, 4, b, 4, c, 2);
  return 0;
}
