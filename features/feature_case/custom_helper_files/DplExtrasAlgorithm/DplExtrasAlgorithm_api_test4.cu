// ====------ DplExtrasAlgorithm_api_test4.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/DplExtrasAlgorithm/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++17
// RUN: grep "IsCalled" %T/DplExtrasAlgorithm/api_test4_out/MainSourceFiles.yaml | wc -l > %T/DplExtrasAlgorithm/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/DplExtrasAlgorithm/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DplExtrasAlgorithm/api_test4_out

// CHECK: 5
// TEST_FEATURE: DplExtrasAlgorithm_copy_if

#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>

int main() {
  int A[10];
  int B[10];
  thrust::copy_if(A, A + 10, B, A,[=] __device__(int idx) { return true; });
  return 0;
}
