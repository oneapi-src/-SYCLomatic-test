// ===------ SparseUtils_api_test1.cu --------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: SparseUtils_matrix_info


#include <hip/hip_runtime.h>
#include "hipsparse.h"

int main() {
  hipsparseMatDescr_t a;
  return 0;
}
