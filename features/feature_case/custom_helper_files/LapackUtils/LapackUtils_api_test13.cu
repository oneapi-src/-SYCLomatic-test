// ===------ LapackUtils_api_test13.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_potrf


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;
  void* device_ws_s;
  void* host_ws_s;
  int *info;

  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s, 3, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  return 0;
}
