// ===------ LapackUtils_api_test7.cu --------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_getrs


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  int64_t* ipiv_s;
  float* b_s;
  hipsolverHandle_t handle;
  hipsolverDnParams_t params;
  int *info;

  hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_R_32F, a_s, 2, ipiv_s, HIP_R_32F, b_s, 2, info);
  return 0;
}
