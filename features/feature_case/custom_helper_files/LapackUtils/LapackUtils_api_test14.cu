// ===------ LapackUtils_api_test14.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_potrs


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  float* b_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;
  void* device_ws_s;
  void* host_ws_s;
  int *info;

  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_32F, a_s, 3, HIP_R_32F, b_s, 3, info);
  return 0;
}
