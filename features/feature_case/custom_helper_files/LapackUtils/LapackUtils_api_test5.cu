// ===------ LapackUtils_api_test5.cu --------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_getrf_scratchpad_size


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  int64_t* ipiv_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;

  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_R_32F, a_s, 2, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  return 0;
}
