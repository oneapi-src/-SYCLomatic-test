// ===------ LapackUtils_api_test27.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //


// TEST_FEATURE: LapackUtils_trtri


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float *a_s;
  hipsolverHandle_t handle;
  size_t lwork_s;
  size_t lwork_host_s;
  void* device_ws_s;
  void* host_ws_s;
  int *info;
  cusolverDnXtrtri(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_R_32F, a_s, 2, device_ws_s, lwork_s, host_ws_s, lwork_host_s, info);
  return 0;
}
