// ===------ LapackUtils_api_test1.cu --------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_sygvd


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  float *a_s, *b_s, *w_s, *work_s;
  int lwork_s;
  int *devInfo;
  hipsolverDnSsygvd(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s, 3, b_s, 3, w_s, work_s, lwork_s, devInfo);
  return 0;
}
