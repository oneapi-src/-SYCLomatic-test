// ===------ LapackUtils_api_test9.cu --------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_geqrf


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  int64_t* tau_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;
  void* device_ws_s;
  void* host_ws_s;
  int *info;

  hipsolverDnXgeqrf(handle, params, 2, 2, HIP_R_32F, a_s, 2, HIP_R_32F, tau_s, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  return 0;
}
