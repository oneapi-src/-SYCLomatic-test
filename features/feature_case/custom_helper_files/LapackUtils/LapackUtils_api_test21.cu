// ===------ LapackUtils_api_test21.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //


// TEST_FEATURE: LapackUtils_syhegvd_scratchpad_size


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverEigType_t itype;
  hipsolverEigMode_t jobz;
  hipblasFillMode_t uplo;
  int n;
  const float *A;
  int lda;
  const float *B;
  int ldb;
  const float *W;
  int *lwork;
  hipsolverSyevjInfo_t params;

  hipsolverDnSsygvj_bufferSize(handle, itype, jobz, uplo, n, A, lda, B, ldb, W,
                              lwork, params);
  return 0;
}
