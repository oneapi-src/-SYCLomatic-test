// ===------ LapackUtils_api_test17.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //


// TEST_FEATURE: LapackUtils_syheevx_scratchpad_size


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverDnParams_t params;
  hipsolverEigMode_t jobz;
  hipsolverEigRange_t range;
  hipblasFillMode_t uplo;
  int64_t n;
  hipDataType dataTypeA;
  const void *A;
  int64_t lda;
  void *vl;
  void *vu;
  int64_t il;
  int64_t iu;
  int64_t *h_meig;
  hipDataType dataTypeW;
  const void *W;
  hipDataType computeType;
  size_t *workspaceInBytesOnDevice;
  size_t *workspaceInBytesOnHost;

  cusolverDnXsyevdx_bufferSize(handle, params, jobz, range, uplo, n, dataTypeA,
                               A, lda, vl, vu, il, iu, h_meig, dataTypeW, W,
                               computeType, workspaceInBytesOnDevice,
                               workspaceInBytesOnHost);
  return 0;
}
