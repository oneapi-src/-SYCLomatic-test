// ===------ LapackUtils_api_test25.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //


// TEST_FEATURE: LapackUtils_syheevd


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float *a_s;
  float *w_s;
  hipsolverHandle_t handle;
  hipsolverDnParams_t params;
  size_t lwork_s;
  size_t lwork_host_s;
  void *device_ws_s;
  void *host_ws_s;
  int *info;
  cusolverDnXsyevd(handle, params, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, HIP_R_32F, a_s, 2, HIP_R_32F, w_s, HIP_R_32F, device_ws_s, lwork_s, host_ws_s, lwork_host_s, info);
  return 0;
}
