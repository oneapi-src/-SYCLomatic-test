// ===------ LapackUtils_api_test16.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //


// TEST_FEATURE: LapackUtils_syheevx_T


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverEigMode_t jobz;
  hipsolverEigRange_t range;
  hipblasFillMode_t uplo;
  int n;
  float *A;
  int lda;
  float vl;
  float vu;
  int il;
  int iu;
  int *h_meig;
  float *W;
  float *work;
  int lwork;
  int *devInfo;

  hipsolverDnSsyevdx(handle, jobz, range, uplo, n, A, lda, vl, vu, il, iu,
                    h_meig, W, work, lwork, devInfo);
  return 0;
}
