// ===------ LapackUtils_api_test18.cu -------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //


// TEST_FEATURE: LapackUtils_syheevx


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverDnParams_t params;
  hipsolverEigMode_t jobz;
  hipsolverEigRange_t range;
  hipblasFillMode_t uplo;
  int64_t n;
  hipDataType dataTypeA;
  void *A;
  int64_t lda;
  void *vl;
  void *vu;
  int64_t il;
  int64_t iu;
  int64_t *meig64;
  hipDataType dataTypeW;
  void *W;
  hipDataType computeType;
  void *bufferOnDevice;
  size_t workspaceInBytesOnDevice;
  void *bufferOnHost;
  size_t workspaceInBytesOnHost;
  int *info;

  cusolverDnXsyevdx(handle, params, jobz, range, uplo, n, dataTypeA, A, lda, vl,
                    vu, il, iu, meig64, dataTypeW, W, computeType,
                    bufferOnDevice, workspaceInBytesOnDevice, bufferOnHost,
                    workspaceInBytesOnHost, info);
  return 0;
}
