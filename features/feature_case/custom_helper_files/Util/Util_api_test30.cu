
#include <hip/hip_runtime.h>
// ===------- Util_api_test30.cu --------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//

// TEST_FEATURE: Util_calculate_max_active_wg_per_xecore

__global__ void k() {}

int main() {
  int num_blocks;
  int block_size = 128;
  size_t dynamic_shared_memory_size = 0;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, k, block_size, dynamic_shared_memory_size);
  return 0;
}
