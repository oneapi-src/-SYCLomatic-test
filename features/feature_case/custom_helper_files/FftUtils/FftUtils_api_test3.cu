// ===------ FftUtils_api_test3.cu------------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: FftUtils_fft_solver


#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

int main() {
  hipfftHandle plan;
  float2* odata;
  float2* idata;
  hipfftPlan1d(&plan, 10, HIPFFT_C2C, 3);
  hipfftExecC2C(plan, idata, odata, HIPFFT_FORWARD);
  return 0;
}
