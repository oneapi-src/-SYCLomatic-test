// ===------ FftUtils_api_test1.cu------------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: FftUtils_fft_dir


#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

int main() {
  int a = HIPFFT_FORWARD;
  return 0;
}
