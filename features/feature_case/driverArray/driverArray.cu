
#include <hip/hip_runtime.h>
int main(){
    int *data;
    size_t width, height, depth, pitch, woffset, hoffset;
    hipStream_t cs;
    hipArray_t acu;
    hipMemcpyHtoA(acu, woffset, data, width);
    hipMemcpyAtoH(data, acu, woffset, width);
    hipMemcpyHtoAAsync(acu, woffset, data, width, cs);
    hipMemcpyAtoHAsync(data, acu, woffset, width, cs);

    hipDeviceptr_t data2;
    hipMalloc(&data2, sizeof(int) * 30);
    hipMemcpyDtoA(acu, woffset, data2, width);
    hipMemcpyAtoD(data2, acu, woffset, width);

    hipArray_t acu2;
    hipMemcpyAtoA(acu, woffset, acu2, woffset, width);
}