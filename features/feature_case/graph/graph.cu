// ===------- graph.cu ------------------------------------ *- CUDA -* ---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <stdio.h>

const int blockSize = 256;
const int numBlocks = (10 + blockSize - 1) / blockSize;

__global__ void init(float *a) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < 10) {
    a[id] = 1.0f;
  }
}

__global__ void incrementA(float *a) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < 10) {
    a[id] += 1.0f;
  }
}

int main() {

  hipGraph_t graph;

  hipStream_t stream;

  hipStreamCreate(&stream);

  float *d_a, h_a[10];

  hipMalloc(&d_a, 10 * sizeof(float));

  init<<<numBlocks, blockSize, 0, stream>>>(d_a);

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  incrementA<<<numBlocks, blockSize, 0, stream>>>(d_a);

  hipStreamEndCapture(stream, &graph);
  hipGraphExec_t execGraph;
  hipGraphInstantiate(&execGraph, graph, NULL, NULL, 0);

  hipGraphLaunch(execGraph, stream);

  hipStreamSynchronize(stream); // Ensure the graph has completed execution

  hipMemcpy(h_a, d_a, 10 * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < 10; i++) {
    if (h_a[i] != 2.0f) {
      printf("Results do not match\n");
      return -1;
    }
  }

  printf("Passed\n");

  hipStreamDestroy(stream);
  hipFree(d_a);
  hipGraphExecDestroy(execGraph);

  return 0;
}
