#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  int cur_device;
  hipGetDevice(&cur_device);
  
  int can_access = 0;
  hipDeviceCanAccessPeer(&can_access, cur_device, cur_device);
  hipDeviceDisablePeerAccess(cur_device);
  hipDeviceEnablePeerAccess(cur_device, 0);

  std::cout << "test passed" << std::endl;

  return 0;
}