// ====------ cudnn-activation.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

template<cudnnDataType_t T>
struct dt_trait{
    typedef void type;
};
template<>
struct dt_trait<CUDNN_DATA_FLOAT>{
    typedef float type;
};

template<>
struct dt_trait<CUDNN_DATA_INT32>{
    typedef int type;
};
template<>
struct dt_trait<CUDNN_DATA_HALF>{
    typedef float type;
};


template<typename T>
void check(std::vector<T> &expect, std::vector<T> &actual, int num, float precision) {
  for(int i = 0; i < num; i++){
      if(std::abs(expect[i] - actual[i]) > precision) {
          std::cout << "test failed" << std::endl;
          std::cout << "expect:" << expect[i] << std::endl;
          std::cout << "actual:" << actual[i] << std::endl;
          exit(-1);
      }
  }
}

template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test1() {

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor;

    cudnnCreate(&handle);



    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);

    HT *data, *out;
    std::vector<HT> host_data(ele_num);
    std::vector<HT> host_out(ele_num);

    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i;
        host_out[i] = i;
    }

    hipMalloc(&data, ele_num * sizeof(HT));
    hipMalloc(&out, ele_num * sizeof(HT));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);

    cudnnActivationDescriptor_t desc;
    cudnnCreateActivationDescriptor(&desc);
    cudnnSetActivationDescriptor(desc, CUDNN_ACTIVATION_SIGMOID, CUDNN_NOT_PROPAGATE_NAN, 0.f);

    float alpha = 2.f, beta = 1.5f;
    auto s = cudnnActivationForward(handle, desc, &alpha, dataTensor, data, &beta, outTensor, out);
    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, ele_num * sizeof(HT), hipMemcpyDeviceToHost);

    std::vector<float> expect = {
        1, 2.96212, 4.76159, 6.40515, 7.96403,
        9.48661, 10.9951, 12.4982, 13.9993, 15.4998,
        16.9999, 18.5, 20, 21.5, 23,
        24.5, 26, 27.5, 29, 30.5,
        32, 33.5, 35, 36.5, 38,
        39.5, 41, 42.5, 44, 45.5,
        47, 48.5, 50, 51.5, 53,
        54.5, 56, 57.5, 59, 60.5,
        62, 63.5, 65, 66.5, 68,
        69.5, 71, 72.5, 74, 75.5    
      };
    check(expect, host_out, expect.size(), 1e-3);
    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
}

template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test2() {

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, diffdataTensor, diffoutTensor;

    cudnnCreate(&handle);



    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    HT *data, *out, *diffdata, *diffout;
    std::vector<HT> host_data(ele_num);
    std::vector<HT> host_out(ele_num);
    std::vector<HT> host_diffdata(ele_num);
    std::vector<HT> host_diffout(ele_num);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i * 0.1f;
        host_out[i] = i;
        host_diffdata[i] = i;
        host_diffout[i] = 1.f;
    }

    hipMalloc(&data, ele_num * sizeof(HT));
    hipMalloc(&out, ele_num * sizeof(HT));
    hipMalloc(&diffdata, ele_num * sizeof(HT));
    hipMalloc(&diffout, ele_num * sizeof(HT));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);

    cudnnActivationDescriptor_t desc;
    cudnnCreateActivationDescriptor(&desc);
    cudnnSetActivationDescriptor(desc, CUDNN_ACTIVATION_SIGMOID, CUDNN_PROPAGATE_NAN, 0.f);

    float alpha = 1.5f, beta = 0.f;
    cudnnActivationForward(handle, desc, &alpha, dataTensor, data, &beta, outTensor, out);

    alpha = 2.f, beta = 0.f;

    auto s = cudnnActivationBackward(handle, desc, &alpha, outTensor, out, diffoutTensor, diffout, dataTensor, data, &beta, diffdataTensor, diffdata);
    hipDeviceSynchronize();
    hipMemcpy(host_diffdata.data(), diffdata, ele_num * sizeof(HT), hipMemcpyDeviceToHost);

    std::vector<float> expect = {
        0.375, 0.334723, 0.289074, 0.238399, 0.183142,
        0.123828, 0.0610447, -0.00457374, -0.072368, -0.141673,
        -0.211834, -0.282226, -0.352262, -0.42141, -0.489194,
        -0.555202, -0.61909, -0.680577, -0.739441, -0.795526,
        -0.848724, -0.898978, -0.946273, -0.990628, -1.03209,
        -1.07075, -1.10668, -1.14001, -1.17084, -1.19932,
        -1.22557, -1.24972, -1.27191, -1.29227, -1.31092,
        -1.32799, -1.3436, -1.35786, -1.37087, -1.38273,
        -1.39354, -1.40338, -1.41234, -1.42049, -1.42789,
        -1.43462, -1.44073, -1.44629, -1.45132, -1.4559             
    };
    check(expect, host_diffdata, expect.size(), 1e-3);
    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    hipFree(diffdata);
    hipFree(diffout);
}

int main() {
    test1<CUDNN_DATA_FLOAT>();
    test2<CUDNN_DATA_FLOAT>();
    std::cout << "test passed" << std::endl;
    return 0;
}