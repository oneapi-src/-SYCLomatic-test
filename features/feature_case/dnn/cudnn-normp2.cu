// ====------ cudnn-normp2.cu ---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

template<cudnnDataType_t T>
struct dt_trait{
    typedef void type;
};
template<>
struct dt_trait<CUDNN_DATA_FLOAT>{
    typedef float type;
};

template<>
struct dt_trait<CUDNN_DATA_INT32>{
    typedef int type;
};
template<>
struct dt_trait<CUDNN_DATA_HALF>{
    typedef float type;
};

template<typename T>
void check(std::vector<T> &expect, std::vector<T> &actual, int num, float precision) {
  for(int i = 0; i < num; i++){
      if(std::abs(expect[i] - actual[i]) > precision) {
          std::cout << "test failed" << std::endl;
          std::cout << "expect:" << expect[i] << std::endl;
          std::cout << "actual:" << actual[i] << std::endl;
          exit(-1);
      }
  }
}

template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test4() {
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, scalebiasTensor;
    cudnnCreate(&handle);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&scalebiasTensor);

    int in = 2, ic = 4, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 5, ow = 5;
    int sbn = 1, sbc = 4, sbh = 5, sbw = 5;
    int ele_num = in* ic * ih * iw;
    int oele_num = on* oc * oh * ow;
    int sele_num = sbn*sbc * sbh * sbw;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    cudnnSetTensor4dDescriptor(scalebiasTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, sbn, sbc, sbh, sbw);

    int save = 1;
    float *data, *out, *scale, *bias, *rmean, *rvar, *smean, *svar, *z;
    std::vector<float> host_data(ele_num, 1.0f);
    std::vector<float> host_z(oele_num, 1.0f);
    std::vector<float> host_out(oele_num, 0.0f);
    std::vector<float> host_scale(sele_num, 1.0f);
    std::vector<float> host_bias(sele_num, 0.0f);
    std::vector<float> host_rmean(sele_num, 0.0f);
    std::vector<float> host_rvar(sele_num, 0.0f);
    std::vector<float> host_smean(save * sele_num, 0.0f);
    std::vector<float> host_svar(save * sele_num, 0.0f);

    for(int i = 0; i < ele_num; i++) {
        host_data[i] =  i + 4.f;
        host_out[i] = 1.f;
        host_z[i] = 10;
    }
    for(int i = 0; i < sele_num; i++) {
        host_scale[i] = i;
        host_bias[i] = i;
        host_rmean[i] = i;
        host_rvar[i] = i;
        host_smean[i] = i;
        host_svar[i] = i;
    }

    hipMalloc(&data, sizeof(float) * ele_num);
    hipMalloc(&z, sizeof(float) * oele_num);
    hipMalloc(&out, sizeof(float) * oele_num);
    hipMalloc(&scale, sizeof(float) * sele_num);
    hipMalloc(&bias, sizeof(float) * sele_num);
    hipMalloc(&rmean, sizeof(float) * sele_num);
    hipMalloc(&rvar, sizeof(float) * sele_num);
    hipMalloc(&smean, sizeof(float) * save*sele_num);
    hipMalloc(&svar, sizeof(float)  * save*sele_num);

    hipMemcpy(data, host_data.data(), sizeof(float) * ele_num, hipMemcpyHostToDevice);
    hipMemcpy(z, host_z.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(scale, host_scale.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(bias, host_bias.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rmean, host_rmean.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rvar, host_rvar.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(smean, host_smean.data(),  sizeof(float) * save * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(svar, host_svar.data(), sizeof(float) * save * sele_num, hipMemcpyHostToDevice);

    float alpha = 2.5f, beta = 1.5f, eps = 1.f;
    double factor = 0.5f;
    cudnnActivationDescriptor_t ActivationDesc;
    cudnnCreateActivationDescriptor(&ActivationDesc);
    cudnnSetActivationDescriptor(ActivationDesc, CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0.0f);

    float *workspace, *reservespace;
    size_t workspace_size, reservespace_size;

    cudnnGetNormalizationForwardTrainingWorkspaceSize(
        handle, 
        CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_OPS_NORM,
        CUDNN_NORM_ALGO_STANDARD,
        dataTensor,
        dataTensor,
        outTensor,
        scalebiasTensor,
        ActivationDesc,
        scalebiasTensor,
        &workspace_size,
        1
    );
    cudnnGetNormalizationTrainingReserveSpaceSize(
        handle,
        CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_OPS_NORM,
        CUDNN_NORM_ALGO_STANDARD,
        NULL,
        dataTensor,
        &reservespace_size,
        1
    );
    hipMalloc(&workspace, workspace_size);
    hipMalloc(&reservespace,  reservespace_size);
    auto status = cudnnNormalizationForwardTraining(
        handle, 
        CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_OPS_NORM,
        CUDNN_NORM_ALGO_STANDARD,
        &alpha,
        &beta,
        dataTensor,
        data,
        scalebiasTensor,
        scale,
        bias,
        factor,
        scalebiasTensor,
        rmean,
        rvar,
        eps,
        smean,
        svar,
        ActivationDesc,
        dataTensor,
        z,
        outTensor,
        out,
        workspace,
        workspace_size,
        reservespace,
        reservespace_size,
        1);

    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, sizeof(float) * oele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rmean.data(), rmean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rvar.data(), rvar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);

    std::vector<float> expect = {
        1.5, 1.5005, 1.501, 1.5015, 1.502,
        1.5025, 1.503, 1.5035, 1.504, 1.5045,
        1.505, 1.5055, 1.506, 1.5065, 1.507,
        1.5075, 1.508, 1.5085, 1.509, 1.5095,
        1.51, 1.5105, 1.511, 1.5115, 1.512,
        
        1.5125, 1.513, 1.5135, 1.514, 1.5145,
        1.515, 1.5155, 1.516, 1.5165, 1.517,
        1.5175, 1.518, 1.5185, 1.519, 1.5195,
        1.52, 1.5205, 1.521, 1.5215, 1.522,
        1.5225, 1.523, 1.5235, 1.524, 1.5245,
        
        1.525, 1.5255, 1.526, 1.5265, 1.527,
        1.5275, 1.528, 1.5285, 1.529, 1.5295,
        1.53, 1.5305, 1.531, 1.5315, 1.532,
        1.5325, 1.533, 1.5335, 1.534, 1.5345,
        1.535, 1.5355, 1.536, 1.5365, 1.537,
        
        1.5375, 1.538, 1.5385, 1.539, 1.5395,
        1.54, 1.5405, 1.541, 1.5415, 1.542,
        1.5425, 1.543, 1.5435, 1.544, 1.54449,
        1.54499, 1.54549, 1.54599, 1.54649, 1.54699,
        1.54749, 1.54799, 1.54849, 1.54899, 1.54949,

        1.5, 6.4995, 11.499, 16.4985, 21.498,
        26.4975, 31.497, 36.4965, 41.496, 46.4955,
        51.495, 56.4945, 61.494, 66.4935, 71.493,
        76.4925, 81.492, 86.4915, 91.491, 96.4905,
        101.49, 106.49, 111.489, 116.488, 121.488,
        
        126.487, 131.487, 136.487, 141.486, 146.486,
        151.485, 156.484, 161.484, 166.484, 171.483,
        176.483, 181.482, 186.482, 191.481, 196.48,
        201.48, 206.479, 211.479, 216.479, 221.478,
        226.477, 231.477, 236.477, 241.476, 246.476,
        
        251.475, 256.474, 261.474, 266.474, 271.473,
        276.473, 281.472, 286.471, 291.471, 296.471,
        301.47, 306.47, 311.469, 316.469, 321.468,
        326.467, 331.467, 336.466, 341.466, 346.466,
        351.465, 356.464, 361.464, 366.464, 371.463,
        
        376.462, 381.462, 386.462, 391.461, 396.461,
        401.46, 406.459, 411.459, 416.458, 421.458,
        426.458, 431.457, 436.457, 441.456, 446.456,
        451.455, 456.454, 461.454, 466.453, 471.453,
        476.453, 481.452, 486.452, 491.451, 496.451,        
        };
    std::vector<float> expect_rmean = {
        27, 28, 29, 30, 31,
        32, 33, 34, 35, 36,
        37, 38, 39, 40, 41,
        42, 43, 44, 45, 46,
        47, 48, 49, 50, 51,
        
        52, 53, 54, 55, 56,
        57, 58, 59, 60, 61,
        62, 63, 64, 65, 66,
        67, 68, 69, 70, 71,
        72, 73, 74, 75, 76,
        
        77, 78, 79, 80, 81,
        82, 83, 84, 85, 86,
        87, 88, 89, 90, 91,
        92, 93, 94, 95, 96,
        97, 98, 99, 100, 101,
        
        102, 103, 104, 105, 106,
        107, 108, 109, 110, 111,
        112, 113, 114, 115, 116,
        117, 118, 119, 120, 121,
        122, 123, 124, 125, 126,        
        };

    std::vector<float> expect_rvar = {
        2500, 2500.5, 2501, 2501.5, 2502,
        2502.5, 2503, 2503.5, 2504, 2504.5,
        2505, 2505.5, 2506, 2506.5, 2507,
        2507.5, 2508, 2508.5, 2509, 2509.5,
        2510, 2510.5, 2511, 2511.5, 2512,
        
        2512.5, 2513, 2513.5, 2514, 2514.5,
        2515, 2515.5, 2516, 2516.5, 2517,
        2517.5, 2518, 2518.5, 2519, 2519.5,
        2520, 2520.5, 2521, 2521.5, 2522,
        2522.5, 2523, 2523.5, 2524, 2524.5,
        
        2525, 2525.5, 2526, 2526.5, 2527,
        2527.5, 2528, 2528.5, 2529, 2529.5,
        2530, 2530.5, 2531, 2531.5, 2532,
        2532.5, 2533, 2533.5, 2534, 2534.5,
        2535, 2535.5, 2536, 2536.5, 2537,
        
        2537.5, 2538, 2538.5, 2539, 2539.5,
        2540, 2540.5, 2541, 2541.5, 2542,
        2542.5, 2543, 2543.5, 2544, 2544.5,
        2545, 2545.5, 2546, 2546.5, 2547,
        2547.5, 2548, 2548.5, 2549, 2549.5,                     
        };
    check(expect, host_out, expect.size(), 1e-1);
    check(expect_rmean, host_rmean, expect_rmean.size(), 1e-1);
    check(expect_rvar, host_rvar, expect_rvar.size(), 1e-1);
    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
}

template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test5() {
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, scalebiasTensor;
    cudnnCreate(&handle);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&scalebiasTensor);

    int in = 2, ic = 4, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 5, ow = 5;
    int sbn = 1, sbc = 4, sbh = 1, sbw = 1;
    int ele_num = in* ic * ih * iw;
    int oele_num = on* oc * oh * ow;
    int sele_num = sbn*sbc * sbh * sbw;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    cudnnSetTensor4dDescriptor(scalebiasTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, sbn, sbc, sbh, sbw);

    int save = 1;
    float *data, *out, *scale, *bias, *rmean, *rvar, *smean, *svar, *z;
    std::vector<float> host_data(ele_num, 1.0f);
    std::vector<float> host_z(oele_num, 1.0f);
    std::vector<float> host_out(oele_num, 0.0f);
    std::vector<float> host_scale(sele_num, 1.0f);
    std::vector<float> host_bias(sele_num, 0.0f);
    std::vector<float> host_rmean(sele_num, 0.0f);
    std::vector<float> host_rvar(sele_num, 0.0f);
    std::vector<float> host_smean(save * sele_num, 0.0f);
    std::vector<float> host_svar(save * sele_num, 0.0f);

    for(int i = 0; i < ele_num; i++) {
        host_data[i] =  i + 4.f;
        host_out[i] = 1.f;
        host_z[i] = 10;
    }
    for(int i = 0; i < sele_num; i++) {
        host_scale[i] = i;
        host_bias[i] = i;
        host_rmean[i] = i;
        host_rvar[i] = i;
        host_smean[i] = i;
        host_svar[i] = i;
    }

    hipMalloc(&data, sizeof(float) * ele_num);
    hipMalloc(&z, sizeof(float) * oele_num);
    hipMalloc(&out, sizeof(float) * oele_num);
    hipMalloc(&scale, sizeof(float) * sele_num);
    hipMalloc(&bias, sizeof(float) * sele_num);
    hipMalloc(&rmean, sizeof(float) * sele_num);
    hipMalloc(&rvar, sizeof(float) * sele_num);
    hipMalloc(&smean, sizeof(float) * save*sele_num);
    hipMalloc(&svar, sizeof(float)  * save*sele_num);

    hipMemcpy(data, host_data.data(), sizeof(float) * ele_num, hipMemcpyHostToDevice);
    hipMemcpy(z, host_z.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(scale, host_scale.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(bias, host_bias.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rmean, host_rmean.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rvar, host_rvar.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(smean, host_smean.data(),  sizeof(float) * save * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(svar, host_svar.data(), sizeof(float) * save * sele_num, hipMemcpyHostToDevice);

    float alpha = 2.5f, beta = 1.5f, eps = 1.f;
    double factor = 0.5f;
    cudnnActivationDescriptor_t ActivationDesc;
    cudnnCreateActivationDescriptor(&ActivationDesc);
    cudnnSetActivationDescriptor(ActivationDesc, CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0.0f);

    float *workspace, *reservespace;
    size_t workspace_size, reservespace_size;

    cudnnGetNormalizationForwardTrainingWorkspaceSize(
        handle,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        CUDNN_NORM_ALGO_STANDARD,
        dataTensor,
        dataTensor,
        outTensor,
        scalebiasTensor,
        ActivationDesc,
        scalebiasTensor,
        &workspace_size,
        1
    );
    cudnnGetNormalizationTrainingReserveSpaceSize(
        handle,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        CUDNN_NORM_ALGO_STANDARD,
        NULL,
        dataTensor,
        &reservespace_size,
        1
    );
    hipMalloc(&workspace, workspace_size);
    hipMalloc(&reservespace,  reservespace_size);
    auto status = cudnnNormalizationForwardTraining(
        handle, 
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        CUDNN_NORM_ALGO_STANDARD,
        &alpha,
        &beta,
        dataTensor,
        data,
        scalebiasTensor,
        scale,
        bias,
        factor,
        scalebiasTensor,
        rmean,
        rvar,
        eps,
        smean,
        svar,
        ActivationDesc,
        dataTensor,
        z,
        outTensor,
        out,
        workspace,
        workspace_size,
        reservespace,
        reservespace_size,
        1);

    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, sizeof(float) * oele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rmean.data(), rmean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rvar.data(), rvar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);

    std::vector<float> expect = {
        1.5, 1.5, 1.5, 1.5, 1.5,
        1.5, 1.5, 1.5, 1.5, 1.5,
        1.5, 1.5, 1.5, 1.5, 1.5,
        1.5, 1.5, 1.5, 1.5, 1.5,
        1.5, 1.5, 1.5, 1.5, 1.5,
        
        0.932347, 0.981825, 1.0313, 1.08078, 1.13026,
        1.17974, 1.22922, 1.27869, 1.32817, 1.37765,
        1.42713, 1.47661, 1.52609, 1.57556, 1.62504,
        1.67452, 1.724, 1.77348, 1.82296, 1.87243,
        1.92191, 1.97139, 2.02087, 2.07035, 2.11983,
        
        0.364693, 0.46365, 0.562607, 0.661563, 0.76052,
        0.859476, 0.958433, 1.05739, 1.15635, 1.2553,
        1.35426, 1.45322, 1.55217, 1.65113, 1.75009,
        1.84904, 1.948, 2.04695, 2.14591, 2.24487,
        2.34382, 2.44278, 2.54174, 2.64069, 2.73965,
        
        -0.202961, -0.0545259, 0.093909, 0.242344, 0.390779,
        0.539214, 0.687648, 0.836083, 0.984518, 1.13295,
        1.28139, 1.42982, 1.57826, 1.72669, 1.87513,
        2.02356, 2.172, 2.32043, 2.46887, 2.6173,
        2.76574, 2.91417, 3.06261, 3.21104, 3.35948,
        
        1.5, 1.5, 1.5, 1.5, 1.5,
        1.5, 1.5, 1.5, 1.5, 1.5,
        1.5, 1.5, 1.5, 1.5, 1.5,
        1.5, 1.5, 1.5, 1.5, 1.5,
        1.5, 1.5, 1.5, 1.5, 1.5,
        
        5.88017, 5.92965, 5.97913, 6.02861, 6.07809,
        6.12757, 6.17704, 6.22652, 6.276, 6.32548,
        6.37496, 6.42444, 6.47391, 6.52339, 6.57287,
        6.62235, 6.67183, 6.72131, 6.77078, 6.82026,
        6.86974, 6.91922, 6.9687, 7.01818, 7.06765,
        
        10.2603, 10.3593, 10.4583, 10.5572, 10.6562,
        10.7551, 10.8541, 10.953, 11.052, 11.151,
        11.2499, 11.3489, 11.4478, 11.5468, 11.6457,
        11.7447, 11.8437, 11.9426, 12.0416, 12.1405,
        12.2395, 12.3384, 12.4374, 12.5364, 12.6353,
        
        14.6405, 14.789, 14.9374, 15.0858, 15.2343,
        15.3827, 15.5311, 15.6796, 15.828, 15.9764,
        16.1249, 16.2733, 16.4217, 16.5702, 16.7186,
        16.867, 17.0155, 17.1639, 17.3124, 17.4608,
        17.6092, 17.7577, 17.9061, 18.0545, 18.203,
        };
    std::vector<float> expect_rmean = {
        33,

        46,
        
        59,
        
        72,             
        };

    std::vector<float> expect_rvar = {
        1302.04,

        1302.54,
        
        1303.04,
        
        1303.54,                            
        };
    check(expect, host_out, expect.size(), 1e-1);
    check(expect_rmean, host_rmean, expect_rmean.size(), 1e-1);
    check(expect_rvar, host_rvar, expect_rvar.size(), 1e-1);
    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
}

int main() {
    test4<CUDNN_DATA_FLOAT>();
    test5<CUDNN_DATA_FLOAT>();
    std::cout << "test passed" << std::endl;
    return 0;
}
