
#include <hip/hip_runtime.h>
#include "cudnn.h"
#include <iostream>

int main() {
  cudnnStatus_t status{CUDNN_STATUS_SUCCESS};

  const char *msg=cudnnGetErrorString(status);

  if (!msg)
    return 1;

  std::cout << "string = " << msg << "\n";

  return 0;
}
