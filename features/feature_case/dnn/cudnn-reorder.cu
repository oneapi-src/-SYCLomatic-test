// ====------ cudnn-reorder.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

template<cudnnDataType_t T>
struct dt_trait{
    typedef void type;
};
template<>
struct dt_trait<CUDNN_DATA_FLOAT>{
    typedef float type;
};

template<>
struct dt_trait<CUDNN_DATA_INT32>{
    typedef int type;
};
template<>
struct dt_trait<CUDNN_DATA_HALF>{
    typedef float type;
};

template<typename T>
void check(std::vector<T> &expect, std::vector<T> &actual, int num, float precision) {
  for(int i = 0; i < num; i++){
      if(std::abs(expect[i] - actual[i]) > precision) {
          std::cout << "test failed" << std::endl;
          std::cout << "expect:" << expect[i] << std::endl;
          std::cout << "actual:" << actual[i] << std::endl;
          exit(-1);
      }
  }
}

template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test() {

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor;

    cudnnCreate(&handle);



    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NHWC, T, n, c, h, w);

    HT *data, *out;
    std::vector<HT> host_data(ele_num, 0);
    std::vector<HT> host_out(ele_num, 0);

    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i;
        host_out[i] = 0;
    }

    hipMalloc(&data, ele_num * sizeof(HT));
    hipMalloc(&out, ele_num * sizeof(HT));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);

    float alpha = 3.f, beta = 1.f;
    auto s = cudnnTransformTensor(handle, &alpha, dataTensor, data, &beta, outTensor, out);
    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, ele_num * sizeof(HT), hipMemcpyDeviceToHost);
    std::vector<float> expect = {
      0, 75, 3, 78, 6,
      81, 9, 84, 12, 87,
      15, 90, 18, 93, 21,
      96, 24, 99, 27, 102,
      30, 105, 33, 108, 36,
      111, 39, 114, 42, 117,
      45, 120, 48, 123, 51,
      126, 54, 129, 57, 132,
      60, 135, 63, 138, 66,
      141, 69, 144, 72, 147
    };
    check(expect, host_out, expect.size(), 1e-3);
    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
}

int main() {
    test<CUDNN_DATA_FLOAT>();
    std::cout << "test passed" << std::endl;
    return 0;
}