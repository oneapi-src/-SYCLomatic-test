// ====------ driverCtx.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
int main(){
  unsigned int ver;
  hipDevice_t device;
  hipCtx_t ctx;
  hipDevice_t* dev_ptr;

  hipDeviceGet(&device, 0);
  hipCtxCreate(&ctx, 0, device);
  hipCtxGetApiVersion(ctx, &ver);
  hipCtxGetDevice(dev_ptr);
  return 0;
}

