// ===-------- text_experimental_obj_mipmap.cu ------- *- CUDA -* ---------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iostream>

#define PRINT_PASS 1

using namespace std;

int passed = 0;
int failed = 0;

void checkResult(string name, bool IsPassed) {
  cout << name;
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

template <typename T, typename EleT>
__global__ void kernel4(EleT *output, hipTextureObject_t tex, int w, int h,
                        float level) {
  for (int i = 0; i < h; ++i) {
    for (int j = 0; j < w; ++j) {
      auto ret = tex2DLod<T>(tex, j, i, level);
      output[8 * (w * i + j)] = ret.x;
      output[8 * (w * i + j) + 1] = ret.y;
      output[8 * (w * i + j) + 2] = ret.z;
      output[8 * (w * i + j) + 3] = ret.w;
      auto ret1 = tex2DLod<T>(tex, j + 0.3, i + 0.3, level);
      output[8 * (w * i + j) + 5] = ret.x;
      output[8 * (w * i + j) + 6] = ret.y;
      output[8 * (w * i + j) + 7] = ret.z;
      output[8 * (w * i + j) + 8] = ret.w;
    }
  }
}

template <typename T, typename ArrT1, typename ArrT2>
hipMipmappedArray_t getInput(ArrT1 &mipmap1, ArrT2 &mipmap2, size_t w,
                              size_t h, const hipChannelFormatDesc &desc) {
  hipMipmappedArray_t input;
  hipMallocMipmappedArray(&input, &desc, {w, h, 0}, 2);
  hipArray_t temp;
  hipGetMipmappedArrayLevel(&temp, input, 0);
  hipMemcpy2DToArray(temp, 0, 0, mipmap1, sizeof(T) * w, sizeof(T) * w, h,
                      hipMemcpyHostToDevice);
  hipArray_t temp1;
  hipGetMipmappedArrayLevel(&temp1, input, 1);
  hipMemcpy2DToArray(temp1, 0, 0, mipmap2, sizeof(T) * w / 2,
                      sizeof(T) * w / 2, h / 2, hipMemcpyHostToDevice);
  return input;
}

hipTextureObject_t
getTex(hipMipmappedArray_t input, float minMipmapLevelClamp,
       float maxMipmapLevelClamp, float maxAnisotropy = 0,
       hipTextureFilterMode mipmapFilterMode = hipFilterModePoint) {
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeMipmappedArray;
  resDesc.res.mipmap.mipmap = input;

  hipTextureDesc texDesc;
  // memset(&texDesc, 0, sizeof(texDesc)); // TODO: Need open after bug fixing.
  texDesc.maxAnisotropy = maxAnisotropy;
  texDesc.mipmapFilterMode = mipmapFilterMode;
  texDesc.minMipmapLevelClamp = minMipmapLevelClamp;
  texDesc.maxMipmapLevelClamp = maxMipmapLevelClamp;

  hipTextureObject_t tex;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  return tex;
}

int main() {
  bool pass = true;

  {
    const int short4H = 2;
    const int short4W = 4;
    short4 short4MimMap1[short4H * short4W] = {
        {1, 2, 3, 4},     {5, 6, 7, 8},
        {9, 10, 11, 12},  {13, 14, 15, 16}, // 1
        {17, 18, 19, 20}, {21, 22, 23, 24},
        {25, 26, 27, 28}, {29, 30, 31, 32}, // 2
    };
    short4 short4MimMap2[short4H * short4W / 4] = {
        {11, 22, 33, 44}, {55, 66, 77, 88}, // 1
    };
    auto short4Input =
        getInput<short4>(short4MimMap1, short4MimMap2, short4W, short4H,
                         hipCreateChannelDesc<short4>());
    auto short4Tex = getTex(short4Input, 0.1, 0.9);

    {
      short4 short4Expect0[short4H * short4W * 2] = {
          {1, 2, 3, 4},     {0, 1, 2, 3},     {13, 14, 15, 16},
          {0, 13, 14, 15},  {13, 14, 15, 16}, {0, 13, 14, 15},
          {13, 14, 15, 16}, {0, 13, 14, 15}, // 1
          {17, 18, 19, 20}, {0, 17, 18, 19},  {29, 30, 31, 32},
          {0, 29, 30, 31},  {29, 30, 31, 32}, {0, 29, 30, 31},
          {29, 30, 31, 32}, {0, 29, 30, 31}, // 2
      };
      short *short4Output0;
      hipMallocManaged(&short4Output0, sizeof(short4Expect0));
      kernel4<short4><<<1, 1>>>(short4Output0, short4Tex, short4W, short4H, 0);
      hipDeviceSynchronize();
      for (int i = 0; i < short4W * short4H * 2; ++i) {
        if (short4Output0[4 * i] != short4Expect0[i].x ||
            short4Output0[4 * i + 1] != short4Expect0[i].y ||
            short4Output0[4 * i + 2] != short4Expect0[i].z ||
            short4Output0[4 * i + 3] != short4Expect0[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("short4:0", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < short4H; ++i) {
          for (int j = 0; j < short4W; ++j)
            cout << "{" << short4Output0[8 * (short4W * i + j)] << ", "
                 << short4Output0[8 * (short4W * i + j) + 1] << ", "
                 << short4Output0[8 * (short4W * i + j) + 2] << ", "
                 << short4Output0[8 * (short4W * i + j) + 3] << "}, {"
                 << short4Output0[8 * (short4W * i + j) + 4] << ", "
                 << short4Output0[8 * (short4W * i + j) + 5] << ", "
                 << short4Output0[8 * (short4W * i + j) + 6] << ", "
                 << short4Output0[8 * (short4W * i + j) + 7] << "}, ";
          cout << endl;
        }
      pass = true;
    }
    {
      short4 short4Expect0_3[short4H * short4W * 2] = {
          {1, 2, 3, 4},     {0, 1, 2, 3},     {13, 14, 15, 16},
          {0, 13, 14, 15},  {13, 14, 15, 16}, {0, 13, 14, 15},
          {13, 14, 15, 16}, {0, 13, 14, 15}, // 1
          {17, 18, 19, 20}, {0, 17, 18, 19},  {29, 30, 31, 32},
          {0, 29, 30, 31},  {29, 30, 31, 32}, {0, 29, 30, 31},
          {29, 30, 31, 32}, {0, 29, 30, 31}, // 2
      };
      short *short4Output0_3;
      hipMallocManaged(&short4Output0_3, sizeof(short4Expect0_3));
      kernel4<short4>
          <<<1, 1>>>(short4Output0_3, short4Tex, short4W, short4H, 0.3);
      hipDeviceSynchronize();
      for (int i = 0; i < short4W * short4H * 2; ++i) {
        if (short4Output0_3[4 * i] != short4Expect0_3[i].x ||
            short4Output0_3[4 * i + 1] != short4Expect0_3[i].y ||
            short4Output0_3[4 * i + 2] != short4Expect0_3[i].z ||
            short4Output0_3[4 * i + 3] != short4Expect0_3[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("short4:0.3", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < short4H; ++i) {
          for (int j = 0; j < short4W; ++j)
            cout << "{" << short4Output0_3[8 * (short4W * i + j)] << ", "
                 << short4Output0_3[8 * (short4W * i + j) + 1] << ", "
                 << short4Output0_3[8 * (short4W * i + j) + 2] << ", "
                 << short4Output0_3[8 * (short4W * i + j) + 3] << "}, {"
                 << short4Output0_3[8 * (short4W * i + j) + 4] << ", "
                 << short4Output0_3[8 * (short4W * i + j) + 5] << ", "
                 << short4Output0_3[8 * (short4W * i + j) + 6] << ", "
                 << short4Output0_3[8 * (short4W * i + j) + 7] << "}, ";
          cout << endl;
        }
      pass = true;
    }
    {
      short4 short4Expect1[short4H * short4W * 2] = {
          {11, 22, 33, 44}, {0, 11, 22, 33},  {55, 66, 77, 88},
          {0, 55, 66, 77},  {55, 66, 77, 88}, {0, 55, 66, 77},
          {55, 66, 77, 88}, {0, 55, 66, 77}, // 1
          {11, 22, 33, 44}, {0, 11, 22, 33},  {55, 66, 77, 88},
          {0, 55, 66, 77},  {55, 66, 77, 88}, {0, 55, 66, 77},
          {55, 66, 77, 88}, {0, 55, 66, 77}, // 2
      };
      short *short4Output1;
      hipMallocManaged(&short4Output1, sizeof(short4Expect1));
      kernel4<short4><<<1, 1>>>(short4Output1, short4Tex, short4W, short4H, 1);
      hipDeviceSynchronize();
      for (int i = 0; i < short4W * short4H * 2; ++i) {
        if (short4Output1[4 * i] != short4Expect1[i].x ||
            short4Output1[4 * i + 1] != short4Expect1[i].y ||
            short4Output1[4 * i + 2] != short4Expect1[i].z ||
            short4Output1[4 * i + 3] != short4Expect1[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("short4:1", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < short4H; ++i) {
          for (int j = 0; j < short4W; ++j)
            cout << "{" << short4Output1[8 * (short4W * i + j)] << ", "
                 << short4Output1[8 * (short4W * i + j) + 1] << ", "
                 << short4Output1[8 * (short4W * i + j) + 2] << ", "
                 << short4Output1[8 * (short4W * i + j) + 3] << "}, {"
                 << short4Output1[8 * (short4W * i + j) + 4] << ", "
                 << short4Output1[8 * (short4W * i + j) + 5] << ", "
                 << short4Output1[8 * (short4W * i + j) + 6] << ", "
                 << short4Output1[8 * (short4W * i + j) + 7] << "}, ";
          cout << endl;
        }
      pass = true;
    }

    hipDestroyTextureObject(short4Tex);
    hipFreeMipmappedArray(short4Input);
  }

  {
    const int float4H = 4;
    const int float4W = 2;
    float4 float4MimMap1[float4H * float4W] = {
        {1, 2, 3, 4},     {5, 6, 7, 8},     // 1
        {9, 10, 11, 12},  {13, 14, 15, 16}, // 2
        {17, 18, 19, 20}, {21, 22, 23, 24}, // 3
        {25, 26, 27, 28}, {29, 30, 31, 32}, // 4
    };
    float4 float4MimMap2[float4H * float4W / 4] = {
        {11, 22, 33, 44}, // 1
        {55, 66, 77, 88}, // 2
    };
    auto *float4Input =
        getInput<float4>(float4MimMap1, float4MimMap2, float4W, float4H,
                         hipCreateChannelDesc<float4>());

    {
      auto float4Tex = getTex(float4Input, 0.1, 0.9);

      {
        float4 float4Expect0[float4H * float4W * 2] = {
            {1, 2, 3, 4},     {0, 1, 2, 3},
            {5, 6, 7, 8},     {0, 5, 6, 7}, // 1
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 2
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 3
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0[i].x - precision ||
               float4Output0[4 * i] > float4Expect0[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|0|Point:0", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect0_3[float4H * float4W * 2] = {
            {1, 2, 3, 4},     {0, 1, 2, 3},
            {5, 6, 7, 8},     {0, 5, 6, 7}, // 1
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 2
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 3
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0_3));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0.3);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0_3[i].x - precision ||
               float4Output0[4 * i] > float4Expect0_3[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0_3[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0_3[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0_3[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0_3[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0_3[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0_3[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|0|Point:0.3", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect1[float4H * float4W * 2] = {
            {11, 22, 33, 44}, {0, 11, 22, 33},
            {11, 22, 33, 44}, {0, 11, 22, 33}, // 1
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 2
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 3
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect1));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 1);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect1[i].x - precision ||
               float4Output0[4 * i] > float4Expect1[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect1[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect1[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect1[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect1[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect1[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect1[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|0|Point:1", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }

      hipDestroyTextureObject(float4Tex);
    }
    {
      auto float4Tex = getTex(float4Input, 0.1, 0.9, 0, hipFilterModeLinear);

      {
        float4 float4Expect0[float4H * float4W * 2] = {
            {1.97656, 3.95312, 5.92969, 7.90625},
            {0, 1.97656, 3.95312, 5.92969},
            {5.58594, 7.5625, 9.53906, 11.5156},
            {0, 5.58594, 7.5625, 9.53906}, // 1
            {27.9297, 29.9062, 31.8828, 33.8594},
            {0, 27.9297, 29.9062, 31.8828},
            {31.5391, 33.5156, 35.4922, 37.4688},
            {0, 31.5391, 33.5156, 35.4922}, // 2
            {27.9297, 29.9062, 31.8828, 33.8594},
            {0, 27.9297, 29.9062, 31.8828},
            {31.5391, 33.5156, 35.4922, 37.4688},
            {0, 31.5391, 33.5156, 35.4922}, // 3
            {27.9297, 29.9062, 31.8828, 33.8594},
            {0, 27.9297, 29.9062, 31.8828},
            {31.5391, 33.5156, 35.4922, 37.4688},
            {0, 31.5391, 33.5156, 35.4922}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0[i].x - precision ||
               float4Output0[4 * i] > float4Expect0[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|0|Linear:0", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect0_3[float4H * float4W * 2] = {
            {3.96875, 7.9375, 11.9062, 15.875},
            {0, 3.96875, 7.9375, 11.9062},
            {6.78125, 10.75, 14.7188, 18.6875},
            {0, 6.78125, 10.75, 14.7188}, // 1
            {33.9062, 37.875, 41.8438, 45.8125},
            {0, 33.9062, 37.875, 41.8438},
            {36.7188, 40.6875, 44.6562, 48.625},
            {0, 36.7188, 40.6875, 44.6562}, // 2
            {33.9062, 37.875, 41.8438, 45.8125},
            {0, 33.9062, 37.875, 41.8438},
            {36.7188, 40.6875, 44.6562, 48.625},
            {0, 36.7188, 40.6875, 44.6562}, // 3
            {33.9062, 37.875, 41.8438, 45.8125},
            {0, 33.9062, 37.875, 41.8438},
            {36.7188, 40.6875, 44.6562, 48.625},
            {0, 36.7188, 40.6875, 44.6562}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0_3));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0.3);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0_3[i].x - precision ||
               float4Output0[4 * i] > float4Expect0_3[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0_3[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0_3[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0_3[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0_3[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0_3[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0_3[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|0|Linear:0.3", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect1[float4H * float4W * 2] = {
            {9.98438, 19.9688, 29.9531, 39.9375},
            {0, 9.98438, 19.9688, 29.9531},
            {10.3906, 20.375, 30.3594, 40.3438},
            {0, 10.3906, 20.375, 30.3594}, // 1
            {51.9531, 61.9375, 71.9219, 81.9062},
            {0, 51.9531, 61.9375, 71.9219},
            {52.3594, 62.3438, 72.3281, 82.3125},
            {0, 52.3594, 62.3438, 72.3281}, // 2
            {51.9531, 61.9375, 71.9219, 81.9062},
            {0, 51.9531, 61.9375, 71.9219},
            {52.3594, 62.3438, 72.3281, 82.3125},
            {0, 52.3594, 62.3438, 72.3281}, // 3
            {51.9531, 61.9375, 71.9219, 81.9062},
            {0, 51.9531, 61.9375, 71.9219},
            {52.3594, 62.3438, 72.3281, 82.3125},
            {0, 52.3594, 62.3438, 72.3281}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect1));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 1);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect1[i].x - precision ||
               float4Output0[4 * i] > float4Expect1[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect1[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect1[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect1[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect1[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect1[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect1[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|0|Linear:1", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }

      hipDestroyTextureObject(float4Tex);
    }
    {
      auto float4Tex = getTex(float4Input, 0.1, 0.9, 6, hipFilterModeLinear);

      {
        float4 float4Expect0[float4H * float4W * 2] = {
            {1.97656, 3.95312, 5.92969, 7.90625},
            {0, 1.97656, 3.95312, 5.92969},
            {5.58594, 7.5625, 9.53906, 11.5156},
            {0, 5.58594, 7.5625, 9.53906}, // 1
            {27.9297, 29.9062, 31.8828, 33.8594},
            {0, 27.9297, 29.9062, 31.8828},
            {31.5391, 33.5156, 35.4922, 37.4688},
            {0, 31.5391, 33.5156, 35.4922}, // 2
            {27.9297, 29.9062, 31.8828, 33.8594},
            {0, 27.9297, 29.9062, 31.8828},
            {31.5391, 33.5156, 35.4922, 37.4688},
            {0, 31.5391, 33.5156, 35.4922}, // 3
            {27.9297, 29.9062, 31.8828, 33.8594},
            {0, 27.9297, 29.9062, 31.8828},
            {31.5391, 33.5156, 35.4922, 37.4688},
            {0, 31.5391, 33.5156, 35.4922}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0[i].x - precision ||
               float4Output0[4 * i] > float4Expect0[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|6|Linear:0", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect0_3[float4H * float4W * 2] = {
            {2.48438, 4.96875, 7.45312, 9.9375},
            {0, 2.48438, 4.96875, 7.45312},
            {5.89062, 8.375, 10.8594, 13.3438},
            {0, 5.89062, 8.375, 10.8594}, // 1
            {29.4531, 31.9375, 34.4219, 36.9062},
            {0, 29.4531, 31.9375, 34.4219},
            {32.8594, 35.3438, 37.8281, 40.3125},
            {0, 32.8594, 35.3438, 37.8281}, // 2
            {29.4531, 31.9375, 34.4219, 36.9062},
            {0, 29.4531, 31.9375, 34.4219},
            {32.8594, 35.3438, 37.8281, 40.3125},
            {0, 32.8594, 35.3438, 37.8281}, // 3
            {29.4531, 31.9375, 34.4219, 36.9062},
            {0, 29.4531, 31.9375, 34.4219},
            {32.8594, 35.3438, 37.8281, 40.3125},
            {0, 32.8594, 35.3438, 37.8281}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0_3));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0.3);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0_3[i].x - precision ||
               float4Output0[4 * i] > float4Expect0_3[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0_3[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0_3[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0_3[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0_3[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0_3[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0_3[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|6|Linear:0.3", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect1[float4H * float4W * 2] = {
            {9.98438, 19.9688, 29.9531, 39.9375},
            {0, 9.98438, 19.9688, 29.9531},
            {10.3906, 20.375, 30.3594, 40.3438},
            {0, 10.3906, 20.375, 30.3594}, // 1
            {51.9531, 61.9375, 71.9219, 81.9062},
            {0, 51.9531, 61.9375, 71.9219},
            {52.3594, 62.3438, 72.3281, 82.3125},
            {0, 52.3594, 62.3438, 72.3281}, // 2
            {51.9531, 61.9375, 71.9219, 81.9062},
            {0, 51.9531, 61.9375, 71.9219},
            {52.3594, 62.3438, 72.3281, 82.3125},
            {0, 52.3594, 62.3438, 72.3281}, // 3
            {51.9531, 61.9375, 71.9219, 81.9062},
            {0, 51.9531, 61.9375, 71.9219},
            {52.3594, 62.3438, 72.3281, 82.3125},
            {0, 52.3594, 62.3438, 72.3281}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect1));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 1);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect1[i].x - precision ||
               float4Output0[4 * i] > float4Expect1[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect1[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect1[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect1[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect1[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect1[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect1[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.9|6|Linear:1", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }

      hipDestroyTextureObject(float4Tex);
    }
    {
      auto float4Tex = getTex(float4Input, 0.5, 0.9);

      {
        float4 float4Expect0[float4H * float4W * 2] = {
            {11, 22, 33, 44}, {0, 11, 22, 33},
            {11, 22, 33, 44}, {0, 11, 22, 33}, // 1
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 2
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 3
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0[i].x - precision ||
               float4Output0[4 * i] > float4Expect0[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.5|0.9|0|Point:0", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect0_3[float4H * float4W * 2] = {
            {11, 22, 33, 44}, {0, 11, 22, 33},
            {11, 22, 33, 44}, {0, 11, 22, 33}, // 1
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 2
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 3
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0_3));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0.3);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0_3[i].x - precision ||
               float4Output0[4 * i] > float4Expect0_3[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0_3[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0_3[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0_3[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0_3[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0_3[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0_3[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.5|0.9|0|Point:0.3", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect1[float4H * float4W * 2] = {
            {11, 22, 33, 44}, {0, 11, 22, 33},
            {11, 22, 33, 44}, {0, 11, 22, 33}, // 1
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 2
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 3
            {55, 66, 77, 88}, {0, 55, 66, 77},
            {55, 66, 77, 88}, {0, 55, 66, 77}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect1));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 1);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect1[i].x - precision ||
               float4Output0[4 * i] > float4Expect1[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect1[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect1[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect1[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect1[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect1[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect1[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.5|0.9|0|Point:1", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }

      hipDestroyTextureObject(float4Tex);
    }
    {
      auto float4Tex = getTex(float4Input, 0.1, 0.2);

      {
        float4 float4Expect0[float4H * float4W * 2] = {
            {1, 2, 3, 4},     {0, 1, 2, 3},
            {5, 6, 7, 8},     {0, 5, 6, 7}, // 1
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 2
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 3
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0[i].x - precision ||
               float4Output0[4 * i] > float4Expect0[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.2|0|Point:0", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect0_3[float4H * float4W * 2] = {
            {1, 2, 3, 4},     {0, 1, 2, 3},
            {5, 6, 7, 8},     {0, 5, 6, 7}, // 1
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 2
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 3
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect0_3));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 0.3);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect0_3[i].x - precision ||
               float4Output0[4 * i] > float4Expect0_3[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect0_3[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect0_3[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect0_3[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect0_3[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect0_3[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect0_3[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.2|0|Point:0.3", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }
      {
        float4 float4Expect1[float4H * float4W * 2] = {
            {1, 2, 3, 4},     {0, 1, 2, 3},
            {5, 6, 7, 8},     {0, 5, 6, 7}, // 1
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 2
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 3
            {25, 26, 27, 28}, {0, 25, 26, 27},
            {29, 30, 31, 32}, {0, 29, 30, 31}, // 4
        };
        float *float4Output0;
        hipMallocManaged(&float4Output0, sizeof(float4Expect1));
        kernel4<float4>
            <<<1, 1>>>(float4Output0, float4Tex, float4W, float4H, 1);
        hipDeviceSynchronize();
        float precision = 0.0001;
        for (int i = 0; i < float4H * float4W * 2; ++i) {
          if ((float4Output0[4 * i] < float4Expect1[i].x - precision ||
               float4Output0[4 * i] > float4Expect1[i].x + precision) ||
              (float4Output0[4 * i + 1] < float4Expect1[i].y - precision ||
               float4Output0[4 * i + 1] > float4Expect1[i].y + precision) ||
              (float4Output0[4 * i + 2] < float4Expect1[i].z - precision ||
               float4Output0[4 * i + 2] > float4Expect1[i].z + precision) ||
              (float4Output0[4 * i + 3] < float4Expect1[i].w - precision ||
               float4Output0[4 * i + 3] > float4Expect1[i].w + precision)) {
            pass = false;
            break;
          }
        }
        checkResult("float4|0.1|0.2|0|Point:1", pass);
        if (PRINT_PASS || !pass)
          for (int i = 0; i < float4H; ++i) {
            for (int j = 0; j < float4W; ++j)
              cout << "{" << float4Output0[8 * (float4W * i + j)] << ", "
                   << float4Output0[8 * (float4W * i + j) + 1] << ", "
                   << float4Output0[8 * (float4W * i + j) + 2] << ", "
                   << float4Output0[8 * (float4W * i + j) + 3] << "}, {"
                   << float4Output0[8 * (float4W * i + j) + 4] << ", "
                   << float4Output0[8 * (float4W * i + j) + 5] << ", "
                   << float4Output0[8 * (float4W * i + j) + 6] << ", "
                   << float4Output0[8 * (float4W * i + j) + 7] << "}, ";
            cout << endl;
          }
        pass = true;
      }

      hipDestroyTextureObject(float4Tex);
    }

    hipFreeMipmappedArray(float4Input);
  }

  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
