// ====------------ text_obj_pitch2d.cu---------- *- CUDA -* ------------===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define WIDTH 4 // Must be multiple of 2, Need investigation.
#define HEIGHT 2

__global__ void kernal(int *output, hipTextureObject_t tex) {
  for (int i = 0; i < HEIGHT; ++i) {
    for (int j = 0; j < WIDTH; ++j) {
      auto ret = tex2D<int4>(tex, j, i);
      output[4 * (WIDTH * i + j)] = ret.x;
      output[4 * (WIDTH * i + j) + 1] = ret.y;
      output[4 * (WIDTH * i + j) + 2] = ret.z;
      output[4 * (WIDTH * i + j) + 3] = ret.w;
    }
  }
}

int main() {
  int4 expect[WIDTH * HEIGHT];
  for (int i = 0; i < WIDTH * HEIGHT; i++)
    expect[i] = {i * 4, i * 4 + 1, i * 4 + 2, i * 4 + 3};
  int4 *input;
  hipMalloc(&input, sizeof(expect));
  hipMemcpy(input, &expect, sizeof(expect), hipMemcpyHostToDevice);

  int *output;
  hipMallocManaged(&output, sizeof(expect));

  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = input;
  resDesc.res.pitch2D.width = WIDTH;
  resDesc.res.pitch2D.height = HEIGHT;
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<int4>();
  resDesc.res.pitch2D.pitchInBytes = sizeof(int4) * WIDTH;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;

  hipTextureObject_t tex;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  kernal<<<1, 1>>>(output, tex);
  hipDeviceSynchronize();
  hipDestroyTextureObject(tex);
  hipFree(input);
  for (int i = 0; i < HEIGHT; ++i) {
    for (int j = 0; j < WIDTH; ++j) {
      cout << "{" << output[4 * (WIDTH * i + j)] << ", "
           << output[4 * (WIDTH * i + j) + 1] << ", "
           << output[4 * (WIDTH * i + j) + 2] << ", "
           << output[4 * (WIDTH * i + j) + 3] << "}" << endl;
    }
  }
  for (int i = 0; i < WIDTH * HEIGHT; ++i) {
    if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
        output[4 * i + 2] != expect[i].z || output[4 * i + 3] != expect[i].w) {
      cout << "test failed" << endl;
      return 1;
    }
  }
  return 0;
}
