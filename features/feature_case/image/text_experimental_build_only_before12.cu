#include "hip/hip_runtime.h"
// ===----- text_experimental_build_only_before12.cu ----- *- CUDA -* -----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//

#include <hip/hip_runtime.h>

static texture<float4, 2> r;

void Runtime_MemoryManagement() {
  hipArray_t a = nullptr;
  size_t s = 1;
  hipMemcpyKind k = hipMemcpyDefault;
  void *v = nullptr;
  cudaMemcpyArrayToArray(a, s, s, a, s, s, s, k);
  hipMemcpyFromArray(v, a, s, s, s, k);
  cudaMemcpyFromArrayAsync(v, a, s, s, s, k);
  hipMemcpyToArray(a, s, s, v, s, k);
  cudaMemcpyToArrayAsync(a, s, s, v, s, k);
}

void Runtime_TextureReferenceManagement() {
  size_t s = 1;
  void *v;
  hipChannelFormatDesc d;
  hipArray_t a = nullptr;
  hipMipmappedArray_t ma = nullptr;
  hipBindTexture(&s, &r, v, &d);
  hipBindTexture2D(&s, &r, v, &d, s, s, s);
  hipBindTextureToArray(&r, a, &d);
  hipBindTextureToMipmappedArray(&r, ma, &d);
  hipUnbindTexture(&r);
}

void Driver_TextureReferenceManagement() {
  HIPaddress_mode am;
  hipTexRef r;
  int i = 1;
  HIPfilter_mode fm;
  unsigned int u;
  size_t s = 1;
  hipDeviceptr_t d;
  HIP_ARRAY_DESCRIPTOR D;
  hipArray_t a;
  hipArray_Format f;
  hipTexRefGetAddressMode(&am, r, i);
  hipTexRefGetFilterMode(&fm, r);
  hipTexRefGetFlags(&u, r);
  hipTexRefSetAddress(&s, r, d, s);
  hipTexRefSetAddress2D(r, &D, d, s);
  hipTexRefSetAddressMode(r, i, am);
  hipTexRefSetArray(r, a, u);
  hipTexRefSetFilterMode(r, fm);
  hipTexRefSetFlags(r, u);
  hipTexRefSetFormat(r, f, i);
}

int main() {
  Runtime_MemoryManagement();
  Runtime_TextureReferenceManagement();
  Driver_TextureReferenceManagement();
  return 0;
}
