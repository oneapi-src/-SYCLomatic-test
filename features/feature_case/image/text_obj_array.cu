// ===------------- text_obj_array.cu ---------- *- CUDA -* ---------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

template <typename T, typename EleT>
__global__ void kernel4(EleT *output, hipTextureObject_t tex, int w, int h) {
  for (int i = 0; i < h; ++i) {
    for (int j = 0; j < w; ++j) {
      auto ret = tex2D<T>(tex, j, i);
      output[4 * (w * i + j)] = ret.x;
      output[4 * (w * i + j) + 1] = ret.y;
      output[4 * (w * i + j) + 2] = ret.z;
      output[4 * (w * i + j) + 3] = ret.w;
    }
  }
}

template <typename T, typename ArrT>
hipArray *getInput(ArrT &expect, size_t w, size_t h,
                    const hipChannelFormatDesc &desc) {
  hipArray *input;
  hipMallocArray(&input, &desc, w, h);
  hipMemcpy2DToArray(input, 0, 0, expect, sizeof(T) * w, sizeof(T) * w, h,
                      hipMemcpyHostToDevice);
  return input;
}

hipTextureObject_t
getTex(hipArray_t input,
       hipTextureAddressMode addressMode = hipAddressModeWrap,
       hipTextureFilterMode textureFilterMode = hipFilterModePoint,
       int normalizedCoords = 0) {
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = input;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));

  hipTextureObject_t tex;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  return tex;
}

int main() {
  const int h = 2;
  const int w = 4;
  short4 expect[h * w] = {
      {1, 2, 3, 4},     {5, 6, 7, 8},     {9, 10, 11, 12},  {13, 14, 15, 16},
      {17, 18, 19, 20}, {21, 22, 23, 24}, {25, 26, 27, 28}, {29, 30, 31, 32},
  };
  auto *short4Input =
      getInput<short4>(expect, w, h, hipCreateChannelDesc<short4>());
  short *output;
  hipMallocManaged(&output, sizeof(expect));
  auto short4Tex = getTex(short4Input);
  kernel4<short4><<<1, 1>>>(output, short4Tex, w, h);
  hipDeviceSynchronize();
  hipDestroyTextureObject(short4Tex);
  hipFreeArray(short4Input);

  for (int i = 0; i < h; ++i) {
    for (int j = 0; j < w; ++j) {
      cout << "{" << output[4 * (w * i + j)] << ", "
           << output[4 * (w * i + j) + 1] << ", " << output[4 * (w * i + j) + 2]
           << ", " << output[4 * (w * i + j) + 3] << "}," << endl;
    }
  }
  cout << "short4 test ";
  for (int i = 0; i < w * h; ++i) {
    if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
        output[4 * i + 2] != expect[i].z || output[4 * i + 3] != expect[i].w) {
      cout << "failed!" << endl;
      return 1;
    }
  }
  cout << "passed!" << endl;
  return 0;
}
