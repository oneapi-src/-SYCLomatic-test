// ====------ cublas_64.cu ------------------------------- *- CUDA -* ---==== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ====------------------------------------------------------------------==== //


#include <hip/hip_runtime.h>
#include "hipblas.h"

void foo() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  int64_t m;
  int64_t n;
  int64_t k;
  const float *alpha_s;
  const double *alpha_d;
  const float2 *alpha_c;
  const double2 *alpha_z;
  const float *A_s;
  const double *A_d;
  const float2 *A_c;
  const double2 *A_z;
  int64_t lda;
  const float *B_s;
  const double *B_d;
  const float2 *B_c;
  const double2 *B_z;
  int64_t ldb;
  const float *beta_s;
  const double *beta_d;
  const float2 *beta_c;
  const double2 *beta_z;
  float *C_s;
  double *C_d;
  float2 *C_c;
  double2 *C_z;
  float *C1_s;
  double *C1_d;
  float2 *C1_c;
  double2 *C1_z;
  int64_t ldc;
  hipblasFillMode_t uplo;
  hipblasSideMode_t side;
  hipblasDiagType_t diag;
  int64_t result;
  float result_s;
  double result_d;
  float2 result_c;
  double2 result_z;
  int64_t incx;
  int64_t incy;

  status = hipblasIsamax_64(handle, n, A_s, lda, &result);
  status = hipblasIdamax_64(handle, n, A_d, lda, &result);
  status = hipblasIcamax_64(handle, n, A_c, lda, &result);
  status = hipblasIzamax_64(handle, n, A_z, lda, &result);

  status = hipblasIsamin_64(handle, n, A_s, lda, &result);
  status = hipblasIdamin_64(handle, n, A_d, lda, &result);
  status = hipblasIcamin_64(handle, n, A_c, lda, &result);
  status = hipblasIzamin_64(handle, n, A_z, lda, &result);

  status = hipblasSnrm2_64(handle, n, A_s, incx, &result_s);
  status = hipblasDnrm2_64(handle, n, A_d, incx, &result_d);
  status = hipblasScnrm2_64(handle, n, A_c, incx, &result_s);
  status = hipblasDznrm2_64(handle, n, A_z, incx, &result_d);

  status = hipblasSdot_64(handle, n, A_s, incx, B_s, incy, &result_s);
  status = hipblasDdot_64(handle, n, A_d, incx, B_d, incy, &result_d);
  status = hipblasCdotu_64(handle, n, A_c, incx, B_c, incy, &result_c);
  status = hipblasCdotc_64(handle, n, A_c, incx, B_c, incy, &result_c);
  status = hipblasZdotu_64(handle, n, A_z, incx, B_z, incy, &result_z);
  status = hipblasZdotc_64(handle, n, A_z, incx, B_z, incy, &result_z);

  status = hipblasSscal_64(handle, n, alpha_s, C_s, incx);
  status = hipblasDscal_64(handle, n, alpha_d, C_d, incx);
  status = hipblasCscal_64(handle, n, alpha_c, C_c, incx);
  status = hipblasZscal_64(handle, n, alpha_z, C_z, incx);
  status = hipblasCsscal_64(handle, n, alpha_s, C_c, incx);
  status = hipblasZdscal_64(handle, n, alpha_d, C_z, incx);

  status = hipblasSaxpy_64(handle, n, alpha_s, A_s, incx, C_s, incy);
  status = hipblasDaxpy_64(handle, n, alpha_d, A_d, incx, C_d, incy);
  status = hipblasCaxpy_64(handle, n, alpha_c, A_c, incx, C_c, incy);
  status = hipblasZaxpy_64(handle, n, alpha_z, A_z, incx, C_z, incy);

  status = hipblasScopy_64(handle, n, A_s, incx, C_s, incy);
  status = hipblasDcopy_64(handle, n, A_d, incx, C_d, incy);
  status = hipblasCcopy_64(handle, n, A_c, incx, C_c, incy);
  status = hipblasZcopy_64(handle, n, A_z, incx, C_z, incy);

  status = hipblasSswap_64(handle, n, C_s, incx, C1_s, incy);
  status = hipblasDswap_64(handle, n, C_d, incx, C1_d, incy);
  status = hipblasCswap_64(handle, n, C_c, incx, C1_c, incy);
  status = hipblasZswap_64(handle, n, C_z, incx, C1_z, incy);

  status = hipblasSasum_64(handle, n, A_s, incx, &result_s);
  status = hipblasDasum_64(handle, n, A_d, incx, &result_d);
  status = hipblasScasum_64(handle, n, A_c, incx, &result_s);
  status = hipblasDzasum_64(handle, n, A_z, incx, &result_d);

  status = hipblasSgemm_64(handle, transa, transb, m, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDgemm_64(handle, transa, transb, m, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCgemm_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZgemm_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasSsyrk_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, beta_s, C_s, ldc);
  status = hipblasDsyrk_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, beta_d, C_d, ldc);
  status = hipblasCsyrk_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  status = hipblasZsyrk_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  status = hipblasSsymm_64(handle, side, uplo, m, n, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsymm_64(handle, side, uplo, m, n, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsymm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsymm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasStrsm_64(handle, side, uplo, transa, diag, m, n, alpha_s, A_s, lda, C_s, ldc);
  status = hipblasDtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_d, A_d, lda, C_d, ldc);
  status = hipblasCtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_c, A_c, lda, C_c, ldc);
  status = hipblasZtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_z, A_z, lda, C_z, ldc);

  status = hipblasChemm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZhemm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasCherk_64(handle, uplo, transa, n, k, alpha_s, A_c, lda, beta_s, C_c, ldc);
  status = hipblasZherk_64(handle, uplo, transa, n, k, alpha_d, A_z, lda, beta_d, C_z, ldc);

  status = hipblasSsyr2k_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsyr2k_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsyr2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsyr2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasCher2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_s, C_c, ldc);
  status = hipblasZher2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_d, C_z, ldc);

  status = hipblasSgeam_64(handle, transa, transb, m, n, alpha_s, A_s, lda, beta_s, B_s, ldb, C_s, ldc);
  status = hipblasDgeam_64(handle, transa, transb, m, n, alpha_d, A_d, lda, beta_d, B_d, ldb, C_d, ldc);
  status = hipblasCgeam_64(handle, transa, transb, m, n, alpha_c, A_c, lda, beta_c, B_c, ldb, C_c, ldc);
  status = hipblasZgeam_64(handle, transa, transb, m, n, alpha_z, A_z, lda, beta_z, B_z, ldb, C_z, ldc);

  status = hipblasSdgmm_64(handle, side, m, n, A_s, lda, B_s, ldb, C_s, ldc);
  status = hipblasDdgmm_64(handle, side, m, n, A_d, lda, B_d, ldb, C_d, ldc);
  status = hipblasCdgmm_64(handle, side, m, n, A_c, lda, B_c, ldb, C_c, ldc);
  status = hipblasZdgmm_64(handle, side, m, n, A_z, lda, B_z, ldb, C_z, ldc);
}
