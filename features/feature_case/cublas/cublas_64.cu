// ====------ cublas_64.cu ------------------------------- *- CUDA -* ---==== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ====------------------------------------------------------------------==== //


#include <hip/hip_runtime.h>
#include "hipblas.h"

void foo() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  int64_t m;
  int64_t n;
  int64_t k;
  const float *alpha_s;
  const double *alpha_d;
  const float2 *alpha_c;
  const double2 *alpha_z;
  const float *A_s;
  const double *A_d;
  const float2 *A_c;
  const double2 *A_z;
  int64_t lda;
  const float *B_s;
  const double *B_d;
  const float2 *B_c;
  const double2 *B_z;
  int64_t ldb;
  const float *beta_s;
  const double *beta_d;
  const float2 *beta_c;
  const double2 *beta_z;
  float *C_s;
  double *C_d;
  float2 *C_c;
  double2 *C_z;
  int64_t ldc;
  hipblasFillMode_t uplo;
  hipblasSideMode_t side;
  hipblasDiagType_t diag;

  status = hipblasSgemm_64(handle, transa, transb, m, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDgemm_64(handle, transa, transb, m, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCgemm_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZgemm_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasSsyrk_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, beta_s, C_s, ldc);
  status = hipblasDsyrk_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, beta_d, C_d, ldc);
  status = hipblasCsyrk_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  status = hipblasZsyrk_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  status = hipblasSsymm_64(handle, side, uplo, m, n, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsymm_64(handle, side, uplo, m, n, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsymm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsymm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasStrsm_64(handle, side, uplo, transa, diag, m, n, alpha_s, A_s, lda, C_s, ldc);
  status = hipblasDtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_d, A_d, lda, C_d, ldc);
  status = hipblasCtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_c, A_c, lda, C_c, ldc);
  status = hipblasZtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_z, A_z, lda, C_z, ldc);

  status = hipblasChemm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZhemm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasCherk_64(handle, uplo, transa, n, k, alpha_s, A_c, lda, beta_s, C_c, ldc);
  status = hipblasZherk_64(handle, uplo, transa, n, k, alpha_d, A_z, lda, beta_d, C_z, ldc);

  status = hipblasSsyr2k_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsyr2k_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsyr2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsyr2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  status = hipblasCher2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_s, C_c, ldc);
  status = hipblasZher2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_d, C_z, ldc);

  status = hipblasSgeam_64(handle, transa, transb, m, n, alpha_s, A_s, lda, beta_s, B_s, ldb, C_s, ldc);
  status = hipblasDgeam_64(handle, transa, transb, m, n, alpha_d, A_d, lda, beta_d, B_d, ldb, C_d, ldc);
  status = hipblasCgeam_64(handle, transa, transb, m, n, alpha_c, A_c, lda, beta_c, B_c, ldb, C_c, ldc);
  status = hipblasZgeam_64(handle, transa, transb, m, n, alpha_z, A_z, lda, beta_z, B_z, ldb, C_z, ldc);

  status = hipblasSdgmm_64(handle, side, m, n, A_s, lda, B_s, ldb, C_s, ldc);
  status = hipblasDdgmm_64(handle, side, m, n, A_d, lda, B_d, ldb, C_d, ldc);
  status = hipblasCdgmm_64(handle, side, m, n, A_c, lda, B_c, ldb, C_c, ldc);
  status = hipblasZdgmm_64(handle, side, m, n, A_z, lda, B_z, ldb, C_z, ldc);
}
