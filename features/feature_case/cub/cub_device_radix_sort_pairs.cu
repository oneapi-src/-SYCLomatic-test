#include <hipcub/hipcub.hpp>
#include <initializer_list>
#include <stdio.h>
#include <vector>

template <typename T> T *init(std::initializer_list<T> list) {
  T *arr = nullptr;
  hipMalloc(&arr, sizeof(T) * list.size());
  hipMemcpy(arr, list.begin(), sizeof(T) * list.size(),
             hipMemcpyHostToDevice);
  return arr;
}

bool test() {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  int *d_values_in = init({0, 1, 2, 3, 4, 5, 6});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{0, 3, 5, 6, 7, 8, 9};
  std::vector<int> expected_values_out{5, 4, 3, 1, 2, 0, 6};

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in,
                                  d_keys_out, d_values_in, d_values_out,
                                  num_items);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in,
                                  d_keys_out, d_values_in, d_values_out,
                                  num_items);
  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin()) &&
         std::equal(expected_values_out.begin(), expected_values_out.end(),
                    values_out.begin());
}

bool test1() {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  int *d_values_in = init({0, 1, 2, 3, 4, 5, 6});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{3, 0, 6, 7, 5, 8, 9};
  std::vector<int> expected_values_out{4, 5, 1, 2, 3, 0, };

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in,
                                  d_keys_out, d_values_in, d_values_out,
                                  num_items, 2);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in,
                                  d_keys_out, d_values_in, d_values_out,
                                  num_items, 2);
  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin()) &&
         std::equal(expected_values_out.begin(), expected_values_out.end(),
                    values_out.begin());
}

bool test2() {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  int *d_values_in = init({0, 1, 2, 3, 4, 5, 6});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{3, 0, 6, 7, 5, 8, 9};
  std::vector<int> expected_values_out{4, 5, 1, 2, 3, 0, 6};

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in,
                                  d_keys_out, d_values_in, d_values_out,
                                  num_items, 2, 4);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in,
                                  d_keys_out, d_values_in, d_values_out,
                                  num_items, 2, 4);
  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin()) &&
         std::equal(expected_values_out.begin(), expected_values_out.end(),
                    values_out.begin());
}

bool test3() {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  int *d_values_in = init({0, 1, 2, 3, 4, 5, 6});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{3, 0, 6, 7, 5, 8, 9};
  std::vector<int> expected_values_out{4, 5, 1, 2, 3, 0, 6};

  hipStream_t s = nullptr;
  hipStreamCreate(&s);
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in,
                                  d_keys_out, d_values_in, d_values_out,
                                  num_items, 2, 4, s);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in,
                                  d_keys_out, d_values_in, d_values_out,
                                  num_items, 2, 4, s);
  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin()) &&
         std::equal(expected_values_out.begin(), expected_values_out.end(),
                    values_out.begin());
}

bool test_descending() {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  int *d_values_in = init({0, 1, 2, 3, 4, 5, 6});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{9, 8, 7, 6, 5, 3, 0};
  std::vector<int> expected_values_out{6, 0, 2, 1, 3, 4, 5};

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_keys_in, d_keys_out, d_values_in,
                                            d_values_out, num_items);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_keys_in, d_keys_out, d_values_in,
                                            d_values_out, num_items);
  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin()) &&
         std::equal(expected_values_out.begin(), expected_values_out.end(),
                    values_out.begin());
}

bool test_descending1() {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  int *d_values_in = init({0, 1, 2, 3, 4, 5, 6});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 9, 6, 7, 5, 3, 0};
  std::vector<int> expected_values_out{0, 6, 1, 2, 3, 4, 5};

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_keys_in, d_keys_out, d_values_in,
                                            d_values_out, num_items, 2);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_keys_in, d_keys_out, d_values_in,
                                            d_values_out, num_items, 2);
  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin()) &&
         std::equal(expected_values_out.begin(), expected_values_out.end(),
                    values_out.begin());
}

bool test_descending2() {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  int *d_values_in = init({0, 1, 2, 3, 4, 5, 6});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 9, 6, 7, 5, 3, 0};
  std::vector<int> expected_values_out{0, 6, 1, 2, 3, 4, 5};

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_keys_in, d_keys_out, d_values_in,
                                            d_values_out, num_items, 2, 4);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_keys_in, d_keys_out, d_values_in,
                                            d_values_out, num_items, 2, 4);
  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin()) &&
         std::equal(expected_values_out.begin(), expected_values_out.end(),
                    values_out.begin());
}

bool test_descending3() {
  int num_items = 7;
  int *d_keys_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_keys_out = init({0, 0, 0, 0, 0, 0, 0});
  int *d_values_in = init({0, 1, 2, 3, 4, 5, 6});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 9, 6, 7, 5, 3, 0};
  std::vector<int> expected_values_out{0, 6, 1, 2, 3, 4, 5};
  hipStream_t s = nullptr;
  hipStreamCreate(&s);
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_keys_in, d_keys_out, d_values_in,
                                            d_values_out, num_items, 2, 4, s);
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes,
                                            d_keys_in, d_keys_out, d_values_in,
                                            d_values_out, num_items, 2, 4, s);
  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);
  
  return std::equal(expected_keys_out.begin(), expected_keys_out.end(),
                    keys_out.begin()) &&
         std::equal(expected_values_out.begin(), expected_values_out.end(),
                    values_out.begin());
}

int main() {
  int res = 0;
  if (!test()) {
    printf("hipcub::DeviceRadixSort::SortPairs failed\n");
    res = 1;
  }
  if (!test1()) {
    printf("hipcub::DeviceRadixSort::SortPairs failed\n");
    res = 1;
  }
  if (!test2()) {
    printf("hipcub::DeviceRadixSort::SortPairs failed\n");
    res = 1;
  }
  if (!test3()) {
    printf("hipcub::DeviceRadixSort::SortPairs failed\n");
    res = 1;
  }

  if (!test_descending()) {
    printf("hipcub::DeviceRadixSort::SortPairsDescending failed\n");
    res = 1;
  }

  if (!test_descending1()) {
    printf("hipcub::DeviceRadixSort::SortPairsDescending failed\n");
    res = 1;
  }

  if (!test_descending2()) {
    printf("hipcub::DeviceRadixSort::SortPairsDescending failed\n");
    res = 1;
  }

  if (!test_descending3()) {
    printf("hipcub::DeviceRadixSort::SortPairsDescending failed\n");
    res = 1;
  }

  return res;
}