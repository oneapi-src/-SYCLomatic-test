// ====------ cub_device_run_length_encode_encode.cu------- *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//


#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define DATA_NUM 100

template<typename T = int>
void init_data(T* data, int num) {
  T host_data[DATA_NUM];
  for(int i = 0; i < num; i++)
    host_data[i] = i;
  hipMemcpy(data, host_data, num * sizeof(T), hipMemcpyHostToDevice);
}
template<typename T = int>
bool verify_data(T* data, T* expect, int num, int step = 1) {
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(T), hipMemcpyDeviceToHost);
  for(int i = 0; i < num; i = i + step) {
    if(host_data[i] != expect[i]) {
      return false;
    }
  }
  return true;
}
template<typename T = int>
void print_data(T* data, int num, bool IsHost = false) {
  if(IsHost) {
    for (int i = 0; i < num; i++) {
      std::cout << data[i] << ", ";
      if((i+1)%32 == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    return;
  }
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < num; i++) {
    std::cout << host_data[i] << ", ";
    if((i+1)%32 == 0)
        std::cout << std::endl;
  }
  std::cout << std::endl;
}

// hipcub::DeviceRunLengthEncode::Encode
bool test_device_encode() {
  static const int N = 8;
  int data[N] = {0, 2, 2, 9, 5, 5, 5, 8};
  int *d_in = nullptr;
  int *d_temp = nullptr;
  int *d_unique = nullptr;
  int *d_counts = nullptr;
  int *d_selected_num = nullptr;
  int h_selected_num = 0;
  size_t d_temp_size = 0;

  hipMalloc((void **)&d_in, sizeof(int) * N);
  hipMalloc((void **)&d_unique, sizeof(int) * N);
  hipMalloc((void **)&d_counts, sizeof(int) * N);
  hipMalloc((void **)&d_selected_num, sizeof(int));
  hipMemcpy((void *)d_in, (void *)data, sizeof(data), hipMemcpyHostToDevice);
  hipcub::DeviceRunLengthEncode::Encode(nullptr, d_temp_size, d_in, d_unique, d_counts, d_selected_num, N);
  hipMalloc((void **)&d_temp, d_temp_size);
  hipcub::DeviceRunLengthEncode::Encode(d_temp, d_temp_size, d_in, d_unique, d_counts, d_selected_num, N);
  hipDeviceSynchronize();

  int expect_select_num = 5;
  int expect_unique[] = {0, 2, 9, 5, 8};
  int expect_counts[] = {1, 2, 1, 3, 1};

  
  if (!verify_data(d_selected_num, &expect_select_num, 1)) {
    std::cout << "hipcub::DeviceRunLengthEncode::Encode select_num verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(&expect_select_num, 1, true);
    std::cout << "current result:\n";
    print_data<int>(d_selected_num, 1);
    return false;
  }

  if (!verify_data(d_unique, (int *)expect_unique, expect_select_num)) {
    std::cout << "hipcub::DeviceRunLengthEncode::Encode output unique data verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect_unique, 1, true);
    std::cout << "current result:\n";
    print_data<int>(d_unique, 1);
    return false;
  }

   if (!verify_data(d_counts, (int *)expect_counts, expect_select_num)) {
    std::cout << "hipcub::DeviceRunLengthEncode::Encode output counts data verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect_counts, 1, true);
    std::cout << "current result:\n";
    print_data<int>(d_counts, 1);
    return false;
  }

  return true;
}

int main() {
  if (test_device_encode()) {
    std::cout << "hipcub::DeviceRunLengthEncode::Encode Pass\n";
    return 0;
  }
  return 1;
}
