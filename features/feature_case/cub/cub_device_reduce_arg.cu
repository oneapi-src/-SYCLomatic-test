#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

template <typename T> T *init(std::initializer_list<T> list) {
  T *p = nullptr;
  hipMalloc<T>(&p, sizeof(T) * list.size());
  hipMemcpy(p, list.begin(), sizeof(T) * list.size(), hipMemcpyHostToDevice);
  return p;
}

bool test_arg_max() {
  int num_items = 7;
  int *d_in = init({8, 6, 7, 5, 3, 0, 9});
  hipcub::KeyValuePair<int, int> *d_out =
                                  init<hipcub::KeyValuePair<int, int>>({{-1, -1}}),
                              out;
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(out), hipMemcpyDeviceToHost);
  return out.key == 6 && out.value == 9;
}

bool test_arg_max_non_defaule_stream() {
  int num_items = 7;
  int *d_in = init({8, 6, 7, 5, 3, 0, 9});
  hipcub::KeyValuePair<int, int> *d_out =
                                  init<hipcub::KeyValuePair<int, int>>({{-1, -1}}),
                              out;
  hipStream_t s = nullptr;
  hipStreamCreate(&s);
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items, s);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items, s);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(out), hipMemcpyDeviceToHost);
  hipStreamDestroy(s);
  return out.key == 6 && out.value == 9;
}

bool test_arg_min() {
  int num_items = 7;
  int *d_in = init({8, 6, 7, 5, 3, 0, 9});
  hipcub::KeyValuePair<int, int> *d_out =
                                  init<hipcub::KeyValuePair<int, int>>({{-1, -1}}),
                              out;
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(out), hipMemcpyDeviceToHost);
  return out.key == 5 && out.value == 0;
}

bool test_arg_min_non_default_stream() {
  int num_items = 7;
  int *d_in = init({8, 6, 7, 5, 3, 0, 9});
  hipcub::KeyValuePair<int, int> *d_out =
                                  init<hipcub::KeyValuePair<int, int>>({{-1, -1}}),
                              out;
  hipStream_t s = nullptr;
  hipStreamCreate(&s);
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items, s);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items, s);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(out), hipMemcpyDeviceToHost);
  hipStreamDestroy(s);
  return out.key == 5 && out.value == 0;
}

__global__ void test_arg_min_max_op_in_device(int *Res) {
  hipcub::KeyValuePair<int, int> LHS{1, 3}, RHS{2, 4};
  hipcub::ArgMin MinOp;
  hipcub::ArgMax MaxOp;
  auto Min = MinOp(LHS, RHS);
  auto Max = MaxOp(LHS, RHS);
  *Res = Min.key == 1 && Min.value == 3 && Max.key == 2 && Max.value == 4;
}

bool test_arg_min_max_op_in_host() {
  hipcub::KeyValuePair<int, int> LHS{1, 3}, RHS{2, 4};
  hipcub::ArgMin MinOp;
  hipcub::ArgMax MaxOp;
  auto Min = MinOp(LHS, RHS);
  auto Max = MaxOp(LHS, RHS);
  return Min.key == 1 && Min.value == 3 && Max.key == 2 && Max.value == 4;
}

bool test_arg_min_max_op() {
  int *Res;
  hipMallocManaged(&Res, sizeof(int));
  *Res = 0;
  test_arg_min_max_op_in_device<<<1, 1>>>(Res);
  hipDeviceSynchronize();
  bool Val = *Res;
  hipFree(Res);
  return Val && test_arg_min_max_op_in_host();
}

int main() {
  int res = 0;
  if (!test_arg_max()) {
    res = 1;
    std::cout << "hipcub::DeviceReduce::ArgMax test failed\n";
  }

  if (!test_arg_max_non_defaule_stream()) {
    res = 1;
    std::cout << "hipcub::DeviceReduce::ArgMax(Non default stream) test failed\n";
  }

  if (!test_arg_min()) {
    res = 1;
    std::cout << "hipcub::DeviceReduce::ArgMin test failed\n";
  }

  if (!test_arg_min_non_default_stream()) {
    res = 1;
    std::cout << "hipcub::DeviceReduce::ArgMin(Non default stream) test failed\n";
  }

  if (!test_arg_min_max_op()) {
    res = 1;
    std::cout << "cub::{ArgMin, ArgMax} binary operator test failed\n";
  }

  return res;
}
