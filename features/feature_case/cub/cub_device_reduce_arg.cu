#include <hipcub/hipcub.hpp>

template <typename T> T *init(std::initializer_list<T> list) {
  T *p = nullptr;
  hipMalloc<T>(&p, sizeof(T) * list.size());
  hipMemcpy(p, list.begin(), sizeof(T) * list.size(), hipMemcpyHostToDevice);
  return p;
}

bool test_arg_max() {
  int num_items = 7;
  int *d_in = init({8, 6, 7, 5, 3, 0, 9});
  hipcub::KeyValuePair<int, int> *d_out =
                                  init<hipcub::KeyValuePair<int, int>>({{-1, -1}}),
                              out;
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(out), hipMemcpyDeviceToHost);
  return out.key == 6 && out.value == 9;
}

bool test_arg_max_non_defaule_stream() {
  int num_items = 7;
  int *d_in = init({8, 6, 7, 5, 3, 0, 9});
  hipcub::KeyValuePair<int, int> *d_out =
                                  init<hipcub::KeyValuePair<int, int>>({{-1, -1}}),
                              out;
  hipStream_t s = nullptr;
  hipStreamCreate(&s);
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items, s);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items, s);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(out), hipMemcpyDeviceToHost);
  hipStreamDestroy(s);
  return out.key == 6 && out.value == 9;
}

bool test_arg_min() {
  int num_items = 7;
  int *d_in = init({8, 6, 7, 5, 3, 0, 9});
  hipcub::KeyValuePair<int, int> *d_out =
                                  init<hipcub::KeyValuePair<int, int>>({{-1, -1}}),
                              out;
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(out), hipMemcpyDeviceToHost);
  return out.key == 5 && out.value == 0;
}

bool test_arg_min_non_default_stream() {
  int num_items = 7;
  int *d_in = init({8, 6, 7, 5, 3, 0, 9});
  hipcub::KeyValuePair<int, int> *d_out =
                                  init<hipcub::KeyValuePair<int, int>>({{-1, -1}}),
                              out;
  hipStream_t s = nullptr;
  hipStreamCreate(&s);
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items, s);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out,
                            num_items, s);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(out), hipMemcpyDeviceToHost);
  hipStreamDestroy(s);
  return out.key == 5 && out.value == 0;
}

int main() {
  int res = 0;
  if (!test_arg_max()) {
    res = 1;
    std::cout << "hipcub::DeviceReduce::ArgMax test failed\n";
  }

  if (!test_arg_max_non_defaule_stream()) {
    res = 1;
    std::cout << "hipcub::DeviceReduce::ArgMax(Non default stream) test failed\n";
  }

  if (!test_arg_min()) {
    res = 1;
    std::cout << "hipcub::DeviceReduce::ArgMin test failed\n";
  }

  if (!test_arg_min_non_default_stream()) {
    res = 1;
    std::cout << "hipcub::DeviceReduce::ArgMin(Non default stream) test failed\n";
  }

  return res;
}
