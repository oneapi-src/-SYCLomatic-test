// ====------ cub_device_reduce_max.cu-------------------- *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hipcub/hipcub.hpp>
#include <initializer_list>
#include <cstddef>

template <typename T> T *init(std::initializer_list<T> list) {
  T *p = nullptr;
  hipMalloc<T>(&p, sizeof(T) * list.size());
  hipMemcpy(p, list.begin(), sizeof(T) * list.size(), hipMemcpyHostToDevice);
  return p;
}

int num_items = 7;
int *d_in;
int *d_out;
int out;

bool test1() {
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(int), hipMemcpyDeviceToHost);
  return out == 9;
}

bool test2() {
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipStream_t s;
  hipStreamCreate(&s);
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, s);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, s);
  hipStreamDestroy(s);
  hipFree(d_temp_storage);
  hipMemcpy(&out, d_out, sizeof(int), hipMemcpyDeviceToHost);
  return out == 9;
}

int main() {
  d_in = init({8, 6, 7, 5, -3, 0, 9});
  d_out = init({0});
  bool res = test1();
  res =  test2() && res;
  hipFree(d_in);
  hipFree(d_out);

  if (!res) {
    printf("hipcub::DeviceReduce::Max test failed!\n");
    return 1;
  }
  return 0;
}
