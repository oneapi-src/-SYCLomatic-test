#include <cstdlib>
#include <iostream>
#include <vector>

#include <hipcub/hipcub.hpp>
#include <cub/device/device_segmented_sort.cuh>

bool testStableSortPairs() {
  // Declare, allocate, and initialize device-accessible pointers
  // for sorting data
  int num_items;     // e.g., 7
  int num_segments;  // e.g., 3
  int *d_offsets;    // e.g., [0, 3, 3, 7]
  int *d_keys_in;    // e.g., [8, 6, 7, 5, 3, 0, 9]
  int *d_keys_out;   // e.g., [-, -, -, -, -, -, -]
  int *d_values_in;  // e.g., [0, 1, 2, 3, 4, 5, 6]
  int *d_values_out; // e.g., [-, -, -, -, -, -, -]

  num_items = 7;
  num_segments = 3;
  hipMallocManaged(&d_offsets, (num_segments + 1) * sizeof(*d_offsets));
  hipMallocManaged(&d_keys_in, num_items * sizeof(*d_keys_in));
  hipMallocManaged(&d_keys_out, num_items * sizeof(*d_keys_out));
  hipMallocManaged(&d_values_in, num_items * sizeof(*d_values_in));
  hipMallocManaged(&d_values_out, num_items * sizeof(*d_values_out));

  d_offsets[0] = 0;
  d_offsets[1] = 3;
  d_offsets[2] = 3;
  d_offsets[3] = 7;

  d_keys_in[0] = 8;
  d_keys_in[1] = 6;
  d_keys_in[2] = 7;
  d_keys_in[3] = 5;
  d_keys_in[4] = 3;
  d_keys_in[5] = 0;
  d_keys_in[6] = 9;

  d_values_in[0] = 0;
  d_values_in[1] = 1;
  d_values_in[2] = 2;
  d_values_in[3] = 3;
  d_values_in[4] = 4;
  d_values_in[5] = 5;
  d_values_in[6] = 6;

  hipDeviceSynchronize();

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::StableSortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1);

  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  // Run sorting operation
  hipcub::DeviceSegmentedSort::StableSortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1);

  // d_keys_out            <-- [6, 7, 8, 0, 3, 5, 9]
  // d_values_out          <-- [1, 2, 0, 5, 4, 3, 6]

  std::vector<int> d_keys_out_expected = {6, 7, 8, 0, 3, 5, 9};
  std::vector<int> d_values_out_expected = {1, 2, 0, 5, 4, 3, 6};

  hipDeviceSynchronize();
  for (int i = 0; i < num_items; i++) {
    if (d_keys_out[i] != d_keys_out_expected[i] ||
        d_values_out[i] != d_values_out_expected[i]) {
      return false;
    }
  }

  return true;
}

bool testStableSortPairsDB() {
  // Declare, allocate, and initialize device-accessible pointers
  // for sorting data
  int num_items;     // e.g., 7
  int num_segments;  // e.g., 3
  int *d_offsets;    // e.g., [0, 3, 3, 7]
  int *d_keys_in;    // e.g., [8, 6, 7, 5, 3, 0, 9]
  int *d_keys_out;   // e.g., [-, -, -, -, -, -, -]
  int *d_values_in;  // e.g., [0, 1, 2, 3, 4, 5, 6]
  int *d_values_out; // e.g., [-, -, -, -, -, -, -]

  num_items = 7;
  num_segments = 3;
  hipMallocManaged(&d_offsets, (num_segments + 1) * sizeof(*d_offsets));
  hipMallocManaged(&d_keys_in, num_items * sizeof(*d_keys_in));
  hipMallocManaged(&d_keys_out, num_items * sizeof(*d_keys_out));
  hipMallocManaged(&d_values_in, num_items * sizeof(*d_values_in));
  hipMallocManaged(&d_values_out, num_items * sizeof(*d_values_out));

  d_offsets[0] = 0;
  d_offsets[1] = 3;
  d_offsets[2] = 3;
  d_offsets[3] = 7;

  d_keys_in[0] = 8;
  d_keys_in[1] = 6;
  d_keys_in[2] = 7;
  d_keys_in[3] = 5;
  d_keys_in[4] = 3;
  d_keys_in[5] = 0;
  d_keys_in[6] = 9;

  d_values_in[0] = 0;
  d_values_in[1] = 1;
  d_values_in[2] = 2;
  d_values_in[3] = 3;
  d_values_in[4] = 4;
  d_values_in[5] = 5;
  d_values_in[6] = 6;

  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  hipcub::DoubleBuffer<int> d_values(d_values_in, d_values_out);

  hipDeviceSynchronize();

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::StableSortPairs(
      d_temp_storage, temp_storage_bytes, d_keys, d_values, num_items,
      num_segments, d_offsets, d_offsets + 1);

  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  // Run sorting operation
  hipcub::DeviceSegmentedSort::StableSortPairs(
      d_temp_storage, temp_storage_bytes, d_keys, d_values, num_items,
      num_segments, d_offsets, d_offsets + 1);

  // d_keys.Current()      <-- [6, 7, 8, 0, 3, 5, 9]
  // d_values.Current()    <-- [1, 2, 0, 5, 4, 3, 6]

  std::vector<int> d_keys_out_expected = {6, 7, 8, 0, 3, 5, 9};
  std::vector<int> d_values_out_expected = {1, 2, 0, 5, 4, 3, 6};

  hipDeviceSynchronize();
  for (int i = 0; i < num_items; i++) {
    if (d_keys.Current()[i] != d_keys_out_expected[i] ||
        d_values.Current()[i] != d_values_out_expected[i]) {
      return false;
    }
  }

  return true;
}

bool testStableSortPairsDescending() {
  // Declare, allocate, and initialize device-accessible pointers
  // for sorting data
  int num_items;     // e.g., 7
  int num_segments;  // e.g., 3
  int *d_offsets;    // e.g., [0, 3, 3, 7]
  int *d_keys_in;    // e.g., [8, 6, 7, 5, 3, 0, 9]
  int *d_keys_out;   // e.g., [-, -, -, -, -, -, -]
  int *d_values_in;  // e.g., [0, 1, 2, 3, 4, 5, 6]
  int *d_values_out; // e.g., [-, -, -, -, -, -, -]

  num_items = 7;
  num_segments = 3;
  hipMallocManaged(&d_offsets, (num_segments + 1) * sizeof(*d_offsets));
  hipMallocManaged(&d_keys_in, num_items * sizeof(*d_keys_in));
  hipMallocManaged(&d_keys_out, num_items * sizeof(*d_keys_out));
  hipMallocManaged(&d_values_in, num_items * sizeof(*d_values_in));
  hipMallocManaged(&d_values_out, num_items * sizeof(*d_values_out));

  d_offsets[0] = 0;
  d_offsets[1] = 3;
  d_offsets[2] = 3;
  d_offsets[3] = 7;

  d_keys_in[0] = 8;
  d_keys_in[1] = 6;
  d_keys_in[2] = 7;
  d_keys_in[3] = 5;
  d_keys_in[4] = 3;
  d_keys_in[5] = 0;
  d_keys_in[6] = 9;

  d_values_in[0] = 0;
  d_values_in[1] = 1;
  d_values_in[2] = 2;
  d_values_in[3] = 3;
  d_values_in[4] = 4;
  d_values_in[5] = 5;
  d_values_in[6] = 6;

  hipDeviceSynchronize();

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::StableSortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1);

  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  // Run sorting operation
  hipcub::DeviceSegmentedSort::StableSortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1);

  // d_keys_out            <-- [8, 7, 6, 9, 5, 3, 0]
  // d_values_out          <-- [0, 2, 1, 6, 3, 4, 5]

  std::vector<int> d_keys_out_expected = {8, 7, 6, 9, 5, 3, 0};
  std::vector<int> d_values_out_expected = {0, 2, 1, 6, 3, 4, 5};

  hipDeviceSynchronize();
  for (int i = 0; i < num_items; i++) {
    if (d_keys_out[i] != d_keys_out_expected[i] ||
        d_values_out[i] != d_values_out_expected[i]) {
      return false;
    }
  }

  return true;
}

bool testStableSortPairsDescendingDB() {
  // Declare, allocate, and initialize device-accessible pointers
  // for sorting data
  int num_items;     // e.g., 7
  int num_segments;  // e.g., 3
  int *d_offsets;    // e.g., [0, 3, 3, 7]
  int *d_keys_in;    // e.g., [8, 6, 7, 5, 3, 0, 9]
  int *d_keys_out;   // e.g., [-, -, -, -, -, -, -]
  int *d_values_in;  // e.g., [0, 1, 2, 3, 4, 5, 6]
  int *d_values_out; // e.g., [-, -, -, -, -, -, -]

  num_items = 7;
  num_segments = 3;
  hipMallocManaged(&d_offsets, (num_segments + 1) * sizeof(*d_offsets));
  hipMallocManaged(&d_keys_in, num_items * sizeof(*d_keys_in));
  hipMallocManaged(&d_keys_out, num_items * sizeof(*d_keys_out));
  hipMallocManaged(&d_values_in, num_items * sizeof(*d_values_in));
  hipMallocManaged(&d_values_out, num_items * sizeof(*d_values_out));

  d_offsets[0] = 0;
  d_offsets[1] = 3;
  d_offsets[2] = 3;
  d_offsets[3] = 7;

  d_keys_in[0] = 8;
  d_keys_in[1] = 6;
  d_keys_in[2] = 7;
  d_keys_in[3] = 5;
  d_keys_in[4] = 3;
  d_keys_in[5] = 0;
  d_keys_in[6] = 9;

  d_values_in[0] = 0;
  d_values_in[1] = 1;
  d_values_in[2] = 2;
  d_values_in[3] = 3;
  d_values_in[4] = 4;
  d_values_in[5] = 5;
  d_values_in[6] = 6;

  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  hipcub::DoubleBuffer<int> d_values(d_values_in, d_values_out);

  hipDeviceSynchronize();

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::StableSortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys, d_values, num_items,
      num_segments, d_offsets, d_offsets + 1);

  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);

  // Run sorting operation
  hipcub::DeviceSegmentedSort::StableSortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys, d_values, num_items,
      num_segments, d_offsets, d_offsets + 1);

  // d_keys.Current()      <-- [8, 7, 6, 9, 5, 3, 0]
  // d_values.Current()    <-- [0, 2, 1, 6, 3, 4, 5]

  std::vector<int> d_keys_out_expected = {8, 7, 6, 9, 5, 3, 0};
  std::vector<int> d_values_out_expected = {0, 2, 1, 6, 3, 4, 5};

  hipDeviceSynchronize();
  for (int i = 0; i < num_items; i++) {
    if (d_keys.Current()[i] != d_keys_out_expected[i] ||
        d_values.Current()[i] != d_values_out_expected[i]) {
      return false;
    }
  }

  return true;
}

int main() {
  if (!testStableSortPairs()) {
    std::cerr << "StableSortPairs test failed" << std::endl;
    return EXIT_FAILURE;
  }

  if (!testStableSortPairsDB()) {
    std::cerr << "StableSortPairs (DoubleBuffer) test failed" << std::endl;
    return EXIT_FAILURE;
  }

  if (!testStableSortPairsDescending()) {
    std::cerr << "StableSortPairsDescending test failed" << std::endl;
    return EXIT_FAILURE;
  }

  if (!testStableSortPairsDescendingDB()) {
    std::cerr << "StableSortPairsDescending (DoubleBuffer) test failed"
              << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Tests passed" << std::endl;
  return EXIT_SUCCESS;
}
