// ====------ cub_device_scan_inclusive_scan.cu----------- *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define DATA_NUM 100

struct ScanOp {
  template <typename T, typename = typename std::enable_if<
                            std::is_arithmetic<T>::value>::type>
  __device__ T operator()(const T &lhs, const T &rhs) const {
    return lhs + rhs;
  }
};

template<typename T = int>
void init_data(T* data, int num) {
  T host_data[DATA_NUM];
  for(int i = 0; i < num; i++)
    host_data[i] = i;
  hipMemcpy(data, host_data, num * sizeof(T), hipMemcpyHostToDevice);
}
template<typename T = int>
bool verify_data(T* data, T* expect, int num, int step = 1) {
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(T), hipMemcpyDeviceToHost);
  for(int i = 0; i < num; i = i + step) {
    if(host_data[i] != expect[i]) {
      return false;
    }
  }
  return true;
}
template<typename T = int>
void print_data(T* data, int num, bool IsHost = false) {
  if(IsHost) {
    for (int i = 0; i < num; i++) {
      std::cout << data[i] << ", ";
      if((i+1)%32 == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    return;
  }
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < num; i++) {
    std::cout << host_data[i] << ", ";
    if((i+1)%32 == 0)
        std::cout << std::endl;
  }
  std::cout << std::endl;
}

// hipcub::DeviceScan::InclusiveScan
bool test_device_inclusive_scan() {
  static const int n = 10;
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect[n] = {0, 1, 3, 6, 10, 15, 21, 28, 36, 45};
  hipMalloc((void **)&device_in, sizeof(int) * n);
  hipMalloc((void **)&device_out, sizeof(int) * n);
  init_data(device_in, n);
  ScanOp scan_op;
  hipcub::DeviceScan::InclusiveScan(temp_storage, temp_storage_size, device_in,
                                device_out, scan_op, n);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceScan::InclusiveScan(temp_storage, temp_storage_size, device_in,
                                device_out, scan_op, n);
  hipDeviceSynchronize();
  if (!verify_data(device_out, expect, n)) {
    std::cout << "hipcub::DeviceScan::InclusiveScan verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, n);
    return false;
  }
  return true;
}

int main() {
  if (test_device_inclusive_scan()) {
    std::cout << "hipcub::DeviceScan::InclusiveScan Pass\n";
    return 0;
  }
  return 1;
}
