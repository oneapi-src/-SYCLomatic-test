#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include <vector>

// CustomMin functor
struct CustomMin {
  template <typename T>
  __device__ __forceinline__ T operator()(const T &a, const T &b) const {
    return (b < a) ? b : a;
  }
};
// CustomEqual functor
struct CustomEqual {
  template <typename T>
  __device__ __forceinline__ T operator()(const T &a, const T &b) const {
    return a == b;
  }
};

template <typename T> T *init(std::initializer_list<T> list) {
  T *arr = nullptr;
  hipMalloc(&arr, sizeof(T) * list.size());
  hipMemcpy(arr, list.begin(), sizeof(T) * list.size(),
             hipMemcpyHostToDevice);
  return arr;
}

bool test() {
  int num_items = 7;
  int *d_keys_in = init({0, 0, 1, 1, 1, 2, 2});
  int *d_values_in = init({8, 6, 7, 5, 3, 0, 9});
  int *d_values_out = init({0, 0, 0, 0, 0, 0, 0});
  CustomMin min_op;
  CustomEqual equality_op;
  std::vector<int> expected({INT_MAX, 8, INT_MAX, 7, 5, INT_MAX, 0});
  // Determine temporary device storage requirements for exclusive prefix scan
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveScanByKey(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_values_in, d_values_out,
      min_op, (int)INT_MAX, num_items, equality_op);
  // Allocate temporary storage for exclusive prefix scan
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  // Run exclusive prefix min-scan
  hipcub::DeviceScan::ExclusiveScanByKey(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_values_in, d_values_out,
      min_op, (int)INT_MAX, num_items, equality_op);
  std::vector<int> output(num_items, 0);
  hipMemcpy(output.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_keys_in);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  return std::equal(expected.begin(), expected.end(), output.begin());
}

int main() {
  if (!test()) {
    printf("hipcub::DeviceScan::ExclusiveScanByKey test failed!\n");
    return 1;
  }
  return 0;
}
