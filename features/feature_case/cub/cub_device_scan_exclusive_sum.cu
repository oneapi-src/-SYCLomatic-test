// ====------ cub_device_scan_exclusive_sum.cu------------ *- CUDA -* ------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define DATA_NUM 100

template<typename T = int>
void init_data(T* data, int num) {
  T host_data[DATA_NUM];
  for(int i = 0; i < num; i++)
    host_data[i] = i;
  hipMemcpy(data, host_data, num * sizeof(T), hipMemcpyHostToDevice);
}
template<typename T = int>
bool verify_data(T* data, T* expect, int num, int step = 1) {
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(T), hipMemcpyDeviceToHost);
  for(int i = 0; i < num; i = i + step) {
    if(host_data[i] != expect[i]) {
      return false;
    }
  }
  return true;
}
template<typename T = int>
void print_data(T* data, int num, bool IsHost = false) {
  if(IsHost) {
    for (int i = 0; i < num; i++) {
      std::cout << data[i] << ", ";
      if((i+1)%32 == 0)
        std::cout << std::endl;
    }
    std::cout << std::endl;
    return;
  }
  T host_data[DATA_NUM];
  hipMemcpy(host_data, data, num * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < num; i++) {
    std::cout << host_data[i] << ", ";
    if((i+1)%32 == 0)
        std::cout << std::endl;
  }
  std::cout << std::endl;
}

// hipcub::DeviceScan::ExclusiveSum
bool test_device_scan_exclusive_sum() {
  static const int n = 10;
  int *device_in;
  int *device_out;
  void *temp_storage = NULL;
  size_t temp_storage_size = 0;
  int expect[n] = {0, 0, 1, 3, 6, 10, 15, 21, 28, 36};
  hipMalloc((void **)&device_in, sizeof(int) * n);
  hipMalloc((void **)&device_out, sizeof(int) * n);
  init_data(device_in, n);
  hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipMalloc((void **)&temp_storage, temp_storage_size);
  hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_size, device_in,
                                device_out, n);
  hipDeviceSynchronize();
  if (!verify_data(device_out, expect, n)) {
    std::cout << "hipcub::DeviceScan::ExclusiveSum verify failed\n";
    std::cout << "expect:\n";
    print_data<int>(expect, 1, true);
    std::cout << "current result:\n";
    print_data<int>(device_out, n);
    return false;
  }
  return true;
}

int main() {
  if (test_device_scan_exclusive_sum()) {
    std::cout << "hipcub::DeviceScan::ExclusiveSum Pass\n";
    return 0;
  }
  return 1;
}
