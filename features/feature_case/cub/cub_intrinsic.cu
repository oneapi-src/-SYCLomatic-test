#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

template <typename T> T *init(std::initializer_list<T> list) {
  T *p = nullptr;
  hipMalloc<T>(&p, sizeof(T) * list.size());
  hipMemcpy(p, list.begin(), sizeof(T) * list.size(), hipMemcpyHostToDevice);
  return p;
}

__global__ void iadd3_kernel(int x, int y, int z, int *output) {
  *output = hipcub::IADD3(x, y, z);
}

bool iadd3(int x, int y, int z) {
  int output, *d_output = init({0});
  iadd3_kernel<<<1, 1>>>(x, y, z, d_output);
  hipMemcpy(&output, d_output, sizeof(int), hipMemcpyDeviceToHost);
  if (output != x + y + z) {
    std::cout << "hipcub::IADD3 test failed"
                 "\n";
    std::cout << "input: " << x << " " << y << " " << z << "\n";
    std::cout << "expected: " << output << "\n";
    std::cout << "result: " << x + y + z << "\n";
    return false;
  }
  return true;
}

bool test_iadd3() {
  return iadd3(1, 2, 3) && iadd3(4, 5, 6) && iadd3(9991, 12, 7) &&
         iadd3(0, 1, 0);
}

int main() {
  if (!test_iadd3()) {
    return 1;
  }
  return 0;
}
