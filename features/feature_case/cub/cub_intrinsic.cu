#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

template <typename T> T *init(std::initializer_list<T> list) {
  T *p = nullptr;
  hipMalloc<T>(&p, sizeof(T) * list.size());
  hipMemcpy(p, list.begin(), sizeof(T) * list.size(), hipMemcpyHostToDevice);
  return p;
}

__global__ void iadd3_kernel(int x, int y, int z, int *output) {
  *output = hipcub::IADD3(x, y, z);
}

bool iadd3(int x, int y, int z) {
  int output, *d_output = init({0});
  iadd3_kernel<<<1, 1>>>(x, y, z, d_output);
  hipMemcpy(&output, d_output, sizeof(int), hipMemcpyDeviceToHost);
  if (output != x + y + z) {
    std::cout << "hipcub::IADD3 test failed"
                 "\n";
    std::cout << "input: " << x << " " << y << " " << z << "\n";
    std::cout << "expected: " << output << "\n";
    std::cout << "result: " << x + y + z << "\n";
    return false;
  }
  return true;
}

bool test_iadd3() {
  return iadd3(1, 2, 3) && iadd3(4, 5, 6) && iadd3(9991, 12, 7) &&
         iadd3(0, 1, 0);
}

__global__ void laneid_and_warpid(int *laneids, int *warpids) {
  unsigned tid =
      ((blockIdx.x + (blockIdx.y * gridDim.x)) * (blockDim.x * blockDim.y)) +
      (threadIdx.x + (threadIdx.y * blockDim.x));
  laneids[tid] = hipcub::LaneId();
  warpids[tid] = hipcub::WarpId();
}

bool test_laneid_warpid() {
  int *d_warpids, *d_laneids;
  hipMalloc(&d_laneids, sizeof(int) * 66);
  hipMalloc(&d_warpids, sizeof(int) * 66);
  laneid_and_warpid<<<2, 33>>>(d_laneids, d_warpids);
  hipDeviceSynchronize();
  int laneids[66] = {0}, warpids[66] = {0};
  hipMemcpy(laneids, d_laneids, sizeof(int) * 66, hipMemcpyDeviceToHost);
  hipMemcpy(warpids, d_warpids, sizeof(int) * 66, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  std::map<int, int> cnt_laneid, cnt_warpid, cnt_laneid_num;
  for (int I = 0; I < 66; ++I) {
    cnt_warpid[warpids[I]]++;
    cnt_laneid[laneids[I]]++;
  }

  int total_warpid = 0;
  for (const auto &[k, v] : cnt_warpid)
    total_warpid += v;
  for (const auto &[k, v] : cnt_laneid)
    cnt_laneid_num[v]++;

  auto check_laneid_num = [&]() {
    if (cnt_laneid_num.size() != 2)
      return false;
    const auto first = *cnt_laneid_num.begin();
    const auto second = *std::next(cnt_laneid_num.begin());
    return first.first + 2 == second.first;
  };

  hipFree(d_laneids);
  hipFree(d_warpids);
  return total_warpid == 66 && check_laneid_num();
}

bool test_current_device() {
  unsigned CurDev = hipcub::CurrentDevice();
  return true;
}

bool test_device_count() {
  unsigned device_count = hipcub::DeviceCount();
  device_count = hipcub::DeviceCountCachedValue();
  device_count = hipcub::DeviceCountUncached();
  (void) device_count;
  return true;
}

bool test_sync_stream() {
  hipcub::SyncStream(0);
  hipcub::SyncStream((hipStream_t)(uintptr_t)1);
  hipcub::SyncStream((hipStream_t)(uintptr_t)2);
  hipStream_t NewS;
  hipStreamCreate(&NewS);
  hipcub::SyncStream(NewS);
  hipStreamDestroy(NewS);
  (void) NewS;
  return true;
}

bool test_ptx_version() {
  int ver = 0;
  hipcub::PtxVersion(ver);
  hipcub::PtxVersion(ver, 0);
  hipcub::PtxVersionUncached(ver);
  hipcub::PtxVersionUncached(ver, 0);
  (void) ver;
  return true;
}

__global__ void bfe_kernel(int *res) {
  if (hipcub::BFE((uint8_t)0xF0, 4, 8) != 15) {
    *res = 1;
    return;
  }
  if (hipcub::BFE((uint16_t)0x0FF0u, 4, 12) != 255) {
    *res = 2;
    return;
  }
  if (hipcub::BFE(0x00FFFF00u, 8, 16) != 65535u) {
    *res = 3;
    return;
  }
  if (hipcub::BFE(0x000000FFull, 0, 9) != 255) {
    *res = 4;
    return;
  }
  *res = 0;
}

__global__ void bfi_kernel(int *res) {
  unsigned d = 0;
  hipcub::BFI(d, 0x00FF0000u, 0x0000FFFFu, 0, 16);
  if (d != 0x00FFFFFFu) {
    *res = 1;
    return;
  }

  hipcub::BFI(d, 0x00FF0000u, 0x000000FFu, 0, 8);
  if (d != 0x00FF00FFu) {
    *res = 2;
    return;
  }
  *res = 0;
}

bool test_bfe() {
  int *res;
  hipMallocManaged(&res, sizeof(int));
  bfe_kernel<<<1, 1>>>(res);
  hipDeviceSynchronize();
  return *res == 0;
}

bool test_bfi() {
  int *res;
  hipMallocManaged(&res, sizeof(int));
  bfi_kernel<<<1, 1>>>(res);
  hipDeviceSynchronize();
  return *res == 0;
}

#define TEST(FUNC)                                                             \
  if (!FUNC()) {                                                               \
    printf(#FUNC " failed\n");                                                 \
    return 1;                                                                  \
  }

int main() {
  TEST(test_iadd3);
  TEST(test_laneid_warpid);
  TEST(test_current_device);
  TEST(test_device_count);
  TEST(test_sync_stream);
  TEST(test_ptx_version);
  TEST(test_bfe);
  TEST(test_bfi);
  return 0;
}
