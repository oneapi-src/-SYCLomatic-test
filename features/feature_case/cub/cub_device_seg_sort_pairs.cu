#include <hipcub/hipcub.hpp>
#include <initializer_list>
#include <stdio.h>
#include <vector>

template <typename T> T *init(std::initializer_list<T> list) {
  T *arr = nullptr;
  hipMalloc(&arr, sizeof(T) * list.size());
  hipMemcpy(arr, list.begin(), sizeof(T) * list.size(),
             hipMemcpyHostToDevice);
  return arr;
}

bool test() {
  // clang-format off
  
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  int  *d_values_in       = init({0, 1, 2, 3, 4, 5, 6});
  int  *d_values_out      = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{6, 7, 8, 0, 3, 5, 9};
  std::vector<int> expected_values_out{1, 2, 0, 5, 4, 3, 6};
  // clang-format on

  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1);
  
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  
  hipcub::DeviceSegmentedSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin()) &&
         std::equal(values_out.begin(), values_out.end(),
                    expected_values_out.begin());
}

bool test2() {
  // clang-format off
  
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  int  *d_values_in       = init({0, 1, 2, 3, 4, 5, 6});
  int  *d_values_out      = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{6, 7, 8, 0, 3, 5, 9};
  std::vector<int> expected_values_out{1, 2, 0, 5, 4, 3, 6};
  // clang-format on
  hipStream_t s;
  hipStreamCreate(&s);
  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1, s);
  
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  
  hipcub::DeviceSegmentedSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1, s);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);
  
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin()) &&
         std::equal(values_out.begin(), values_out.end(),
                    expected_values_out.begin());
}

bool test_descending() {
  // clang-format off
  
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  int  *d_values_in       = init({0, 1, 2, 3, 4, 5, 6});
  int  *d_values_out      = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 7, 6, 9, 5, 3, 0};
  std::vector<int> expected_values_out{0, 2, 1, 6, 3, 4, 5};
  // clang-format on

  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1);
  
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  
  hipcub::DeviceSegmentedSort::SortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin()) &&
         std::equal(values_out.begin(), values_out.end(),
                    expected_values_out.begin());
}

bool test_descending2() {
  // clang-format off
  
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  int  *d_values_in       = init({0, 1, 2, 3, 4, 5, 6});
  int  *d_values_out      = init({0, 0, 0, 0, 0, 0, 0});
  std::vector<int> expected_keys_out{8, 7, 6, 9, 5, 3, 0};
  std::vector<int> expected_values_out{0, 2, 1, 6, 3, 4, 5};
  // clang-format on
  hipStream_t s;
  hipStreamCreate(&s);
  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1, s);
  
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  
  hipcub::DeviceSegmentedSort::SortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, num_segments, d_offsets, d_offsets + 1, s);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_values_out, sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);
  
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin()) &&
         std::equal(values_out.begin(), values_out.end(),
                    expected_values_out.begin());
}

bool test_double_buffer() {
  // clang-format off
  
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  int  *d_values_in       = init({0, 1, 2, 3, 4, 5, 6});
  int  *d_values_out      = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  hipcub::DoubleBuffer<int> d_vals(d_values_in, d_values_out);
  std::vector<int> expected_keys_out{6, 7, 8, 0, 3, 5, 9};
  std::vector<int> expected_values_out{1, 2, 0, 5, 4, 3, 6};
  // clang-format on

  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items, num_segments, d_offsets, d_offsets + 1);
  
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  
  hipcub::DeviceSegmentedSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items, num_segments, d_offsets, d_offsets + 1);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_vals.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin()) &&
         std::equal(values_out.begin(), values_out.end(),
                    expected_values_out.begin());
}

bool test_double_buffer2() {
  // clang-format off
  
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  int  *d_values_in       = init({0, 1, 2, 3, 4, 5, 6});
  int  *d_values_out      = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  hipcub::DoubleBuffer<int> d_vals(d_values_in, d_values_out);
  std::vector<int> expected_keys_out{6, 7, 8, 0, 3, 5, 9};
  std::vector<int> expected_values_out{1, 2, 0, 5, 4, 3, 6};
  // clang-format on
  hipStream_t s;
  hipStreamCreate(&s);
  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items, num_segments, d_offsets, d_offsets + 1, s);
  
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  
  hipcub::DeviceSegmentedSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys,  d_vals, num_items, num_segments, d_offsets, d_offsets + 1, s);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_vals.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);
  
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin()) &&
         std::equal(values_out.begin(), values_out.end(),
                    expected_values_out.begin());
}

bool test_double_buffer_descending() {
  // clang-format off
  
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  int  *d_values_in       = init({0, 1, 2, 3, 4, 5, 6});
  int  *d_values_out      = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  hipcub::DoubleBuffer<int> d_vals(d_values_in, d_values_out);
  std::vector<int> expected_keys_out{8, 7, 6, 9, 5, 3, 0};
  std::vector<int> expected_values_out{0, 2, 1, 6, 3, 4, 5};
  // clang-format on

  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items, num_segments, d_offsets, d_offsets + 1);
  
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  
  hipcub::DeviceSegmentedSort::SortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items, num_segments, d_offsets, d_offsets + 1);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_vals.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin()) &&
         std::equal(values_out.begin(), values_out.end(),
                    expected_values_out.begin());
}

bool test_double_buffer_descending2() {
  // clang-format off
  
  int  num_items          = 7;
  int  num_segments       = 3;
  int  *d_offsets         = init({0, 3, 3, 7});
  int  *d_keys_in         = init({8, 6, 7, 5, 3, 0, 9});
  int  *d_keys_out        = init({0, 0, 0, 0, 0, 0, 0});
  int  *d_values_in       = init({0, 1, 2, 3, 4, 5, 6});
  int  *d_values_out      = init({0, 0, 0, 0, 0, 0, 0});
  hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out);
  hipcub::DoubleBuffer<int> d_vals(d_values_in, d_values_out);
  std::vector<int> expected_keys_out{8, 7, 6, 9, 5, 3, 0};
  std::vector<int> expected_values_out{0, 2, 1, 6, 3, 4, 5};
  // clang-format on
  hipStream_t s;
  hipStreamCreate(&s);
  
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items, num_segments, d_offsets, d_offsets + 1, s);
  
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  
  hipcub::DeviceSegmentedSort::SortPairsDescending(
      d_temp_storage, temp_storage_bytes, d_keys, d_vals, num_items, num_segments, d_offsets, d_offsets + 1, s);

  std::vector<int> keys_out(num_items), values_out(num_items);
  hipMemcpy(keys_out.data(), d_keys.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipMemcpy(values_out.data(), d_vals.Current(), sizeof(int) * num_items,
             hipMemcpyDeviceToHost);
  hipFree(d_offsets);
  hipFree(d_keys_in);
  hipFree(d_keys_out);
  hipFree(d_values_in);
  hipFree(d_values_out);
  hipFree(d_temp_storage);
  hipStreamDestroy(s);
  
  return std::equal(keys_out.begin(), keys_out.end(),
                    expected_keys_out.begin()) &&
         std::equal(values_out.begin(), values_out.end(),
                    expected_values_out.begin());
}

int main() {
  int res = 0;
  if (!test()) {
    printf("hipcub::DeviceSegmentedSort::SortPairs failed\n");
    res = 1;
  }
  if (!test2()) {
    printf("hipcub::DeviceSegmentedSort::SortPairs failed\n");
    res = 1;
  }

  if (!test_descending()) {
    printf("hipcub::DeviceSegmentedSort::SortPairsDescending failed\n");
    res = 1;
  }

  if (!test_descending2()) {
    printf("hipcub::DeviceSegmentedSort::SortPairsDescending failed\n");
    res = 1;
  }

  if (!test_double_buffer()) {
    printf("hipcub::DeviceSegmentedSort::SortPairs (DoubleBuffer) failed\n");
    res = 1;
  }

  if (!test_double_buffer2()) {
    printf("hipcub::DeviceSegmentedSort::SortPairs (DoubleBuffer) failed\n");
    res = 1;
  }

  if (!test_double_buffer_descending()) {
    printf("hipcub::DeviceSegmentedSort::SortPairsDescending (DoubleBuffer) failed\n");
    res = 1;
  }

  if (!test_double_buffer_descending2()) {
    printf("hipcub::DeviceSegmentedSort::SortPairsDescending (DoubleBuffer) failed\n");
    res = 1;
  }

  return res;
}
