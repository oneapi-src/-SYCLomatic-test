// ====------ in_order_queue_events.cu--------------- *- CUDA -*---------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include<hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(const int *A, int *B, int *C, int N) {
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      if (i < N) {
          C[i] = A[i] + B[i];
      }
}

void test1() {
  hipStream_t s1;
  hipStreamCreate(&s1);

  int N = 10;
  size_t size = N * sizeof(int);

  int *h_A = new int[N];
  int *h_B = new int[N];
  int *h_C = new int[N];

  for (int i = 0; i < N; ++i) {
    h_A[i] = static_cast<int>(i);
    h_B[i] = static_cast<int>(i * 2);
  }

  int *d_A, *d_B, *d_C;
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, s1);
  hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, s1);

  vectorAdd<<<1, N>>>(d_A, d_B, d_C, N);
  hipDeviceSynchronize();
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  std::cout << "Result: " << std::endl;
  for (int i = 0; i < N; ++i) {
    if (h_A[i] + h_B[i] != h_C[i]) {
      std::cout << "test failed" << std::endl;
      exit(-1);
    }
    std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
  }
}

void test2() {
  hipStream_t s1;
  hipStreamCreate(&s1);

  int N = 10;
  size_t size = N * sizeof(int);

  int *h_A = new int[N];
  int *h_B = new int[N];
  int *h_C = new int[N];

  for (int i = 0; i < N; ++i) {
    h_A[i] = static_cast<int>(i);
    h_B[i] = static_cast<int>(i * 2);
  }

  int *d_A, *d_B, *d_C;
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, 0);

  vectorAdd<<<1, N, 0, s1>>>(d_A, d_B, d_C, N);

  hipDeviceSynchronize();
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  std::cout << "Result: " << std::endl;
  for (int i = 0; i < N; ++i) {
    if (h_A[i] + h_B[i] != h_C[i]) {
      std::cout << "test failed" << std::endl;
      exit(-1);
    }
    std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
  }
}

int main() {
  test1();
  test2();
  std::cout << "test pass" << std::endl;
  return 0;
}