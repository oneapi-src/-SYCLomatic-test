#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDevice_t device1, device2;
    hipCtx_t context1, context2;

    // Initialize CUDA Driver API
    hipError_t result = hipInit(0);

    hipDeviceGet(&device1, 0); // Device 0
    hipDeviceGet(&device2, 1); // Device 1

    // Create contexts for the devices
    hipCtxCreate(&context1, 0, device1);
    hipCtxCreate(&context2, 0, device2);

    // Enable peer access between the two contexts
    hipCtxSetCurrent(context1);
    result = hipCtxEnablePeerAccess(context2, 0);
    if (result != hipSuccess) {
        std::cerr << "Failed to enable peer access from device 0 to device 1\n";
    }

    int accessEnabled;
    result = hipDeviceCanAccessPeer(&accessEnabled, device1, device2);

    std::cout << "Peer-to-peer device access enabled.\n";

    return 0;
}

