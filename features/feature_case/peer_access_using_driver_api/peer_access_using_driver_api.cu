//===--- peer_access_driver_api_tests.cu -------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
//===------------------------------------------------------ -===//
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDevice_t device1, device2;
    hipCtx_t context1, context2;

    // Initialize CUDA Driver API
    hipError_t result = hipInit(0);

    hipDeviceGet(&device1, 0); // Device 0
    hipDeviceGet(&device2, 1); // Device 1

    // Create contexts for the devices
    hipCtxCreate(&context1, 0, device1);
    hipCtxCreate(&context2, 0, device2);

    // Enable peer access between the two contexts
    hipCtxSetCurrent(context1);
    result = hipCtxEnablePeerAccess(context2, 0);
    if (result != hipSuccess) {
        std::cerr << "Failed to enable peer access from device 0 to device 1\n";
    }

    int accessEnabled;
    result = hipDeviceCanAccessPeer(&accessEnabled, device1, device2);

    std::cout << "Peer-to-peer device access enabled.\n";

    return 0;
}

