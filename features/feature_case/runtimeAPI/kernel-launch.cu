// ====------ kernel-launch.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <stdio.h>
#define VECTOR_SIZE 256

__global__ void VectorAddKernel(float* A, float* B, float* C)
{
    A[threadIdx.x] = threadIdx.x + 1.0f;
    B[threadIdx.x] = threadIdx.x + 1.0f;
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

int main(void)
{
    float *d_A, *d_B, *d_C;
	
    hipMalloc(&d_A, VECTOR_SIZE*sizeof(float));
    hipMalloc(&d_B, VECTOR_SIZE*sizeof(float));
    hipMalloc(&d_C, VECTOR_SIZE*sizeof(float));
    
    void **args = (void **)malloc(sizeof(float **) * 3);
    args[0] = &d_A;
    args[1] = &d_B;
    args[2] = &d_C;

    int threadsPerBlock = VECTOR_SIZE;
    int blocksPerGrid = 1;

    hipLaunchKernel(reinterpret_cast<const void*>((const void *)VectorAddKernel), blocksPerGrid, threadsPerBlock, args, 0, 0);    

    hipEvent_t e;
    hipEventCreate(&e);
    hipEventRecord(e, 0);

    hipError_t ret = hipEventQuery(e);

    while(ret != hipSuccess)
      ret = hipEventQuery(e);

    hipEventDestroy(e);

    float Result[VECTOR_SIZE] = { };
    hipMemcpy(Result, d_C, VECTOR_SIZE*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(args);

    for (int i = 0; i < VECTOR_SIZE; i++) {
        if (i % 16 == 0) {
            printf("\n");
        }
        printf("%f ", Result[i]);    
    }

    return 0;
}

