// ====------ cuda_stream_query.cu---------- *- CUDA -* ----------------===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  hipStream_t stream;
  hipStreamCreate(&stream);

  float *d_data;
  hipMalloc(&d_data, sizeof(float));
  hipMemsetAsync(d_data, 0, sizeof(float), stream);

  hipError_t status = hipStreamQuery(stream);
  if (status == hipSuccess) {
    std::cout << "Stream operations have completed." << std::endl;
  } else if (status == hipErrorNotReady) {
    std::cout << "Stream operations are still in progress." << std::endl;
  } else {
    std::cerr << "An error occurred while querying the stream status."
              << std::endl;
  }
  hipFree(d_data);
  hipStreamDestroy(stream);

  return 0;
}
