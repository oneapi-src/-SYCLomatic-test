// ====------------ math-ext-double.cu---------- *- CUDA -* -------------===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

using namespace std;

typedef vector<double> d_vector;
typedef pair<double, int> di_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

template <typename T = double>
void checkResult(const string &FuncName, const vector<T> &Inputs,
                 const double &Expect, const double &DeviceResult,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision) << DeviceResult
       << " (expect " << Expect - pow(10, -precision) << " ~ "
       << Expect + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(DeviceResult - Expect) < pow(10, -precision));
}

// Double Precision Mathematical Functions

__global__ void cylBesselI0(double *const Result, double Input1) {
  *Result = cyl_bessel_i0(Input1);
}

void testCylBesselI0Cases(const vector<pair<double, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    cylBesselI0<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("cyl_bessel_i0", {TestCase.first}, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void cylBesselI1(double *const Result, double Input1) {
  *Result = cyl_bessel_i1(Input1);
}

void testCylBesselI1Cases(const vector<pair<double, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    cylBesselI1<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("cyl_bessel_i1", {TestCase.first}, TestCase.second.first,
                *Result, TestCase.second.second);
  }
}

__global__ void _erfcinv(double *const DeviceResult, double Input) {
  *DeviceResult = erfcinv(Input);
}

void testErfcinv(double *const DeviceResult, double Input) {
  _erfcinv<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // TODO: Need test host side.
}

void testErfcinvCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testErfcinv(DeviceResult, 0);
  cout << "erfcinv(" << 0 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  testErfcinv(DeviceResult, 2);
  cout << "erfcinv(" << 2 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testErfcinv(DeviceResult, TestCase.first);
    checkResult("erfcinv", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _erfinv(double *const DeviceResult, double Input) {
  *DeviceResult = erfinv(Input);
}

void testErfinv(double *const DeviceResult, double Input) {
  _erfinv<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testErfinvCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testErfinv(DeviceResult, -1);
  cout << "erfinv(" << -1 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  testErfinv(DeviceResult, 1);
  cout << "erfinv(" << 1 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testErfinv(DeviceResult, TestCase.first);
    checkResult("erfinv", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _j0(double *const Result, double Input1) {
  *Result = j0(Input1);
}

void testJ0Cases(const vector<pair<double, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _j0<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("j0", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _j1(double *const Result, double Input1) {
  *Result = j1(Input1);
}

void testJ1Cases(const vector<pair<double, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _j1<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("j1", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _jn(double *const Result, int Input1, double Input2) {
  *Result = jn(Input1, Input2);
}

void testJnCases(const vector<pair<pair<int, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _jn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("jn", {(double)TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void setVecValue(double *Input1, const double Input2) {
  *Input1 = Input2;
}

__global__ void _norm(double *const DeviceResult, int Input1,
                      const double *Input2) {
  *DeviceResult = norm(Input1, Input2);
}

void testNorm(double *const DeviceResult, int Input1, const double *Input2) {
  _norm<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormCases(const vector<pair<d_vector, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    double *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testNorm(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("norm", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

__global__ void _normcdf(double *const DeviceResult, double Input) {
  *DeviceResult = normcdf(Input);
}

void testNormcdf(double *const DeviceResult, double Input) {
  _normcdf<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormcdfCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    testNormcdf(DeviceResult, TestCase.first);
    checkResult("normcdf", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _normcdfinv(double *const DeviceResult, double Input) {
  *DeviceResult = normcdfinv(Input);
}

void testNormcdfinv(double *const DeviceResult, double Input) {
  _normcdfinv<<<1, 1>>>(DeviceResult, Input);
  hipDeviceSynchronize();
  // Call from host.
}

void testNormcdfinvCases(const vector<pair<double, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  // Boundary values.
  testNormcdfinv(DeviceResult, 0);
  cout << "normcdfinv(" << 0 << ") = " << *DeviceResult << " (expect -inf)";
  check(*DeviceResult < -999999.9);
  testNormcdfinv(DeviceResult, 1);
  cout << "normcdfinv(" << 1 << ") = " << *DeviceResult << " (expect inf)";
  check(*DeviceResult > 999999.9);
  // Other test values.
  for (const auto &TestCase : TestCases) {
    testNormcdfinv(DeviceResult, TestCase.first);
    checkResult("normcdfinv", {TestCase.first}, TestCase.second.first,
                *DeviceResult, TestCase.second.second);
  }
}

__global__ void _rnorm(double *const DeviceResult, int Input1,
                       const double *Input2) {
  *DeviceResult = rnorm(Input1, Input2);
}

void testRnorm(double *const DeviceResult, int Input1, const double *Input2) {
  _rnorm<<<1, 1>>>(DeviceResult, Input1, Input2);
  hipDeviceSynchronize();
  // Call from host.
}

void testRnormCases(const vector<pair<d_vector, di_pair>> &TestCases) {
  double *DeviceResult;
  hipMallocManaged(&DeviceResult, sizeof(*DeviceResult));
  for (const auto &TestCase : TestCases) {
    double *Input;
    hipMallocManaged(&Input, TestCase.first.size() * sizeof(*Input));
    for (size_t i = 0; i < TestCase.first.size(); ++i) {
      // Notice: cannot set value from host!
      setVecValue<<<1, 1>>>(Input + i, TestCase.first[i]);
      hipDeviceSynchronize();
    }
    testRnorm(DeviceResult, TestCase.first.size(), Input);
    string arg = "&{";
    for (size_t i = 0; i < TestCase.first.size() - 1; ++i) {
      arg += to_string(TestCase.first[i]) + ", ";
    }
    arg += to_string(TestCase.first.back()) + "}";
    checkResult<string>("rnorm", {to_string(TestCase.first.size()), arg},
                        TestCase.second.first, *DeviceResult,
                        TestCase.second.second);
  }
}

__global__ void _y0(double *const Result, double Input1) {
  *Result = y0(Input1);
}

void testY0Cases(const vector<pair<double, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _y0<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("y0", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _y1(double *const Result, double Input1) {
  *Result = y1(Input1);
}

void testY1Cases(const vector<pair<double, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _y1<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("y1", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _yn(double *const Result, int Input1, double Input2) {
  *Result = yn(Input1, Input2);
}

void testYnCases(const vector<pair<pair<int, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _yn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("yn", {(double)TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

// Double Precision Intrinsics

__global__ void dadd_rd(double *const Result, double Input1, double Input2) {
  *Result = __dadd_rd(Input1, Input2);
}

void testDadd_rdCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dadd_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dadd_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dadd_rn(double *const Result, double Input1, double Input2) {
  *Result = __dadd_rn(Input1, Input2);
}

void testDadd_rnCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dadd_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dadd_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dadd_ru(double *const Result, double Input1, double Input2) {
  *Result = __dadd_ru(Input1, Input2);
}

void testDadd_ruCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dadd_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dadd_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dadd_rz(double *const Result, double Input1, double Input2) {
  *Result = __dadd_rz(Input1, Input2);
}

void testDadd_rzCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dadd_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dadd_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dmul_rd(double *const Result, double Input1, double Input2) {
  *Result = __dmul_rd(Input1, Input2);
}

void testDmul_rdCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dmul_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dmul_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dmul_rn(double *const Result, double Input1, double Input2) {
  *Result = __dmul_rn(Input1, Input2);
}

void testDmul_rnCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dmul_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dmul_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dmul_ru(double *const Result, double Input1, double Input2) {
  *Result = __dmul_ru(Input1, Input2);
}

void testDmul_ruCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dmul_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dmul_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dmul_rz(double *const Result, double Input1, double Input2) {
  *Result = __dmul_rz(Input1, Input2);
}

void testDmul_rzCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dmul_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dmul_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dsub_rd(double *const Result, double Input1, double Input2) {
  *Result = __dsub_rd(Input1, Input2);
}

void testDsub_rdCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dsub_rd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dsub_rd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dsub_rn(double *const Result, double Input1, double Input2) {
  *Result = __dsub_rn(Input1, Input2);
}

void testDsub_rnCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dsub_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dsub_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dsub_ru(double *const Result, double Input1, double Input2) {
  *Result = __dsub_ru(Input1, Input2);
}

void testDsub_ruCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dsub_ru<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dsub_ru", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void dsub_rz(double *const Result, double Input1, double Input2) {
  *Result = __dsub_rz(Input1, Input2);
}

void testDsub_rzCases(
    const vector<pair<pair<double, double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    dsub_rz<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__dsub_rz", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void fma_rd(double *const Result, double Input1, double Input2,
                       double Input3) {
  *Result = __fma_rd(Input1, Input2, Input3);
}

void testFma_rdCases(const vector<pair<vector<double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fma_rd<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                     TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fma_rd", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fma_rn(double *const Result, double Input1, double Input2,
                       double Input3) {
  *Result = __fma_rn(Input1, Input2, Input3);
}

void testFma_rnCases(const vector<pair<vector<double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fma_rn<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                     TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fma_rn", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fma_ru(double *const Result, double Input1, double Input2,
                       double Input3) {
  *Result = __fma_ru(Input1, Input2, Input3);
}

void testFma_ruCases(const vector<pair<vector<double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fma_ru<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                     TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fma_ru", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void fma_rz(double *const Result, double Input1, double Input2,
                       double Input3) {
  *Result = __fma_rz(Input1, Input2, Input3);
}

void testFma_rzCases(const vector<pair<vector<double>, di_pair>> &TestCases) {
  double *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    fma_rz<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                     TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__fma_rz", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

int main() {
  testCylBesselI0Cases({
      {0.3, {1.022626879351597, 15}},
      {0.5, {1.063483370741324, 15}},
      {0.8, {1.166514922869803, 15}},
      {1.6, {1.749980639738909, 15}},
      {-5, {27.23987182360445, 14}},
  });
  testCylBesselI1Cases({
      {0.3, {0.1516938400035928, 16}},
      {0.5, {0.2578943053908963, 16}},
      {0.8, {0.4328648026206398, 16}},
      {1.6, {1.08481063512988, 15}},
      {-5, {-24.33564214245052, 14}},
  });
  testErfcinvCases({
      {0.3, {0.732869077959217, 15}},
      {0.5, {0.4769362762044698, 16}},
      {0.8, {0.1791434546212916, 16}},
      {1.6, {-0.595116081449995, 15}},
  });
  testErfinvCases({
      {-0.3, {-0.2724627147267544, 16}},
      {-0.5, {-0.4769362762044698, 16}},
      {0, {0, 37}},
      {0.5, {0.4769362762044698, 16}},
  });
  testJ0Cases({
      {0.3, {0.977626246538296, 15}},
      {0.5, {0.938469807240813, 15}},
      {0.8, {0.8462873527504802, 16}},
      {1.6, {0.4554021676393806, 16}},
      {-5, {-0.1775967713143383, 16}},
  });
  testJ1Cases({
      {0.3, {0.148318816273104, 16}},
      {0.5, {0.2422684576748739, 16}},
      {0.8, {0.36884204609417, 16}},
      {1.6, {0.56989593526168, 15}},
      {-5, {0.327579137591465, 15}},
  });
  testJnCases({
      {{1, 0.3}, {0.148318816273104, 16}},
      {{2, 0.5}, {0.03060402345868264, 17}},
      {{3, 0.8}, {0.010246766330553604, 18}},
      {{4, 1.6}, {0.014995161059601511, 18}},
      {{5, -5}, {-0.2611405461201702, 16}},
  });
  testNormCases({
      {{-0.3, -0.34, -0.98}, {1.079814798935447, 15}},
      {{0.3, 0.34, 0.98}, {1.079814798935447, 15}},
      {{0.5}, {0.5, 16}},
      {{23, 432, 23, 456, 23}, {629.4020972319682, 13}},
  });
  testNormcdfCases({
      {-5, {0.0000002866515718791939, 22}},
      {-3, {0.00134989803163009458, 20}},
      {0, {0.5, 16}},
      {1, {0.841344746068543, 15}},
      {5, {0.9999997133484281, 16}},
  });
  testNormcdfinvCases({
      {0.3, {-0.524400512708041, 15}},
      {0.5, {0, 37}},
      {0.8, {0.841621233572915, 15}},
  });
  testRnormCases({
      {{-0.3, -0.34, -0.98}, {0.926084733220795, 15}},
      {{0.3, 0.34, 0.98}, {0.926084733220795, 15}},
      {{0.5}, {2, 16}},
      {{23, 432, 23, 456, 23}, {0.001588809450108087, 18}},
  });
  testY0Cases({
      {0.3, {-0.8072735778045195, 16}},
      {0.5, {-0.4445187335067065, 16}},
      {0.8, {-0.0868022796566067, 16}},
      {1.6, {0.420426896415748, 15}},
      {5, {-0.308517625249034, 15}},
  });
  testY1Cases({
      {0.3, {-2.293105138388529, 15}},
      {0.5, {-1.471472392670243, 15}},
      {0.8, {-0.978144176683359, 15}},
      {1.6, {-0.3475780082651325, 16}},
      {5, {0.1478631433912269, 16}},
  });
  testYnCases({
      {{1, 0.3}, {-2.293105138388529, 15}},
      {{2, 0.5}, {-5.441370837174267, 15}},
      {{3, 0.8}, {-10.8146466335756, 14}},
      {{4, 1.6}, {-5.856365000513249, 15}},
      {{0, 5}, {-0.308517625249034, 15}},
  });
  testDadd_rdCases({
      {{-0.3, -0.4}, {-0.7, 7}},
      {{0.3, -0.4}, {-0.1, 8}},
      {{0.3, 0.4}, {0.7, 7}},
      {{0.3, 0.8}, {1.1, 7}},
      {{3, 4}, {7, 37}},
  });
  testDadd_rnCases({
      {{-0.3, -0.4}, {-0.7, 7}},
      {{0.3, -0.4}, {-0.1, 8}},
      {{0.3, 0.4}, {0.7, 7}},
      {{0.3, 0.8}, {1.1, 7}},
      {{3, 4}, {7, 37}},
  });
  testDadd_ruCases({
      {{-0.3, -0.4}, {-0.7, 7}},
      {{0.3, -0.4}, {-0.1, 8}},
      {{0.3, 0.4}, {0.7, 7}},
      {{0.3, 0.8}, {1.1, 7}},
      {{3, 4}, {7, 37}},
  });
  testDadd_rzCases({
      {{-0.3, -0.4}, {-0.7, 7}},
      {{0.3, -0.4}, {-0.1, 8}},
      {{0.3, 0.4}, {0.7, 7}},
      {{0.3, 0.8}, {1.1, 7}},
      {{3, 4}, {7, 37}},
  });
  testDmul_rdCases({
      {{-0.3, -0.4}, {0.12, 8}},
      {{0.3, -0.4}, {-0.12, 8}},
      {{0.3, 0.4}, {0.12, 8}},
      {{0.3, 0.8}, {0.24, 8}},
      {{3, 4}, {12, 37}},
  });
  testDmul_rnCases({
      {{-0.3, -0.4}, {0.12, 8}},
      {{0.3, -0.4}, {-0.12, 8}},
      {{0.3, 0.4}, {0.12, 8}},
      {{0.3, 0.8}, {0.24, 8}},
      {{3, 4}, {12, 37}},
  });
  testDmul_ruCases({
      {{-0.3, -0.4}, {0.12, 8}},
      {{0.3, -0.4}, {-0.12, 8}},
      {{0.3, 0.4}, {0.12, 8}},
      {{0.3, 0.8}, {0.24, 8}},
      {{3, 4}, {12, 37}},
  });
  testDmul_rzCases({
      {{-0.3, -0.4}, {0.12, 8}},
      {{0.3, -0.4}, {-0.12, 8}},
      {{0.3, 0.4}, {0.12, 8}},
      {{0.3, 0.8}, {0.24, 8}},
      {{3, 4}, {12, 37}},
  });
  testDsub_rdCases({
      {{-0.3, -0.4}, {0.1, 8}},
      {{0.3, -0.4}, {0.7, 7}},
      {{0.3, 0.4}, {-0.1, 8}},
      {{0.3, 0.8}, {-0.5, 15}},
      {{3, 4}, {-1, 37}},
  });
  testDsub_rnCases({
      {{-0.3, -0.4}, {0.1, 8}},
      {{0.3, -0.4}, {0.7, 7}},
      {{0.3, 0.4}, {-0.1, 8}},
      {{0.3, 0.8}, {-0.5, 37}},
      {{3, 4}, {-1, 37}},
  });
  testDsub_ruCases({
      {{-0.3, -0.4}, {0.1, 8}},
      {{0.3, -0.4}, {0.7, 7}},
      {{0.3, 0.4}, {-0.1, 8}},
      {{0.3, 0.8}, {-0.5, 37}},
      {{3, 4}, {-1, 37}},
  });
  testDsub_rzCases({
      {{-0.3, -0.4}, {0.1, 8}},
      {{0.3, -0.4}, {0.7, 7}},
      {{0.3, 0.4}, {-0.1, 8}},
      {{0.3, 0.8}, {-0.5, 37}},
      {{3, 4}, {-1, 37}},
  });
  testFma_rdCases({
      {{-0.3, -0.4, -0.2}, {-0.08000000000000002, 17}},
      {{0.3, -0.4, -0.1}, {-0.22, 16}},
      {{0.3, 0.4, 0.1}, {0.22, 16}},
      {{0.3, 0.4, 0}, {0.12, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFma_rnCases({
      {{-0.3, -0.4, -0.2}, {-0.08000000000000002, 17}},
      {{0.3, -0.4, -0.1}, {-0.22, 16}},
      {{0.3, 0.4, 0.1}, {0.22, 16}},
      {{0.3, 0.4, 0}, {0.12, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFma_ruCases({
      {{-0.3, -0.4, -0.2}, {-0.08, 17}},
      {{0.3, -0.4, -0.1}, {-0.22, 16}},
      {{0.3, 0.4, 0.1}, {0.22, 16}},
      {{0.3, 0.4, 0}, {0.12000000000000001, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testFma_rzCases({
      {{-0.3, -0.4, -0.2}, {-0.08, 17}},
      {{0.3, -0.4, -0.1}, {-0.22, 16}},
      {{0.3, 0.4, 0.1}, {0.22, 16}},
      {{0.3, 0.4, 0}, {0.12, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
