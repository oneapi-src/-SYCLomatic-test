//====---------- math-emu-bf16.cu ----------- *- CUDA -* ------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "hip/hip_bf16.h"

using namespace std;

typedef vector<__hip_bfloat16> bf16_vector;
typedef pair<__hip_bfloat16, int> bf16i_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

void checkResult(const string &FuncName, const vector<float> &Inputs,
                 const float &Expect, const float &Result,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision < 0 ? 0 : precision)
       << Result << " (expect " << Expect - pow(10, -precision) << " ~ "
       << Expect + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(Result - Expect) < pow(10, -precision));
}

void checkResult(const string &FuncName, const vector<__hip_bfloat16> &Inputs,
                 const __hip_bfloat16 &Expect, const float &Result,
                 const int precision) {
  vector<float> FInputs;
  for (const auto &it : Inputs) {
    FInputs.push_back(__bfloat162float(it));
  }
  float FExpect{__bfloat162float(Expect)};
  checkResult(FuncName, FInputs, FExpect, Result, precision);
}

// Bfloat16 Arithmetic Functions

__global__ void habs(float *const Result, __hip_bfloat16 Input1) {
  *Result = __habs(Input1);
}

void testHabsCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    habs<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__habs", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void hadd(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hadd(Input1, Input2);
}

void testHaddCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hadd<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hadd", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hadd_rn(float *const Result, __hip_bfloat16 Input1,
                        __hip_bfloat16 Input2) {
  *Result = __hadd_rn(Input1, Input2);
}

void testHadd_rnCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hadd_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hadd_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hadd_sat(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2) {
  *Result = __hadd_sat(Input1, Input2);
}

void testHadd_satCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hadd_sat<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hadd_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hdiv(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hdiv(Input1, Input2);
}

void testHdivCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hdiv<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hdiv", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hfma(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2, __hip_bfloat16 Input3) {
  *Result = __hfma(Input1, Input2, Input3);
}

void testHfmaCases(const vector<pair<bf16_vector, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hfma<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                   TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__hfma", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
    if (TestCase.first.size() != 3) {
      failed++;
      cout << " ---- failed" << endl;
      return;
    }
  }
}

__global__ void hfma_sat(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2, __hip_bfloat16 Input3) {
  *Result = __hfma_sat(Input1, Input2, Input3);
}

void testHfma_satCases(const vector<pair<bf16_vector, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hfma_sat<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                       TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__hfma_sat", TestCase.first, TestCase.second.first, *Result,
                TestCase.second.second);
    if (TestCase.first.size() != 3) {
      failed++;
      cout << " ---- failed" << endl;
      return;
    }
  }
}

__global__ void hmul(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hmul(Input1, Input2);
}

void testHmulCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hmul<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmul", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hmul_rn(float *const Result, __hip_bfloat16 Input1,
                        __hip_bfloat16 Input2) {
  *Result = __hmul_rn(Input1, Input2);
}

void testHmul_rnCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hmul_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmul_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hmul_sat(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2) {
  *Result = __hmul_sat(Input1, Input2);
}

void testHmul_satCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hmul_sat<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hmul_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hneg(float *const Result, __hip_bfloat16 Input1) {
  *Result = __hneg(Input1);
}

void testHnegCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hneg<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__hneg", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void hsub(float *const Result, __hip_bfloat16 Input1,
                     __hip_bfloat16 Input2) {
  *Result = __hsub(Input1, Input2);
}

void testHsubCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hsub<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hsub", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hsub_rn(float *const Result, __hip_bfloat16 Input1,
                        __hip_bfloat16 Input2) {
  *Result = __hsub_rn(Input1, Input2);
}

void testHsub_rnCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hsub_rn<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hsub_rn", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

__global__ void hsub_sat(float *const Result, __hip_bfloat16 Input1,
                         __hip_bfloat16 Input2) {
  *Result = __hsub_sat(Input1, Input2);
}

void testHsub_satCases(
    const vector<pair<pair<__hip_bfloat16, __hip_bfloat16>, bf16i_pair>>
        &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hsub_sat<<<1, 1>>>(Result, TestCase.first.first, TestCase.first.second);
    hipDeviceSynchronize();
    checkResult("__hsub_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, *Result, TestCase.second.second);
  }
}

// Bfloat16 Math Functions

__global__ void _hceil(float *const Result, __hip_bfloat16 Input1) {
  *Result = hceil(Input1);
}

void testHceilCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hceil<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hceil", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hcos(float *const Result, __hip_bfloat16 Input1) {
  *Result = hcos(Input1);
}

void testHcosCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hcos<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hcos", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hexp(float *const Result, __hip_bfloat16 Input1) {
  *Result = hexp(Input1);
}

void testHexpCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hexp<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hexp", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hexp10(float *const Result, __hip_bfloat16 Input1) {
  *Result = hexp10(Input1);
}

void testHexp10Cases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hexp10<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hexp10", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hexp2(float *const Result, __hip_bfloat16 Input1) {
  *Result = hexp2(Input1);
}

void testHexp2Cases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hexp2<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hexp2", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hfloor(float *const Result, __hip_bfloat16 Input1) {
  *Result = hfloor(Input1);
}

void testHfloorCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hfloor<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hfloor", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hlog(float *const Result, __hip_bfloat16 Input1) {
  *Result = hlog(Input1);
}

void testHlogCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hlog<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hlog", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hlog10(float *const Result, __hip_bfloat16 Input1) {
  *Result = hlog10(Input1);
}

void testHlog10Cases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hlog10<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hlog10", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hlog2(float *const Result, __hip_bfloat16 Input1) {
  *Result = hlog2(Input1);
}

void testHlog2Cases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hlog2<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hlog2", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hrcp(float *const Result, __hip_bfloat16 Input1) {
  *Result = hrcp(Input1);
}

void testHrcpCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hrcp<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hrcp", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hrint(float *const Result, __hip_bfloat16 Input1) {
  *Result = hrint(Input1);
}

void testHrintCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hrint<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hrint", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hrsqrt(float *const Result, __hip_bfloat16 Input1) {
  *Result = hrsqrt(Input1);
}

void testHrsqrtCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hrsqrt<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hrsqrt", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hsin(float *const Result, __hip_bfloat16 Input1) {
  *Result = hsin(Input1);
}

void testHsinCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hsin<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hsin", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _hsqrt(float *const Result, __hip_bfloat16 Input1) {
  *Result = hsqrt(Input1);
}

void testHsqrtCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _hsqrt<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("hsqrt", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

__global__ void _htrunc(float *const Result, __hip_bfloat16 Input1) {
  *Result = htrunc(Input1);
}

void testHtruncCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    _htrunc<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("htrunc", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

int main() {
  testHabsCases({
      {{-0.3}, {0.30078125, 16}},
      {{0.3}, {0.30078125, 16}},
      {{0.5}, {0.5, 16}},
      {{0.4}, {0.400390625, 16}},
      {{6}, {6, 15}},
  });
  testHaddCases({
      {{-0.3, -0.4}, {-0.703125, 16}},
      {{0.3, -0.4}, {-0.099609375, 17}},
      {{0.3, 0.4}, {0.703125, 16}},
      {{0.3, 0.8}, {1.1015625, 15}},
      {{3, 4}, {7, 15}},
  });
  testHadd_rnCases({
      {{-0.3, -0.4}, {-0.703125, 16}},
      {{0.3, -0.4}, {-0.099609375, 17}},
      {{0.3, 0.4}, {0.703125, 16}},
      {{0.3, 0.8}, {1.1015625, 15}},
      {{3, 4}, {7, 15}},
  });
  testHadd_satCases({
      {{-0.3, -0.4}, {0, 37}},
      {{0.3, -0.4}, {0, 37}},
      {{0.3, 0.4}, {0.703125, 16}},
      {{0.3, 0.8}, {1, 15}},
      {{3, 4}, {1, 15}},
  });
  testHdivCases({
      {{-0.3, -0.4}, {0.75, 16}},
      {{0.3, -0.4}, {-0.75, 16}},
      {{0.3, 0.4}, {0.75, 16}},
      {{0.3, 0.8}, {0.375, 16}},
      {{3, 4}, {0.75, 16}},
  });
  testHfmaCases({
      {{-0.3, -0.4, -0.2}, {-0.07958984375, 17}},
      {{0.3, -0.4, -0.1}, {-0.220703125, 16}},
      {{0.3, 0.4, 0.1}, {0.220703125, 16}},
      {{0.3, 0.4, 0}, {0.12060546875, 17}},
      {{3, 4, 5}, {17, 14}},
  });
  testHfma_satCases({
      {{-0.3, -0.4, -0.2}, {0, 37}},
      {{0.3, -0.4, -0.1}, {0, 37}},
      {{0.3, 0.4, 0.1}, {0.220703125, 16}},
      {{0.3, 0.4, 0}, {0.12060546875, 17}},
      {{3, 4, 5}, {1, 15}},
  });
  testHmulCases({
      {{-0.3, -0.4}, {0.12060546875, 17}},
      {{0.3, -0.4}, {-0.12060546875, 17}},
      {{0.3, 0.4}, {0.12060546875, 17}},
      {{0.3, 0.8}, {0.2412109375, 16}},
      {{3, 4}, {12, 15}},
  });
  testHmul_rnCases({
      {{-0.3, -0.4}, {0.12060546875, 17}},
      {{0.3, -0.4}, {-0.12060546875, 17}},
      {{0.3, 0.4}, {0.12060546875, 17}},
      {{0.3, 0.8}, {0.2412109375, 16}},
      {{3, 4}, {12, 15}},
  });
  testHmul_satCases({
      {{-0.3, -0.4}, {0.12060546875, 17}},
      {{0.3, -0.4}, {0, 37}},
      {{0.3, 0.4}, {0.12060546875, 17}},
      {{0.3, 0.8}, {0.2412109375, 16}},
      {{3, 4}, {1, 15}},
  });
  testHnegCases({
      {{-0.3}, {0.30078125, 16}},
      {{0.3}, {-0.30078125, 16}},
      {{0.5}, {-0.5, 16}},
      {{0.4}, {-0.400390625, 16}},
      {{6}, {-6, 15}},
  });
  testHsubCases({
      {{-0.3, -0.4}, {0.099609375, 17}},
      {{0.3, -0.4}, {0.703125, 16}},
      {{0.3, 0.4}, {-0.099609375, 17}},
      {{0.3, -0.8}, {1.1015625, 15}},
      {{3, 4}, {-1, 15}},
  });
  testHsub_rnCases({
      {{-0.3, -0.4}, {0.099609375, 17}},
      {{0.3, -0.4}, {0.703125, 16}},
      {{0.3, 0.4}, {-0.099609375, 17}},
      {{0.3, -0.8}, {1.1015625, 15}},
      {{3, 4}, {-1, 15}},
  });
  testHsub_satCases({
      {{-0.3, -0.4}, {0.099609375, 17}},
      {{0.3, -0.4}, {0.703125, 16}},
      {{0.3, 0.4}, {0, 37}},
      {{0.3, -0.8}, {1, 15}},
      {{3, 4}, {0, 37}},
  });
  testHceilCases({
      {-0.3, {0, 37}},
      {0.34, {1, 15}},
      {0.8, {1, 15}},
      {23, {23, 14}},
      {-12, {-12, 15}},
  });
  testHcosCases({
      {-0.3, {0.96, 2}},
      {0.34, {0.94, 2}},
      {0.8, {0.695, 3}},
      {23, {-0.53, 2}},
      {-12, {0.844, 3}},
  });
  testHexpCases({
      {-0.3, {0.74, 2}},
      {0.34, {1.41, 2}},
      {0.8, {2.23, 2}},
      {10, {22016, -2}},
      {-12, {0.00000614, 8}},
  });
  testHexp10Cases({
      {-0.3, {0.5, 3}},
      {0.34, {2.188, 3}},
      {0.8, {6.31, 2}},
      {4, {9984, -2}},
      {-12, {0.000000000001, 14}},
  });
  testHexp2Cases({
      {-0.3, {0.812, 3}},
      {0.34, {1.26562, 5}},
      {0.8, {1.742, 3}},
      {12, {4096, 12}},
      {-12, {0.000244140625, 19}},
  });
  testHfloorCases({
      {-0.3, {-1, 15}},
      {0.34, {0, 37}},
      {0.8, {0, 37}},
      {23, {23, 14}},
      {-12, {-12, 15}},
  });
  testHlogCases({
      {0.3, {-1.2, 2}},
      {0.34, {-1.08, 2}},
      {0.8, {-0.222, 3}},
      {23, {3.14, 2}},
      {12, {2.484, 3}},
  });
  testHlog10Cases({
      {0.3, {-0.52, 2}},
      {0.34, {-0.4688, 4}},
      {0.8, {-0.097, 3}},
      {23, {1.36, 2}},
      {12, {1.08, 2}},
  });
  testHlog2Cases({
      {0.3, {-1.73, 2}},
      {0.34, {-1.6, 1}},
      {0.8, {-0.32, 3}},
      {23, {4.53, 2}},
      {12, {3.58, 2}},
  });
  testHrcpCases({
      {-0.3, {-3.33, 2}},
      {0.34, {2.94, 2}},
      {0.8, {1.25, 2}},
      {23, {0.0435, 4}},
      {-12, {-0.083, 3}},
  });
  testHrintCases({
      {-0.3, {0, 37}},
      {0.34, {0., 37}},
      {0.8, {1, 15}},
      {23, {23, 14}},
      {-12, {-12, 14}},
  });
  testHrsqrtCases({
      {0.3, {1.82, 2}},
      {0.34, {1.72, 2}},
      {0.8, {1.117, 3}},
      {23, {0.209, 3}},
      {12, {0.289, 3}},
  });
  testHsinCases({
      {-0.3, {-0.297, 3}},
      {0.34, {0.334, 3}},
      {0.8, {0.719, 3}},
      {23, {-0.85, 2}},
      {-12, {0.54, 2}},
  });
  testHsqrtCases({
      {0.3, {0.55, 2}},
      {0.34, {0.582, 3}},
      {0.8, {0.895, 3}},
      {23, {4.8, 1}},
      {12, {3.47, 2}},
  });
  testHtruncCases({
      {-0.3, {0, 37}},
      {0.34, {0, 37}},
      {0.8, {0, 37}},
      {23, {23, 14}},
      {-12, {-12, 15}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
