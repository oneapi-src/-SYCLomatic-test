// ====---------- math-emu-half2.cu---------- *- CUDA -* ------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "hip/hip_fp16.h"

using namespace std;

typedef pair<__half2, __half2> half2_pair;
typedef vector<__half2> half2_vector;
typedef pair<__half2, int> h2i_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

void checkResult(const string &FuncName, const vector<float2> &Inputs,
                 const float2 &Expect, const float2 &Result,
                 const int precision) {
  cout << FuncName << "({" << Inputs[0].x << ", " << Inputs[0].y << "}";
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", {" << Inputs[i].x << ", " << Inputs[i].y << "}";
  }
  cout << ") = {" << fixed << setprecision(precision) << Result.x << ", "
       << Result.y << "} (expect {" << Expect.x - pow(10, -precision) << " ~ "
       << Expect.x + pow(10, -precision) << ", "
       << Expect.y - pow(10, -precision) << " ~ "
       << Expect.y + pow(10, -precision) << "})";
  cout.unsetf(ios::fixed);
  check(abs(Result.x - Expect.x) < pow(10, -precision) &&
        abs(Result.y - Expect.y) < pow(10, -precision));
}

void checkResult(const string &FuncName, const half2_vector &Inputs,
                 const __half2 &Expect, const float2 &Result,
                 const int precision) {
  vector<float2> FInputs;
  for (const auto &it : Inputs) {
    FInputs.push_back(__half22float2(it));
  }
  float2 FExpect = __half22float2(Expect);
  checkResult(FuncName, FInputs, FExpect, Result, precision);
}

__global__ void hadd2_sat(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hadd2_sat(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHadd2_sat(float *const Result, __half2 Input1, __half2 Input2) {
  hadd2_sat<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHadd2_satCases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHadd2_sat(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hadd2_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hfma2_sat(float *const Result, __half2 Input1, __half2 Input2,
                          __half2 Input3) {
  auto ret = __hfma2_sat(Input1, Input2, Input3);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHfma2_sat(float *const Result, __half2 Input1, __half2 Input2,
                   __half2 Input3) {
  hfma2_sat<<<1, 1>>>(Result, Input1, Input2, Input3);
  hipDeviceSynchronize();
}

void testHfma2_satCases(const vector<pair<half2_vector, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHfma2_sat(Result, TestCase.first[0], TestCase.first[1],
                  TestCase.first[2]);
    checkResult("__hfma2_sat", TestCase.first, TestCase.second.first,
                {Result[0], Result[1]}, TestCase.second.second);
    if (TestCase.first.size() != 3) {
      failed++;
      cout << " ---- failed" << endl;
      return;
    }
  }
}

__global__ void hmul2_sat(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hmul2_sat(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHmul2_sat(float *const Result, __half2 Input1, __half2 Input2) {
  hmul2_sat<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHmul2_satCases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHmul2_sat(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hmul2_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hsub2_sat(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hsub2_sat(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHsub2_sat(float *const Result, __half2 Input1, __half2 Input2) {
  hsub2_sat<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHsub2_satCases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHsub2_sat(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hsub2_sat", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

void printResultBool(const string &FuncName, const half2_vector &Inputs,
                     const bool &Expect, const bool &Result) {
  cout << FuncName << "({" << __low2float(Inputs[0]) << ", "
       << __high2float(Inputs[0]) << "}";
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", {" << __low2float(Inputs[i]) << ", " << __high2float(Inputs[i])
         << "}";
  }
  cout << ") = " << Result << " (expect " << Expect << ")";
  check(Result == Expect);
}

__global__ void hbeq2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbeq2(Input1, Input2);
}

void testHbeq2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbeq2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbeq2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbeq2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbeq2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbequ2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbequ2(Input1, Input2);
}

void testHbequ2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbequ2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbequ2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbequ2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbequ2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbge2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbge2(Input1, Input2);
}

void testHbge2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbge2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbge2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbge2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbge2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbgeu2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbgeu2(Input1, Input2);
}

void testHbgeu2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbgeu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbgeu2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbgeu2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbgeu2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbgt2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbgt2(Input1, Input2);
}

void testHbgt2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbgt2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbgt2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbgt2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbgt2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbgtu2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbgtu2(Input1, Input2);
}

void testHbgtu2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbgtu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbgtu2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbgtu2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbgtu2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hble2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hble2(Input1, Input2);
}

void testHble2(bool *const Result, __half2 Input1, __half2 Input2) {
  hble2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHble2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHble2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hble2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbleu2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbleu2(Input1, Input2);
}

void testHbleu2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbleu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbleu2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbleu2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbleu2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hblt2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hblt2(Input1, Input2);
}

void testHblt2(bool *const Result, __half2 Input1, __half2 Input2) {
  hblt2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHblt2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHblt2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hblt2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbltu2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbltu2(Input1, Input2);
}

void testHbltu2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbltu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbltu2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbltu2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbltu2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbne2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbne2(Input1, Input2);
}

void testHbne2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbne2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbne2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbne2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbne2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void hbneu2(bool *const Result, __half2 Input1, __half2 Input2) {
  *Result = __hbneu2(Input1, Input2);
}

void testHbneu2(bool *const Result, __half2 Input1, __half2 Input2) {
  hbneu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHbneu2Cases(const vector<pair<half2_pair, bool>> &TestCases) {
  bool *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHbneu2(Result, TestCase.first.first, TestCase.first.second);
    printResultBool("__hbneu2", {TestCase.first.first, TestCase.first.second},
                    TestCase.second, *Result);
  }
}

__global__ void heq2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __heq2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHeq2(float *const Result, __half2 Input1, __half2 Input2) {
  heq2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHeq2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHeq2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__heq2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hequ2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hequ2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHequ2(float *const Result, __half2 Input1, __half2 Input2) {
  hequ2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHequ2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHequ2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hequ2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hge2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hge2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHge2(float *const Result, __half2 Input1, __half2 Input2) {
  hge2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHge2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHge2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hge2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hgeu2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hgeu2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHgeu2(float *const Result, __half2 Input1, __half2 Input2) {
  hgeu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHgeu2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHgeu2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hgeu2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hgt2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hgt2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHgt2(float *const Result, __half2 Input1, __half2 Input2) {
  hgt2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHgt2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHgt2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hgt2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hgtu2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hgtu2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHgtu2(float *const Result, __half2 Input1, __half2 Input2) {
  hgtu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHgtu2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHgtu2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hgtu2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hisnan2(float *const Result, __half2 Input1) {
  auto ret = __hisnan2(Input1);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHisnan2(float *const Result, __half2 Input1) {
  hisnan2<<<1, 1>>>(Result, Input1);
  hipDeviceSynchronize();
}

void testHisnan2Cases(const vector<pair<half2, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHisnan2(Result, TestCase.first);
    checkResult("__hisnan2", {TestCase.first}, TestCase.second.first,
                {Result[0], Result[1]}, TestCase.second.second);
  }
}

__global__ void hle2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hle2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHle2(float *const Result, __half2 Input1, __half2 Input2) {
  hle2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHle2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHle2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hle2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hleu2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hleu2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHleu2(float *const Result, __half2 Input1, __half2 Input2) {
  hleu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHleu2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHleu2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hleu2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hlt2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hlt2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHlt2(float *const Result, __half2 Input1, __half2 Input2) {
  hlt2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHlt2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHlt2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hlt2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hltu2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hltu2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHltu2(float *const Result, __half2 Input1, __half2 Input2) {
  hltu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHltu2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHltu2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hltu2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hne2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hne2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHne2(float *const Result, __half2 Input1, __half2 Input2) {
  hne2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHne2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHne2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hne2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

__global__ void hneu2(float *const Result, __half2 Input1, __half2 Input2) {
  auto ret = __hneu2(Input1, Input2);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHneu2(float *const Result, __half2 Input1, __half2 Input2) {
  hneu2<<<1, 1>>>(Result, Input1, Input2);
  hipDeviceSynchronize();
}

void testHneu2Cases(const vector<pair<half2_pair, h2i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    testHneu2(Result, TestCase.first.first, TestCase.first.second);
    checkResult("__hneu2", {TestCase.first.first, TestCase.first.second},
                TestCase.second.first, {Result[0], Result[1]},
                TestCase.second.second);
  }
}

int main() {
  testHadd2_satCases({
      {{{-0.3, -0.5}, {-0.4, -0.6}}, {{0, 0}, 37}},
      {{{0.3, 0.5}, {-0.4, 0.6}}, {{0, 1}, 15}},
      {{{0.3, 0.5}, {0.4, 0.2}}, {{0.7001953125, 0.7001953125}, 16}},
      {{{0.3, 0.5}, {0.4, 0.6}}, {{0.7001953125, 1}, 15}},
      {{{3, 5}, {4, 6}}, {{1, 1}, 15}},
  });
  testHfma2_satCases({
      {{{-0.3, -0.5}, {-0.4, -0.6}, {-0.2, -0.7}}, {{0, 0}, 37}},
      {{{0.3, 0.5}, {-0.4, 0.6}, {-0.1, 0.2}}, {{0, 0.5}, 16}},
      {{{0.3, 0.5}, {0.4, 0.2}, {0.1, 0.1}},
       {{0.219970703125, 0.199951171875}, 16}},
      {{{0.3, 0.5}, {0.4, 0.6}, {0, 0.3}},
       {{0.1199951171875, 0.60009765625}, 16}},
      {{{3, 5}, {4, 6}, {5, 8}}, {{1, 1}, 15}},
  });
  testHmul2_satCases({
      {{{-0.3, -5}, {0.4, 6}}, {{0, 0}, 37}},
      {{{0.3, 5}, {-4, 0.6}}, {{0, 1}, 15}},
      {{{0.3, 0.5}, {0.4, 0.2}}, {{0.1199951171875, 0.0999755859375}, 17}},
      {{{0.3, 0.5}, {0.4, 0.6}}, {{0.1199951171875, 0.300048828125}, 16}},
      {{{3, 5}, {4, 6}}, {{1, 1}, 15}},
  });
  testHsub2_satCases({
      {{{0, 0}, {-0.4, -0.6}}, {{0.39990234375, 0.60009765625}, 16}},
      {{{0, 1}, {0.4, 0.6}}, {{0, 0.39990234375}, 16}},
      {{{0.7, 0.7}, {0.4, 0.2}}, {{0.30029296875, 0.5}, 16}},
      {{{0.7, 2}, {0.4, 0.6}}, {{0.30029296875, 1}, 15}},
      {{{1, 1}, {4, 6}}, {{0, 0}, 37}},
  });
  testHbeq2Cases({
      {{{0, 0}, {-0.4, -0.6}}, false},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, true},
      {{{1, 1}, {4, 6}}, false},
      {{{NAN, 1}, {1, 1}}, false},
  });
  testHbequ2Cases({
      {{{0, 0}, {-0.4, -0.6}}, false},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, true},
      {{{1, 1}, {4, 6}}, false},
      {{{NAN, 1}, {1, 1}}, true},
  });
  testHbge2Cases({
      {{{0, 0}, {-0.4, -0.6}}, true},
      {{{0.7, 0.7}, {0.4, 0.7}}, true},
      {{{0.7, 2}, {0.7, 2}}, true},
      {{{1, 1}, {4, 6}}, false},
      {{{NAN, 1}, {1, 1}}, false},
  });
  testHbgeu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, true},
      {{{0.7, 0.7}, {0.4, 0.7}}, true},
      {{{0.7, 2}, {0.7, 2}}, true},
      {{{1, 1}, {4, 6}}, false},
      {{{NAN, 1}, {1, 1}}, true},
  });
  testHbgt2Cases({
      {{{0, 0}, {-0.4, -0.6}}, true},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, false},
      {{{1, 1}, {4, 6}}, false},
      {{{NAN, 2}, {1, 1}}, false},
  });
  testHbgtu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, true},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, false},
      {{{1, 1}, {4, 6}}, false},
      {{{NAN, 2}, {1, 1}}, true},
  });
  testHble2Cases({
      {{{0, 0}, {-0.4, -0.6}}, false},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, true},
      {{{1, 1}, {4, 6}}, true},
      {{{NAN, 1}, {1, 1}}, false},
  });
  testHbleu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, false},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, true},
      {{{1, 1}, {4, 6}}, true},
      {{{NAN, 1}, {1, 1}}, true},
  });
  testHblt2Cases({
      {{{0, 0}, {-0.4, -0.6}}, false},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, false},
      {{{1, 1}, {4, 6}}, true},
      {{{NAN, 1}, {1, 2}}, false},
  });
  testHbltu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, false},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, false},
      {{{1, 1}, {4, 6}}, true},
      {{{NAN, 1}, {1, 2}}, true},
  });
  testHbne2Cases({
      {{{0, 0}, {-0.4, -0.6}}, true},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, false},
      {{{1, 1}, {4, 6}}, true},
      {{{NAN, 1}, {1, 2}}, false},
  });
  testHbneu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, true},
      {{{0.7, 0.7}, {0.4, 0.7}}, false},
      {{{0.7, 2}, {0.7, 2}}, false},
      {{{1, 1}, {4, 6}}, true},
      {{{NAN, 1}, {1, 2}}, true},
  });
  testHeq2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{0, 0}, 37}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{0, 1}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{1, 1}, 15}},
      {{{1, 1}, {4, 6}}, {{0, 0}, 37}},
      {{{NAN, 1}, {1, 1}}, {{0, 1}, 15}},
  });
  testHequ2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{0, 0}, 37}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{0, 1}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{1, 1}, 15}},
      {{{1, 1}, {4, 6}}, {{0, 0}, 37}},
      {{{NAN, 1}, {1, 1}}, {{1, 1}, 15}},
  });
  testHge2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{1, 1}, 15}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{1, 1}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{1, 1}, 15}},
      {{{1, 1}, {4, 6}}, {{0, 0}, 37}},
      {{{NAN, 1}, {1, 1}}, {{0, 1}, 15}},
  });
  testHgeu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{1, 1}, 15}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{1, 1}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{1, 1}, 15}},
      {{{1, 1}, {4, 6}}, {{0, 0}, 37}},
      {{{NAN, 1}, {1, 1}}, {{1, 1}, 15}},
  });
  testHgt2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{1, 1}, 15}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{1, 0}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{0, 0}, 37}},
      {{{1, 1}, {4, 6}}, {{0, 0}, 37}},
      {{{NAN, 1}, {1, 1}}, {{0, 0}, 37}},
  });
  testHgtu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{1, 1}, 15}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{1, 0}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{0, 0}, 37}},
      {{{1, 1}, {4, 6}}, {{0, 0}, 37}},
      {{{NAN, 1}, {1, 1}}, {{1, 0}, 15}},
  });
  testHisnan2Cases({
      {{0, 0}, {{0, 0}, 37}},
      {{0.7, 2}, {{0, 0}, 37}},
      {{NAN, 1}, {{1, 0}, 15}},
      {{NAN, NAN}, {{1, 1}, 15}},
      {{0, NAN}, {{0, 1}, 15}},
  });
  testHle2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{0, 0}, 37}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{0, 1}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{1, 1}, 15}},
      {{{1, 1}, {4, 6}}, {{1, 1}, 15}},
      {{{NAN, 1}, {1, 1}}, {{0, 1}, 15}},
  });
  testHleu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{0, 0}, 37}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{0, 1}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{1, 1}, 15}},
      {{{1, 1}, {4, 6}}, {{1, 1}, 15}},
      {{{NAN, 1}, {1, 1}}, {{1, 1}, 15}},
  });
  testHlt2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{0, 0}, 37}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{0, 0}, 37}},
      {{{0.7, 2}, {0.7, 2}}, {{0, 0}, 37}},
      {{{1, 1}, {4, 6}}, {{1, 1}, 15}},
      {{{NAN, 1}, {1, 1}}, {{0, 0}, 37}},
  });
  testHltu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{0, 0}, 37}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{0, 0}, 37}},
      {{{0.7, 2}, {0.7, 2}}, {{0, 0}, 37}},
      {{{1, 1}, {4, 6}}, {{1, 1}, 15}},
      {{{NAN, 1}, {1, 1}}, {{1, 0}, 15}},
  });
  testHne2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{1, 1}, 15}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{1, 0}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{0, 0}, 37}},
      {{{1, 1}, {4, 6}}, {{1, 1}, 15}},
      {{{NAN, 1}, {1, 1}}, {{0, 0}, 37}},
  });
  testHneu2Cases({
      {{{0, 0}, {-0.4, -0.6}}, {{1, 1}, 15}},
      {{{0.7, 0.7}, {0.4, 0.7}}, {{1, 0}, 15}},
      {{{0.7, 2}, {0.7, 2}}, {{0, 0}, 37}},
      {{{1, 1}, {4, 6}}, {{1, 1}, 15}},
      {{{NAN, 1}, {1, 1}}, {{1, 0}, 15}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
