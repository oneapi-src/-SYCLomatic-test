// ====------ module-main.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <string>
int main(){
    hipModule_t M;
    hipFunction_t F;
#ifdef _WIN32
    std::string Path{"./module-kernel.dll"};
#else
    std::string Path{"./module-kernel.so"};
#endif
    std::string FunctionName{"foo"}, Data;
    FunctionName = "foo";
    hipModuleLoad(&M, Path.c_str());
    hipModuleGetFunction(&F, M, FunctionName.c_str());
    float **param[2] = {0};
    float *p0, *p1;
    hipMalloc(&p0, sizeof(float));
    hipMalloc(&p1, sizeof(float));
    param[0] = &p0;
    param[1] = &p1;
    hipModuleLaunchKernel(F, 1, 1, 1, 1, 1, 1, 10, 0, (void**)param, nullptr);
    hipTexRef tex;
    hipModuleGetTexRef(&tex, M, "tex");
    hipModuleUnload(M);
    hipDeviceSynchronize();
    hipFree(param[0]);
    hipFree(param[1]);
    return 0;
}
