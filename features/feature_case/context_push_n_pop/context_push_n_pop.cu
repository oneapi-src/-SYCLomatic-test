//===------------- context_push_n_pop.cu -------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
//===------------------------------------------------------ -===//
#include <hip/hip_runtime.h>
#include <iostream>


int main() {
    hipCtx_t ctx1, ctx2;
    hipDevice_t device;

    // Initialize the CUDA Driver API
    hipInit(0);

    // Get the device
    hipDeviceGet(&device, 0);

    // Create the first context
    hipCtxCreate(&ctx1, 0, device);

    // Create the second context
    hipCtxCreate(&ctx2, 0, device);

    // Get the current context and push it onto the stack
    hipCtx_t currentCtx;
    hipCtxGetCurrent(&currentCtx);

    hipCtxPushCurrent(ctx1);

    // Now the current context is ctx1
    std::cout << "Context 1 is now current" << std::endl;

    // Push the current context (ctx1) and switch to ctx2
    hipCtxPushCurrent(ctx2);

    // Now the current context is ctx2
    std::cout << "Context 2 is now current" << std::endl;

    // Pop the context stack to switch back to ctx1
    hipCtxPopCurrent(&currentCtx);

    // currentCtx should be ctx1 now
    std::cout << "Context 1 is back to current" << std::endl;

    // Pop the context stack to switch back to the original context
    hipCtxPopCurrent(&currentCtx);

    // currentCtx should be the original context now
    std::cout << "Original context is back to current" << std::endl;

    // Cleanup
    hipCtxDestroy(ctx1);
    hipCtxDestroy(ctx2);

    return 0;
}
