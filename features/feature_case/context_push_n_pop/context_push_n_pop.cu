#include <hip/hip_runtime.h>
#include <iostream>

void checkCUDAError(hipError_t result) {
    if (result != hipSuccess) {
        const char* errorStr;
        hipDrvGetErrorString(result, &errorStr);
        std::cerr << "CUDA Error: " << errorStr << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    hipCtx_t ctx1, ctx2;
    hipDevice_t device;
    hipError_t result;

    // Initialize the CUDA Driver API
    result = hipInit(0);
    checkCUDAError(result);

    // Get the device
    result = hipDeviceGet(&device, 0);
    checkCUDAError(result);

    // Create the first context
    result = hipCtxCreate(&ctx1, 0, device);
    checkCUDAError(result);

    // Create the second context
    result = hipCtxCreate(&ctx2, 0, device);
    checkCUDAError(result);

    // Get the current context and push it onto the stack
    hipCtx_t currentCtx;
    result = hipCtxGetCurrent(&currentCtx);
    checkCUDAError(result);

    result = hipCtxPushCurrent(ctx1);
    checkCUDAError(result);

    // Now the current context is ctx1
    std::cout << "Context 1 is now current" << std::endl;

    // Push the current context (ctx1) and switch to ctx2
    result = hipCtxPushCurrent(ctx2);
    checkCUDAError(result);

    // Now the current context is ctx2
    std::cout << "Context 2 is now current" << std::endl;

    // Pop the context stack to switch back to ctx1
    result = hipCtxPopCurrent(&currentCtx);
    checkCUDAError(result);

    // currentCtx should be ctx1 now
    std::cout << "Context 1 is back to current" << std::endl;

    // Pop the context stack to switch back to the original context
    result = hipCtxPopCurrent(&currentCtx);
    checkCUDAError(result);

    // currentCtx should be the original context now
    std::cout << "Original context is back to current" << std::endl;

    // Cleanup
    hipCtxDestroy(ctx1);
    hipCtxDestroy(ctx2);

    return 0;
}
