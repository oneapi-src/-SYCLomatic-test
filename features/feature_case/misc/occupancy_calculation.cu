
#include <hip/hip_runtime.h>
// ===------ occupancy_calculation.cu --------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

__global__ void k() {}

int main() {
  int num_blocks;
  int block_size = 128;
  size_t dynamic_shared_memory_size = 0;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, k, block_size, dynamic_shared_memory_size);
  hipFunction_t func;
  hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, func, block_size, dynamic_shared_memory_size);
  return 0;
}
