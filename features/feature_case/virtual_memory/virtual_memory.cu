// ====------ virtual_memory.cu--------------- *- CUDA -*---------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 100

int main() {
    hipInit(0);
    hipDevice_t device;
    hipDeviceGet(&device, 0);
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);

    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;
    size_t granularity;
    hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);   
    size_t POOL_SIZE =  granularity;

    hipDeviceptr_t reserved_addr;
    hipMemGenericAllocationHandle_t allocHandle;
    hipMemAddressReserve(&reserved_addr, POOL_SIZE, 0, 0, 0);
    hipMemCreate(&allocHandle, POOL_SIZE, &prop, 0);
    hipMemMap(reserved_addr, POOL_SIZE, 0, allocHandle, 0);

    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = device;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    hipMemSetAccess(reserved_addr, POOL_SIZE, &accessDesc, 1);
    int* host_data = new int[SIZE];
    int* host_data2 = new int[SIZE];
    for (int i = 0; i < SIZE; ++i) {
        host_data[i] = i;
        host_data2[i] = 0;
    }

    hipMemcpyHtoD(reserved_addr, host_data, SIZE * sizeof(int));
    hipMemcpyDtoH(host_data2, reserved_addr, SIZE * sizeof(int));

    for (int i = 0; i < SIZE; ++i) {
        if(host_data[i] != host_data2[i]) {
          std::cout << "test failed" << std::endl;
          exit(-1);
        }
    }
    std::cout << "test passed" << std::endl;

    hipMemUnmap(reserved_addr, POOL_SIZE);
    hipMemRelease(allocHandle);
    hipMemAddressFree(reserved_addr, POOL_SIZE);

    delete[] host_data;
    delete[] host_data2;

    hipCtxDestroy(context);
    return 0;
}
