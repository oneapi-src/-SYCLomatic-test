
#include <hip/hip_runtime.h>
// ===------- deviceAPI.cu ------------------------------- *- CUDA -* ----=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

int main() {
  hipSetDeviceFlags(hipDeviceMapHost);
  float *d_dst;
  hipStream_t stream;
  // unsupported API
  // cudaMemcpyPeerAsync(d_dst, 1, d_dst, 1, 111, stream);

  int concurrentManagedAccess = 0;
  int p_gpuDevice = 0;
  hipDeviceGetAttribute(&concurrentManagedAccess, hipDeviceAttributeConcurrentManagedAccess, p_gpuDevice);
  int alignment;
  hipDeviceGetAttribute(&alignment, hipDeviceAttributeTextureAlignment, p_gpuDevice);
  return 0;
}
