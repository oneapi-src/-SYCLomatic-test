#include<stdio.h>
#include <malloc.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include "nccl.h"


int main(int argc,char**argv){
  int version, nranks = 2, rank = 3, device_num = -1;
    int device_id = -1;
    
    ncclUniqueId id;
    ncclComm_t comm;
    int size=32;
    ncclCommInitRank(&comm, nranks, id, rank);
    ncclCommCount(comm, &device_num);
    //allocating and initializing device buffers
    float ** sendbuff=(float**)malloc(1*sizeof(float*));
    float ** recvbuff=(float**)malloc(1*sizeof(float*));
    hipStream_t stream = 0;
    hipMalloc(sendbuff,size*sizeof(float));
    hipMalloc(recvbuff,size*sizeof(float));
    hipMemset(sendbuff[0],1,size*sizeof(float));
    hipMemset(recvbuff[0],0,size*sizeof(float));
    float *hostbuff;
    hostbuff = (float *)malloc(size*sizeof(float));
    for (int i=0; i<size; ++i)
      hostbuff[i] = i;
    hipMemcpy(sendbuff[0], hostbuff, size*sizeof(float), hipMemcpyHostToDevice);

    ncclAllReduce(sendbuff, recvbuff, size, ncclFloat, ncclSum, comm, stream);
    hipFree(sendbuff);
    hipFree(recvbuff);
    free(hostbuff);

    printf("TEST PASS\n");
    return 0;

}