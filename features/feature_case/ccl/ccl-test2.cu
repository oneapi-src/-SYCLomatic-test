// ====------ ccl-test2.cu-------------------- *- CUDA -* --////////////--===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <mpi.h>
#include <nccl.h>
#include "hip/hip_runtime.h"

int main() {
    int version, nranks = 2, rank = 1, device_num = -1;
    int device_id = -1;
    int rank_g=0;
    hipStream_t stream=0;
    size_t count = 10*1024;
    float *sendbuff, *recvbuff,*hostbuff = (float *)malloc(count * sizeof(float));
    for(int i =1;i<count+1;++i) *(hostbuff+i-1)=i;

    hipMalloc(&sendbuff, count * sizeof(float));
    hipMemcpy(sendbuff, hostbuff, sizeof(float) * count, hipMemcpyHostToDevice);
    
    ncclUniqueId id;
    ncclComm_t comm;

    ncclGetVersion(&version);

    MPI_Init(NULL, NULL);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if (rank == 0)
      ncclGetUniqueId(&id);

    MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);

    hipSetDevice(0);

    ncclCommInitRank(&comm, nranks, id, rank);

    ncclCommUserRank(comm, &rank_g);

    ncclBroadcast(sendbuff, sendbuff, count, ncclFloat, rank_g, comm, stream);
    hipStreamSynchronize(stream);
    ncclBcast(sendbuff, count, ncclFloat, rank_g, comm, stream);
    hipStreamSynchronize(stream);
    ncclCommDestroy(comm);
    MPI_Finalize();
    hipFree(sendbuff);
    free(hostbuff);

    printf("TEST PASS\n");
}
