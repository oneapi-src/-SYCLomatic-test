// ====------ ccl-test.cu-------------------- *- CUDA -* --////////////--===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include <mpi.h>
#include <nccl.h>
#include "hip/hip_runtime.h"

int main() {
    int version, nranks = 2, rank = 1, device_num = -1;
    int device_id = -1;

    hipStream_t stream=0;
    size_t count = 10*1024*1024;
    float *sendbuff, *recvbuff,*hostbuff = (float *)malloc(count * sizeof(float));
    for(int i =1;i<count+1;++i) *(hostbuff+i-1)=i;

    hipMalloc(&sendbuff, count * sizeof(float));
    hipMalloc(&recvbuff, count * sizeof(float));
    hipMemcpy(sendbuff, hostbuff, sizeof(float) * count, hipMemcpyHostToDevice);
    
    ncclUniqueId id;
    ncclComm_t comm;

    ncclGetVersion(&version);

    MPI_Init(NULL, NULL);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if (rank == 0)
      ncclGetUniqueId(&id);

    MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);

    hipSetDevice(0);

    ncclCommInitRank(&comm, nranks, id, rank);

    ncclCommCount(comm, &device_num);

    ncclCommCuDevice(comm, &device_id);

    ncclAllReduce((void*)sendbuff, (void*)recvbuff, count, ncclFloat, ncclSum, comm, stream);
    hipStreamSynchronize(stream);
    ncclCommDestroy(comm);
    MPI_Finalize();
    hipFree(sendbuff);
    hipFree(recvbuff);
    free(hostbuff);

    printf("TEST PASS\n");
}
