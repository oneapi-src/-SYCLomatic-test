// ===------- profiler.cu ------------------------------- *- CUDA -* ----=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
int main() {
  hipProfilerStart();
  hipError_t result = hipProfilerStart();

  hipProfilerStop();
  hipError_t r2 = hipProfilerStop();
  return 0;
}
