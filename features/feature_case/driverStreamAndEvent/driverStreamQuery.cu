//===-------------- driverStreamQuery.cu -------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
//===------------------------------------------------------ -===//
#include <hip/hip_runtime.h>
#include <iostream>

int driverStreamQuery(void) {
    hipStream_t stream;

    // Initialize the CUDA Driver API
    hipInit(0);

    // Create a CUDA stream
    hipStreamCreateWithFlags(&stream, hipStreamDefault);

    hipError_t queryResult = hipStreamQuery(stream);

    if (queryResult == hipSuccess) {
        std::cout << "Kernel execution has completed." << std::endl;
    } else if (queryResult == hipErrorNotReady) {
        std::cout << "Kernel execution has not yet completed." << std::endl;
    } else {
        std::cerr << "Failed to query the stream status." << std::endl;
    }

    // Clean up resources
    hipStreamDestroy(stream);

    return 0;
}
