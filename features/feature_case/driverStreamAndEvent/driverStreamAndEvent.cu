// ====------ driverStreamAndEvent.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include<vector>
#include<hip/hip_runtime.h>

void process(hipStream_t st, char *data, hipError_t status) {}

template<typename T>
void callback(hipStream_t hStream, hipError_t status, void* userData) {
  T *data = static_cast<T *>(userData);
  process(hStream, data, status);
}

int main(){
  hipFunction_t f;
  hipStream_t s;
  hipStreamCreate(&s);
  hipEvent_t e;
  hipDeviceptr_t  cuPtr;  
  void* data;
  unsigned int flag;
  hipStreamAddCallback(s, callback<char>, data, flag);

  cuFuncSetCacheConfig(f, hipFuncCachePreferNone);

  hipStreamCreateWithFlags(&s, hipStreamDefault);
  hipStreamSynchronize(s);

  hipEventCreateWithFlags(&e, hipEventDefault);
  hipStreamWaitEvent(s, e, 0);

  hipEventRecord(e, s);
  hipEventSynchronize(e);

  hipError_t r;
  r = hipEventQuery(e);

  hipEvent_t start, end;
  hipEventRecord(start, s);
  hipEventRecord(end, s);
  hipEventSynchronize(start);
  hipEventSynchronize(end);
  float result_time;
  hipEventElapsedTime(&result_time, start, end);

  int rr;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, f);

  hipStreamAttachMemAsync(s, cuPtr, std::vector<int>(1,1).front(), flag);

  hipStreamDestroy(s);
  hipEventDestroy(start);
  hipEventDestroy(end);
  return 0;
}

