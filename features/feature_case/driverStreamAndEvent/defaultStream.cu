#include <iostream>
#include <cstdint>
#include <hip/hip_runtime.h>

struct stream_wrapper {
  hipStream_t s;
  stream_wrapper() : s(0) {}
  void set(hipStream_t t) { s = t; }
  void set_int(uintptr_t p) { s = (hipStream_t) p; }
};

int main() {
  stream_wrapper wr {};
  int x, y;
  int res = 0;
  int i = 0;

  auto run = [&]() {
    x = -1;
    y = 42;
    hipMemcpyAsync(&x, &y, sizeof(int), hipMemcpyDefault, wr.s);
    hipStreamSynchronize(wr.s);
    if (x != y) {
      std::cout << "default stream fail " << i << "\n";
      res = 1;
    }
    ++i;
  };

  run();

  wr.set(hipStreamDefault);
  run();
  
  wr.set(hipStreamLegacy);
  run();

  wr.set(hipStreamPerThread);
  run();

  wr.set_int(0);
  run();

  hipStream_t s;
  hipStreamCreate(&s);
  wr.set_int((uintptr_t) s);
  run();

  if (!res) {
    std::cout << "default stream success\n";
  }

  return res;  
}
