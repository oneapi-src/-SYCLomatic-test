// ====------ driverMem.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
int main(){
    size_t result1, result2;
    int size = 32;
    float* f_A;
    hipDeviceptr_t f_D = 0;
    hipDeviceptr_t f_D2 = 0;
    hipError_t r;

    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);

    hipMemAllocHost((void **)&f_A, size);

    hipMalloc(&f_D, size);

    hipMallocManaged(&f_D, size, hipMemAttachHost);


    hipStream_t stream;

    hipMemcpyHtoDAsync(f_D, f_A, size, stream);

    hipMemcpyHtoDAsync(f_D, f_A, size, 0);

    hipMemcpyHtoD(f_D, f_A, size);

    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);

    r = hipMemcpyDtoDAsync(f_D, f_D2, size, stream);

    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);

    r = hipMemcpyDtoDAsync(f_D, f_D2, size, 0);


    hipMemcpyDtoD(f_D, f_D2, size);

    r = hipMemcpyDtoD(f_D, f_D2, size);

    hipMemcpyDtoHAsync(f_A, f_D, size, stream);

    hipMemcpyDtoHAsync(f_A, f_D, size, 0);

    hipMemcpyDtoH(f_A, f_D, size);


    hipHostGetDevicePointer(&f_D, f_A, 0);

    hip_Memcpy2D cpy;

    cpy.dstMemoryType = hipMemoryTypeHost;

    cpy.dstHost = f_A;

    cpy.dstPitch = 20;

    cpy.dstY = 10;

    cpy.dstXInBytes = 15;


    cpy.srcMemoryType = hipMemoryTypeDevice;

    cpy.srcDevice = f_D;

    cpy.srcPitch = 20;

    cpy.srcY = 10;

    cpy.srcXInBytes = 15;


    cpy.WidthInBytes = 4;

    cpy.Height = 7;


    hipMemcpyParam2D(&cpy);

    hipMemcpyParam2DAsync(&cpy, stream);

    HIP_MEMCPY3D cpy2;

    hipArray_t ca;

    cpy2.dstMemoryType = hipMemoryTypeArray;

    cpy2.dstArray = ca;

    cpy2.dstPitch = 5;

    cpy2.dstHeight = 4;

    cpy2.dstY = 3;

    cpy2.dstZ = 2;

    cpy2.dstXInBytes = 1;

    cpy2.dstLOD = 0;


    cpy2.srcMemoryType = hipMemoryTypeHost;

    cpy2.srcHost = f_A;

    cpy2.srcPitch = 5;

    cpy2.srcHeight = 4;

    cpy2.srcY = 3;

    cpy2.srcZ = 2;

    cpy2.srcXInBytes = 1;

    cpy2.srcLOD = 0;


    cpy2.WidthInBytes = 3;

    cpy2.Height = 2;

    cpy2.Depth = 1;

    hipDrvMemcpy3D(&cpy2);

    float *h_A = (float *)malloc(100);
    hipHostFree(h_A);

    unsigned int* pFlags;

    hipMemAllocHost((void **)&f_A, size);

    hipHostGetFlags(pFlags, f_A);

    hipHostRegister((void *)pFlags, size, hipHostRegisterPortable);

    hipHostUnregister((void *)pFlags);

    return 0;
}

