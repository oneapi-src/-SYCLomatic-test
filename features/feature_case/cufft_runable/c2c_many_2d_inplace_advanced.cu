#include "hip/hip_runtime.h"
// ===--- c2c_many_2d_inplace_advanced.cu --------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


// forward
// input
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+         -+
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |          |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+          |
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |          batch0
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+  |  
// |   0   |   0   |   0   |   0   |   0   |   0   |   0   |   0   |   0   |  |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+ -+
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |          |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+          |
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |          batch1
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+  |  
// |   0   |   0   |   0   |   0   |   0   |   0   |   0   |   0   |   0   |  |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+ -+
// |___________n2__________|
// |________nembed2________|
// output
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+         -+
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |          |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+          |
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |          batch0
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+  |  
// |   0   |   0   |   0   |   0   |   0   |   0   |   0   |   0   |   0   |  |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+ -+
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |          |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+          |
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |          batch1
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+  |  
// |   0   |   0   |   0   |   0   |   0   |   0   |   0   |   0   |   0   |  |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+ -+
// |_______________________n2______________________|               |
// |____________________________nembed2____________________________|
bool c2c_many_2d_inplace_advanced() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float2 forward_idata_h[50];
  std::memset(forward_idata_h, 0, sizeof(float2) * 50);
  forward_idata_h[0] = float2{0, 1};
  forward_idata_h[2] = float2{2, 3};
  forward_idata_h[4] = float2{4, 5};
  forward_idata_h[8] = float2{6, 7};
  forward_idata_h[10] = float2{8, 9};
  forward_idata_h[12] = float2{10, 11};
  forward_idata_h[25] = float2{0, 1};
  forward_idata_h[27] = float2{2, 3};
  forward_idata_h[29] = float2{4, 5};
  forward_idata_h[33] = float2{6, 7};
  forward_idata_h[35] = float2{8, 9};
  forward_idata_h[37] = float2{10, 11};

  float2* data_d;
  hipMalloc(&data_d, sizeof(float2) * 50);
  hipMemcpy(data_d, forward_idata_h, sizeof(float2) * 50, hipMemcpyHostToDevice);

  size_t workSize;
  long long int n[2] = {2, 3};
  long long int inembed[2] = {3, 4};
  long long int onembed[2] = {3, 4};
  hipfftMakePlanMany64(plan_fwd, 2, n, inembed, 2, 25, onembed, 2, 25, HIPFFT_C2C, 2, &workSize);
  hipfftExecC2C(plan_fwd, data_d, data_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[50];
  hipMemcpy(forward_odata_h, data_d, sizeof(float2) * 50, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[50];
  forward_odata_ref[0] =  float2{30,36};
  forward_odata_ref[1] =  float2{2,3};
  forward_odata_ref[2] =  float2{-9.4641,-2.5359};
  forward_odata_ref[3] =  float2{6,7};
  forward_odata_ref[4] =  float2{-2.5359,-9.4641};
  forward_odata_ref[5] =  float2{10,11};
  forward_odata_ref[6] =  float2{0,0};
  forward_odata_ref[7] =  float2{0,0};
  forward_odata_ref[8] =  float2{-18,-18};
  forward_odata_ref[9] =  float2{0,0};
  forward_odata_ref[10] = float2{0,0};
  forward_odata_ref[11] = float2{0,0};
  forward_odata_ref[12] = float2{0,0};
  forward_odata_ref[13] = float2{0,0};
  forward_odata_ref[14] = float2{0,0};
  forward_odata_ref[15] = float2{0,0};
  forward_odata_ref[16] = float2{0,0};
  forward_odata_ref[17] = float2{0,0};
  forward_odata_ref[18] = float2{0,0};
  forward_odata_ref[19] = float2{0,0};
  forward_odata_ref[20] = float2{0,0};
  forward_odata_ref[21] = float2{0,0};
  forward_odata_ref[22] = float2{0,0};
  forward_odata_ref[23] = float2{0,0};
  forward_odata_ref[24] = float2{0,0};
  forward_odata_ref[25] = float2{30,36};
  forward_odata_ref[26] = float2{2,3};
  forward_odata_ref[27] = float2{-9.4641,-2.5359};
  forward_odata_ref[28] = float2{6,7};
  forward_odata_ref[29] = float2{-2.5359,-9.4641};
  forward_odata_ref[30] = float2{10,11};
  forward_odata_ref[31] = float2{0,0};
  forward_odata_ref[32] = float2{0,0};
  forward_odata_ref[33] = float2{-18,-18};
  forward_odata_ref[34] = float2{0,0};
  forward_odata_ref[35] = float2{0,0};
  forward_odata_ref[36] = float2{0,0};
  forward_odata_ref[37] = float2{0,0};

  hipfftDestroy(plan_fwd);

  std::vector<int> indices = {0, 2, 4,
                              8, 10, 12,
                              25, 27, 29,
                              33, 35, 37};
  if (!compare(forward_odata_ref, forward_odata_h, indices)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, indices);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, indices);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlanMany64(plan_bwd, 2, n, onembed, 2, 25, inembed, 2, 25, HIPFFT_C2C, 2, &workSize);
  hipfftExecC2C(plan_bwd, data_d, data_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float2 backward_odata_h[50];
  hipMemcpy(backward_odata_h, data_d, sizeof(float2) * 50, hipMemcpyDeviceToHost);

  float2 backward_odata_ref[50];
  backward_odata_ref[0] = float2{0, 6};
  backward_odata_ref[2] = float2{12, 18};
  backward_odata_ref[4] = float2{24, 30};
  backward_odata_ref[8] = float2{36, 42};
  backward_odata_ref[10] = float2{48, 54};
  backward_odata_ref[12] = float2{60, 66};
  backward_odata_ref[25] = float2{0, 6};
  backward_odata_ref[27] = float2{12, 18};
  backward_odata_ref[29] = float2{24, 30};
  backward_odata_ref[33] = float2{36, 42};
  backward_odata_ref[35] = float2{48, 54};
  backward_odata_ref[37] = float2{60, 66};

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  std::vector<int> indices_bwd = {0, 2, 4, 8, 10, 12,
                                  25, 27, 29, 33, 35, 37};
  if (!compare(backward_odata_ref, backward_odata_h, indices_bwd)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices_bwd);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices_bwd);
    return false;
  }
  return true;
}



#ifdef DEBUG_FFT
int main() {
#define FUNC c2c_many_2d_inplace_advanced
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

