#include "hip/hip_runtime.h"
// ===--- z2z_2d_inplace_make_plan.cu ------------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool z2z_2d_inplace_make_plan() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  double2 forward_idata_h[2][5];
  set_value((double*)forward_idata_h, 20);

  double2* data_d;
  hipMalloc(&data_d,sizeof(double2) * 10);
  hipMemcpy(data_d, forward_idata_h, sizeof(double2) * 10, hipMemcpyHostToDevice);

  size_t workSize;
  hipfftMakePlan2d(plan_fwd, 2, 5, HIPFFT_Z2Z, &workSize);
  hipfftExecZ2Z(plan_fwd, data_d, data_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  double2 forward_odata_h[10];
  hipMemcpy(forward_odata_h, data_d, sizeof(double2) * 10, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[10];
  forward_odata_ref[0] =  double2{90,100};
  forward_odata_ref[1] =  double2{-23.7638,3.76382};
  forward_odata_ref[2] =  double2{-13.2492,-6.7508};
  forward_odata_ref[3] =  double2{-6.7508,-13.2492};
  forward_odata_ref[4] =  double2{3.76382,-23.7638};
  forward_odata_ref[5] =  double2{-50,-50};
  forward_odata_ref[6] =  double2{0,0};
  forward_odata_ref[7] =  double2{0,0};
  forward_odata_ref[8] =  double2{0,0};
  forward_odata_ref[9] =  double2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 10)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 10);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 10);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlan2d(plan_bwd, 2, 5, HIPFFT_Z2Z, &workSize);
  hipfftExecZ2Z(plan_bwd, data_d, data_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  double2 backward_odata_h[10];
  hipMemcpy(backward_odata_h, data_d, sizeof(double2) * 10, hipMemcpyDeviceToHost);

  double2 backward_odata_ref[10];
  backward_odata_ref[0] =  double2{0,10};
  backward_odata_ref[1] =  double2{20,30};
  backward_odata_ref[2] =  double2{40,50};
  backward_odata_ref[3] =  double2{60,70};
  backward_odata_ref[4] =  double2{80,90};
  backward_odata_ref[5] =  double2{100,110};
  backward_odata_ref[6] =  double2{120,130};
  backward_odata_ref[7] =  double2{140,150};
  backward_odata_ref[8] =  double2{160,170};
  backward_odata_ref[9] =  double2{180,190};

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 10)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 10);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 10);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC z2z_2d_inplace_make_plan
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

