#include "hip/hip_runtime.h"
// ===--- r2cc2r_2d_outofplace.cu ----------------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>

bool r2cc2r_2d_outofplace() {
  hipfftHandle plan_fwd;
  float forward_idata_h[4][5];
  set_value((float*)forward_idata_h, 20);

  float* forward_idata_d;
  float2* forward_odata_d;
  float* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(float) * 20);
  hipMalloc(&forward_odata_d, sizeof(float2) * (5/2+1) * 4);
  hipMalloc(&backward_odata_d, sizeof(float) * 20);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(float) * 20, hipMemcpyHostToDevice);

  hipfftPlan2d(&plan_fwd, 4, 5, HIPFFT_R2C);
  hipfftExecR2C(plan_fwd, forward_idata_d, forward_odata_d);
  hipDeviceSynchronize();
  float2 forward_odata_h[12];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(float2) * (5/2+1) * 4, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[12];
  forward_odata_ref[0] =  float2{190,0};
  forward_odata_ref[1] =  float2{-10,13.7638};
  forward_odata_ref[2] =  float2{-10,3.2492};
  forward_odata_ref[3] =  float2{-50,50};
  forward_odata_ref[4] =  float2{0,0};
  forward_odata_ref[5] =  float2{0,0};
  forward_odata_ref[6] =  float2{-50,0};
  forward_odata_ref[7] =  float2{0,0};
  forward_odata_ref[8] =  float2{0,0};
  forward_odata_ref[9] =  float2{-50,-50};
  forward_odata_ref[10] = float2{0,0};
  forward_odata_ref[11] = float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 12)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 12);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 12);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftPlan2d(&plan_bwd, 4, 5, HIPFFT_C2R);
  hipfftExecC2R(plan_bwd, forward_odata_d, backward_odata_d);
  hipDeviceSynchronize();
  float backward_odata_h[20];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(float) * 20, hipMemcpyDeviceToHost);

  float backward_odata_ref[20];
  backward_odata_ref[0] =  0;
  backward_odata_ref[1] =  20;
  backward_odata_ref[2] =  40;
  backward_odata_ref[3] =  60;
  backward_odata_ref[4] =  80;
  backward_odata_ref[5] =  100;
  backward_odata_ref[6] =  120;
  backward_odata_ref[7] =  140;
  backward_odata_ref[8] =  160;
  backward_odata_ref[9] =  180;
  backward_odata_ref[10] = 200;
  backward_odata_ref[11] = 220;
  backward_odata_ref[12] = 240;
  backward_odata_ref[13] = 260;
  backward_odata_ref[14] = 280;
  backward_odata_ref[15] = 300;
  backward_odata_ref[16] = 320;
  backward_odata_ref[17] = 340;
  backward_odata_ref[18] = 360;
  backward_odata_ref[19] = 380;

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 20)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 20);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 20);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC r2cc2r_2d_outofplace
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

