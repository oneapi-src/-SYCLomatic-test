#include "hip/hip_runtime.h"
// ===--- r2cc2r_many_3d_inplace_basic.cu --------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool r2cc2r_many_3d_inplace_basic() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float forward_idata_h[64];
  forward_idata_h[0]  = 0;
  forward_idata_h[1]  = 1;
  forward_idata_h[2]  = 2;
  forward_idata_h[4]  = 3;
  forward_idata_h[5]  = 4;
  forward_idata_h[6]  = 5;
  forward_idata_h[8]  = 6;
  forward_idata_h[9]  = 7;
  forward_idata_h[10] = 8;
  forward_idata_h[12] = 9;
  forward_idata_h[13] = 10;
  forward_idata_h[14] = 11;
  forward_idata_h[16] = 12;
  forward_idata_h[17] = 13;
  forward_idata_h[18] = 14;
  forward_idata_h[20] = 15;
  forward_idata_h[21] = 16;
  forward_idata_h[22] = 17;
  forward_idata_h[24] = 18;
  forward_idata_h[25] = 19;
  forward_idata_h[26] = 20;
  forward_idata_h[28] = 21;
  forward_idata_h[29] = 22;
  forward_idata_h[30] = 23;

  forward_idata_h[32]  = 0;
  forward_idata_h[33]  = 1;
  forward_idata_h[34]  = 2;
  forward_idata_h[36]  = 3;
  forward_idata_h[37]  = 4;
  forward_idata_h[38]  = 5;
  forward_idata_h[40]  = 6;
  forward_idata_h[41]  = 7;
  forward_idata_h[42] = 8;
  forward_idata_h[44] = 9;
  forward_idata_h[45] = 10;
  forward_idata_h[46] = 11;
  forward_idata_h[48] = 12;
  forward_idata_h[49] = 13;
  forward_idata_h[50] = 14;
  forward_idata_h[52] = 15;
  forward_idata_h[53] = 16;
  forward_idata_h[54] = 17;
  forward_idata_h[56] = 18;
  forward_idata_h[57] = 19;
  forward_idata_h[58] = 20;
  forward_idata_h[60] = 21;
  forward_idata_h[61] = 22;
  forward_idata_h[62] = 23;

  float* data_d;
  hipMalloc(&data_d, sizeof(float) * 64);
  hipMemcpy(data_d, forward_idata_h, sizeof(float) * 64, hipMemcpyHostToDevice);

  int n[3] = {4, 2, 3};
  size_t workSize;
  hipfftMakePlanMany(plan_fwd, 3, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_R2C, 2, &workSize);
  hipfftExecR2C(plan_fwd, data_d, (float2*)data_d);
  hipDeviceSynchronize();
  float2 forward_odata_h[32];
  hipMemcpy(forward_odata_h, data_d, sizeof(float) * 64, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[32];
  forward_odata_ref[0]  = float2{276,0};
  forward_odata_ref[1]  = float2{-12,6.9282};
  forward_odata_ref[2]  = float2{-36,0};
  forward_odata_ref[3]  = float2{0,0};
  forward_odata_ref[4]  = float2{-72,72};
  forward_odata_ref[5]  = float2{0,0};
  forward_odata_ref[6]  = float2{0,0};
  forward_odata_ref[7]  = float2{0,0};
  forward_odata_ref[8]  = float2{-72,0};
  forward_odata_ref[9]  = float2{0,0};
  forward_odata_ref[10] = float2{0,0};
  forward_odata_ref[11] = float2{0,0};
  forward_odata_ref[12] = float2{-72,-72};
  forward_odata_ref[13] = float2{0,0};
  forward_odata_ref[14] = float2{0,0};
  forward_odata_ref[15] = float2{0,0};
  forward_odata_ref[16] = float2{276,0};
  forward_odata_ref[17] = float2{-12,6.9282};
  forward_odata_ref[18] = float2{-36,0};
  forward_odata_ref[19] = float2{0,0};
  forward_odata_ref[20] = float2{-72,72};
  forward_odata_ref[21] = float2{0,0};
  forward_odata_ref[22] = float2{0,0};
  forward_odata_ref[23] = float2{0,0};
  forward_odata_ref[24] = float2{-72,0};
  forward_odata_ref[25] = float2{0,0};
  forward_odata_ref[26] = float2{0,0};
  forward_odata_ref[27] = float2{0,0};
  forward_odata_ref[28] = float2{-72,-72};
  forward_odata_ref[29] = float2{0,0};
  forward_odata_ref[30] = float2{0,0};
  forward_odata_ref[31] = float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 32)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 32);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 32);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlanMany(plan_bwd, 3, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2R, 2, &workSize);
  hipfftExecC2R(plan_bwd, (float2*)data_d, data_d);
  hipDeviceSynchronize();
  float backward_odata_h[64];
  hipMemcpy(backward_odata_h, data_d, sizeof(float) * 64, hipMemcpyDeviceToHost);

  float backward_odata_ref[64];
  backward_odata_ref[0]  = 0;
  backward_odata_ref[1]  = 24;
  backward_odata_ref[2]  = 48;
  backward_odata_ref[3]  = 6.9282;
  backward_odata_ref[4]  = 72;
  backward_odata_ref[5]  = 96;
  backward_odata_ref[6]  = 120;
  backward_odata_ref[7]  = 6.9282;
  backward_odata_ref[8]  = 144;
  backward_odata_ref[9]  = 168;
  backward_odata_ref[10] = 192;
  backward_odata_ref[11] = 6.9282;
  backward_odata_ref[12] = 216;
  backward_odata_ref[13] = 240;
  backward_odata_ref[14] = 264;
  backward_odata_ref[15] = 6.9282;
  backward_odata_ref[16] = 288;
  backward_odata_ref[17] = 312;
  backward_odata_ref[18] = 336;
  backward_odata_ref[19] = 6.9282;
  backward_odata_ref[20] = 360;
  backward_odata_ref[21] = 384;
  backward_odata_ref[22] = 408;
  backward_odata_ref[23] = 6.9282;
  backward_odata_ref[24] = 432;
  backward_odata_ref[25] = 456;
  backward_odata_ref[26] = 480;
  backward_odata_ref[27] = 6.9282;
  backward_odata_ref[28] = 504;
  backward_odata_ref[29] = 528;
  backward_odata_ref[30] = 552;
  backward_odata_ref[31] = 6.9282;
  backward_odata_ref[32] = 0;
  backward_odata_ref[33] = 24;
  backward_odata_ref[34] = 48;
  backward_odata_ref[35] = 6.9282;
  backward_odata_ref[36] = 72;
  backward_odata_ref[37] = 96;
  backward_odata_ref[38] = 120;
  backward_odata_ref[39] = 6.9282;
  backward_odata_ref[40] = 144;
  backward_odata_ref[41] = 168;
  backward_odata_ref[42] = 192;
  backward_odata_ref[43] = 6.9282;
  backward_odata_ref[44] = 216;
  backward_odata_ref[45] = 240;
  backward_odata_ref[46] = 264;
  backward_odata_ref[47] = 6.9282;
  backward_odata_ref[48] = 288;
  backward_odata_ref[49] = 312;
  backward_odata_ref[50] = 336;
  backward_odata_ref[51] = 6.9282;
  backward_odata_ref[52] = 360;
  backward_odata_ref[53] = 384;
  backward_odata_ref[54] = 408;
  backward_odata_ref[55] = 6.9282;
  backward_odata_ref[56] = 432;
  backward_odata_ref[57] = 456;
  backward_odata_ref[58] = 480;
  backward_odata_ref[59] = 6.9282;
  backward_odata_ref[60] = 504;
  backward_odata_ref[61] = 528;
  backward_odata_ref[62] = 552;
  backward_odata_ref[63] = 6.9282;

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  std::vector<int> indices = {0, 1, 2,
                              4, 5, 6,
                              8, 9, 10,
                              12, 13, 14,
                              16 ,17, 18,
                              20, 21, 22,
                              24, 25, 26,
                              28, 29, 30,
                              32, 33, 34,
                              36, 37, 38,
                              40, 41, 42,
                              44, 45, 46,
                              48, 49, 50,
                              52, 53, 54,
                              56, 57, 58,
                              60, 61, 62};
  if (!compare(backward_odata_ref, backward_odata_h, indices)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC r2cc2r_many_3d_inplace_basic
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

