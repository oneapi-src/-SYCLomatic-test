#include "hip/hip_runtime.h"
// ===--- z2z_many_2d_outofplace_basic.cu --------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool z2z_many_2d_outofplace_basic() {
  hipfftHandle plan_fwd;
  double2 forward_idata_h[2/*n0*/ * 3/*n1*/ * 2/*batch*/];
  set_value((double*)forward_idata_h, 12);
  set_value((double*)forward_idata_h + 12, 12);

  double2* forward_idata_d;
  double2* forward_odata_d;
  double2* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(double2) * 12);
  hipMalloc(&forward_odata_d, sizeof(double2) * 12);
  hipMalloc(&backward_odata_d, sizeof(double2) * 12);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(double2) * 12, hipMemcpyHostToDevice);

  int n[2] = {2, 3};
  hipfftPlanMany(&plan_fwd, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_Z2Z, 2);
  hipfftExecZ2Z(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  double2 forward_odata_h[12];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(double2) * 12, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[12];
  forward_odata_ref[0] =  double2{30,36};
  forward_odata_ref[1] =  double2{-9.4641,-2.5359};
  forward_odata_ref[2] =  double2{-2.5359,-9.4641};
  forward_odata_ref[3] =  double2{-18,-18};
  forward_odata_ref[4] =  double2{0,0};
  forward_odata_ref[5] =  double2{0,0};
  forward_odata_ref[6] =  double2{30,36};
  forward_odata_ref[7] =  double2{-9.4641,-2.5359};
  forward_odata_ref[8] =  double2{-2.5359,-9.4641};
  forward_odata_ref[9] =  double2{-18,-18};
  forward_odata_ref[10] = double2{0,0};
  forward_odata_ref[11] = double2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 12)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 12);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 12);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftPlanMany(&plan_bwd, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_Z2Z, 2);
  hipfftExecZ2Z(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  double2 backward_odata_h[12];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(double2) * 12, hipMemcpyDeviceToHost);

  double2 backward_odata_ref[12];
  backward_odata_ref[0] =  double2{0,6};
  backward_odata_ref[1] =  double2{12,18};
  backward_odata_ref[2] =  double2{24,30};
  backward_odata_ref[3] =  double2{36,42};
  backward_odata_ref[4] =  double2{48,54};
  backward_odata_ref[5] =  double2{60,66};
  backward_odata_ref[6] =  double2{0,6};
  backward_odata_ref[7] =  double2{12,18};
  backward_odata_ref[8] =  double2{24,30};
  backward_odata_ref[9] =  double2{36,42};
  backward_odata_ref[10] = double2{48,54};
  backward_odata_ref[11] = double2{60,66};

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 12)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 12);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 12);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC z2z_many_2d_outofplace_basic
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

