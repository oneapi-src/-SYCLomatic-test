#include "hip/hip_runtime.h"
// ===--- d2zz2d_many_2d_outofplace_basic.cu -----------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>

bool d2zz2d_many_2d_outofplace_basic() {
  hipfftHandle plan_fwd;
  double forward_idata_h[2/*n0*/ * 3/*n1*/ * 2/*batch*/];
  set_value(forward_idata_h, 6);
  set_value(forward_idata_h + 6, 6);

  double* forward_idata_d;
  double2* forward_odata_d;
  double* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(double) * 2 * 3 * 2);
  hipMalloc(&forward_odata_d, 2 * 2 * sizeof(double2) * (3/2+1));
  hipMalloc(&backward_odata_d, sizeof(double) * 2 * 3 * 2);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(double) * 2 * 3 * 2, hipMemcpyHostToDevice);

  int n[2] = {2, 3};
  hipfftPlanMany(&plan_fwd, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_D2Z, 2);
  hipfftExecD2Z(plan_fwd, forward_idata_d, forward_odata_d);
  hipDeviceSynchronize();
  double2 forward_odata_h[8];
  hipMemcpy(forward_odata_h, forward_odata_d, 2 * 2 * sizeof(double2) * (3/2+1), hipMemcpyDeviceToHost);

  double2 forward_odata_ref[8];
  forward_odata_ref[0] =  double2{15,0};
  forward_odata_ref[1] =  double2{-3,1.73205};
  forward_odata_ref[2] =  double2{-9,0};
  forward_odata_ref[3] =  double2{0,0};
  forward_odata_ref[4] =  double2{15,0};
  forward_odata_ref[5] =  double2{-3,1.73205};
  forward_odata_ref[6] =  double2{-9,0};
  forward_odata_ref[7] =  double2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 8)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 8);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 8);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftPlanMany(&plan_bwd, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_Z2D, 2);
  hipfftExecZ2D(plan_bwd, forward_odata_d, backward_odata_d);
  hipDeviceSynchronize();
  double backward_odata_h[12];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(double) * 12, hipMemcpyDeviceToHost);

  double backward_odata_ref[12];
  backward_odata_ref[0] =  0;
  backward_odata_ref[1] =  6;
  backward_odata_ref[2] =  12;
  backward_odata_ref[3] =  18;
  backward_odata_ref[4] =  24;
  backward_odata_ref[5] =  30;
  backward_odata_ref[6] =  0;
  backward_odata_ref[7] =  6;
  backward_odata_ref[8] =  12;
  backward_odata_ref[9] =  18;
  backward_odata_ref[10] = 24;
  backward_odata_ref[11] = 30;

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 12)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 12);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 12);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC d2zz2d_many_2d_outofplace_basic
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

