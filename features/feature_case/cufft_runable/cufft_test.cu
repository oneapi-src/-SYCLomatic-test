// ===--- cufft_test.cu --------------------------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "c2c_1d_inplace.cu"
#include "c2c_1d_inplace_make_plan.cu"
#include "c2c_1d_outofplace.cu"
#include "c2c_1d_outofplace_make_plan.cu"
#include "c2c_2d_inplace.cu"
#include "c2c_2d_inplace_make_plan.cu"
#include "c2c_2d_outofplace.cu"
#include "c2c_2d_outofplace_make_plan.cu"
#include "c2c_3d_inplace.cu"
#include "c2c_3d_inplace_make_plan.cu"
#include "c2c_3d_outofplace.cu"
#include "c2c_3d_outofplace_make_plan.cu"
#include "c2c_many_1d_inplace_advanced.cu"
#include "c2c_many_1d_inplace_basic.cu"
#include "c2c_many_1d_outofplace_advanced.cu"
#include "c2c_many_1d_outofplace_basic.cu"
#include "c2c_many_2d_inplace_advanced.cu"
#include "c2c_many_2d_inplace_basic.cu"
#include "c2c_many_2d_outofplace_advanced.cu"
#include "c2c_many_2d_outofplace_basic.cu"
#include "c2c_many_3d_inplace_advanced.cu"
#include "c2c_many_3d_inplace_basic.cu"
#include "c2c_many_3d_outofplace_advanced.cu"
#include "c2c_many_3d_outofplace_basic.cu"
#include "d2zz2d_1d_inplace.cu"
#include "d2zz2d_1d_inplace_make_plan.cu"
#include "d2zz2d_1d_outofplace.cu"
#include "d2zz2d_1d_outofplace_make_plan.cu"
#include "d2zz2d_2d_inplace.cu"
#include "d2zz2d_2d_inplace_make_plan.cu"
#include "d2zz2d_2d_outofplace.cu"
#include "d2zz2d_2d_outofplace_make_plan.cu"
#include "d2zz2d_3d_inplace.cu"
#include "d2zz2d_3d_inplace_make_plan.cu"
#include "d2zz2d_3d_outofplace.cu"
#include "d2zz2d_3d_outofplace_make_plan.cu"
#include "d2zz2d_many_1d_inplace_advanced.cu"
#include "d2zz2d_many_1d_inplace_basic.cu"
#include "d2zz2d_many_1d_outofplace_advanced.cu"
#include "d2zz2d_many_1d_outofplace_basic.cu"
#include "d2zz2d_many_2d_inplace_advanced.cu"
#include "d2zz2d_many_2d_inplace_basic.cu"
#include "d2zz2d_many_2d_outofplace_advanced.cu"
#include "d2zz2d_many_2d_outofplace_basic.cu"
#include "d2zz2d_many_3d_inplace_advanced.cu"
#include "d2zz2d_many_3d_inplace_basic.cu"
#include "d2zz2d_many_3d_outofplace_advanced.cu"
#include "d2zz2d_many_3d_outofplace_basic.cu"
#include "r2cc2r_1d_inplace.cu"
#include "r2cc2r_1d_inplace_make_plan.cu"
#include "r2cc2r_1d_outofplace.cu"
#include "r2cc2r_1d_outofplace_make_plan.cu"
#include "r2cc2r_2d_inplace.cu"
#include "r2cc2r_2d_inplace_make_plan.cu"
#include "r2cc2r_2d_outofplace.cu"
#include "r2cc2r_2d_outofplace_make_plan.cu"
#include "r2cc2r_3d_inplace.cu"
#include "r2cc2r_3d_inplace_make_plan.cu"
#include "r2cc2r_3d_outofplace.cu"
#include "r2cc2r_3d_outofplace_make_plan.cu"
#include "r2cc2r_many_1d_inplace_advanced.cu"
#include "r2cc2r_many_1d_inplace_basic.cu"
#include "r2cc2r_many_1d_outofplace_advanced.cu"
#include "r2cc2r_many_1d_outofplace_basic.cu"
#include "r2cc2r_many_2d_inplace_advanced.cu"
#include "r2cc2r_many_2d_inplace_basic.cu"
#include "r2cc2r_many_2d_outofplace_advanced.cu"
#include "r2cc2r_many_2d_outofplace_basic.cu"
#include "r2cc2r_many_3d_inplace_advanced.cu"
#include "r2cc2r_many_3d_inplace_basic.cu"
#include "r2cc2r_many_3d_outofplace_advanced.cu"
#include "r2cc2r_many_3d_outofplace_basic.cu"
#include "z2z_1d_inplace.cu"
#include "z2z_1d_inplace_make_plan.cu"
#include "z2z_1d_outofplace.cu"
#include "z2z_1d_outofplace_make_plan.cu"
#include "z2z_2d_inplace.cu"
#include "z2z_2d_inplace_make_plan.cu"
#include "z2z_2d_outofplace.cu"
#include "z2z_2d_outofplace_make_plan.cu"
#include "z2z_3d_inplace.cu"
#include "z2z_3d_inplace_make_plan.cu"
#include "z2z_3d_outofplace.cu"
#include "z2z_3d_outofplace_make_plan.cu"
#include "z2z_many_1d_inplace_advanced.cu"
#include "z2z_many_1d_inplace_basic.cu"
#include "z2z_many_1d_outofplace_advanced.cu"
#include "z2z_many_1d_outofplace_basic.cu"
#include "z2z_many_2d_inplace_advanced.cu"
#include "z2z_many_2d_inplace_basic.cu"
#include "z2z_many_2d_outofplace_advanced.cu"
#include "z2z_many_2d_outofplace_basic.cu"
#include "z2z_many_3d_inplace_advanced.cu"
#include "z2z_many_3d_inplace_basic.cu"
#include "z2z_many_3d_outofplace_advanced.cu"
#include "z2z_many_3d_outofplace_basic.cu"


#define TEST(func)                   \
{                                    \
  bool res = func();                 \
  if (!res) {                        \
  printf("failed case: "#func"\n");  \
  }                                  \
  all_pass = all_pass && res;        \
  hipDeviceSynchronize();           \
}

int main() {
  bool all_pass = true;

  TEST(c2c_1d_inplace);
  TEST(c2c_1d_inplace_make_plan);
  TEST(c2c_1d_outofplace);
  TEST(c2c_1d_outofplace_make_plan);
  TEST(c2c_2d_inplace);
  TEST(c2c_2d_inplace_make_plan);
  TEST(c2c_2d_outofplace);
  TEST(c2c_2d_outofplace_make_plan);
  TEST(c2c_3d_inplace);
  TEST(c2c_3d_inplace_make_plan);
  TEST(c2c_3d_outofplace);
  TEST(c2c_3d_outofplace_make_plan);
  TEST(c2c_many_1d_inplace_advanced);
  TEST(c2c_many_1d_inplace_basic);
  TEST(c2c_many_1d_outofplace_advanced);
  TEST(c2c_many_1d_outofplace_basic);
  TEST(c2c_many_2d_inplace_advanced);
  TEST(c2c_many_2d_inplace_basic);
  TEST(c2c_many_2d_outofplace_advanced);
  TEST(c2c_many_2d_outofplace_basic);
  TEST(c2c_many_3d_inplace_advanced);
  TEST(c2c_many_3d_inplace_basic);
  TEST(c2c_many_3d_outofplace_advanced);
  TEST(c2c_many_3d_outofplace_basic);
  TEST(d2zz2d_1d_inplace);
  TEST(d2zz2d_1d_inplace_make_plan);
  TEST(d2zz2d_1d_outofplace);
  TEST(d2zz2d_1d_outofplace_make_plan);
  TEST(d2zz2d_2d_inplace);
  TEST(d2zz2d_2d_inplace_make_plan);
  TEST(d2zz2d_2d_outofplace);
  TEST(d2zz2d_2d_outofplace_make_plan);
  TEST(d2zz2d_3d_inplace);
  TEST(d2zz2d_3d_inplace_make_plan);
  TEST(d2zz2d_3d_outofplace);
  TEST(d2zz2d_3d_outofplace_make_plan);
  TEST(d2zz2d_many_1d_inplace_advanced);
  TEST(d2zz2d_many_1d_inplace_basic);
  TEST(d2zz2d_many_1d_outofplace_advanced);
  TEST(d2zz2d_many_1d_outofplace_basic);
  TEST(d2zz2d_many_2d_inplace_advanced);
  TEST(d2zz2d_many_2d_inplace_basic);
  TEST(d2zz2d_many_2d_outofplace_advanced);
  TEST(d2zz2d_many_2d_outofplace_basic);
  TEST(d2zz2d_many_3d_inplace_advanced);
  TEST(d2zz2d_many_3d_inplace_basic);
  TEST(d2zz2d_many_3d_outofplace_advanced);
  TEST(d2zz2d_many_3d_outofplace_basic);
  TEST(r2cc2r_1d_inplace);
  TEST(r2cc2r_1d_inplace_make_plan);
  TEST(r2cc2r_1d_outofplace);
  TEST(r2cc2r_1d_outofplace_make_plan);
  TEST(r2cc2r_2d_inplace);
  TEST(r2cc2r_2d_inplace_make_plan);
  TEST(r2cc2r_2d_outofplace);
  TEST(r2cc2r_2d_outofplace_make_plan);
  TEST(r2cc2r_3d_inplace);
  TEST(r2cc2r_3d_inplace_make_plan);
  TEST(r2cc2r_3d_outofplace);
  TEST(r2cc2r_3d_outofplace_make_plan);
  TEST(r2cc2r_many_1d_inplace_advanced);
  TEST(r2cc2r_many_1d_inplace_basic);
  TEST(r2cc2r_many_1d_outofplace_advanced);
  TEST(r2cc2r_many_1d_outofplace_basic);
  TEST(r2cc2r_many_2d_inplace_advanced);
  TEST(r2cc2r_many_2d_inplace_basic);
  TEST(r2cc2r_many_2d_outofplace_advanced);
  TEST(r2cc2r_many_2d_outofplace_basic);
  TEST(r2cc2r_many_3d_inplace_advanced);
  TEST(r2cc2r_many_3d_inplace_basic);
  TEST(r2cc2r_many_3d_outofplace_advanced);
  TEST(r2cc2r_many_3d_outofplace_basic);
  TEST(z2z_1d_inplace);
  TEST(z2z_1d_inplace_make_plan);
  TEST(z2z_1d_outofplace);
  TEST(z2z_1d_outofplace_make_plan);
  TEST(z2z_2d_inplace);
  TEST(z2z_2d_inplace_make_plan);
  TEST(z2z_2d_outofplace);
  TEST(z2z_2d_outofplace_make_plan);
  TEST(z2z_3d_inplace);
  TEST(z2z_3d_inplace_make_plan);
  TEST(z2z_3d_outofplace);
  TEST(z2z_3d_outofplace_make_plan);
  TEST(z2z_many_1d_inplace_advanced);
  TEST(z2z_many_1d_inplace_basic);
  TEST(z2z_many_1d_outofplace_advanced);
  TEST(z2z_many_1d_outofplace_basic);
  TEST(z2z_many_2d_inplace_advanced);
  TEST(z2z_many_2d_inplace_basic);
  TEST(z2z_many_2d_outofplace_advanced);
  TEST(z2z_many_2d_outofplace_basic);
  TEST(z2z_many_3d_inplace_advanced);
  TEST(z2z_many_3d_inplace_basic);
  TEST(z2z_many_3d_outofplace_advanced);
  TEST(z2z_many_3d_outofplace_basic);

  if (all_pass) {
    printf("Pass\n");
    return 0;
  }
  printf("Fail\n");
  return -1;
}
