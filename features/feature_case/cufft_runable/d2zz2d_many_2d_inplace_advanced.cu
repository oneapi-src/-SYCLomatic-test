#include "hip/hip_runtime.h"
// ===--- d2zz2d_many_2d_inplace_advanced.cu -----------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>



// forward
// input
// +---+---+---+---+         -+
// | r | r | r | 0 |          |
// +---+---+---+---+          |
// | r | r | r | 0 |          batch0
// +---+---+---+---+         -+
// | r | r | r | 0 |          |
// +---+---+---+---+          |
// | r | r | r | 0 |          batch1
// +---+---+---+---+         -+
// |______n2_______|
// |___nembed2_____|
// output
// +---+---+---+---+         -+
// |   c   |   c   |          |
// +---+---+---+---+          |
// |   c   |   c   |          batch0
// +---+---+---+---+         -+
// |   c   |   c   |          |
// +---+---+---+---+          |
// |   c   |   c   |          batch1
// +---+---+---+---+         -+
// |______n2_______|
// |___nembed2_____|
bool d2zz2d_many_2d_inplace_advanced() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  double forward_idata_h[16];
  std::memset(forward_idata_h, 0, sizeof(double) * 16);
  forward_idata_h[0]  = 0;
  forward_idata_h[1]  = 1;
  forward_idata_h[2]  = 2;
  forward_idata_h[4]  = 3;
  forward_idata_h[5]  = 4;
  forward_idata_h[6]  = 5;
  forward_idata_h[8]  = 0;
  forward_idata_h[9]  = 1;
  forward_idata_h[10] = 2;
  forward_idata_h[12] = 3;
  forward_idata_h[13] = 4;
  forward_idata_h[14] = 5;

  double* data_d;
  hipMalloc(&data_d, sizeof(double) * 16);
  hipMemcpy(data_d, forward_idata_h, sizeof(double) * 16, hipMemcpyHostToDevice);

  size_t workSize;
  long long int n[2] = {2, 3};
  long long int inembed[2] = {2, 4};
  long long int onembed[2] = {2, 2};
  hipfftMakePlanMany64(plan_fwd, 2, n, inembed, 1, 8, onembed, 1, 4, HIPFFT_D2Z, 2, &workSize);
  hipfftExecD2Z(plan_fwd, data_d, (double2*)data_d);
  hipDeviceSynchronize();
  double2 forward_odata_h[8];
  hipMemcpy(forward_odata_h, data_d, sizeof(double) * 16, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[8];
  forward_odata_ref[0] = double2{15, 0};
  forward_odata_ref[1] = double2{-3, 1.73205};
  forward_odata_ref[2] = double2{-9, 0};
  forward_odata_ref[3] = double2{0, 0};
  forward_odata_ref[4] = double2{15, 0};
  forward_odata_ref[5] = double2{-3, 1.73205};
  forward_odata_ref[6] = double2{-9, 0};
  forward_odata_ref[7] = double2{0, 0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 8)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 8);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 8);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlanMany64(plan_bwd, 2, n, onembed, 1, 4, inembed, 1, 8, HIPFFT_Z2D, 2, &workSize);
  hipfftExecZ2D(plan_bwd, (double2*)data_d, data_d);
  hipDeviceSynchronize();
  double backward_odata_h[16];
  hipMemcpy(backward_odata_h, data_d, sizeof(double) * 16, hipMemcpyDeviceToHost);

  double backward_odata_ref[16];
  backward_odata_ref[0]  = 0;
  backward_odata_ref[1]  = 6;
  backward_odata_ref[2]  = 12;
  backward_odata_ref[4]  = 18;
  backward_odata_ref[5]  = 24;
  backward_odata_ref[6]  = 30;
  backward_odata_ref[8]  = 0;
  backward_odata_ref[9]  = 6;
  backward_odata_ref[10] = 12;
  backward_odata_ref[12] = 18;
  backward_odata_ref[13] = 24;
  backward_odata_ref[14] = 30;

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  std::vector<int> indices_bwd = {0, 1, 2, 4, 5, 6,
                                  8, 9, 10, 12, 13, 14};
  if (!compare(backward_odata_ref, backward_odata_h, indices_bwd)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices_bwd);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices_bwd);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC d2zz2d_many_2d_inplace_advanced
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

