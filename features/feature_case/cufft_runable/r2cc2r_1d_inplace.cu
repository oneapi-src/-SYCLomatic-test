#include "hip/hip_runtime.h"
// ===--- r2cc2r_1d_inplace.cu -------------------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>



bool r2cc2r_1d_inplace() {
  hipfftHandle plan_fwd;
  float forward_idata_h[16];
  set_value(forward_idata_h, 7);
  set_value(forward_idata_h + 8, 7);

  float* data_d;
  hipMalloc(&data_d, sizeof(float) * 16);
  hipMemcpy(data_d, forward_idata_h, sizeof(float) * 16, hipMemcpyHostToDevice);

  hipfftPlan1d(&plan_fwd, 7, HIPFFT_R2C, 2);
  hipfftExecR2C(plan_fwd, data_d, (float2*)data_d);
  hipDeviceSynchronize();
  float2 forward_odata_h[8];
  hipMemcpy(forward_odata_h, data_d, sizeof(float) * 16, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[8];
  forward_odata_ref[0] = float2{21,0};
  forward_odata_ref[1] = float2{-3.5,7.26783};
  forward_odata_ref[2] = float2{-3.5,2.79116};
  forward_odata_ref[3] = float2{-3.5,0.798852};
  forward_odata_ref[4] = float2{21,0};
  forward_odata_ref[5] = float2{-3.5,7.26783};
  forward_odata_ref[6] = float2{-3.5,2.79116};
  forward_odata_ref[7] = float2{-3.5,0.798852};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 8)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 8);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 8);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftPlan1d(&plan_bwd, 7, HIPFFT_C2R, 2);
  hipfftExecC2R(plan_bwd, (float2*)data_d, data_d);
  hipDeviceSynchronize();
  float backward_odata_h[16];
  hipMemcpy(backward_odata_h, data_d, sizeof(float) * 16, hipMemcpyDeviceToHost);

  float backward_odata_ref[16];
  backward_odata_ref[0] = 0;
  backward_odata_ref[1] = 7;
  backward_odata_ref[2] = 14;
  backward_odata_ref[3] = 21;
  backward_odata_ref[4] = 28;
  backward_odata_ref[5] = 35;
  backward_odata_ref[6] = 42;
  backward_odata_ref[7] = 0.798852;
  backward_odata_ref[8] = 0;
  backward_odata_ref[9] = 7;
  backward_odata_ref[10] = 14;
  backward_odata_ref[11] = 21;
  backward_odata_ref[12] = 28;
  backward_odata_ref[13] = 35;
  backward_odata_ref[14] = 42;
  backward_odata_ref[15] = 0.798852;

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  std::vector<int> indices = {0, 1, 2,  3,  4,  5,  6,
                              8, 9, 10, 11, 12, 13, 14};
  if (!compare(backward_odata_ref, backward_odata_h, indices)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices);
    return false;
  }
  return true;
}

#ifdef DEBUG_FFT
int main() {
#define FUNC r2cc2r_1d_inplace
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

