#include "hip/hip_runtime.h"
// ===--- z2z_many_1d_outofplace_advanced.cu -----------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


// forward
// input
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |   0   |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |   0   |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+
// |________________________n______________________|               |       |________________________n______________________|               |       |
// |_____________________________nembed____________________________|       |_____________________________nembed____________________________|       |
// |___________________________________batch0______________________________|___________________________________batch1______________________________|
// output
// +---+---+---+---+---+---+---+---+---+---+---+---+
// |   c   |   c   |   c   |   c   |   c   |   c   |
// +---+---+---+---+---+---+---+---+---+---+---+---+
// |___________n___________|___________n___________|
// |_________nembed________|_________nembed________|
// |_________batch0________|_________batch1________|
bool z2z_many_1d_outofplace_advanced() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  double2 forward_idata_h[18];
  std::memset(forward_idata_h, 0, sizeof(double2) * 18);
  set_value_with_stride(forward_idata_h, 3, 2);
  set_value_with_stride(forward_idata_h + 9, 3, 2);

  double2* forward_idata_d;
  double2* forward_odata_d;
  double2* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(double2) * 18);
  hipMalloc(&forward_odata_d, sizeof(double2) * 6);
  hipMalloc(&backward_odata_d, sizeof(double2) * 18);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(double2) * 18, hipMemcpyHostToDevice);

  size_t workSize;
  long long int n[1] = {3};
  long long int inembed[1] = {4};
  long long int onembed[1] = {3};
  hipfftXtMakePlanMany(plan_fwd, 1, n, inembed, 2, 9, HIP_C_64F, onembed, 1, 3, HIP_C_64F, 2, &workSize, HIP_C_64F);
  hipfftXtExec(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  double2 forward_odata_h[6];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(double2) * 6, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[6];
  forward_odata_ref[0] =  double2{6,9};
  forward_odata_ref[1] =  double2{-4.73205,-1.26795};
  forward_odata_ref[2] =  double2{-1.26795,-4.73205};
  forward_odata_ref[3] =  double2{6,9};
  forward_odata_ref[4] =  double2{-4.73205,-1.26795};
  forward_odata_ref[5] =  double2{-1.26795,-4.73205};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 6)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 6);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 6);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftXtMakePlanMany(plan_bwd, 1, n, onembed, 1, 3, HIP_C_64F, inembed, 2, 9, HIP_C_64F, 2, &workSize, HIP_C_64F);
  hipfftXtExec(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  double2 backward_odata_h[18];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(double2) * 18, hipMemcpyDeviceToHost);

  double2 backward_odata_ref[18];
  backward_odata_ref[0] =  double2{0,3};
  backward_odata_ref[1] =  double2{0,0};
  backward_odata_ref[2] =  double2{6,9};
  backward_odata_ref[3] =  double2{0,0};
  backward_odata_ref[4] =  double2{12,15};
  backward_odata_ref[5] =  double2{0,0};
  backward_odata_ref[6] =  double2{0,0};
  backward_odata_ref[7] =  double2{0,0};
  backward_odata_ref[8] =  double2{0,0};
  backward_odata_ref[9] =  double2{0,3};
  backward_odata_ref[10] = double2{0,0};
  backward_odata_ref[11] = double2{6,9};
  backward_odata_ref[12] = double2{0,0};
  backward_odata_ref[13] = double2{12,15};
  backward_odata_ref[14] = double2{0,0};
  backward_odata_ref[15] = double2{0,0};
  backward_odata_ref[16] = double2{0,0};
  backward_odata_ref[17] = double2{0,0};

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  std::vector<int> indices = {0, 2, 4,
                              9, 11, 13};
  if (!compare(backward_odata_ref, backward_odata_h, indices)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC z2z_many_1d_outofplace_advanced
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

