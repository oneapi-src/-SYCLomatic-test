#include "hip/hip_runtime.h"
// ===--- z2z_many_1d_outofplace_basic.cu --------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool z2z_many_1d_outofplace_basic() {
  hipfftHandle plan_fwd;
  double2 forward_idata_h[10];
  set_value((double*)forward_idata_h, 10);
  set_value((double*)forward_idata_h + 10, 10);

  double2* forward_idata_d;
  double2* forward_odata_d;
  double2* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(double2) * 10);
  hipMalloc(&forward_odata_d, sizeof(double2) * 10);
  hipMalloc(&backward_odata_d, sizeof(double2) * 10);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(double2) * 10, hipMemcpyHostToDevice);

  int n[1] = {5};
  hipfftPlanMany(&plan_fwd, 1, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_Z2Z, 2);
  hipfftExecZ2Z(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  double2 forward_odata_h[10];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(double2) * 10, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[10];
  forward_odata_ref[0] =  double2{20,25};
  forward_odata_ref[1] =  double2{-11.8819,1.88191};
  forward_odata_ref[2] =  double2{-6.6246,-3.3754};
  forward_odata_ref[3] =  double2{-3.3754,-6.6246};
  forward_odata_ref[4] =  double2{1.88191,-11.8819};
  forward_odata_ref[5] =  double2{20,25};
  forward_odata_ref[6] =  double2{-11.8819,1.88191};
  forward_odata_ref[7] =  double2{-6.6246,-3.3754};
  forward_odata_ref[8] =  double2{-3.3754,-6.6246};
  forward_odata_ref[9] =  double2{1.88191,-11.8819};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 10)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 10);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 10);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftPlanMany(&plan_bwd, 1, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_Z2Z, 2);
  hipfftExecZ2Z(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  double2 backward_odata_h[10];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(double2) * 10, hipMemcpyDeviceToHost);

  double2 backward_odata_ref[10];
  backward_odata_ref[0] =  double2{0,5};
  backward_odata_ref[1] =  double2{10,15};
  backward_odata_ref[2] =  double2{20,25};
  backward_odata_ref[3] =  double2{30,35};
  backward_odata_ref[4] =  double2{40,45};
  backward_odata_ref[5] =  double2{0,5};
  backward_odata_ref[6] =  double2{10,15};
  backward_odata_ref[7] =  double2{20,25};
  backward_odata_ref[8] =  double2{30,35};
  backward_odata_ref[9] =  double2{40,45};

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 10)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 10);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 10);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC z2z_many_1d_outofplace_basic
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

