#include "hip/hip_runtime.h"
// ===--- c2c_many_2d_inplace_basic.cu -----------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>

bool c2c_many_2d_inplace_basic() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float2 forward_idata_h[2/*n0*/ * 3/*n1*/ * 2/*batch*/];
  set_value((float*)forward_idata_h, 12);
  set_value((float*)forward_idata_h + 12, 12);

  float2* data_d;
  hipMalloc(&data_d, sizeof(float2) * 12);
  hipMemcpy(data_d, forward_idata_h, sizeof(float2) * 12, hipMemcpyHostToDevice);

  int n[2] = {2, 3};
  size_t workSize;
  hipfftMakePlanMany(plan_fwd, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, 2, &workSize);
  hipfftExecC2C(plan_fwd, data_d, data_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[12];
  hipMemcpy(forward_odata_h, data_d, sizeof(float2) * 12, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[12];
  forward_odata_ref[0] =  float2{30,36};
  forward_odata_ref[1] =  float2{-9.4641,-2.5359};
  forward_odata_ref[2] =  float2{-2.5359,-9.4641};
  forward_odata_ref[3] =  float2{-18,-18};
  forward_odata_ref[4] =  float2{0,0};
  forward_odata_ref[5] =  float2{0,0};
  forward_odata_ref[6] =  float2{30,36};
  forward_odata_ref[7] =  float2{-9.4641,-2.5359};
  forward_odata_ref[8] =  float2{-2.5359,-9.4641};
  forward_odata_ref[9] =  float2{-18,-18};
  forward_odata_ref[10] = float2{0,0};
  forward_odata_ref[11] = float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 12)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 12);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 12);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlanMany(plan_bwd, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, 2, &workSize);
  hipfftExecC2C(plan_bwd, data_d, data_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float2 backward_odata_h[12];
  hipMemcpy(backward_odata_h, data_d, sizeof(float2) * 12, hipMemcpyDeviceToHost);

  float2 backward_odata_ref[12];
  backward_odata_ref[0] =  float2{0,6};
  backward_odata_ref[1] =  float2{12,18};
  backward_odata_ref[2] =  float2{24,30};
  backward_odata_ref[3] =  float2{36,42};
  backward_odata_ref[4] =  float2{48,54};
  backward_odata_ref[5] =  float2{60,66};
  backward_odata_ref[6] =  float2{0,6};
  backward_odata_ref[7] =  float2{12,18};
  backward_odata_ref[8] =  float2{24,30};
  backward_odata_ref[9] =  float2{36,42};
  backward_odata_ref[10] = float2{48,54};
  backward_odata_ref[11] = float2{60,66};

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 12)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 12);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 12);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC c2c_many_2d_inplace_basic
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

