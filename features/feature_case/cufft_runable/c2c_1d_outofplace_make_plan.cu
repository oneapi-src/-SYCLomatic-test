#include "hip/hip_runtime.h"
// ===--- c2c_1d_outofplace_make_plan.cu ---------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool c2c_1d_outofplace_make_plan() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float2 forward_idata_h[14];
  set_value((float*)forward_idata_h, 14);
  set_value((float*)forward_idata_h + 14, 14);

  float2* forward_idata_d;
  float2* forward_odata_d;
  float2* backward_odata_d;
  hipMalloc(&forward_idata_d, 2 * sizeof(float2) * 7);
  hipMalloc(&forward_odata_d, 2 * sizeof(float2) * 7);
  hipMalloc(&backward_odata_d, 2 * sizeof(float2) * 7);
  hipMemcpy(forward_idata_d, forward_idata_h, 2 * sizeof(float2) * 7, hipMemcpyHostToDevice);

  size_t workSize;
  hipfftMakePlan1d(plan_fwd, 7, HIPFFT_C2C, 2, &workSize);
  hipfftExecC2C(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[14];
  hipMemcpy(forward_odata_h, forward_odata_d, 2 * sizeof(float2) * 7, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[14];
  forward_odata_ref[0] =  float2{42,49};
  forward_odata_ref[1] =  float2{-21.5356,7.53565};
  forward_odata_ref[2] =  float2{-12.5823,-1.41769};
  forward_odata_ref[3] =  float2{-8.5977,-5.4023};
  forward_odata_ref[4] =  float2{-5.4023,-8.5977};
  forward_odata_ref[5] =  float2{-1.41769,-12.5823};
  forward_odata_ref[6] =  float2{7.53565,-21.5356};
  forward_odata_ref[7] =  float2{42,49};
  forward_odata_ref[8] =  float2{-21.5356,7.53565};
  forward_odata_ref[9] =  float2{-12.5823,-1.41769};
  forward_odata_ref[10] = float2{-8.5977,-5.4023};
  forward_odata_ref[11] = float2{-5.4023,-8.5977};
  forward_odata_ref[12] = float2{-1.41769,-12.5823};
  forward_odata_ref[13] = float2{7.53565,-21.5356};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 14)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 14);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 14);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlan1d(plan_bwd, 7, HIPFFT_C2C, 2, &workSize);
  hipfftExecC2C(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float2 backward_odata_h[14];
  hipMemcpy(backward_odata_h, backward_odata_d, 2 * sizeof(float2) * 7, hipMemcpyDeviceToHost);

  float2 backward_odata_ref[14];
  backward_odata_ref[0] =  float2{0,7};
  backward_odata_ref[1] =  float2{14,21};
  backward_odata_ref[2] =  float2{28,35};
  backward_odata_ref[3] =  float2{42,49};
  backward_odata_ref[4] =  float2{56,63};
  backward_odata_ref[5] =  float2{70,77};
  backward_odata_ref[6] =  float2{84,91};
  backward_odata_ref[7] =  float2{0,7};
  backward_odata_ref[8] =  float2{14,21};
  backward_odata_ref[9] =  float2{28,35};
  backward_odata_ref[10] = float2{42,49};
  backward_odata_ref[11] = float2{56,63};
  backward_odata_ref[12] = float2{70,77};
  backward_odata_ref[13] = float2{84,91};

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 14)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 14);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 14);
    return false;
  }
  return true;
}

#ifdef DEBUG_FFT
int main() {
#define FUNC c2c_1d_outofplace_make_plan
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

