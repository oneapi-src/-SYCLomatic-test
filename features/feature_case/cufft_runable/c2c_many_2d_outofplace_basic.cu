#include "hip/hip_runtime.h"
// ===--- c2c_many_2d_outofplace_basic.cu --------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>

bool c2c_many_2d_outofplace_basic() {
  hipfftHandle plan_fwd;
  float2 forward_idata_h[2/*n0*/ * 3/*n1*/ * 2/*batch*/];
  set_value((float*)forward_idata_h, 12);
  set_value((float*)forward_idata_h + 12, 12);

  float2* forward_idata_d;
  float2* forward_odata_d;
  float2* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(float2) * 12);
  hipMalloc(&forward_odata_d, sizeof(float2) * 12);
  hipMalloc(&backward_odata_d, sizeof(float2) * 12);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(float2) * 12, hipMemcpyHostToDevice);

  int n[2] = {2, 3};
  hipfftPlanMany(&plan_fwd, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, 2);
  hipfftExecC2C(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[12];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(float2) * 12, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[12];
  forward_odata_ref[0] =  float2{30,36};
  forward_odata_ref[1] =  float2{-9.4641,-2.5359};
  forward_odata_ref[2] =  float2{-2.5359,-9.4641};
  forward_odata_ref[3] =  float2{-18,-18};
  forward_odata_ref[4] =  float2{0,0};
  forward_odata_ref[5] =  float2{0,0};
  forward_odata_ref[6] =  float2{30,36};
  forward_odata_ref[7] =  float2{-9.4641,-2.5359};
  forward_odata_ref[8] =  float2{-2.5359,-9.4641};
  forward_odata_ref[9] =  float2{-18,-18};
  forward_odata_ref[10] = float2{0,0};
  forward_odata_ref[11] = float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 12)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 12);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 12);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftPlanMany(&plan_bwd, 2, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_C2C, 2);
  hipfftExecC2C(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float2 backward_odata_h[12];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(float2) * 12, hipMemcpyDeviceToHost);

  float2 backward_odata_ref[12];
  backward_odata_ref[0] =  float2{0,6};
  backward_odata_ref[1] =  float2{12,18};
  backward_odata_ref[2] =  float2{24,30};
  backward_odata_ref[3] =  float2{36,42};
  backward_odata_ref[4] =  float2{48,54};
  backward_odata_ref[5] =  float2{60,66};
  backward_odata_ref[6] =  float2{0,6};
  backward_odata_ref[7] =  float2{12,18};
  backward_odata_ref[8] =  float2{24,30};
  backward_odata_ref[9] =  float2{36,42};
  backward_odata_ref[10] = float2{48,54};
  backward_odata_ref[11] = float2{60,66};

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 12)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 12);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 12);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC c2c_many_2d_outofplace_basic
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

