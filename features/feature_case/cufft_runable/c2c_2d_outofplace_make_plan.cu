#include "hip/hip_runtime.h"
// ===--- c2c_2d_outofplace_make_plan.cu ---------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool c2c_2d_outofplace_make_plan() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float2 forward_idata_h[2][5];
  set_value((float*)forward_idata_h, 20);

  float2* forward_idata_d;
  float2* forward_odata_d;
  float2* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(float2) * 10);
  hipMalloc(&forward_odata_d, sizeof(float2) * 10);
  hipMalloc(&backward_odata_d, sizeof(float2) * 10);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(float2) * 10, hipMemcpyHostToDevice);

  size_t workSize;
  hipfftMakePlan2d(plan_fwd, 2, 5, HIPFFT_C2C, &workSize);
  hipfftExecC2C(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[10];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(float2) * 10, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[10];
  forward_odata_ref[0] =  float2{90,100};
  forward_odata_ref[1] =  float2{-23.7638,3.76382};
  forward_odata_ref[2] =  float2{-13.2492,-6.7508};
  forward_odata_ref[3] =  float2{-6.7508,-13.2492};
  forward_odata_ref[4] =  float2{3.76382,-23.7638};
  forward_odata_ref[5] =  float2{-50,-50};
  forward_odata_ref[6] =  float2{0,0};
  forward_odata_ref[7] =  float2{0,0};
  forward_odata_ref[8] =  float2{0,0};
  forward_odata_ref[9] =  float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 10)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 10);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 10);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlan2d(plan_bwd, 2, 5, HIPFFT_C2C, &workSize);
  hipfftExecC2C(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float2 backward_odata_h[10];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(float2) * 10, hipMemcpyDeviceToHost);

  float2 backward_odata_ref[10];
  backward_odata_ref[0] =  float2{0,10};
  backward_odata_ref[1] =  float2{20,30};
  backward_odata_ref[2] =  float2{40,50};
  backward_odata_ref[3] =  float2{60,70};
  backward_odata_ref[4] =  float2{80,90};
  backward_odata_ref[5] =  float2{100,110};
  backward_odata_ref[6] =  float2{120,130};
  backward_odata_ref[7] =  float2{140,150};
  backward_odata_ref[8] =  float2{160,170};
  backward_odata_ref[9] =  float2{180,190};

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 10)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 10);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 10);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC c2c_2d_outofplace_make_plan
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

