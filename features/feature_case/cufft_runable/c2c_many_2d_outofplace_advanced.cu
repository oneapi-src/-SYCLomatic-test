#include "hip/hip_runtime.h"
// ===--- c2c_many_2d_outofplace_advanced.cu -----------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>

// forward
// input
// +---+---+---+---+---+---+      -+
// |   c   |   c   |   c   |       |
// +---+---+---+---+---+---+       |
// |   c   |   c   |   c   |       batch0
// +---+---+---+---+---+---+      -+
// |   c   |   c   |   c   |       |
// +---+---+---+---+---+---+       |
// |   c   |   c   |   c   |       batch1
// +---+---+---+---+---+---+      -+
// |___________n2__________|
// |________nembed2________|
// output
// +---+---+---+---+---+---+ -+
// |   c   |   c   |   c   |  |
// +---+---+---+---+---+---+  batch0
// |   c   |   c   |   c   |  |
// +---+---+---+---+---+---+ -+
// |   c   |   c   |   c   |  |
// +---+---+---+---+---+---+  batch1
// |   c   |   c   |   c   |  |
// +---+---+---+---+---+---+ -+
// |__________n2___________|
// |________nembed2________|
bool c2c_many_2d_outofplace_advanced() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float2 forward_idata_h[12];
  std::memset(forward_idata_h, 0, sizeof(float2) * 12);
  forward_idata_h[0] = float2{0, 1};
  forward_idata_h[1] = float2{2, 3};
  forward_idata_h[2] = float2{4, 5};
  forward_idata_h[3] = float2{6, 7};
  forward_idata_h[4] = float2{8, 9};
  forward_idata_h[5] = float2{10, 11};
  forward_idata_h[6] = float2{0, 1};
  forward_idata_h[7] = float2{2, 3};
  forward_idata_h[8] = float2{4, 5};
  forward_idata_h[9] = float2{6, 7};
  forward_idata_h[10] = float2{8, 9};
  forward_idata_h[11] = float2{10, 11};

  float2* forward_idata_d;
  float2* forward_odata_d;
  float2* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(float2) * 12);
  hipMalloc(&forward_odata_d, sizeof(float2) * 12);
  hipMalloc(&backward_odata_d, sizeof(float2) * 12);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(float2) * 12, hipMemcpyHostToDevice);

  size_t workSize;
  long long int n[2] = {2, 3};
  long long int inembed[2] = {2, 3};
  long long int onembed[2] = {2, 3};
  hipfftXtMakePlanMany(plan_fwd, 2, n, inembed, 1, 6, HIP_C_32F, onembed, 1, 6, HIP_C_32F, 2, &workSize, HIP_C_32F);
  hipfftXtExec(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[12];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(float2) * 12, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[12];
  forward_odata_ref[0] =  float2{30,36};
  forward_odata_ref[1] =  float2{-9.4641,-2.5359};
  forward_odata_ref[2] =  float2{-2.5359,-9.4641};
  forward_odata_ref[3] =  float2{-18,-18};
  forward_odata_ref[4] =  float2{0,0};
  forward_odata_ref[5] =  float2{0,0};
  forward_odata_ref[6] =  float2{30,36};
  forward_odata_ref[7] =  float2{-9.4641,-2.5359};
  forward_odata_ref[8] =  float2{-2.5359,-9.4641};
  forward_odata_ref[9] =  float2{-18,-18};
  forward_odata_ref[10] = float2{0,0};
  forward_odata_ref[11] = float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 12)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 12);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 12);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftXtMakePlanMany(plan_bwd, 2, n, onembed, 1, 6, HIP_C_32F, inembed, 1, 6, HIP_C_32F, 2, &workSize, HIP_C_32F);
  hipfftXtExec(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float2 backward_odata_h[12];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(float2) * 12, hipMemcpyDeviceToHost);

  float2 backward_odata_ref[12];
  backward_odata_ref[0] = float2{0, 6};
  backward_odata_ref[1] = float2{12, 18};
  backward_odata_ref[2] = float2{24, 30};
  backward_odata_ref[3] = float2{36, 42};
  backward_odata_ref[4] = float2{48, 54};
  backward_odata_ref[5] = float2{60, 66};
  backward_odata_ref[6] = float2{0, 6};
  backward_odata_ref[7] = float2{12, 18};
  backward_odata_ref[8] = float2{24, 30};
  backward_odata_ref[9] = float2{36, 42};
  backward_odata_ref[10] = float2{48, 54};
  backward_odata_ref[11] = float2{60, 66};

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 12)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 12);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 12);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC c2c_many_2d_outofplace_advanced
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

