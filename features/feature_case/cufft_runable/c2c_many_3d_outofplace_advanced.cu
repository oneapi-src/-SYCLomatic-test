#include "hip/hip_runtime.h"
// ===--- c2c_many_3d_outofplace_advanced.cu -----------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


// forward
// input
// +---+---+---+---+---+---+---+---+  ---+              ----+       
// |   c   |   c   |   c   |   c   |     |                  |       
// +---+---+---+---+---+---+---+---+     n2/nembed2         |     
// |   c   |   c   |   c   |   c   |     |                  |                    
// +---+---+---+---+---+---+---+---+     |                  |            
// |   c   |   c   |   c   |   c   |     |                  |            
// +---+---+---+---+---+---+---+---+  ---+          n1/nembed1/a batch
// |   c   |   c   |   c   |   c   |     |                  |            
// +---+---+---+---+---+---+---+---+     n2/nembed2         |          
// |   c   |   c   |   c   |   c   |     |                  |        
// +---+---+---+---+---+---+---+---+     |                  |        
// |   c   |   c   |   c   |   c   |     |                  |        
// +---+---+---+---+---+---+---+---+  ---+              ----+     
// output
// +---+---+---+---+---+---+---+---+  ---+              ----+       
// |   c   |   c   |   c   |   c   |     |                  |       
// +---+---+---+---+---+---+---+---+     n2/nembed2         |     
// |   c   |   c   |   c   |   c   |     |                  |                    
// +---+---+---+---+---+---+---+---+     |                  |            
// |   c   |   c   |   c   |   c   |     |                  |            
// +---+---+---+---+---+---+---+---+  ---+          n1/nembed1/a batch
// |   c   |   c   |   c   |   c   |     |                  |            
// +---+---+---+---+---+---+---+---+     n2/nembed2         |          
// |   c   |   c   |   c   |   c   |     |                  |        
// +---+---+---+---+---+---+---+---+     |                  |        
// |   c   |   c   |   c   |   c   |     |                  |        
// +---+---+---+---+---+---+---+---+  ---+              ----+     
// |______________n3_______________|
// |____________nembed3____________|
bool c2c_many_3d_outofplace_advanced() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float2 forward_idata_h[48];
  std::memset(forward_idata_h, 0, sizeof(float2) * 48);
  set_value((float*)forward_idata_h, 48);
  set_value((float*)forward_idata_h + 48, 48);

  float2* forward_idata_d;
  float2* forward_odata_d;
  float2* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(float2) * 48);
  hipMalloc(&forward_odata_d, sizeof(float2) * 48);
  hipMalloc(&backward_odata_d, sizeof(float2) * 48);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(float2) * 48, hipMemcpyHostToDevice);

  size_t workSize;
  long long int n[3] = {2, 3, 4};
  long long int inembed[3] = {2, 3, 4};
  long long int onembed[3] = {2, 3, 4};
  hipfftXtMakePlanMany(plan_fwd, 3, n, inembed, 1, 24, HIP_C_32F, onembed, 1, 24, HIP_C_32F, 2, &workSize, HIP_C_32F);
  hipfftXtExec(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[48];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(float2) * 48, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[48];
  forward_odata_ref[0] =  float2{552,576};
  forward_odata_ref[1] =  float2{-48,0};
  forward_odata_ref[2] =  float2{-24,-24};
  forward_odata_ref[3] =  float2{0,-48};
  forward_odata_ref[4] =  float2{-151.426,-40.5744};
  forward_odata_ref[5] =  float2{0,0};
  forward_odata_ref[6] =  float2{0,0};
  forward_odata_ref[7] =  float2{0,0};
  forward_odata_ref[8] =  float2{-40.5744,-151.426};
  forward_odata_ref[9] =  float2{0,0};
  forward_odata_ref[10] = float2{0,0};
  forward_odata_ref[11] = float2{0,0};
  forward_odata_ref[12] = float2{-288,-288};
  forward_odata_ref[13] = float2{0,0};
  forward_odata_ref[14] = float2{0,0};
  forward_odata_ref[15] = float2{0,0};
  forward_odata_ref[16] = float2{0,0};
  forward_odata_ref[17] = float2{0,0};
  forward_odata_ref[18] = float2{0,0};
  forward_odata_ref[19] = float2{0,0};
  forward_odata_ref[20] = float2{0,0};
  forward_odata_ref[21] = float2{0,0};
  forward_odata_ref[22] = float2{0,0};
  forward_odata_ref[23] = float2{0,0};
  forward_odata_ref[24] = float2{552,576};
  forward_odata_ref[25] = float2{-48,0};
  forward_odata_ref[26] = float2{-24,-24};
  forward_odata_ref[27] = float2{0,-48};
  forward_odata_ref[28] = float2{-151.426,-40.5744};
  forward_odata_ref[29] = float2{0,0};
  forward_odata_ref[30] = float2{0,0};
  forward_odata_ref[31] = float2{0,0};
  forward_odata_ref[32] = float2{-40.5744,-151.426};
  forward_odata_ref[33] = float2{0,0};
  forward_odata_ref[34] = float2{0,0};
  forward_odata_ref[35] = float2{0,0};
  forward_odata_ref[36] = float2{-288,-288};
  forward_odata_ref[37] = float2{0,0};
  forward_odata_ref[38] = float2{0,0};
  forward_odata_ref[39] = float2{0,0};
  forward_odata_ref[40] = float2{0,0};
  forward_odata_ref[41] = float2{0,0};
  forward_odata_ref[42] = float2{0,0};
  forward_odata_ref[43] = float2{0,0};
  forward_odata_ref[44] = float2{0,0};
  forward_odata_ref[45] = float2{0,0};
  forward_odata_ref[46] = float2{0,0};
  forward_odata_ref[47] = float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 48)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 48);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 48);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftXtMakePlanMany(plan_bwd, 3, n, onembed, 1, 24, HIP_C_32F, inembed, 1, 24, HIP_C_32F, 2, &workSize, HIP_C_32F);
  hipfftXtExec(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float2 backward_odata_h[48];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(float2) * 48, hipMemcpyDeviceToHost);

  float2 backward_odata_ref[48] = {
    float2{0, 24},
    float2{48, 72},
    float2{96, 120},
    float2{144, 168},
    float2{192, 216},
    float2{240, 264},
    float2{288, 312},
    float2{336, 360},
    float2{384, 408},
    float2{432, 456},
    float2{480, 504},
    float2{528, 552},
    float2{576, 600},
    float2{624, 648},
    float2{672, 696},
    float2{720, 744},
    float2{768, 792},
    float2{816, 840},
    float2{864, 888},
    float2{912, 936},
    float2{960, 984},
    float2{1008, 1032},
    float2{1056, 1080},
    float2{1104, 1128},
    float2{0, 24},
    float2{48, 72},
    float2{96, 120},
    float2{144, 168},
    float2{192, 216},
    float2{240, 264},
    float2{288, 312},
    float2{336, 360},
    float2{384, 408},
    float2{432, 456},
    float2{480, 504},
    float2{528, 552},
    float2{576, 600},
    float2{624, 648},
    float2{672, 696},
    float2{720, 744},
    float2{768, 792},
    float2{816, 840},
    float2{864, 888},
    float2{912, 936},
    float2{960, 984},
    float2{1008, 1032},
    float2{1056, 1080},
    float2{1104, 1128}
  };

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 48)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 48);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 48);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC c2c_many_3d_outofplace_advanced
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

