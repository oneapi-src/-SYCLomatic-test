#include "hip/hip_runtime.h"
// ===--- z2z_many_3d_inplace_basic.cu -----------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool z2z_many_3d_inplace_basic() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  double2 forward_idata_h[2/*n0*/ * 3/*n1*/ * 4/*n2*/ * 2/*batch*/];
  set_value((double*)forward_idata_h, 48);
  set_value((double*)forward_idata_h + 48, 48);

  double2* data_d;
  hipMalloc(&data_d, sizeof(double2) * 48);
  hipMemcpy(data_d, forward_idata_h, sizeof(double2) * 48, hipMemcpyHostToDevice);

  int n[3] = {2 ,3, 4};
  size_t workSize;
  hipfftMakePlanMany(plan_fwd, 3, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_Z2Z, 2, &workSize);
  hipfftExecZ2Z(plan_fwd, data_d, data_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  double2 forward_odata_h[48];
  hipMemcpy(forward_odata_h, data_d, sizeof(double2) * 48, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[48];
  forward_odata_ref[0] =  double2{552,576};
  forward_odata_ref[1] =  double2{-48,0};
  forward_odata_ref[2] =  double2{-24,-24};
  forward_odata_ref[3] =  double2{0,-48};
  forward_odata_ref[4] =  double2{-151.426,-40.5744};
  forward_odata_ref[5] =  double2{0,0};
  forward_odata_ref[6] =  double2{0,0};
  forward_odata_ref[7] =  double2{0,0};
  forward_odata_ref[8] =  double2{-40.5744,-151.426};
  forward_odata_ref[9] =  double2{0,0};
  forward_odata_ref[10] = double2{0,0};
  forward_odata_ref[11] = double2{0,0};
  forward_odata_ref[12] = double2{-288,-288};
  forward_odata_ref[13] = double2{0,0};
  forward_odata_ref[14] = double2{0,0};
  forward_odata_ref[15] = double2{0,0};
  forward_odata_ref[16] = double2{0,0};
  forward_odata_ref[17] = double2{0,0};
  forward_odata_ref[18] = double2{0,0};
  forward_odata_ref[19] = double2{0,0};
  forward_odata_ref[20] = double2{0,0};
  forward_odata_ref[21] = double2{0,0};
  forward_odata_ref[22] = double2{0,0};
  forward_odata_ref[23] = double2{0,0};
  forward_odata_ref[24] = double2{552,576};
  forward_odata_ref[25] = double2{-48,0};
  forward_odata_ref[26] = double2{-24,-24};
  forward_odata_ref[27] = double2{0,-48};
  forward_odata_ref[28] = double2{-151.426,-40.5744};
  forward_odata_ref[29] = double2{0,0};
  forward_odata_ref[30] = double2{0,0};
  forward_odata_ref[31] = double2{0,0};
  forward_odata_ref[32] = double2{-40.5744,-151.426};
  forward_odata_ref[33] = double2{0,0};
  forward_odata_ref[34] = double2{0,0};
  forward_odata_ref[35] = double2{0,0};
  forward_odata_ref[36] = double2{-288,-288};
  forward_odata_ref[37] = double2{0,0};
  forward_odata_ref[38] = double2{0,0};
  forward_odata_ref[39] = double2{0,0};
  forward_odata_ref[40] = double2{0,0};
  forward_odata_ref[41] = double2{0,0};
  forward_odata_ref[42] = double2{0,0};
  forward_odata_ref[43] = double2{0,0};
  forward_odata_ref[44] = double2{0,0};
  forward_odata_ref[45] = double2{0,0};
  forward_odata_ref[46] = double2{0,0};
  forward_odata_ref[47] = double2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 48)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 48);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 48);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlanMany(plan_bwd, 3, n, nullptr, 0, 0, nullptr, 0, 0, HIPFFT_Z2Z, 2, &workSize);
  hipfftExecZ2Z(plan_bwd, data_d, data_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  double2 backward_odata_h[48];
  hipMemcpy(backward_odata_h, data_d, sizeof(double2) * 48, hipMemcpyDeviceToHost);

  double2 backward_odata_ref[48];
  backward_odata_ref[0] =  double2{0,24};
  backward_odata_ref[1] =  double2{48,72};
  backward_odata_ref[2] =  double2{96,120};
  backward_odata_ref[3] =  double2{144,168};
  backward_odata_ref[4] =  double2{192,216};
  backward_odata_ref[5] =  double2{240,264};
  backward_odata_ref[6] =  double2{288,312};
  backward_odata_ref[7] =  double2{336,360};
  backward_odata_ref[8] =  double2{384,408};
  backward_odata_ref[9] =  double2{432,456};
  backward_odata_ref[10] = double2{480,504};
  backward_odata_ref[11] = double2{528,552};
  backward_odata_ref[12] = double2{576,600};
  backward_odata_ref[13] = double2{624,648};
  backward_odata_ref[14] = double2{672,696};
  backward_odata_ref[15] = double2{720,744};
  backward_odata_ref[16] = double2{768,792};
  backward_odata_ref[17] = double2{816,840};
  backward_odata_ref[18] = double2{864,888};
  backward_odata_ref[19] = double2{912,936};
  backward_odata_ref[20] = double2{960,984};
  backward_odata_ref[21] = double2{1008,1032};
  backward_odata_ref[22] = double2{1056,1080};
  backward_odata_ref[23] = double2{1104,1128};
  backward_odata_ref[24] = double2{0,24};
  backward_odata_ref[25] = double2{48,72};
  backward_odata_ref[26] = double2{96,120};
  backward_odata_ref[27] = double2{144,168};
  backward_odata_ref[28] = double2{192,216};
  backward_odata_ref[29] = double2{240,264};
  backward_odata_ref[30] = double2{288,312};
  backward_odata_ref[31] = double2{336,360};
  backward_odata_ref[32] = double2{384,408};
  backward_odata_ref[33] = double2{432,456};
  backward_odata_ref[34] = double2{480,504};
  backward_odata_ref[35] = double2{528,552};
  backward_odata_ref[36] = double2{576,600};
  backward_odata_ref[37] = double2{624,648};
  backward_odata_ref[38] = double2{672,696};
  backward_odata_ref[39] = double2{720,744};
  backward_odata_ref[40] = double2{768,792};
  backward_odata_ref[41] = double2{816,840};
  backward_odata_ref[42] = double2{864,888};
  backward_odata_ref[43] = double2{912,936};
  backward_odata_ref[44] = double2{960,984};
  backward_odata_ref[45] = double2{1008,1032};
  backward_odata_ref[46] = double2{1056,1080};
  backward_odata_ref[47] = double2{1104,1128};

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 48)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 48);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 48);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC z2z_many_3d_inplace_basic
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

