#include "hip/hip_runtime.h"
// ===--- r2cc2r_many_2d_outofplace_advanced.cu --------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>



// forward
// input
// +---+---+---+---+---+---+     -+
// | r | 0 | r | 0 | r | 0 |      |
// +---+---+---+---+---+---+      |
// | r | 0 | r | 0 | r | 0 |      batch0
// +---+---+---+---+---+---+     -+
// | r | 0 | r | 0 | r | 0 |      |
// +---+---+---+---+---+---+      |
// | r | 0 | r | 0 | r | 0 |      batch1
// +---+---+---+---+---+---+     -+
// |__________n2___________|
// |________nembed2________|
// output
// +---+---+---+---+ -+
// |   c   |   c   |  |
// +---+---+---+---+  batch0
// |   c   |   c   |  |
// +---+---+---+---+ -+
// |   c   |   c   |  |
// +---+---+---+---+  batch1
// |   c   |   c   |  |
// +---+---+---+---+ -+
// |______n2_______|
// |____nembed2____|
bool r2cc2r_many_2d_outofplace_advanced() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float forward_idata_h[24];
  std::memset(forward_idata_h, 0, sizeof(float) * 24);
  forward_idata_h[0]  = 0;
  forward_idata_h[2]  = 1;
  forward_idata_h[4]  = 2;
  forward_idata_h[6]  = 3;
  forward_idata_h[8]  = 4;
  forward_idata_h[10] = 5;
  forward_idata_h[12] = 0;
  forward_idata_h[14] = 1;
  forward_idata_h[16] = 2;
  forward_idata_h[18] = 3;
  forward_idata_h[20] = 4;
  forward_idata_h[22] = 5;

  float* forward_idata_d;
  float2* forward_odata_d;
  float* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(float) * 24);
  hipMalloc(&forward_odata_d, sizeof(float2) * 8);
  hipMalloc(&backward_odata_d, sizeof(float) * 24);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(float) * 24, hipMemcpyHostToDevice);

  long long int n[2] = {2, 3};
  long long int inembed[2] = {2, 3};
  long long int onembed[2] = {2, 2};
  size_t workSize;
  hipfftXtMakePlanMany(plan_fwd, 2, n, inembed, 2, 12, HIP_R_32F, onembed, 1, 4, HIP_C_32F, 2, &workSize, HIP_C_32F);
  hipfftXtExec(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[8];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(float2) * 8, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[8];
  forward_odata_ref[0] =  float2{15,0};
  forward_odata_ref[1] =  float2{-3,1.73205};
  forward_odata_ref[2] =  float2{-9,0};
  forward_odata_ref[3] =  float2{0,0};
  forward_odata_ref[4] =  float2{15,0};
  forward_odata_ref[5] =  float2{-3,1.73205};
  forward_odata_ref[6] =  float2{-9,0};
  forward_odata_ref[7] =  float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 8)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 8);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 8);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftXtMakePlanMany(plan_bwd, 2, n, onembed, 1, 4, HIP_C_32F, inembed, 2, 12, HIP_R_32F, 2, &workSize, HIP_C_32F);
  hipfftXtExec(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float backward_odata_h[24];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(float) * 24, hipMemcpyDeviceToHost);

  float backward_odata_ref[24];
  backward_odata_ref[0]  = 0;
  backward_odata_ref[2]  = 6;
  backward_odata_ref[4]  = 12;
  backward_odata_ref[6]  = 18;
  backward_odata_ref[8]  = 24;
  backward_odata_ref[10] = 30;
  backward_odata_ref[12] = 0;
  backward_odata_ref[14] = 6;
  backward_odata_ref[16] = 12;
  backward_odata_ref[18] = 18;
  backward_odata_ref[20] = 24;
  backward_odata_ref[22] = 30;

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  std::vector<int> indices = {0, 2, 4,
                              6, 8, 10,
                              12, 14, 16,
                              18, 20, 22};
  if (!compare(backward_odata_ref, backward_odata_h, indices)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices);
    return false;
  }
  return true;
}

#ifdef DEBUG_FFT
int main() {
#define FUNC r2cc2r_many_2d_outofplace_advanced
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

