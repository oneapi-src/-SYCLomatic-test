#include "hip/hip_runtime.h"
// ===--- z2z_1d_inplace.cu ----------------------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool z2z_1d_inplace() {
  hipfftHandle plan_fwd;
  double2 forward_idata_h[14];
  set_value((double*)forward_idata_h, 14);
  set_value((double*)forward_idata_h + 14, 14);

  double2* data_d;
  hipMalloc(&data_d, 2 * sizeof(double2) * 7);
  hipMemcpy(data_d, forward_idata_h, 2 * sizeof(double2) * 7, hipMemcpyHostToDevice);

  hipfftPlan1d(&plan_fwd, 7, HIPFFT_Z2Z, 2);
  hipfftExecZ2Z(plan_fwd, data_d, data_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  double2 forward_odata_h[14];
  hipMemcpy(forward_odata_h, data_d, 2 * sizeof(double2) * 7, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[14];
  forward_odata_ref[0] =  double2{42,49};
  forward_odata_ref[1] =  double2{-21.5356,7.53565};
  forward_odata_ref[2] =  double2{-12.5823,-1.41769};
  forward_odata_ref[3] =  double2{-8.5977,-5.4023};
  forward_odata_ref[4] =  double2{-5.4023,-8.5977};
  forward_odata_ref[5] =  double2{-1.41769,-12.5823};
  forward_odata_ref[6] =  double2{7.53565,-21.5356};
  forward_odata_ref[7] =  double2{42,49};
  forward_odata_ref[8] =  double2{-21.5356,7.53565};
  forward_odata_ref[9] =  double2{-12.5823,-1.41769};
  forward_odata_ref[10] = double2{-8.5977,-5.4023};
  forward_odata_ref[11] = double2{-5.4023,-8.5977};
  forward_odata_ref[12] = double2{-1.41769,-12.5823};
  forward_odata_ref[13] = double2{7.53565,-21.5356};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 14)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 14);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 14);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftPlan1d(&plan_bwd, 7, HIPFFT_Z2Z, 2);
  hipfftExecZ2Z(plan_bwd, data_d, data_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  double2 backward_odata_h[14];
  hipMemcpy(backward_odata_h, data_d, 2 * sizeof(double2) * 7, hipMemcpyDeviceToHost);

  double2 backward_odata_ref[14];
  backward_odata_ref[0] =  double2{0,7};
  backward_odata_ref[1] =  double2{14,21};
  backward_odata_ref[2] =  double2{28,35};
  backward_odata_ref[3] =  double2{42,49};
  backward_odata_ref[4] =  double2{56,63};
  backward_odata_ref[5] =  double2{70,77};
  backward_odata_ref[6] =  double2{84,91};
  backward_odata_ref[7] =  double2{0,7};
  backward_odata_ref[8] =  double2{14,21};
  backward_odata_ref[9] =  double2{28,35};
  backward_odata_ref[10] = double2{42,49};
  backward_odata_ref[11] = double2{56,63};
  backward_odata_ref[12] = double2{70,77};
  backward_odata_ref[13] = double2{84,91};

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 14)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 14);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 14);
    return false;
  }
  return true;
}



#ifdef DEBUG_FFT
int main() {
#define FUNC z2z_1d_inplace
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

