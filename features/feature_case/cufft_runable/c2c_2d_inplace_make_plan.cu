#include "hip/hip_runtime.h"
// ===--- c2c_2d_inplace_make_plan.cu ------------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>

bool c2c_2d_inplace_make_plan() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float2 forward_idata_h[2][5];
  set_value((float*)forward_idata_h, 20);

  float2* data_d;
  hipMalloc(&data_d,sizeof(float2) * 10);
  hipMemcpy(data_d, forward_idata_h, sizeof(float2) * 10, hipMemcpyHostToDevice);

  size_t workSize;
  hipfftMakePlan2d(plan_fwd, 2, 5, HIPFFT_C2C, &workSize);
  hipfftExecC2C(plan_fwd, data_d, data_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[10];
  hipMemcpy(forward_odata_h, data_d, sizeof(float2) * 10, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[10];
  forward_odata_ref[0] =  float2{90,100};
  forward_odata_ref[1] =  float2{-23.7638,3.76382};
  forward_odata_ref[2] =  float2{-13.2492,-6.7508};
  forward_odata_ref[3] =  float2{-6.7508,-13.2492};
  forward_odata_ref[4] =  float2{3.76382,-23.7638};
  forward_odata_ref[5] =  float2{-50,-50};
  forward_odata_ref[6] =  float2{0,0};
  forward_odata_ref[7] =  float2{0,0};
  forward_odata_ref[8] =  float2{0,0};
  forward_odata_ref[9] =  float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 10)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 10);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 10);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlan2d(plan_bwd, 2, 5, HIPFFT_C2C, &workSize);
  hipfftExecC2C(plan_bwd, data_d, data_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float2 backward_odata_h[10];
  hipMemcpy(backward_odata_h, data_d, sizeof(float2) * 10, hipMemcpyDeviceToHost);

  float2 backward_odata_ref[10];
  backward_odata_ref[0] =  float2{0,10};
  backward_odata_ref[1] =  float2{20,30};
  backward_odata_ref[2] =  float2{40,50};
  backward_odata_ref[3] =  float2{60,70};
  backward_odata_ref[4] =  float2{80,90};
  backward_odata_ref[5] =  float2{100,110};
  backward_odata_ref[6] =  float2{120,130};
  backward_odata_ref[7] =  float2{140,150};
  backward_odata_ref[8] =  float2{160,170};
  backward_odata_ref[9] =  float2{180,190};

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 10)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 10);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 10);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC c2c_2d_inplace_make_plan 
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

