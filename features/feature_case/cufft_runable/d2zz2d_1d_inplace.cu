#include "hip/hip_runtime.h"
// ===--- d2zz2d_1d_inplace.cu -------------------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool d2zz2d_1d_inplace() {
  hipfftHandle plan_fwd;
  double forward_idata_h[16];
  set_value(forward_idata_h, 7);
  set_value(forward_idata_h + 8, 7);

  double* data_d;
  hipMalloc(&data_d, sizeof(double) * 16);
  hipMemcpy(data_d, forward_idata_h, sizeof(double) * 16, hipMemcpyHostToDevice);

  hipfftPlan1d(&plan_fwd, 7, HIPFFT_D2Z, 2);
  hipfftExecD2Z(plan_fwd, data_d, (double2*)data_d);
  hipDeviceSynchronize();
  double2 forward_odata_h[8];
  hipMemcpy(forward_odata_h, data_d, sizeof(double) * 16, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[8];
  forward_odata_ref[0] = double2{21,0};
  forward_odata_ref[1] = double2{-3.5,7.26783};
  forward_odata_ref[2] = double2{-3.5,2.79116};
  forward_odata_ref[3] = double2{-3.5,0.798852};
  forward_odata_ref[4] = double2{21,0};
  forward_odata_ref[5] = double2{-3.5,7.26783};
  forward_odata_ref[6] = double2{-3.5,2.79116};
  forward_odata_ref[7] = double2{-3.5,0.798852};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 8)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 8);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 8);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftPlan1d(&plan_bwd, 7, HIPFFT_Z2D, 2);
  hipfftExecZ2D(plan_bwd, (double2*)data_d, data_d);
  hipDeviceSynchronize();
  double backward_odata_h[16];
  hipMemcpy(backward_odata_h, data_d, sizeof(double) * 16, hipMemcpyDeviceToHost);

  double backward_odata_ref[16];
  backward_odata_ref[0] = 0;
  backward_odata_ref[1] = 7;
  backward_odata_ref[2] = 14;
  backward_odata_ref[3] = 21;
  backward_odata_ref[4] = 28;
  backward_odata_ref[5] = 35;
  backward_odata_ref[6] = 42;
  backward_odata_ref[7] = 0.798852;
  backward_odata_ref[8] = 0;
  backward_odata_ref[9] = 7;
  backward_odata_ref[10] = 14;
  backward_odata_ref[11] = 21;
  backward_odata_ref[12] = 28;
  backward_odata_ref[13] = 35;
  backward_odata_ref[14] = 42;
  backward_odata_ref[15] = 0.798852;

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  std::vector<int> indices = {0, 1, 2,  3,  4,  5,  6,
                              8, 9, 10, 11, 12, 13, 14};
  if (!compare(backward_odata_ref, backward_odata_h, indices)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC d2zz2d_1d_inplace
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

