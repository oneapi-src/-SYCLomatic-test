#include "hip/hip_runtime.h"
// ===--- r2cc2r_1d_outofplace_make_plan.cu ------------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


bool r2cc2r_1d_outofplace_make_plan() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float forward_idata_h[14];
  set_value(forward_idata_h, 7);
  set_value(forward_idata_h + 7, 7);

  float* forward_idata_d;
  float2* forward_odata_d;
  float* backward_odata_d;
  hipMalloc(&forward_idata_d, 2 * sizeof(float) * 7);
  hipMalloc(&forward_odata_d, 2 * sizeof(float2) * (7/2+1));
  hipMalloc(&backward_odata_d, 2 * sizeof(float) * 7);
  hipMemcpy(forward_idata_d, forward_idata_h, 2 * sizeof(float) * 7, hipMemcpyHostToDevice);

  size_t workSize;
  hipfftMakePlan1d(plan_fwd, 7, HIPFFT_R2C, 2, &workSize);
  hipfftExecR2C(plan_fwd, forward_idata_d, forward_odata_d);
  hipDeviceSynchronize();
  float2 forward_odata_h[8];
  hipMemcpy(forward_odata_h, forward_odata_d, 2 * sizeof(float2) * (7/2+1), hipMemcpyDeviceToHost);

  float2 forward_odata_ref[8];
  forward_odata_ref[0] = float2{21,0};
  forward_odata_ref[1] = float2{-3.5,7.26783};
  forward_odata_ref[2] = float2{-3.5,2.79116};
  forward_odata_ref[3] = float2{-3.5,0.798852};
  forward_odata_ref[4] = float2{21,0};
  forward_odata_ref[5] = float2{-3.5,7.26783};
  forward_odata_ref[6] = float2{-3.5,2.79116};
  forward_odata_ref[7] = float2{-3.5,0.798852};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 8)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 8);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 8);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlan1d(plan_bwd, 7, HIPFFT_C2R, 2, &workSize);
  hipfftExecC2R(plan_bwd, forward_odata_d, backward_odata_d);
  hipDeviceSynchronize();
  float backward_odata_h[14];
  hipMemcpy(backward_odata_h, backward_odata_d, 2 * sizeof(float) * 7, hipMemcpyDeviceToHost);

  float backward_odata_ref[14];
  backward_odata_ref[0] = 0;
  backward_odata_ref[1] = 7;
  backward_odata_ref[2] = 14;
  backward_odata_ref[3] = 21;
  backward_odata_ref[4] = 28;
  backward_odata_ref[5] = 35;
  backward_odata_ref[6] = 42;
  backward_odata_ref[7] = 0;
  backward_odata_ref[8] = 7;
  backward_odata_ref[9] = 14;
  backward_odata_ref[10] = 21;
  backward_odata_ref[11] = 28;
  backward_odata_ref[12] = 35;
  backward_odata_ref[13] = 42;

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  if (!compare(backward_odata_ref, backward_odata_h, 14)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, 14);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, 14);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC r2cc2r_1d_outofplace_make_plan
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

