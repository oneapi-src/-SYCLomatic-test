#include "hip/hip_runtime.h"
// ===--- d2zz2d_many_1d_inplace_advanced.cu -----------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


// forward
// input
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+
// | r | 0 | r | 0 | r | 0 | r | 0 | r | 0 | 0 | 0 | 0 | 0 | 0 | 0 | 0 | 0 | r | 0 | r | 0 | r | 0 | r | 0 | r | 0 | 0 | 0 | 0 | 0 | 0 | 0 | 0 | 0 |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+
// |___________________n___________________|                               |___________________n___________________|                               |
// |_________________nembed________________|                               |_________________nembed________________|                               |
// |___________________________________batch0______________________________|___________________________________batch1______________________________|
// output
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+
// |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |   0   |   c   |   0   |   c   |   0   |   c   |   0   |   0   |   0   |   0   |
// +---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+---+
// |________________________n______________________|               |       |________________________n______________________|               |       |
// |_____________________________nembed____________________________|       |_____________________________nembed____________________________|       |
// |___________________________________batch0______________________________|___________________________________batch1______________________________|
bool d2zz2d_many_1d_inplace_advanced() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  double forward_idata_h[36];
  std::memset(forward_idata_h, 0, sizeof(double) * 36);
  forward_idata_h[0] = 0;
  forward_idata_h[2] = 1;
  forward_idata_h[4] = 2;
  forward_idata_h[6] = 3;
  forward_idata_h[8] = 4;
  forward_idata_h[18] = 0;
  forward_idata_h[20] = 1;
  forward_idata_h[22] = 2;
  forward_idata_h[24] = 3;
  forward_idata_h[26] = 4;

  double* data_d;
  hipMalloc(&data_d, sizeof(double) * 36);
  hipMemcpy(data_d, forward_idata_h, sizeof(double) * 36, hipMemcpyHostToDevice);

  size_t workSize;
  long long int n[1] = {5};
  long long int inembed[1] = {5};
  long long int onembed[1] = {4};
  hipfftMakePlanMany64(plan_fwd, 1, n, inembed, 2, 18, onembed, 2, 9, HIPFFT_D2Z, 2, &workSize);
  hipfftExecD2Z(plan_fwd, data_d, (double2*)data_d);
  hipDeviceSynchronize();
  double2 forward_odata_h[18];
  hipMemcpy(forward_odata_h, data_d, sizeof(double) * 36, hipMemcpyDeviceToHost);

  double2 forward_odata_ref[18];
  forward_odata_ref[0] =  double2{10,0};
  forward_odata_ref[1] =  double2{2,3};
  forward_odata_ref[2] =  double2{-2.5,3.44095};
  forward_odata_ref[3] =  double2{1,2};
  forward_odata_ref[4] =  double2{-2.5,0.812299};
  forward_odata_ref[5] =  double2{0,0};
  forward_odata_ref[6] =  double2{0,0};
  forward_odata_ref[7] =  double2{0,0};
  forward_odata_ref[8] =  double2{0,0};
  forward_odata_ref[9] =  double2{10,0};
  forward_odata_ref[10] = double2{0,0};
  forward_odata_ref[11] = double2{-2.5,3.44095};
  forward_odata_ref[12] = double2{0,0};
  forward_odata_ref[13] = double2{-2.5,0.812299};
  forward_odata_ref[14] = double2{0,0};
  forward_odata_ref[15] = double2{0,0};
  forward_odata_ref[16] = double2{0,0};
  forward_odata_ref[17] = double2{0,0};

  hipfftDestroy(plan_fwd);

  std::vector<int> indices = {0, 2, 4,
                              9, 11, 13};
  if (!compare(forward_odata_ref, forward_odata_h, indices)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, indices);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, indices);

    hipFree(data_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftMakePlanMany64(plan_bwd, 1, n, onembed, 2, 9, inembed, 2, 18, HIPFFT_Z2D, 2, &workSize);
  hipfftExecZ2D(plan_bwd, (double2*)data_d, data_d);
  hipDeviceSynchronize();
  double backward_odata_h[36];
  hipMemcpy(backward_odata_h, data_d, sizeof(double) * 36, hipMemcpyDeviceToHost);

  double backward_odata_ref[36];
  backward_odata_ref[0] = 0;
  backward_odata_ref[2] = 5;
  backward_odata_ref[4] = 10;
  backward_odata_ref[6] = 15;
  backward_odata_ref[8] = 20;
  backward_odata_ref[18] = 0;
  backward_odata_ref[20] = 5;
  backward_odata_ref[22] = 10;
  backward_odata_ref[24] = 15;
  backward_odata_ref[26] = 20;

  hipFree(data_d);
  hipfftDestroy(plan_bwd);

  std::vector<int> indices_bwd = {0, 2, 4, 6, 8,
                                  18, 20, 22, 24, 26};
  if (!compare(backward_odata_ref, backward_odata_h, indices_bwd)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices_bwd);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices_bwd);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC d2zz2d_many_1d_inplace_advanced
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

