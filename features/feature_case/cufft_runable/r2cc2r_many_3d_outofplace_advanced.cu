#include "hip/hip_runtime.h"
// ===--- r2cc2r_many_3d_outofplace_advanced.cu --------------*- CUDA -*---===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===---------------------------------------------------------------------===//

#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"
#include "common.h"
#include <cstring>
#include <iostream>


// forward
// input
// +---+---+---+---+---+---+         -+ -+          ---+-------+     -----------+
// | r |   | r |   | r |   |          |  |             |       |                |
// +---+---+---+---+---+---+          n2 |             |       |                |
// | r |   | r |   | r |   |          |  nembed2       |       |                |
// +---+---+---+---+---+---+         ----+             |       |                |
// | r |   | r |   | r |   |          |  |             |       |                |
// +---+---+---+---+---+---+          n2 |             |       |                |
// | r |   | r |   | r |   |          |  nembed2       |       |                |
// +---+---+---+---+---+---+         ----+             n1      |                a batch
// | r |   | r |   | r |   |          |  |             |       |                |
// +---+---+---+---+---+---+          n2 |             |    nembed1             |
// | r |   | r |   | r |   |          |  nembed2       |       |                |
// +---+---+---+---+---+---+         ----+             |       |                |
// | r |   | r |   | r |   |          |  |             |       |                |
// +---+---+---+---+---+---+          n2 |             |       |                |
// | r |   | r |   | r |   |          |  nembed2       |       |                |
// +---+---+---+---+---+---+         -+--+          ---+-------+     -----------+
// |__________n3___________|
// |________nembed3________|
// output
// +---+---+---+---+ -+          ---+
// |   c   |   c   |  |             |
// +---+---+---+---+  n2/nembed2    |
// |   c   |   c   |  |             |
// +---+---+---+---+ -+             |
// |   c   |   c   |  |             |
// +---+---+---+---+  n2/nembed2    |
// |   c   |   c   |  |             |
// +---+---+---+---+ -+        n1/nembed1/a batch
// |   c   |   c   |  |             |
// +---+---+---+---+  n2/nembed2    |
// |   c   |   c   |  |             |
// +---+---+---+---+ -+             |
// |   c   |   c   |  |             |
// +---+---+---+---+  n2/nembed2    |
// |   c   |   c   |  |             |
// +---+---+---+---+ -+          ---+
// |______n3_______|
// |____nembed3____|
bool r2cc2r_many_3d_outofplace_advanced() {
  hipfftHandle plan_fwd;
  hipfftCreate(&plan_fwd);
  float forward_idata_h[96];
  std::memset(forward_idata_h, 0, sizeof(float) * 96);
  forward_idata_h[0]  = 0;
  forward_idata_h[2]  = 1;
  forward_idata_h[4]  = 2;
  forward_idata_h[6]  = 3;
  forward_idata_h[8]  = 4;
  forward_idata_h[10] = 5;
  forward_idata_h[12] = 6;
  forward_idata_h[14] = 7;
  forward_idata_h[16] = 8;
  forward_idata_h[18] = 9;
  forward_idata_h[20] = 10;
  forward_idata_h[22] = 11;
  forward_idata_h[24] = 12;
  forward_idata_h[26] = 13;
  forward_idata_h[28] = 14;
  forward_idata_h[30] = 15;
  forward_idata_h[32] = 16;
  forward_idata_h[34] = 17;
  forward_idata_h[36] = 18;
  forward_idata_h[38] = 19;
  forward_idata_h[40] = 20;
  forward_idata_h[42] = 21;
  forward_idata_h[44] = 22;
  forward_idata_h[46] = 23;
  std::memcpy(forward_idata_h + 48, forward_idata_h, sizeof(float) * 48);

  float* forward_idata_d;
  float2* forward_odata_d;
  float* backward_odata_d;
  hipMalloc(&forward_idata_d, sizeof(float) * 96);
  hipMalloc(&forward_odata_d, sizeof(float2) * 32);
  hipMalloc(&backward_odata_d, sizeof(float) * 96);
  hipMemcpy(forward_idata_d, forward_idata_h, sizeof(float) * 96, hipMemcpyHostToDevice);

  long long int n[3] = {4, 2, 3};
  long long int inembed[3] = {4, 2, 3};
  long long int onembed[3] = {4, 2, 2};
  size_t workSize;
  hipfftXtMakePlanMany(plan_fwd, 3, n, inembed, 2, 48, HIP_R_32F, onembed, 1, 16, HIP_C_32F, 2, &workSize, HIP_C_32F);
  hipfftXtExec(plan_fwd, forward_idata_d, forward_odata_d, HIPFFT_FORWARD);
  hipDeviceSynchronize();
  float2 forward_odata_h[32];
  hipMemcpy(forward_odata_h, forward_odata_d, sizeof(float2) * 32, hipMemcpyDeviceToHost);

  float2 forward_odata_ref[32];
  forward_odata_ref[0] =  float2{276,0};
  forward_odata_ref[1] =  float2{-12,6.9282};
  forward_odata_ref[2] =  float2{-36,0};
  forward_odata_ref[3] =  float2{0,0};
  forward_odata_ref[4] =  float2{-72,72};
  forward_odata_ref[5] =  float2{0,0};
  forward_odata_ref[6] =  float2{0,0};
  forward_odata_ref[7] =  float2{0,0};
  forward_odata_ref[8] =  float2{-72,0};
  forward_odata_ref[9] =  float2{0,0};
  forward_odata_ref[10] = float2{0,0};
  forward_odata_ref[11] = float2{0,0};
  forward_odata_ref[12] = float2{-72,-72};
  forward_odata_ref[13] = float2{0,0};
  forward_odata_ref[14] = float2{0,0};
  forward_odata_ref[15] = float2{0,0};
  forward_odata_ref[16] = float2{276,0};
  forward_odata_ref[17] = float2{-12,6.9282};
  forward_odata_ref[18] = float2{-36,0};
  forward_odata_ref[19] = float2{0,0};
  forward_odata_ref[20] = float2{-72,72};
  forward_odata_ref[21] = float2{0,0};
  forward_odata_ref[22] = float2{0,0};
  forward_odata_ref[23] = float2{0,0};
  forward_odata_ref[24] = float2{-72,0};
  forward_odata_ref[25] = float2{0,0};
  forward_odata_ref[26] = float2{0,0};
  forward_odata_ref[27] = float2{0,0};
  forward_odata_ref[28] = float2{-72,-72};
  forward_odata_ref[29] = float2{0,0};
  forward_odata_ref[30] = float2{0,0};
  forward_odata_ref[31] = float2{0,0};

  hipfftDestroy(plan_fwd);

  if (!compare(forward_odata_ref, forward_odata_h, 32)) {
    std::cout << "forward_odata_h:" << std::endl;
    print_values(forward_odata_h, 32);
    std::cout << "forward_odata_ref:" << std::endl;
    print_values(forward_odata_ref, 32);

    hipFree(forward_idata_d);
    hipFree(forward_odata_d);
    hipFree(backward_odata_d);

    return false;
  }

  hipfftHandle plan_bwd;
  hipfftCreate(&plan_bwd);
  hipfftXtMakePlanMany(plan_bwd, 3, n, onembed, 1, 16, HIP_C_32F, inembed, 2, 48, HIP_R_32F, 2, &workSize, HIP_C_32F);
  hipfftXtExec(plan_bwd, forward_odata_d, backward_odata_d, HIPFFT_BACKWARD);
  hipDeviceSynchronize();
  float backward_odata_h[96];
  hipMemcpy(backward_odata_h, backward_odata_d, sizeof(float) * 96, hipMemcpyDeviceToHost);

  float backward_odata_ref[96];
  backward_odata_ref[0] = 0;
  backward_odata_ref[2] = 24;
  backward_odata_ref[4] = 48;
  backward_odata_ref[6] = 72;
  backward_odata_ref[8] = 96;
  backward_odata_ref[10] = 120;
  backward_odata_ref[12] = 144;
  backward_odata_ref[14] = 168;
  backward_odata_ref[16] = 192;
  backward_odata_ref[18] = 216;
  backward_odata_ref[20] = 240;
  backward_odata_ref[22] = 264;
  backward_odata_ref[24] = 288;
  backward_odata_ref[26] = 312;
  backward_odata_ref[28] = 336;
  backward_odata_ref[30] = 360;
  backward_odata_ref[32] = 384;
  backward_odata_ref[34] = 408;
  backward_odata_ref[36] = 432;
  backward_odata_ref[38] = 456;
  backward_odata_ref[40] = 480;
  backward_odata_ref[42] = 504;
  backward_odata_ref[44] = 528;
  backward_odata_ref[46] = 552;
  std::memcpy(backward_odata_ref + 48, backward_odata_ref, sizeof(float) * 48);

  hipFree(forward_idata_d);
  hipFree(forward_odata_d);
  hipFree(backward_odata_d);

  hipfftDestroy(plan_bwd);

  std::vector<int> indices = {0, 2, 4, 6, 8,
                              10, 12, 14, 16, 18,
                              20, 12, 14, 16, 18,
                              30, 22, 24, 26, 28,
                              40, 32, 34, 36, 38,
                              50, 42, 44, 46, 48,
                              60, 52, 54, 56, 58,
                              70, 62, 64, 66, 68,
                              80, 72, 74, 76, 78,
                              90, 82, 84, 86, 88,
                              10, 92, 94};
  if (!compare(backward_odata_ref, backward_odata_h, indices)) {
    std::cout << "backward_odata_h:" << std::endl;
    print_values(backward_odata_h, indices);
    std::cout << "backward_odata_ref:" << std::endl;
    print_values(backward_odata_ref, indices);
    return false;
  }
  return true;
}


#ifdef DEBUG_FFT
int main() {
#define FUNC r2cc2r_many_3d_outofplace_advanced
  bool res = FUNC();
  hipDeviceSynchronize();
  if (!res) {
    std::cout << "Fail" << std::endl;
    return -1;
  }
  std::cout << "Pass" << std::endl;
  return 0;
}
#endif

