// ====------ device_info.cu---------- *- CUDA -* -----------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===--------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

void test0() {
  // no need to use `cudaSetDevice`
  size_t free_mem, total_mem;
  hipMemGetInfo(&free_mem, &total_mem);

  printf("total_mem : [%lu]\n", total_mem);
  printf("free_mem  : [%lu]\n", free_mem);
}

void test1() {
  hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, 0);

  const int id = properties.pciDeviceID;
  const hipUUID uuid = properties.uuid;

  properties.pciDeviceID = id;
  properties.uuid = uuid;

  std::cout << "Device ID: " << id << std::endl;
  std::cout << "Device UUID: ";
  for (int i = 0; i < 16; i++) {
    std::cout << std::hex
              << static_cast<int>(static_cast<unsigned char>(uuid.bytes[i]))
              << " ";
  }
  std::cout << std::endl;
}

int main() {
  test0();
  test1();
}
