#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  hipError_t e;
  const char *err_s;
  hipDrvGetErrorString(e, &err_s);
  std::cout << err_s << std::endl;
  return 0;
}
