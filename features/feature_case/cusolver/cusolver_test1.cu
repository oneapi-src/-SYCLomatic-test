// ===------ cusolver_test1.cu ------------------------------*- CUDA -*-----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsolver.h"

#include <cmath>
#include <vector>
#include <cstdio>
#include <complex>

template<class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float* input_data, int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num, hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num, hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }
private:
  inline void from_float_convert(float* in, d_data_t* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t* in, float* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float* in, float2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float* in, double2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float* expect, float* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i]-expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float* expect, float* result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]]-expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

void test_cusolverDnTsygvd() {
  std::vector<float> a = {1, 2, 3, 2, 1, 2, 3, 2, 1};
  std::vector<float> b = {2, -1, 0, -1, 2, -1, 0, -1, 2};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float> b_s(b.data(), 9);
  Data<double> b_d(b.data(), 9);
  Data<float> w_s(3);
  Data<double> w_d(3);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  b_s.H2D();
  b_d.H2D();

  int lwork_s;
  int lwork_d;
  hipsolverDnSsygvd_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s.d_data, 3, b_s.d_data, 3, w_s.d_data, &lwork_s);
  hipsolverDnDsygvd_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_d.d_data, 3, b_d.d_data, 3, w_d.d_data, &lwork_d);

  float* work_s;
  double* work_d;
  int *devInfo;
  hipMalloc(&work_s, sizeof(float) * lwork_s);
  hipMalloc(&work_d, sizeof(double) * lwork_d);
  hipMalloc(&devInfo, sizeof(int));

  hipsolverDnSsygvd(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s.d_data, 3, b_s.d_data, 3, w_s.d_data, work_s, lwork_s, devInfo);
  hipsolverDnDsygvd(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_d.d_data, 3, b_d.d_data, 3, w_d.d_data, work_d, lwork_d, devInfo);

  a_s.D2H();
  a_d.D2H();
  b_s.D2H();
  b_d.D2H();
  w_s.D2H();
  w_d.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroy(handle);
  hipFree(work_s);
  hipFree(work_d);
  hipFree(devInfo);

  float expect_a[9] = {0.500000,-0.000000,-0.500000,0.194937,-0.484769,0.194937,0.679705,0.874642,0.679705};
  float expect_b[9] = {1.414214,-1.000000,0.000000,-0.707107,1.224745,-1.000000,0.000000,-0.816497,1.154701};
  float expect_w[3] = {-1.000000,-0.216991,9.216990};
  if (compare_result(expect_a, a_s.h_data, 9)
      && compare_result(expect_b, b_s.h_data, 9)
      && compare_result(expect_w, w_s.h_data, 3)
      && compare_result(expect_a, a_d.h_data, 9)
      && compare_result(expect_b, b_d.h_data, 9)
      && compare_result(expect_w, w_d.h_data, 3))
    printf("DnTsygvd pass\n");
  else {
    printf("DnTsygvd fail\n");
    test_passed = false;
  }
}

void test_cusolverDnThegvd() {
  std::vector<float> a = {1, 2, 3, 2, 1, 2, 3, 2, 1};
  std::vector<float> b = {2, -1, 0, -1, 2, -1, 0, -1, 2};
  Data<float2> a_s(a.data(), 9);
  Data<double2> a_d(a.data(), 9);
  Data<float2> b_s(b.data(), 9);
  Data<double2> b_d(b.data(), 9);
  Data<float> w_s(3);
  Data<double> w_d(3);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  b_s.H2D();
  b_d.H2D();

  int lwork_s;
  int lwork_d;
  hipsolverDnChegvd_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s.d_data, 3, b_s.d_data, 3, w_s.d_data, &lwork_s);
  hipsolverDnZhegvd_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_d.d_data, 3, b_d.d_data, 3, w_d.d_data, &lwork_d);

  float2* work_s;
  double2* work_d;
  int *devInfo;
  hipMalloc(&work_s, sizeof(float2) * lwork_s);
  hipMalloc(&work_d, sizeof(double2) * lwork_d);
  hipMalloc(&devInfo, sizeof(int));

  hipsolverDnChegvd(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s.d_data, 3, b_s.d_data, 3, w_s.d_data, work_s, lwork_s, devInfo);
  hipsolverDnZhegvd(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_d.d_data, 3, b_d.d_data, 3, w_d.d_data, work_d, lwork_d, devInfo);

  a_s.D2H();
  a_d.D2H();
  b_s.D2H();
  b_d.D2H();
  w_s.D2H();
  w_d.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroy(handle);
  hipFree(work_s);
  hipFree(work_d);
  hipFree(devInfo);

  float expect_a[9] = {0.500000,-0.000000,-0.500000,0.194937,-0.484769,0.194937,0.679705,0.874642,0.679705};
  float expect_b[9] = {1.414214,-1.000000,0.000000,-0.707107,1.224745,-1.000000,0.000000,-0.816497,1.154701};
  float expect_w[3] = {-1.000000,-0.216991,9.216990};
  if (compare_result(expect_a, a_s.h_data, 9)
      && compare_result(expect_b, b_s.h_data, 9)
      && compare_result(expect_w, w_s.h_data, 3)
      && compare_result(expect_a, a_d.h_data, 9)
      && compare_result(expect_b, b_d.h_data, 9)
      && compare_result(expect_w, w_d.h_data, 3))
    printf("DnThegvd pass\n");
  else {
    printf("DnThegvd fail\n");
    test_passed = false;
  }
}

void test_helper() {
  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);
  hipStream_t stream;
  hipsolverGetStream(handle, &stream);
  hipsolverSetStream(handle, stream);
}

void test_cusolverDnTsyheevd() {
  std::vector<float> a = {1, 2, 2, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  Data<float> w_s(2);
  Data<double> w_d(2);
  Data<float> w_c(2);
  Data<double> w_z(2);

  hipsolverHandle_t handle;
  int s = hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  int lwork_s;
  int lwork_d;
  int lwork_c;
  int lwork_z;
  hipsolverDnSsyevd_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_s.d_data, 2, w_s.d_data, &lwork_s);
  hipsolverDnDsyevd_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_d.d_data, 2, w_d.d_data, &lwork_d);
  hipsolverDnCheevd_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_c.d_data, 2, w_c.d_data, &lwork_c);
  hipsolverDnZheevd_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_z.d_data, 2, w_z.d_data, &lwork_z);

  float* work_s;
  double* work_d;
  float2* work_c;
  double2* work_z;
  int *devInfo;
  hipMalloc(&work_s, sizeof(float) * lwork_s);
  hipMalloc(&work_d, sizeof(double) * lwork_d);
  hipMalloc(&work_c, sizeof(float2) * lwork_c);
  hipMalloc(&work_z, sizeof(double2) * lwork_z);
  hipMalloc(&devInfo, sizeof(int));

  hipsolverDnSsyevd(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_s.d_data, 2, w_s.d_data, work_s, lwork_s, devInfo);
  hipsolverDnDsyevd(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_d.d_data, 2, w_d.d_data, work_d, lwork_d, devInfo);
  hipsolverDnCheevd(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_c.d_data, 2, w_c.d_data, work_c, lwork_c, devInfo);
  hipsolverDnZheevd(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_z.d_data, 2, w_z.d_data, work_z, lwork_z, devInfo);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();
  w_s.D2H();
  w_d.D2H();
  w_c.D2H();
  w_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroy(handle);
  hipFree(work_s);
  hipFree(work_d);
  hipFree(work_c);
  hipFree(work_z);
  hipFree(devInfo);

  float expect_a[4] = {-0.894427,0.447214,0.447214,0.894427};
  float expect_w[2] = {0.000000,5.000000};
  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(expect_w, w_s.h_data, 2) &&
      compare_result(expect_w, w_d.h_data, 2) &&
      compare_result(expect_w, w_c.h_data, 2) &&
      compare_result(expect_w, w_z.h_data, 2))
    printf("DnTsyheevd pass\n");
  else {
    printf("DnTsyheevd fail\n");
    test_passed = false;
  }
}

int main() {
  test_helper();
  test_cusolverDnTsygvd();
  test_cusolverDnThegvd();
  test_cusolverDnTsyheevd();

  if (test_passed)
    return 0;
  return -1;
}
