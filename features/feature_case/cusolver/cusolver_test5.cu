// ===------ cusolver_test5.cu ------------------------------*- CUDA -*-----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsolver.h"

#include <cmath>
#include <vector>
#include <cstdio>
#include <complex>

template<class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float* input_data, int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num, hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num, hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }
private:
  inline void from_float_convert(float* in, d_data_t* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t* in, float* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float* in, float2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float* in, double2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float* expect, float* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i]-expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(int* expect, int* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (result[i] != expect[i]) {
      return false;
    }
  }
  return true;
}

bool compare_result(int64_t* expect, int64_t* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (result[i] != expect[i]) {
      return false;
    }
  }
  return true;
}

bool compare_result(float* expect, float* result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]]-expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

void test_cusolverDnXtrtri() {
  std::vector<float> a = {1, 2, 2, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t lwork_s;
  size_t lwork_d;
  size_t lwork_c;
  size_t lwork_z;
  size_t lwork_host_s;
  size_t lwork_host_d;
  size_t lwork_host_c;
  size_t lwork_host_z;

  cusolverDnXtrtri_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_R_32F, a_s.d_data, 2, &lwork_s, &lwork_host_s);
  cusolverDnXtrtri_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_R_64F, a_d.d_data, 2, &lwork_d, &lwork_host_d);
  cusolverDnXtrtri_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_C_32F, a_c.d_data, 2, &lwork_c, &lwork_host_c);
  cusolverDnXtrtri_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_C_64F, a_z.d_data, 2, &lwork_z, &lwork_host_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  hipMalloc(&device_ws_s, lwork_s);
  hipMalloc(&device_ws_d, lwork_d);
  hipMalloc(&device_ws_c, lwork_c);
  hipMalloc(&device_ws_z, lwork_z);
  void* host_ws_s;
  void* host_ws_d;
  void* host_ws_c;
  void* host_ws_z;
  host_ws_s = malloc(lwork_host_s);
  host_ws_d = malloc(lwork_host_d);
  host_ws_c = malloc(lwork_host_c);
  host_ws_z = malloc(lwork_host_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnXtrtri(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_R_32F, a_s.d_data, 2, device_ws_s, lwork_s, host_ws_s, lwork_host_s, info);
  cusolverDnXtrtri(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_R_64F, a_d.d_data, 2, device_ws_d, lwork_d, host_ws_d, lwork_host_d, info);
  cusolverDnXtrtri(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_C_32F, a_c.d_data, 2, device_ws_c, lwork_c, host_ws_c, lwork_host_c, info);
  cusolverDnXtrtri(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_DIAG_NON_UNIT, 2, HIP_C_64F, a_z.d_data, 2, device_ws_z, lwork_z, host_ws_z, lwork_host_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  free(host_ws_s);
  free(host_ws_d);
  free(host_ws_c);
  free(host_ws_z);
  hipFree(info);

  float expect_a[4] = {1.000000,2.000000,-0.500000,0.250000};
  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4))
    printf("DnXtrtri pass\n");
  else {
    printf("DnXtrtri fail\n");
    test_passed = false;
  }
}

int main() {
#ifndef DPCT_USM_LEVEL_NONE
  test_cusolverDnXtrtri();
#endif

  if (test_passed)
    return 0;
  return -1;
}
