#include "hip/hip_runtime.h"
// ===------ cusolver_test4.cu ------------------------------*- CUDA -*-----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//

#include "hipsolver.h"

#include <cmath>
#include <vector>
#include <cstdio>
#include <complex>

template<class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float* input_data, int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num, hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num, hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }
private:
  inline void from_float_convert(float* in, d_data_t* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t* in, float* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float* in, float2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float* in, double2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float* expect, float* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i]-expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float* expect, float* result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]]-expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

void test_cusolverDnXgesvd() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);

  Data<float> s_s(2);
  Data<double> s_d(2);
  Data<float> s_c(2);
  Data<double> s_z(2);

  Data<float> u_s(4);
  Data<double> u_d(4);
  Data<float2> u_c(4);
  Data<double2> u_z(4);

  Data<float> vt_s(4);
  Data<double> vt_d(4);
  Data<float2> vt_c(4);
  Data<double2> vt_z(4);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;
  size_t host_ws_size_s;
  size_t host_ws_size_d;
  size_t host_ws_size_c;
  size_t host_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  cusolverDnXgesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, s_s.d_data, HIP_R_32F, u_s.d_data, 2, HIP_R_32F, vt_s.d_data, 2, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  cusolverDnXgesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, s_d.d_data, HIP_R_64F, u_d.d_data, 2, HIP_R_64F, vt_d.d_data, 2, HIP_R_64F, &device_ws_size_d, &host_ws_size_d);
  cusolverDnXgesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_R_32F, s_c.d_data, HIP_C_32F, u_c.d_data, 2, HIP_C_32F, vt_c.d_data, 2, HIP_C_32F, &device_ws_size_c, &host_ws_size_c);
  cusolverDnXgesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_R_64F, s_z.d_data, HIP_C_64F, u_z.d_data, 2, HIP_C_64F, vt_z.d_data, 2, HIP_C_64F, &device_ws_size_z, &host_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  void* host_ws_s;
  void* host_ws_d;
  void* host_ws_c;
  void* host_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);
  hipMalloc(&host_ws_s, host_ws_size_s);
  hipMalloc(&host_ws_d, host_ws_size_d);
  hipMalloc(&host_ws_c, host_ws_size_c);
  hipMalloc(&host_ws_z, host_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, s_s.d_data, HIP_R_32F, u_s.d_data, 2, HIP_R_32F, vt_s.d_data, 2, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, s_d.d_data, HIP_R_64F, u_d.d_data, 2, HIP_R_64F, vt_d.d_data, 2, HIP_R_64F, device_ws_d, device_ws_size_d, host_ws_d, host_ws_size_d, info);
  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_R_32F, s_c.d_data, HIP_C_32F, u_c.d_data, 2, HIP_C_32F, vt_c.d_data, 2, HIP_C_32F, device_ws_c, device_ws_size_c, host_ws_c, host_ws_size_c, info);
  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_R_64F, s_z.d_data, HIP_C_64F, u_z.d_data, 2, HIP_C_64F, vt_z.d_data, 2, HIP_C_64F, device_ws_z, device_ws_size_z, host_ws_z, host_ws_size_z, info);

  s_s.D2H();
  s_d.D2H();
  s_c.D2H();
  s_z.D2H();

  u_s.D2H();
  u_d.D2H();
  u_c.D2H();
  u_z.D2H();

  vt_s.D2H();
  vt_d.D2H();
  vt_c.D2H();
  vt_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(host_ws_s);
  hipFree(host_ws_d);
  hipFree(host_ws_c);
  hipFree(host_ws_z);
  hipFree(info);

  float expect_s[4] = {5.464985,0.365966};
  float expect_u[4] = {0.576048,0.817416,-0.817416,0.576048};
  float expect_vt[4] = {0.404554,0.914514,0.914514,-0.404554};

  if (compare_result(expect_s, s_s.h_data, 2) &&
      compare_result(expect_s, s_d.h_data, 2) &&
      compare_result(expect_s, s_c.h_data, 2) &&
      compare_result(expect_s, s_z.h_data, 2) &&
      compare_result(expect_u, u_s.h_data, 4) &&
      compare_result(expect_u, u_d.h_data, 4) &&
      compare_result(expect_u, u_c.h_data, 4) &&
      compare_result(expect_u, u_z.h_data, 4) &&
      compare_result(expect_vt, vt_s.h_data, 4) &&
      compare_result(expect_vt, vt_d.h_data, 4) &&
      compare_result(expect_vt, vt_c.h_data, 4) &&
      compare_result(expect_vt, vt_z.h_data, 4))
    printf("DnXgesvd pass\n");
  else {
    printf("DnXgesvd fail\n");
    test_passed = false;
  }
}

void test_cusolverDnGesvd() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);

  Data<float> s_s(2);
  Data<double> s_d(2);
  Data<float> s_c(2);
  Data<double> s_z(2);

  Data<float> u_s(4);
  Data<double> u_d(4);
  Data<float2> u_c(4);
  Data<double2> u_z(4);

  Data<float> vt_s(4);
  Data<double> vt_d(4);
  Data<float2> vt_c(4);
  Data<double2> vt_z(4);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  cusolverDnGesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, s_s.d_data, HIP_R_32F, u_s.d_data, 2, HIP_R_32F, vt_s.d_data, 2, HIP_R_32F, &device_ws_size_s);
  cusolverDnGesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, s_d.d_data, HIP_R_64F, u_d.d_data, 2, HIP_R_64F, vt_d.d_data, 2, HIP_R_64F, &device_ws_size_d);
  cusolverDnGesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_R_32F, s_c.d_data, HIP_C_32F, u_c.d_data, 2, HIP_C_32F, vt_c.d_data, 2, HIP_C_32F, &device_ws_size_c);
  cusolverDnGesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_R_64F, s_z.d_data, HIP_C_64F, u_z.d_data, 2, HIP_C_64F, vt_z.d_data, 2, HIP_C_64F, &device_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnGesvd(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, s_s.d_data, HIP_R_32F, u_s.d_data, 2, HIP_R_32F, vt_s.d_data, 2, HIP_R_32F, device_ws_s, device_ws_size_s, info);
  cusolverDnGesvd(handle, params, 'A', 'A', 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, s_d.d_data, HIP_R_64F, u_d.d_data, 2, HIP_R_64F, vt_d.d_data, 2, HIP_R_64F, device_ws_d, device_ws_size_d, info);
  cusolverDnGesvd(handle, params, 'A', 'A', 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_R_32F, s_c.d_data, HIP_C_32F, u_c.d_data, 2, HIP_C_32F, vt_c.d_data, 2, HIP_C_32F, device_ws_c, device_ws_size_c, info);
  cusolverDnGesvd(handle, params, 'A', 'A', 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_R_64F, s_z.d_data, HIP_C_64F, u_z.d_data, 2, HIP_C_64F, vt_z.d_data, 2, HIP_C_64F, device_ws_z, device_ws_size_z, info);

  s_s.D2H();
  s_d.D2H();
  s_c.D2H();
  s_z.D2H();

  u_s.D2H();
  u_d.D2H();
  u_c.D2H();
  u_z.D2H();

  vt_s.D2H();
  vt_d.D2H();
  vt_c.D2H();
  vt_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_s[4] = {5.464985,0.365966};
  float expect_u[4] = {0.576048,0.817416,-0.817416,0.576048};
  float expect_vt[4] = {0.404554,0.914514,0.914514,-0.404554};

  if (compare_result(expect_s, s_s.h_data, 2) &&
      compare_result(expect_s, s_d.h_data, 2) &&
      compare_result(expect_s, s_c.h_data, 2) &&
      compare_result(expect_s, s_z.h_data, 2) &&
      compare_result(expect_u, u_s.h_data, 4) &&
      compare_result(expect_u, u_d.h_data, 4) &&
      compare_result(expect_u, u_c.h_data, 4) &&
      compare_result(expect_u, u_z.h_data, 4) &&
      compare_result(expect_vt, vt_s.h_data, 4) &&
      compare_result(expect_vt, vt_d.h_data, 4) &&
      compare_result(expect_vt, vt_c.h_data, 4) &&
      compare_result(expect_vt, vt_z.h_data, 4))
    printf("DnGesvd pass\n");
  else {
    printf("DnGesvd fail\n");
    test_passed = false;
  }
}

#if 0
void test_cusolverDnTgesvd() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);

  Data<float> s_s(2);
  Data<double> s_d(2);
  Data<float> s_c(2);
  Data<double> s_z(2);

  Data<float> u_s(4);
  Data<double> u_d(4);
  Data<float2> u_c(4);
  Data<double2> u_z(4);

  Data<float> vt_s(4);
  Data<double> vt_d(4);
  Data<float2> vt_c(4);
  Data<double2> vt_z(4);

  Data<float> rwork_s(1);
  Data<double> rwork_d(1);
  Data<float> rwork_c(1);
  Data<double> rwork_z(1);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  int device_ws_size_s;
  int device_ws_size_d;
  int device_ws_size_c;
  int device_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  hipsolverDnSgesvd_bufferSize(handle, 2, 2, &device_ws_size_s);
  hipsolverDnDgesvd_bufferSize(handle, 2, 2, &device_ws_size_d);
  hipsolverDnCgesvd_bufferSize(handle, 2, 2, &device_ws_size_c);
  hipsolverDnZgesvd_bufferSize(handle, 2, 2, &device_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s * sizeof(float));
  hipMalloc(&device_ws_d, device_ws_size_d * sizeof(double));
  hipMalloc(&device_ws_c, device_ws_size_c * sizeof(float2));
  hipMalloc(&device_ws_z, device_ws_size_z * sizeof(double2));

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnSgesvd(handle, 'A', 'A', 2, 2, (float*)a_s.d_data, 2, (float*)s_s.d_data, (float*)u_s.d_data, 2, (float*)vt_s.d_data, 2, (float*)device_ws_s, device_ws_size_s, (float*)rwork_s.d_data, info);
  hipsolverDnDgesvd(handle, 'A', 'A', 2, 2, (double*)a_d.d_data, 2, (double*)s_d.d_data, (double*)u_d.d_data, 2, (double*)vt_d.d_data, 2, (double*)device_ws_d, device_ws_size_d, (double*)rwork_d.d_data, info);
  hipsolverDnCgesvd(handle, 'A', 'A', 2, 2, (float2*)a_c.d_data, 2, (float*)s_c.d_data, (float2*)u_c.d_data, 2, (float2*)vt_c.d_data, 2, (float2*)device_ws_c, device_ws_size_c, (float*)rwork_c.d_data, info);
  hipsolverDnZgesvd(handle, 'A', 'A', 2, 2, (double2*)a_z.d_data, 2, (double*)s_z.d_data, (double2*)u_z.d_data, 2, (double2*)vt_z.d_data, 2, (double2*)device_ws_z, device_ws_size_z, (double*)rwork_z.d_data, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  s_s.D2H();
  s_d.D2H();
  s_c.D2H();
  s_z.D2H();

  u_s.D2H();
  u_d.D2H();
  u_c.D2H();
  u_z.D2H();

  vt_s.D2H();
  vt_d.D2H();
  vt_c.D2H();
  vt_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_a[4] = {-2.236068,0.618034,-4.919349,-0.894427};
  float expect_s[4] = {5.464985,0.365966};
  float expect_u[4] = {-0.576048,-0.817416,-0.817416,0.576048};
  float expect_vt[4] = {-0.404554,0.914514,-0.914514,-0.404554};

  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(expect_s, s_s.h_data, 2) &&
      compare_result(expect_s, s_d.h_data, 2) &&
      compare_result(expect_s, s_c.h_data, 2) &&
      compare_result(expect_s, s_z.h_data, 2) &&
      compare_result(expect_u, u_s.h_data, 4) &&
      compare_result(expect_u, u_d.h_data, 4) &&
      compare_result(expect_u, u_c.h_data, 4) &&
      compare_result(expect_u, u_z.h_data, 4) &&
      compare_result(expect_vt, vt_s.h_data, 4) &&
      compare_result(expect_vt, vt_d.h_data, 4) &&
      compare_result(expect_vt, vt_c.h_data, 4) &&
      compare_result(expect_vt, vt_z.h_data, 4))
    printf("DnTgesvd pass\n");
  else {
    printf("DnTgesvd fail\n");
    test_passed = false;
  }
}
#endif

void test_cusolverDnXpotrf() {
  std::vector<float> a = {2, -1, 0, -1, 2, -1, 0, -1, 2};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float2> a_c(a.data(), 9);
  Data<double2> a_z(a.data(), 9);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;
  size_t host_ws_size_s;
  size_t host_ws_size_d;
  size_t host_ws_size_c;
  size_t host_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, &device_ws_size_d, &host_ws_size_d);
  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_32F, a_c.d_data, 3, HIP_R_32F, &device_ws_size_c, &host_ws_size_c);
  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_64F, a_z.d_data, 3, HIP_R_64F, &device_ws_size_z, &host_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  void* host_ws_s;
  void* host_ws_d;
  void* host_ws_c;
  void* host_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);
  hipMalloc(&host_ws_s, host_ws_size_s);
  hipMalloc(&host_ws_d, host_ws_size_d);
  hipMalloc(&host_ws_c, host_ws_size_c);
  hipMalloc(&host_ws_z, host_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, device_ws_d, device_ws_size_d, host_ws_d, host_ws_size_d, info);
  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_32F, a_c.d_data, 3, HIP_C_32F, device_ws_c, device_ws_size_c, host_ws_c, host_ws_size_c, info);
  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_64F, a_z.d_data, 3, HIP_C_64F, device_ws_z, device_ws_size_z, host_ws_z, host_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(host_ws_s);
  hipFree(host_ws_d);
  hipFree(host_ws_c);
  hipFree(host_ws_z);
  hipFree(info);

  float expect_a[9] = {1.414214,-0.707107,0.000000,-1.000000,1.224745,-0.816497,0.000000,-1.000000,1.154701};
  if (compare_result(expect_a, a_s.h_data, 9) &&
      compare_result(expect_a, a_d.h_data, 9) &&
      compare_result(expect_a, a_c.h_data, 9) &&
      compare_result(expect_a, a_z.h_data, 9))
    printf("DnXpotrf pass\n");
  else {
    printf("DnXpotrf fail\n");
    test_passed = false;
  }
}

void test_cusolverDnPotrf() {
  std::vector<float> a = {2, -1, 0, -1, 2, -1, 0, -1, 2};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float2> a_c(a.data(), 9);
  Data<double2> a_z(a.data(), 9);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, &device_ws_size_s);
  cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, &device_ws_size_d);
  cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_32F, a_c.d_data, 3, HIP_R_32F, &device_ws_size_c);
  cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_64F, a_z.d_data, 3, HIP_R_64F, &device_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, device_ws_s, device_ws_size_s, info);
  cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, device_ws_d, device_ws_size_d, info);
  cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_32F, a_c.d_data, 3, HIP_C_32F, device_ws_c, device_ws_size_c, info);
  cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_64F, a_z.d_data, 3, HIP_C_64F, device_ws_z, device_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_a[9] = {1.414214,-0.707107,0.000000,-1.000000,1.224745,-0.816497,0.000000,-1.000000,1.154701};
  if (compare_result(expect_a, a_s.h_data, 9) &&
      compare_result(expect_a, a_d.h_data, 9) &&
      compare_result(expect_a, a_c.h_data, 9) &&
      compare_result(expect_a, a_z.h_data, 9))
    printf("DnPotrf pass\n");
  else {
    printf("DnPotrf fail\n");
    test_passed = false;
  }
}

void test_cusolverDnXpotrs() {
  std::vector<float> a = {1.414214,-0.707107,0.000000,-0.707107,1.224745,-0.816497,0.000000,-0.816497,1.154701};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float2> a_c(a.data(), 9);
  Data<double2> a_z(a.data(), 9);
  std::vector<float> b = {0, 0, 4};
  Data<float> b_s(b.data(), 3);
  Data<double> b_d(b.data(), 3);
  Data<float2> b_c(b.data(), 3);
  Data<double2> b_z(b.data(), 3);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, b_s.d_data, 3, info);
  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, b_d.d_data, 3, info);
  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_C_32F, a_c.d_data, 3, HIP_C_32F, b_c.d_data, 3, info);
  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_C_64F, a_z.d_data, 3, HIP_C_64F, b_z.d_data, 3, info);

  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(info);

  float expect_b[3] = {1,2,3};
  if (compare_result(expect_b, b_s.h_data, 3) &&
      compare_result(expect_b, b_d.h_data, 3) &&
      compare_result(expect_b, b_c.h_data, 3) &&
      compare_result(expect_b, b_z.h_data, 3))
    printf("DnXpotrs pass\n");
  else {
    printf("DnXpotrs fail\n");
    test_passed = false;
  }
}

void test_cusolverDnPotrs() {
  std::vector<float> a = {1.414214,-0.707107,0.000000,-0.707107,1.224745,-0.816497,0.000000,-0.816497,1.154701};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float2> a_c(a.data(), 9);
  Data<double2> a_z(a.data(), 9);
  std::vector<float> b = {0, 0, 4};
  Data<float> b_s(b.data(), 3);
  Data<double> b_d(b.data(), 3);
  Data<float2> b_c(b.data(), 3);
  Data<double2> b_z(b.data(), 3);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnPotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, b_s.d_data, 3, info);
  cusolverDnPotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, b_d.d_data, 3, info);
  cusolverDnPotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_C_32F, a_c.d_data, 3, HIP_C_32F, b_c.d_data, 3, info);
  cusolverDnPotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_C_64F, a_z.d_data, 3, HIP_C_64F, b_z.d_data, 3, info);

  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(info);

  float expect_b[3] = {1,2,3};
  if (compare_result(expect_b, b_s.h_data, 3) &&
      compare_result(expect_b, b_d.h_data, 3) &&
      compare_result(expect_b, b_c.h_data, 3) &&
      compare_result(expect_b, b_z.h_data, 3))
    printf("DnPotrs pass\n");
  else {
    printf("DnPotrs fail\n");
    test_passed = false;
  }
}

int main() {
  test_cusolverDnXgesvd();
  test_cusolverDnGesvd();
#if 0
  test_cusolverDnTgesvd();
#endif
  test_cusolverDnXpotrf();
  test_cusolverDnPotrf();
  test_cusolverDnXpotrs();
  test_cusolverDnPotrs();

  if (test_passed)
    return 0;
  return -1;
}
