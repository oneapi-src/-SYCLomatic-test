// ===------ cusolver_test4.cu ------------------------------*- CUDA -*-----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsolver.h"

#include <cmath>
#include <vector>
#include <cstdio>
#include <complex>

template<class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float* input_data, int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num, hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num, hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }
private:
  inline void from_float_convert(float* in, d_data_t* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t* in, float* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float* in, float2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float* in, double2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float* expect, float* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i]-expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(int* expect, int* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (result[i] != expect[i]) {
      return false;
    }
  }
  return true;
}

bool compare_result(int64_t* expect, int64_t* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (result[i] != expect[i]) {
      return false;
    }
  }
  return true;
}

bool compare_result(float* expect, float* result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]]-expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

void test_cusolverDnTsyevdx_cusolverDnTheevdx() {
  std::vector<float> a = {1, 2, 2, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  Data<float> w_s(2);
  Data<double> w_d(2);
  Data<float> w_c(2);
  Data<double> w_z(2);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  int lwork_s;
  int lwork_d;
  int lwork_c;
  int lwork_z;

  int h_meig_s;
  int h_meig_d;
  int h_meig_c;
  int h_meig_z;

  hipsolverDnSsyevdx_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_s.d_data, 2, 0, 0, 0, 0, &h_meig_s, w_s.d_data, &lwork_s);
  hipsolverDnDsyevdx_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_d.d_data, 2, 0, 0, 0, 0, &h_meig_d, w_d.d_data, &lwork_d);
  hipsolverDnCheevdx_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_c.d_data, 2, 0, 0, 0, 0, &h_meig_c, w_c.d_data, &lwork_c);
  hipsolverDnZheevdx_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_z.d_data, 2, 0, 0, 0, 0, &h_meig_z, w_z.d_data, &lwork_z);

  float* device_ws_s;
  double* device_ws_d;
  float2* device_ws_c;
  double2* device_ws_z;
  hipMalloc(&device_ws_s, lwork_s * sizeof(float));
  hipMalloc(&device_ws_d, lwork_d * sizeof(double));
  hipMalloc(&device_ws_c, lwork_c * sizeof(float2));
  hipMalloc(&device_ws_z, lwork_z * sizeof(double2));

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnSsyevdx(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_s.d_data, 2, 0, 0, 0, 0, &h_meig_s, w_s.d_data, device_ws_s, lwork_s, info);
  hipsolverDnDsyevdx(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_d.d_data, 2, 0, 0, 0, 0, &h_meig_d, w_d.d_data, device_ws_d, lwork_d, info);
  hipsolverDnCheevdx(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_c.d_data, 2, 0, 0, 0, 0, &h_meig_c, w_c.d_data, device_ws_c, lwork_c, info);
  hipsolverDnZheevdx(handle, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_z.d_data, 2, 0, 0, 0, 0, &h_meig_z, w_z.d_data, device_ws_z, lwork_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();
  w_s.D2H();
  w_d.D2H();
  w_c.D2H();
  w_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_a[4] = {0.894427,-0.447214,0.447214,0.894427};
  int expect_h_meig = 2;
  float expect_w[2] = {0.000000,5.000000};
  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(&expect_h_meig, &h_meig_s, 1) &&
      compare_result(&expect_h_meig, &h_meig_d, 1) &&
      compare_result(&expect_h_meig, &h_meig_c, 1) &&
      compare_result(&expect_h_meig, &h_meig_z, 1) &&
      compare_result(expect_w, w_s.h_data, 2) &&
      compare_result(expect_w, w_d.h_data, 2) &&
      compare_result(expect_w, w_c.h_data, 2) &&
      compare_result(expect_w, w_z.h_data, 2))
    printf("DnTsyevdx/DnCheevdx pass\n");
  else {
    printf("DnTsyevdx/DnCheevdx fail\n");
    test_passed = false;
  }
}

void test_cusolverDnTsygvdx_cusolverDnThegvdx() {
  std::vector<float> a = {1, 2, 2, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  std::vector<float> b = {1, 0, 0, 1};
  Data<float> b_s(b.data(), 4);
  Data<double> b_d(b.data(), 4);
  Data<float2> b_c(b.data(), 4);
  Data<double2> b_z(b.data(), 4);
  Data<float> w_s(2);
  Data<double> w_d(2);
  Data<float> w_c(2);
  Data<double> w_z(2);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  int lwork_s;
  int lwork_d;
  int lwork_c;
  int lwork_z;

  int h_meig_s;
  int h_meig_d;
  int h_meig_c;
  int h_meig_z;

  hipsolverDnSsygvdx_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_s.d_data, 2, b_s.d_data, 2, 0, 0, 0, 0, &h_meig_s, w_s.d_data, &lwork_s);
  hipsolverDnDsygvdx_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_d.d_data, 2, b_d.d_data, 2, 0, 0, 0, 0, &h_meig_d, w_d.d_data, &lwork_d);
  hipsolverDnChegvdx_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_c.d_data, 2, b_c.d_data, 2, 0, 0, 0, 0, &h_meig_c, w_c.d_data, &lwork_c);
  hipsolverDnZhegvdx_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_z.d_data, 2, b_z.d_data, 2, 0, 0, 0, 0, &h_meig_z, w_z.d_data, &lwork_z);

  float* device_ws_s;
  double* device_ws_d;
  float2* device_ws_c;
  double2* device_ws_z;
  hipMalloc(&device_ws_s, lwork_s * sizeof(float));
  hipMalloc(&device_ws_d, lwork_d * sizeof(double));
  hipMalloc(&device_ws_c, lwork_c * sizeof(float2));
  hipMalloc(&device_ws_z, lwork_z * sizeof(double2));

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnSsygvdx(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_s.d_data, 2, b_s.d_data, 2, 0, 0, 0, 0, &h_meig_s, w_s.d_data, device_ws_s, lwork_s, info);
  hipsolverDnDsygvdx(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_d.d_data, 2, b_d.d_data, 2, 0, 0, 0, 0, &h_meig_d, w_d.d_data, device_ws_d, lwork_d, info);
  hipsolverDnChegvdx(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_c.d_data, 2, b_c.d_data, 2, 0, 0, 0, 0, &h_meig_c, w_c.d_data, device_ws_c, lwork_c, info);
  hipsolverDnZhegvdx(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPSOLVER_EIG_RANGE_ALL, HIPBLAS_FILL_MODE_UPPER, 2, a_z.d_data, 2, b_z.d_data, 2, 0, 0, 0, 0, &h_meig_z, w_z.d_data, device_ws_z, lwork_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();
  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();
  w_s.D2H();
  w_d.D2H();
  w_c.D2H();
  w_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_a[4] = {0.894427,-0.447214,0.447214,0.894427};
  float expect_b[4] = {1,0,0,1};
  int expect_h_meig = 2;
  float expect_w[2] = {0.000000,5.000000};
  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(expect_b, b_s.h_data, 4) &&
      compare_result(expect_b, b_d.h_data, 4) &&
      compare_result(expect_b, b_c.h_data, 4) &&
      compare_result(expect_b, b_z.h_data, 4) &&
      compare_result(&expect_h_meig, &h_meig_s, 1) &&
      compare_result(&expect_h_meig, &h_meig_d, 1) &&
      compare_result(&expect_h_meig, &h_meig_c, 1) &&
      compare_result(&expect_h_meig, &h_meig_z, 1) &&
      compare_result(expect_w, w_s.h_data, 2) &&
      compare_result(expect_w, w_d.h_data, 2) &&
      compare_result(expect_w, w_c.h_data, 2) &&
      compare_result(expect_w, w_z.h_data, 2))
    printf("DnTsygvdx/DnChegvdx pass\n");
  else {
    printf("DnTsygvdx/DnChegvdx fail\n");
    test_passed = false;
  }
}

void test_cusolverDnTsygvj_cusolverDnThegvj() {
  std::vector<float> a = {1, 2, 2, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  std::vector<float> b = {1, 0, 0, 1};
  Data<float> b_s(b.data(), 4);
  Data<double> b_d(b.data(), 4);
  Data<float2> b_c(b.data(), 4);
  Data<double2> b_z(b.data(), 4);
  Data<float> w_s(2);
  Data<double> w_d(2);
  Data<float> w_c(2);
  Data<double> w_z(2);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  hipsolverSyevjInfo_t params;
  hipsolverDnCreateSyevjInfo(&params);

  int lwork_s;
  int lwork_d;
  int lwork_c;
  int lwork_z;

  hipsolverDnSsygvj_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_s.d_data, 2, b_s.d_data, 2, w_s.d_data, &lwork_s, params);
  hipsolverDnDsygvj_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_d.d_data, 2, b_d.d_data, 2, w_d.d_data, &lwork_d, params);
  hipsolverDnChegvj_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_c.d_data, 2, b_c.d_data, 2, w_c.d_data, &lwork_c, params);
  hipsolverDnZhegvj_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_z.d_data, 2, b_z.d_data, 2, w_z.d_data, &lwork_z, params);

  float* device_ws_s;
  double* device_ws_d;
  float2* device_ws_c;
  double2* device_ws_z;
  hipMalloc(&device_ws_s, lwork_s * sizeof(float));
  hipMalloc(&device_ws_d, lwork_d * sizeof(double));
  hipMalloc(&device_ws_c, lwork_c * sizeof(float2));
  hipMalloc(&device_ws_z, lwork_z * sizeof(double2));

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnSsygvj(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_s.d_data, 2, b_s.d_data, 2, w_s.d_data, device_ws_s, lwork_s, info, params);
  hipsolverDnDsygvj(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_d.d_data, 2, b_d.d_data, 2, w_d.d_data, device_ws_d, lwork_d, info, params);
  hipsolverDnChegvj(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_c.d_data, 2, b_c.d_data, 2, w_c.d_data, device_ws_c, lwork_c, info, params);
  hipsolverDnZhegvj(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 2, a_z.d_data, 2, b_z.d_data, 2, w_z.d_data, device_ws_z, lwork_z, info, params);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();
  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();
  w_s.D2H();
  w_d.D2H();
  w_c.D2H();
  w_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroySyevjInfo(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_a[4] = {-0.894427,0.447214,0.447214,0.894427};
  float expect_b[4] = {1,0,0,1};
  float expect_w[2] = {0.000000,5.000000};
  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(expect_b, b_s.h_data, 4) &&
      compare_result(expect_b, b_d.h_data, 4) &&
      compare_result(expect_b, b_c.h_data, 4) &&
      compare_result(expect_b, b_z.h_data, 4) &&
      compare_result(expect_w, w_s.h_data, 2) &&
      compare_result(expect_w, w_d.h_data, 2) &&
      compare_result(expect_w, w_c.h_data, 2) &&
      compare_result(expect_w, w_z.h_data, 2))
    printf("DnTsygvj/DnChegvj pass\n");
  else {
    printf("DnTsygvj/DnChegvj fail\n");
    test_passed = false;
  }
}

int main() {
  test_cusolverDnTsyevdx_cusolverDnTheevdx();
  test_cusolverDnTsygvdx_cusolverDnThegvdx();
  test_cusolverDnTsygvj_cusolverDnThegvj();

  if (test_passed)
    return 0;
  return -1;
}
