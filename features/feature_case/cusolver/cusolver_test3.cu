#include "hip/hip_runtime.h"
// ===------ cusolver_test3.cu ------------------------------*- CUDA -*-----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//

#include "hipsolver.h"

#include <cmath>
#include <vector>
#include <cstdio>
#include <complex>

template<class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float* input_data, int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num, hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num, hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }
private:
  inline void from_float_convert(float* in, d_data_t* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t* in, float* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float* in, float2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float* in, double2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float* expect, float* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i]-expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float* expect, float* result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]]-expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

void test_cusolverDnXgetrf() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  Data<int64_t> ipiv_s(2);
  Data<int64_t> ipiv_d(2);
  Data<int64_t> ipiv_c(2);
  Data<int64_t> ipiv_z(2);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  ipiv_s.H2D();
  ipiv_d.H2D();
  ipiv_c.H2D();
  ipiv_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;
  size_t host_ws_size_s;
  size_t host_ws_size_d;
  size_t host_ws_size_c;
  size_t host_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, &device_ws_size_d, &host_ws_size_d);
  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_C_32F, &device_ws_size_c, &host_ws_size_c);
  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_C_64F, &device_ws_size_z, &host_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  void* host_ws_s;
  void* host_ws_d;
  void* host_ws_c;
  void* host_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);
  hipMalloc(&host_ws_s, host_ws_size_s);
  hipMalloc(&host_ws_d, host_ws_size_d);
  hipMalloc(&host_ws_c, host_ws_size_c);
  hipMalloc(&host_ws_z, host_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnXgetrf(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, ipiv_s.d_data, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  hipsolverDnXgetrf(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, ipiv_d.d_data, HIP_R_64F, device_ws_d, device_ws_size_d, host_ws_d, host_ws_size_d, info);
  hipsolverDnXgetrf(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, ipiv_c.d_data, HIP_C_32F, device_ws_c, device_ws_size_c, host_ws_c, host_ws_size_c, info);
  hipsolverDnXgetrf(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, ipiv_z.d_data, HIP_C_64F, device_ws_z, device_ws_size_z, host_ws_z, host_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();
  ipiv_s.D2H();
  ipiv_d.D2H();
  ipiv_c.D2H();
  ipiv_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(host_ws_s);
  hipFree(host_ws_d);
  hipFree(host_ws_c);
  hipFree(host_ws_z);
  hipFree(info);

  float expect_a[4] = {2, 0.5, 4, 1};
  float expect_ipiv[2] = {2, 2};
  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(expect_ipiv, ipiv_s.h_data, 2) &&
      compare_result(expect_ipiv, ipiv_d.h_data, 2) &&
      compare_result(expect_ipiv, ipiv_c.h_data, 2) &&
      compare_result(expect_ipiv, ipiv_z.h_data, 2))
    printf("DnXgetrf pass\n");
  else {
    printf("DnXgetrf fail\n");
    test_passed = false;
  }
}

void test_cusolverDnXgetrfnp() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;
  size_t host_ws_size_s;
  size_t host_ws_size_d;
  size_t host_ws_size_c;
  size_t host_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, &device_ws_size_d, &host_ws_size_d);
  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_C_32F, &device_ws_size_c, &host_ws_size_c);
  hipsolverDnXgetrf_bufferSize(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_C_64F, &device_ws_size_z, &host_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  void* host_ws_s;
  void* host_ws_d;
  void* host_ws_c;
  void* host_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);
  hipMalloc(&host_ws_s, host_ws_size_s);
  hipMalloc(&host_ws_d, host_ws_size_d);
  hipMalloc(&host_ws_c, host_ws_size_c);
  hipMalloc(&host_ws_z, host_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnXgetrf(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, nullptr, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  hipsolverDnXgetrf(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, nullptr, HIP_R_64F, device_ws_d, device_ws_size_d, host_ws_d, host_ws_size_d, info);
  hipsolverDnXgetrf(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, nullptr, HIP_C_32F, device_ws_c, device_ws_size_c, host_ws_c, host_ws_size_c, info);
  hipsolverDnXgetrf(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, nullptr, HIP_C_64F, device_ws_z, device_ws_size_z, host_ws_z, host_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(host_ws_s);
  hipFree(host_ws_d);
  hipFree(host_ws_c);
  hipFree(host_ws_z);
  hipFree(info);

  float expect_a[4] = {1, 2, 3, -2};
  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4))
    printf("DnXgetrfnp pass\n");
  else {
    printf("DnXgetrfnp fail\n");
    test_passed = false;
  }
}

void test_cusolverDnGetrf() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  Data<int64_t> ipiv_s(2);
  Data<int64_t> ipiv_d(2);
  Data<int64_t> ipiv_c(2);
  Data<int64_t> ipiv_z(2);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  ipiv_s.H2D();
  ipiv_d.H2D();
  ipiv_c.H2D();
  ipiv_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  cusolverDnGetrf_bufferSize(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, &device_ws_size_s);
  cusolverDnGetrf_bufferSize(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, &device_ws_size_d);
  cusolverDnGetrf_bufferSize(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_C_32F, &device_ws_size_c);
  cusolverDnGetrf_bufferSize(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_C_64F, &device_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;

  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnGetrf(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, ipiv_s.d_data, HIP_R_32F, device_ws_s, device_ws_size_s, info);
  cusolverDnGetrf(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, ipiv_d.d_data, HIP_R_64F, device_ws_d, device_ws_size_d, info);
  cusolverDnGetrf(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, ipiv_c.d_data, HIP_C_32F, device_ws_c, device_ws_size_c, info);
  cusolverDnGetrf(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, ipiv_z.d_data, HIP_C_64F, device_ws_z, device_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();
  ipiv_s.D2H();
  ipiv_d.D2H();
  ipiv_c.D2H();
  ipiv_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_a[4] = {2, 0.5, 4, 1};
  float expect_ipiv[2] = {2, 2};
  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(expect_ipiv, ipiv_s.h_data, 2) &&
      compare_result(expect_ipiv, ipiv_d.h_data, 2) &&
      compare_result(expect_ipiv, ipiv_c.h_data, 2) &&
      compare_result(expect_ipiv, ipiv_z.h_data, 2))
    printf("DnGetrf pass\n");
  else {
    printf("DnGetrf fail\n");
    test_passed = false;
  }
}

void test_cusolverDnXgetrs() {
  std::vector<float> a = {2, 0.5, 4, 1};
  std::vector<float> ipiv = {2, 2};
  std::vector<float> b = {23, 34, 31, 46, 39, 58};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  Data<int64_t> ipiv_s(ipiv.data(), 2);
  Data<int64_t> ipiv_d(ipiv.data(), 2);
  Data<int64_t> ipiv_c(ipiv.data(), 2);
  Data<int64_t> ipiv_z(ipiv.data(), 2);
  Data<float> b_s(b.data(), 6);
  Data<double> b_d(b.data(), 6);
  Data<float2> b_c(b.data(), 6);
  Data<double2> b_z(b.data(), 6);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  ipiv_s.H2D();
  ipiv_d.H2D();
  ipiv_c.H2D();
  ipiv_z.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_R_32F, a_s.d_data, 2, ipiv_s.d_data, HIP_R_32F, b_s.d_data, 2, info);
  hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_R_64F, a_d.d_data, 2, ipiv_d.d_data, HIP_R_64F, b_d.d_data, 2, info);
  hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_C_32F, a_c.d_data, 2, ipiv_c.d_data, HIP_C_32F, b_c.d_data, 2, info);
  hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_C_64F, a_z.d_data, 2, ipiv_z.d_data, HIP_C_64F, b_z.d_data, 2, info);

  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(info);

  float expect_b[6] = {5, 6, 7, 8, 9, 10};
  if (compare_result(expect_b, b_s.h_data, 6) &&
      compare_result(expect_b, b_d.h_data, 6) &&
      compare_result(expect_b, b_c.h_data, 6) &&
      compare_result(expect_b, b_z.h_data, 6))
    printf("DnXgetrs pass\n");
  else {
    printf("DnXgetrs fail\n");
    test_passed = false;
  }
}

void test_cusolverDnGetrs() {
  std::vector<float> a = {2, 0.5, 4, 1};
  std::vector<float> ipiv = {2, 2};
  std::vector<float> b = {23, 34, 31, 46, 39, 58};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  Data<int64_t> ipiv_s(ipiv.data(), 2);
  Data<int64_t> ipiv_d(ipiv.data(), 2);
  Data<int64_t> ipiv_c(ipiv.data(), 2);
  Data<int64_t> ipiv_z(ipiv.data(), 2);
  Data<float> b_s(b.data(), 6);
  Data<double> b_d(b.data(), 6);
  Data<float2> b_c(b.data(), 6);
  Data<double2> b_z(b.data(), 6);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  ipiv_s.H2D();
  ipiv_d.H2D();
  ipiv_c.H2D();
  ipiv_z.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnGetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_R_32F, a_s.d_data, 2, ipiv_s.d_data, HIP_R_32F, b_s.d_data, 2, info);
  cusolverDnGetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_R_64F, a_d.d_data, 2, ipiv_d.d_data, HIP_R_64F, b_d.d_data, 2, info);
  cusolverDnGetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_C_32F, a_c.d_data, 2, ipiv_c.d_data, HIP_C_32F, b_c.d_data, 2, info);
  cusolverDnGetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_C_64F, a_z.d_data, 2, ipiv_z.d_data, HIP_C_64F, b_z.d_data, 2, info);

  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(info);

  float expect_b[6] = {5, 6, 7, 8, 9, 10};
  if (compare_result(expect_b, b_s.h_data, 6) &&
      compare_result(expect_b, b_d.h_data, 6) &&
      compare_result(expect_b, b_c.h_data, 6) &&
      compare_result(expect_b, b_z.h_data, 6))
    printf("DnGetrs pass\n");
  else {
    printf("DnGetrs fail\n");
    test_passed = false;
  }
}

void test_cusolverDnXgeqrf() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  Data<float> tau_s(2);
  Data<double> tau_d(2);
  Data<float2> tau_c(2);
  Data<double2> tau_z(2);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  tau_s.H2D();
  tau_d.H2D();
  tau_c.H2D();
  tau_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;
  size_t host_ws_size_s;
  size_t host_ws_size_d;
  size_t host_ws_size_c;
  size_t host_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  hipsolverDnXgeqrf_bufferSize(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, tau_s.d_data, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  hipsolverDnXgeqrf_bufferSize(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, tau_d.d_data, HIP_R_64F, &device_ws_size_d, &host_ws_size_d);
  hipsolverDnXgeqrf_bufferSize(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_C_32F, tau_c.d_data, HIP_C_32F, &device_ws_size_c, &host_ws_size_c);
  hipsolverDnXgeqrf_bufferSize(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_C_64F, tau_z.d_data, HIP_C_64F, &device_ws_size_z, &host_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  void* host_ws_s;
  void* host_ws_d;
  void* host_ws_c;
  void* host_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);
  hipMalloc(&host_ws_s, host_ws_size_s);
  hipMalloc(&host_ws_d, host_ws_size_d);
  hipMalloc(&host_ws_c, host_ws_size_c);
  hipMalloc(&host_ws_z, host_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnXgeqrf(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, tau_s.d_data, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  hipsolverDnXgeqrf(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, tau_d.d_data, HIP_R_64F, device_ws_d, device_ws_size_d, host_ws_d, host_ws_size_d, info);
  hipsolverDnXgeqrf(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_C_32F, tau_c.d_data, HIP_C_32F, device_ws_c, device_ws_size_c, host_ws_c, host_ws_size_c, info);
  hipsolverDnXgeqrf(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_C_64F, tau_z.d_data, HIP_C_64F, device_ws_z, device_ws_size_z, host_ws_z, host_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();
  tau_s.D2H();
  tau_d.D2H();
  tau_c.D2H();
  tau_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(host_ws_s);
  hipFree(host_ws_d);
  hipFree(host_ws_c);
  hipFree(host_ws_z);
  hipFree(info);

  float expect_a[4] = {-2.236068, 0.618034, -4.919349, -0.894427};
  float expect_tau[2] = {1.447214, 0};

  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(expect_tau, tau_s.h_data, 2) &&
      compare_result(expect_tau, tau_d.h_data, 2) &&
      compare_result(expect_tau, tau_c.h_data, 2) &&
      compare_result(expect_tau, tau_z.h_data, 2))
    printf("DnXgeqrf pass\n");
  else {
    printf("DnXgeqrf fail\n");
    test_passed = false;
  }
}

void test_cusolverDnGeqrf() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);
  Data<float> tau_s(2);
  Data<double> tau_d(2);
  Data<float2> tau_c(2);
  Data<double2> tau_z(2);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  tau_s.H2D();
  tau_d.H2D();
  tau_c.H2D();
  tau_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  cusolverDnGeqrf_bufferSize(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, tau_s.d_data, HIP_R_32F, &device_ws_size_s);
  cusolverDnGeqrf_bufferSize(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, tau_d.d_data, HIP_R_64F, &device_ws_size_d);
  cusolverDnGeqrf_bufferSize(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_C_32F, tau_c.d_data, HIP_C_32F, &device_ws_size_c);
  cusolverDnGeqrf_bufferSize(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_C_64F, tau_z.d_data, HIP_C_64F, &device_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnGeqrf(handle, params, 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, tau_s.d_data, HIP_R_32F, device_ws_s, device_ws_size_s, info);
  cusolverDnGeqrf(handle, params, 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, tau_d.d_data, HIP_R_64F, device_ws_d, device_ws_size_d, info);
  cusolverDnGeqrf(handle, params, 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_C_32F, tau_c.d_data, HIP_C_32F, device_ws_c, device_ws_size_c, info);
  cusolverDnGeqrf(handle, params, 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_C_64F, tau_z.d_data, HIP_C_64F, device_ws_z, device_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();
  tau_s.D2H();
  tau_d.D2H();
  tau_c.D2H();
  tau_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_a[4] = {-2.236068, 0.618034, -4.919349, -0.894427};
  float expect_tau[2] = {1.447214, 0};

  if (compare_result(expect_a, a_s.h_data, 4) &&
      compare_result(expect_a, a_d.h_data, 4) &&
      compare_result(expect_a, a_c.h_data, 4) &&
      compare_result(expect_a, a_z.h_data, 4) &&
      compare_result(expect_tau, tau_s.h_data, 2) &&
      compare_result(expect_tau, tau_d.h_data, 2) &&
      compare_result(expect_tau, tau_c.h_data, 2) &&
      compare_result(expect_tau, tau_z.h_data, 2))
    printf("DnGeqrf pass\n");
  else {
    printf("DnGeqrf fail\n");
    test_passed = false;
  }
}

void test_cusolverDnXgesvd() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);

  Data<float> s_s(2);
  Data<double> s_d(2);
  Data<float> s_c(2);
  Data<double> s_z(2);

  Data<float> u_s(4);
  Data<double> u_d(4);
  Data<float2> u_c(4);
  Data<double2> u_z(4);

  Data<float> vt_s(4);
  Data<double> vt_d(4);
  Data<float2> vt_c(4);
  Data<double2> vt_z(4);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;
  size_t host_ws_size_s;
  size_t host_ws_size_d;
  size_t host_ws_size_c;
  size_t host_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  cusolverDnXgesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, s_s.d_data, HIP_R_32F, u_s.d_data, 2, HIP_R_32F, vt_s.d_data, 2, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  cusolverDnXgesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, s_d.d_data, HIP_R_64F, u_d.d_data, 2, HIP_R_64F, vt_d.d_data, 2, HIP_R_64F, &device_ws_size_d, &host_ws_size_d);
  cusolverDnXgesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_R_32F, s_c.d_data, HIP_C_32F, u_c.d_data, 2, HIP_C_32F, vt_c.d_data, 2, HIP_C_32F, &device_ws_size_c, &host_ws_size_c);
  cusolverDnXgesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_R_64F, s_z.d_data, HIP_C_64F, u_z.d_data, 2, HIP_C_64F, vt_z.d_data, 2, HIP_C_64F, &device_ws_size_z, &host_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  void* host_ws_s;
  void* host_ws_d;
  void* host_ws_c;
  void* host_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);
  hipMalloc(&host_ws_s, host_ws_size_s);
  hipMalloc(&host_ws_d, host_ws_size_d);
  hipMalloc(&host_ws_c, host_ws_size_c);
  hipMalloc(&host_ws_z, host_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, s_s.d_data, HIP_R_32F, u_s.d_data, 2, HIP_R_32F, vt_s.d_data, 2, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, s_d.d_data, HIP_R_64F, u_d.d_data, 2, HIP_R_64F, vt_d.d_data, 2, HIP_R_64F, device_ws_d, device_ws_size_d, host_ws_d, host_ws_size_d, info);
  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_R_32F, s_c.d_data, HIP_C_32F, u_c.d_data, 2, HIP_C_32F, vt_c.d_data, 2, HIP_C_32F, device_ws_c, device_ws_size_c, host_ws_c, host_ws_size_c, info);
  cusolverDnXgesvd(handle, params, 'A', 'A', 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_R_64F, s_z.d_data, HIP_C_64F, u_z.d_data, 2, HIP_C_64F, vt_z.d_data, 2, HIP_C_64F, device_ws_z, device_ws_size_z, host_ws_z, host_ws_size_z, info);

  s_s.D2H();
  s_d.D2H();
  s_c.D2H();
  s_z.D2H();

  u_s.D2H();
  u_d.D2H();
  u_c.D2H();
  u_z.D2H();

  vt_s.D2H();
  vt_d.D2H();
  vt_c.D2H();
  vt_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(host_ws_s);
  hipFree(host_ws_d);
  hipFree(host_ws_c);
  hipFree(host_ws_z);
  hipFree(info);

  float expect_s[2] = {5.464985,0.365966};
  float expect_u[4] = {0.576048,0.817416,-0.817416,0.576048};
  float expect_vt[4] = {0.404554,0.914514,0.914514,-0.404554};

  if (compare_result(expect_s, s_s.h_data, 2) &&
      compare_result(expect_s, s_d.h_data, 2) &&
      compare_result(expect_s, s_c.h_data, 2) &&
      compare_result(expect_s, s_z.h_data, 2) &&
      compare_result(expect_u, u_s.h_data, 4) &&
      compare_result(expect_u, u_d.h_data, 4) &&
      compare_result(expect_u, u_c.h_data, 4) &&
      compare_result(expect_u, u_z.h_data, 4) &&
      compare_result(expect_vt, vt_s.h_data, 4) &&
      compare_result(expect_vt, vt_d.h_data, 4) &&
      compare_result(expect_vt, vt_c.h_data, 4) &&
      compare_result(expect_vt, vt_z.h_data, 4))
    printf("DnXgesvd pass\n");
  else {
    printf("DnXgesvd fail\n");
    test_passed = false;
  }
}

void test_cusolverDnGesvd() {
  std::vector<float> a = {1, 2, 3, 4};
  Data<float> a_s(a.data(), 4);
  Data<double> a_d(a.data(), 4);
  Data<float2> a_c(a.data(), 4);
  Data<double2> a_z(a.data(), 4);

  Data<float> s_s(2);
  Data<double> s_d(2);
  Data<float> s_c(2);
  Data<double> s_z(2);

  Data<float> u_s(4);
  Data<double> u_d(4);
  Data<float2> u_c(4);
  Data<double2> u_z(4);

  Data<float> vt_s(4);
  Data<double> vt_d(4);
  Data<float2> vt_c(4);
  Data<double2> vt_z(4);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  cusolverDnGesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, s_s.d_data, HIP_R_32F, u_s.d_data, 2, HIP_R_32F, vt_s.d_data, 2, HIP_R_32F, &device_ws_size_s);
  cusolverDnGesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, s_d.d_data, HIP_R_64F, u_d.d_data, 2, HIP_R_64F, vt_d.d_data, 2, HIP_R_64F, &device_ws_size_d);
  cusolverDnGesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_R_32F, s_c.d_data, HIP_C_32F, u_c.d_data, 2, HIP_C_32F, vt_c.d_data, 2, HIP_C_32F, &device_ws_size_c);
  cusolverDnGesvd_bufferSize(handle, params, 'A', 'A', 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_R_64F, s_z.d_data, HIP_C_64F, u_z.d_data, 2, HIP_C_64F, vt_z.d_data, 2, HIP_C_64F, &device_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnGesvd(handle, params, 'A', 'A', 2, 2, HIP_R_32F, a_s.d_data, 2, HIP_R_32F, s_s.d_data, HIP_R_32F, u_s.d_data, 2, HIP_R_32F, vt_s.d_data, 2, HIP_R_32F, device_ws_s, device_ws_size_s, info);
  cusolverDnGesvd(handle, params, 'A', 'A', 2, 2, HIP_R_64F, a_d.d_data, 2, HIP_R_64F, s_d.d_data, HIP_R_64F, u_d.d_data, 2, HIP_R_64F, vt_d.d_data, 2, HIP_R_64F, device_ws_d, device_ws_size_d, info);
  cusolverDnGesvd(handle, params, 'A', 'A', 2, 2, HIP_C_32F, a_c.d_data, 2, HIP_R_32F, s_c.d_data, HIP_C_32F, u_c.d_data, 2, HIP_C_32F, vt_c.d_data, 2, HIP_C_32F, device_ws_c, device_ws_size_c, info);
  cusolverDnGesvd(handle, params, 'A', 'A', 2, 2, HIP_C_64F, a_z.d_data, 2, HIP_R_64F, s_z.d_data, HIP_C_64F, u_z.d_data, 2, HIP_C_64F, vt_z.d_data, 2, HIP_C_64F, device_ws_z, device_ws_size_z, info);

  s_s.D2H();
  s_d.D2H();
  s_c.D2H();
  s_z.D2H();

  u_s.D2H();
  u_d.D2H();
  u_c.D2H();
  u_z.D2H();

  vt_s.D2H();
  vt_d.D2H();
  vt_c.D2H();
  vt_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_s[2] = {5.464985,0.365966};
  float expect_u[4] = {0.576048,0.817416,-0.817416,0.576048};
  float expect_vt[4] = {0.404554,0.914514,0.914514,-0.404554};

  if (compare_result(expect_s, s_s.h_data, 2) &&
      compare_result(expect_s, s_d.h_data, 2) &&
      compare_result(expect_s, s_c.h_data, 2) &&
      compare_result(expect_s, s_z.h_data, 2) &&
      compare_result(expect_u, u_s.h_data, 4) &&
      compare_result(expect_u, u_d.h_data, 4) &&
      compare_result(expect_u, u_c.h_data, 4) &&
      compare_result(expect_u, u_z.h_data, 4) &&
      compare_result(expect_vt, vt_s.h_data, 4) &&
      compare_result(expect_vt, vt_d.h_data, 4) &&
      compare_result(expect_vt, vt_c.h_data, 4) &&
      compare_result(expect_vt, vt_z.h_data, 4))
    printf("DnGesvd pass\n");
  else {
    printf("DnGesvd fail\n");
    test_passed = false;
  }
}

void test_cusolverDnXpotrf() {
  std::vector<float> a = {2, -1, 0, -1, 2, -1, 0, -1, 2};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float2> a_c(a.data(), 9);
  Data<double2> a_z(a.data(), 9);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;
  size_t host_ws_size_s;
  size_t host_ws_size_d;
  size_t host_ws_size_c;
  size_t host_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, &device_ws_size_s, &host_ws_size_s);
  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, &device_ws_size_d, &host_ws_size_d);
  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_32F, a_c.d_data, 3, HIP_R_32F, &device_ws_size_c, &host_ws_size_c);
  hipsolverDnXpotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_64F, a_z.d_data, 3, HIP_R_64F, &device_ws_size_z, &host_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  void* host_ws_s;
  void* host_ws_d;
  void* host_ws_c;
  void* host_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);
  hipMalloc(&host_ws_s, host_ws_size_s);
  hipMalloc(&host_ws_d, host_ws_size_d);
  hipMalloc(&host_ws_c, host_ws_size_c);
  hipMalloc(&host_ws_z, host_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, device_ws_s, device_ws_size_s, host_ws_s, host_ws_size_s, info);
  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, device_ws_d, device_ws_size_d, host_ws_d, host_ws_size_d, info);
  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_32F, a_c.d_data, 3, HIP_C_32F, device_ws_c, device_ws_size_c, host_ws_c, host_ws_size_c, info);
  hipsolverDnXpotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_64F, a_z.d_data, 3, HIP_C_64F, device_ws_z, device_ws_size_z, host_ws_z, host_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(host_ws_s);
  hipFree(host_ws_d);
  hipFree(host_ws_c);
  hipFree(host_ws_z);
  hipFree(info);

  float expect_a[9] = {1.414214,-0.707107,0.000000,-1.000000,1.224745,-0.816497,0.000000,-1.000000,1.154701};
  if (compare_result(expect_a, a_s.h_data, 9) &&
      compare_result(expect_a, a_d.h_data, 9) &&
      compare_result(expect_a, a_c.h_data, 9) &&
      compare_result(expect_a, a_z.h_data, 9))
    printf("DnXpotrf pass\n");
  else {
    printf("DnXpotrf fail\n");
    test_passed = false;
  }
}

void test_cusolverDnPotrf() {
  std::vector<float> a = {2, -1, 0, -1, 2, -1, 0, -1, 2};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float2> a_c(a.data(), 9);
  Data<double2> a_z(a.data(), 9);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  size_t device_ws_size_s;
  size_t device_ws_size_d;
  size_t device_ws_size_c;
  size_t device_ws_size_z;

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, &device_ws_size_s);
  cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, &device_ws_size_d);
  cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_32F, a_c.d_data, 3, HIP_R_32F, &device_ws_size_c);
  cusolverDnPotrf_bufferSize(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_64F, a_z.d_data, 3, HIP_R_64F, &device_ws_size_z);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  hipMalloc(&device_ws_s, device_ws_size_s);
  hipMalloc(&device_ws_d, device_ws_size_d);
  hipMalloc(&device_ws_c, device_ws_size_c);
  hipMalloc(&device_ws_z, device_ws_size_z);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, device_ws_s, device_ws_size_s, info);
  cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, device_ws_d, device_ws_size_d, info);
  cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_32F, a_c.d_data, 3, HIP_C_32F, device_ws_c, device_ws_size_c, info);
  cusolverDnPotrf(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, HIP_C_64F, a_z.d_data, 3, HIP_C_64F, device_ws_z, device_ws_size_z, info);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(device_ws_s);
  hipFree(device_ws_d);
  hipFree(device_ws_c);
  hipFree(device_ws_z);
  hipFree(info);

  float expect_a[9] = {1.414214,-0.707107,0.000000,-1.000000,1.224745,-0.816497,0.000000,-1.000000,1.154701};
  if (compare_result(expect_a, a_s.h_data, 9) &&
      compare_result(expect_a, a_d.h_data, 9) &&
      compare_result(expect_a, a_c.h_data, 9) &&
      compare_result(expect_a, a_z.h_data, 9))
    printf("DnPotrf pass\n");
  else {
    printf("DnPotrf fail\n");
    test_passed = false;
  }
}

void test_cusolverDnXpotrs() {
  std::vector<float> a = {1.414214,-0.707107,0.000000,-0.707107,1.224745,-0.816497,0.000000,-0.816497,1.154701};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float2> a_c(a.data(), 9);
  Data<double2> a_z(a.data(), 9);
  std::vector<float> b = {0, 0, 4};
  Data<float> b_s(b.data(), 3);
  Data<double> b_d(b.data(), 3);
  Data<float2> b_c(b.data(), 3);
  Data<double2> b_z(b.data(), 3);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  int *info;
  hipMalloc(&info, sizeof(int));

  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, b_s.d_data, 3, info);
  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, b_d.d_data, 3, info);
  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_C_32F, a_c.d_data, 3, HIP_C_32F, b_c.d_data, 3, info);
  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_C_64F, a_z.d_data, 3, HIP_C_64F, b_z.d_data, 3, info);

  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(info);

  float expect_b[3] = {1,2,3};
  if (compare_result(expect_b, b_s.h_data, 3) &&
      compare_result(expect_b, b_d.h_data, 3) &&
      compare_result(expect_b, b_c.h_data, 3) &&
      compare_result(expect_b, b_z.h_data, 3))
    printf("DnXpotrs pass\n");
  else {
    printf("DnXpotrs fail\n");
    test_passed = false;
  }
}

void test_cusolverDnPotrs() {
  std::vector<float> a = {1.414214,-0.707107,0.000000,-0.707107,1.224745,-0.816497,0.000000,-0.816497,1.154701};
  Data<float> a_s(a.data(), 9);
  Data<double> a_d(a.data(), 9);
  Data<float2> a_c(a.data(), 9);
  Data<double2> a_z(a.data(), 9);
  std::vector<float> b = {0, 0, 4};
  Data<float> b_s(b.data(), 3);
  Data<double> b_d(b.data(), 3);
  Data<float2> b_c(b.data(), 3);
  Data<double2> b_z(b.data(), 3);

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  hipsolverDnParams_t params;
  hipsolverDnCreateParams(&params);

  int *info;
  hipMalloc(&info, sizeof(int));

  cusolverDnPotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_32F, a_s.d_data, 3, HIP_R_32F, b_s.d_data, 3, info);
  cusolverDnPotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_64F, a_d.d_data, 3, HIP_R_64F, b_d.d_data, 3, info);
  cusolverDnPotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_C_32F, a_c.d_data, 3, HIP_C_32F, b_c.d_data, 3, info);
  cusolverDnPotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_C_64F, a_z.d_data, 3, HIP_C_64F, b_z.d_data, 3, info);

  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroyParams(params);
  hipsolverDnDestroy(handle);
  hipFree(info);

  float expect_b[3] = {1,2,3};
  if (compare_result(expect_b, b_s.h_data, 3) &&
      compare_result(expect_b, b_d.h_data, 3) &&
      compare_result(expect_b, b_c.h_data, 3) &&
      compare_result(expect_b, b_z.h_data, 3))
    printf("DnPotrs pass\n");
  else {
    printf("DnPotrs fail\n");
    test_passed = false;
  }
}

int main() {
  test_cusolverDnXgetrf();
  test_cusolverDnXgetrfnp();
  test_cusolverDnGetrf();
  test_cusolverDnXgetrs();
  test_cusolverDnGetrs();
  test_cusolverDnXgeqrf();
  test_cusolverDnGeqrf();
  test_cusolverDnXgesvd();
  test_cusolverDnGesvd();
  test_cusolverDnXpotrf();
  test_cusolverDnPotrf();
  test_cusolverDnXpotrs();
  test_cusolverDnPotrs();

  if (test_passed)
    return 0;
  return -1;
}
