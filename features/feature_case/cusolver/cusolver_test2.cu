// ===------ cusolver_test2.cu ------------------------------*- CUDA -*-----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsolver.h"

#include <cmath>
#include <vector>
#include <cstdio>
#include <complex>

template<class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float* input_data, int element_num) : element_num(element_num) {
    h_data = (float*)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num, hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t* h_temp = (d_data_t*)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num, hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }
private:
  inline void from_float_convert(float* in, d_data_t* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t* in, float* out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float* in, float2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float* in, double2* out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2* in, float* out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float* expect, float* result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i]-expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float* expect, float* result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]]-expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

struct Ptr_Data {
  int group_num;
  void** h_data;
  void** d_data;
  Ptr_Data(int group_num) : group_num(group_num) {
    h_data = (void**)malloc(group_num * sizeof(void*));
    memset(h_data, 0, group_num * sizeof(void*));
    hipMalloc(&d_data, group_num * sizeof(void*));
    hipMemset(d_data, 0, group_num * sizeof(void*));
  }
  ~Ptr_Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    hipMemcpy(d_data, h_data, group_num * sizeof(void*), hipMemcpyHostToDevice);
  }
};

#ifndef DPCT_USM_LEVEL_NONE
void test_cusolverDnTpotrfBatched() {
  std::vector<float> a = {2, -1, 0, -1, 2, -1, 0, -1, 2,
                          2, -1, 0, -1, 2, -1, 0, -1, 2};
  Data<float> a_s(a.data(), 18);
  Data<double> a_d(a.data(), 18);
  Data<float2> a_c(a.data(), 18);
  Data<double2> a_z(a.data(), 18);

  Ptr_Data a_s_ptrs(2); a_s_ptrs.h_data[0] = a_s.d_data; a_s_ptrs.h_data[1] = a_s.d_data + 9;
  Ptr_Data a_d_ptrs(2); a_d_ptrs.h_data[0] = a_d.d_data; a_d_ptrs.h_data[1] = a_d.d_data + 9;
  Ptr_Data a_c_ptrs(2); a_c_ptrs.h_data[0] = a_c.d_data; a_c_ptrs.h_data[1] = a_c.d_data + 9;
  Ptr_Data a_z_ptrs(2); a_z_ptrs.h_data[0] = a_z.d_data; a_z_ptrs.h_data[1] = a_z.d_data + 9;

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  a_s_ptrs.H2D();
  a_d_ptrs.H2D();
  a_c_ptrs.H2D();
  a_z_ptrs.H2D();

  int *infoArray;
  hipMalloc(&infoArray, 2 * sizeof(int));

  hipsolverDnSpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, (float **)a_s_ptrs.d_data, 3, infoArray, 2);
  hipsolverDnDpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, (double **)a_d_ptrs.d_data, 3, infoArray, 2);
  hipsolverDnCpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, (float2 **)a_c_ptrs.d_data, 3, infoArray, 2);
  hipsolverDnZpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, (double2 **)a_z_ptrs.d_data, 3, infoArray, 2);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroy(handle);

  std::vector<int> indeces = {0, 3, 4, 6, 7, 8,
                              9,12,13,15,16,17 };
  float expect[18] = { 1.414214,-0.707107,0.000000,-0.707107,1.224745,-0.816497,0.000000,-0.816497,1.154701,
                       1.414214,-0.707107,0.000000,-0.707107,1.224745,-0.816497,0.000000,-0.816497,1.154701 };
  if (compare_result(expect, a_s.h_data, indeces) &&
      compare_result(expect, a_d.h_data, indeces) &&
      compare_result(expect, a_c.h_data, indeces) &&
      compare_result(expect, a_z.h_data, indeces))
    printf("DnTpotrfBatched pass\n");
  else {
    printf("DnTpotrfBatched fail\n");
    test_passed = false;
  }
}

void test_cusolverDnTpotrsBatched() {
  std::vector<float> a = {1.414214,-0.707107,0.000000,-0.707107,1.224745,-0.816497,0.000000,-0.816497,1.154701,
                          1.414214,-0.707107,0.000000,-0.707107,1.224745,-0.816497,0.000000,-0.816497,1.154701 };
  Data<float> a_s(a.data(), 18);
  Data<double> a_d(a.data(), 18);
  Data<float2> a_c(a.data(), 18);
  Data<double2> a_z(a.data(), 18);

  Ptr_Data a_s_ptrs(2); a_s_ptrs.h_data[0] = a_s.d_data; a_s_ptrs.h_data[1] = a_s.d_data + 9;
  Ptr_Data a_d_ptrs(2); a_d_ptrs.h_data[0] = a_d.d_data; a_d_ptrs.h_data[1] = a_d.d_data + 9;
  Ptr_Data a_c_ptrs(2); a_c_ptrs.h_data[0] = a_c.d_data; a_c_ptrs.h_data[1] = a_c.d_data + 9;
  Ptr_Data a_z_ptrs(2); a_z_ptrs.h_data[0] = a_z.d_data; a_z_ptrs.h_data[1] = a_z.d_data + 9;

  std::vector<float> b = {0, 0, 4,
                          0, 0, 4};
  Data<float> b_s(b.data(), 6);
  Data<double> b_d(b.data(), 6);
  Data<float2> b_c(b.data(), 6);
  Data<double2> b_z(b.data(), 6);

  Ptr_Data b_s_ptrs(2); b_s_ptrs.h_data[0] = b_s.d_data; b_s_ptrs.h_data[1] = b_s.d_data + 3;
  Ptr_Data b_d_ptrs(2); b_d_ptrs.h_data[0] = b_d.d_data; b_d_ptrs.h_data[1] = b_d.d_data + 3;
  Ptr_Data b_c_ptrs(2); b_c_ptrs.h_data[0] = b_c.d_data; b_c_ptrs.h_data[1] = b_c.d_data + 3;
  Ptr_Data b_z_ptrs(2); b_z_ptrs.h_data[0] = b_z.d_data; b_z_ptrs.h_data[1] = b_z.d_data + 3;

  hipsolverHandle_t handle;
  hipsolverDnCreate(&handle);

  a_s.H2D();
  a_d.H2D();
  a_c.H2D();
  a_z.H2D();

  a_s_ptrs.H2D();
  a_d_ptrs.H2D();
  a_c_ptrs.H2D();
  a_z_ptrs.H2D();

  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();

  b_s_ptrs.H2D();
  b_d_ptrs.H2D();
  b_c_ptrs.H2D();
  b_z_ptrs.H2D();

  int *infoArray;
  hipMalloc(&infoArray, 2 * sizeof(int));

  hipsolverDnSpotrsBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, 1, (float **)a_s_ptrs.d_data, 3, (float **)b_s_ptrs.d_data, 3, infoArray, 2);
  hipsolverDnDpotrsBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, 1, (double **)a_d_ptrs.d_data, 3, (double **)b_d_ptrs.d_data, 3, infoArray, 2);
  hipsolverDnCpotrsBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, 1, (float2 **)a_c_ptrs.d_data, 3, (float2 **)b_c_ptrs.d_data, 3, infoArray, 2);
  hipsolverDnZpotrsBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, 1, (double2 **)a_z_ptrs.d_data, 3, (double2 **)b_z_ptrs.d_data, 3, infoArray, 2);

  a_s.D2H();
  a_d.D2H();
  a_c.D2H();
  a_z.D2H();

  b_s.D2H();
  b_d.D2H();
  b_c.D2H();
  b_z.D2H();

  hipStreamSynchronize(0);

  hipsolverDnDestroy(handle);

  float expect[6] = { 1,2,3,
                      1,2,3 };
  if (compare_result(expect, b_s.h_data, 6) &&
      compare_result(expect, b_d.h_data, 6) &&
      compare_result(expect, b_c.h_data, 6) &&
      compare_result(expect, b_z.h_data, 6))
    printf("DnTpotrsBatched pass\n");
  else {
    printf("DnTpotrsBatched fail\n");
    test_passed = false;
  }
}
#endif

int main() {
#ifndef DPCT_USM_LEVEL_NONE
  test_cusolverDnTpotrfBatched();
  test_cusolverDnTpotrsBatched();
#endif

  if (test_passed)
    return 0;
  return -1;
}
