// ====------ cusparse-helper.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int foo(int aaaaa){
  int m, n, nnz, k, ldb, ldc;
  double alpha;
  const double* csrValA;
  const int* csrRowPtrA;
  const int* csrColIndA;
  const double* x;
  double beta;
  double* y;
  hipsparseHandle_t handle;
  hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseMatDescr_t descrA;

  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)aaaaa);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)aaaaa);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

  hipDoubleComplex alpha_Z, beta_Z, *csrValA_Z, *x_Z, *y_Z;

  hipsparseStatus_t status;

  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

int foo(hipsparseMatDescr_t descrB){}

