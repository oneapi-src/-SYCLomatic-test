#include "hip/hip_runtime.h"
// ===------- cusparse_2.cu -------------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//

#include "hipsparse.h"

#include <cmath>
#include <complex>
#include <cstdio>
#include <vector>

template <class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float *input_data, int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num,
               hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num,
               hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }

private:
  inline void from_float_convert(float *in, d_data_t *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t *in, float *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float *in, float2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float *in, double2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float *expect, float *result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i] - expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float *expect, float *result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]] - expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

const bool run_complex_datatype = true;

void test_cusparseSetGetStream() {
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  hipStream_t stream;
  hipsparseGetStream(handle, &stream);
  hipsparseSetStream(handle, stream);
  hipsparseDestroy(handle);
  printf("SetGetStream pass\n");
}

void test_cusparseTcsrmv_ge() {
  std::vector<float> a_val_vec = {1, 4, 2, 3, 5, 7, 8, 9, 6};
  Data<float> a_s_val(a_val_vec.data(), 9);
  Data<double> a_d_val(a_val_vec.data(), 9);
  Data<float2> a_c_val(a_val_vec.data(), 9);
  Data<double2> a_z_val(a_val_vec.data(), 9);
  std::vector<float> a_row_ptr_vec = {0, 2, 4, 7, 9};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 5);
  std::vector<float> a_col_ind_vec = {0, 1, 1, 2, 0, 3, 4, 2, 4};
  Data<int> a_col_ind(a_col_ind_vec.data(), 9);

  std::vector<float> b_vec = {1, 2, 3, 4, 5};
  Data<float> b_s(b_vec.data(), 5);
  Data<double> b_d(b_vec.data(), 5);
  Data<float2> b_c(b_vec.data(), 5);
  Data<double2> b_z(b_vec.data(), 5);

  Data<float> c_s(4);
  Data<double> c_d(4);
  Data<float2> c_c(4);
  Data<double2> c_z(4);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, (float *)alpha_s.d_data, descrA, (float *)a_s_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float *)b_s.d_data, (float *)beta_s.d_data, (float *)c_s.d_data);
  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, (double *)alpha_d.d_data, descrA, (double *)a_d_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double *)b_d.d_data, (double *)beta_d.d_data, (double *)c_d.d_data);
  if (run_complex_datatype) {
    hipsparseCcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, (float2 *)alpha_c.d_data, descrA, (float2 *)a_c_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float2 *)b_c.d_data, (float2 *)beta_c.d_data, (float2 *)c_c.d_data);
    hipsparseZcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, (double2 *)alpha_z.d_data, descrA, (double2 *)a_z_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double2 *)b_z.d_data, (double2 *)beta_z.d_data, (double2 *)c_z.d_data);
  }

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);

  float expect_c[4] = {90, 130, 730, 570};
  if (compare_result(expect_c, c_s.h_data, 4) &&
      compare_result(expect_c, c_d.h_data, 4) &&
      compare_result(expect_c, c_c.h_data, 4) &&
      compare_result(expect_c, c_z.h_data, 4))
    printf("Tcsrmv_ge pass\n");
  else {
    printf("Tcsrmv_ge fail\n");
    test_passed = false;
  }
}


//  alpha  *  A          *  B      =  C
//     10  * | 1 4 0 1 |    | 1 |    | 130 |
//           | 4 2 3 0 |    | 2 |    | 170 |
//           | 0 3 0 7 |    | 3 |    | 340 |
//           | 1 0 7 0 |    | 4 |    | 220 |
void test_cusparseTcsrmv_sy() {
  std::vector<float> a_val_vec = {1, 4, 1, 4, 2, 3, 3, 7, 1, 7};
  Data<float> a_s_val(a_val_vec.data(), 10);
  Data<double> a_d_val(a_val_vec.data(), 10);
  Data<float2> a_c_val(a_val_vec.data(), 10);
  Data<double2> a_z_val(a_val_vec.data(), 10);
  std::vector<float> a_row_ptr_vec = {0, 3, 6, 8, 10};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 5);
  std::vector<float> a_col_ind_vec = {0, 1, 3, 0, 1, 2, 1, 3, 0, 2};
  Data<int> a_col_ind(a_col_ind_vec.data(), 10);

  std::vector<float> b_vec = {1, 2, 3, 4};
  Data<float> b_s(b_vec.data(), 4);
  Data<double> b_d(b_vec.data(), 4);
  Data<float2> b_c(b_vec.data(), 4);
  Data<double2> b_z(b_vec.data(), 4);

  Data<float> c_s(4);
  Data<double> c_d(4);
  Data<float2> c_c(4);
  Data<double2> c_z(4);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 10, (float *)alpha_s.d_data, descrA, (float *)a_s_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float *)b_s.d_data, (float *)beta_s.d_data, (float *)c_s.d_data);
  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 10, (double *)alpha_d.d_data, descrA, (double *)a_d_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double *)b_d.d_data, (double *)beta_d.d_data, (double *)c_d.d_data);
  if (run_complex_datatype) {
    hipsparseCcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 10, (float2 *)alpha_c.d_data, descrA, (float2 *)a_c_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float2 *)b_c.d_data, (float2 *)beta_c.d_data, (float2 *)c_c.d_data);
    hipsparseZcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 10, (double2 *)alpha_z.d_data, descrA, (double2 *)a_z_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double2 *)b_z.d_data, (double2 *)beta_z.d_data, (double2 *)c_z.d_data);
  }

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);

  float expect_c[4] = {130, 170, 340, 220};
  if (compare_result(expect_c, c_s.h_data, 4) &&
      compare_result(expect_c, c_d.h_data, 4) &&
      compare_result(expect_c, c_c.h_data, 4) &&
      compare_result(expect_c, c_z.h_data, 4))
    printf("Tcsrmv_sy pass\n");
  else {
    printf("Tcsrmv_sy fail\n");
    test_passed = false;
  }
}

//  alpha  *  A          *  B      =  C
//     10  * | 1 4 0 1 |    | 1 |    | 130 |
//           | 0 2 3 0 |    | 2 |    | 130 |
//           | 0 0 0 7 |    | 3 |    | 280 |
//           | 0 0 0 1 |    | 4 |    | 40  |

// Note: this matrix type is not supported in CUDA but supported in oneMKL
void test_cusparseTcsrmv_tr() {
  std::vector<float> a_val_vec = {1, 4, 1, 2, 3, 7, 1};
  Data<float> a_s_val(a_val_vec.data(), 7);
  Data<double> a_d_val(a_val_vec.data(), 7);
  Data<float2> a_c_val(a_val_vec.data(), 7);
  Data<double2> a_z_val(a_val_vec.data(), 7);
  std::vector<float> a_row_ptr_vec = {0, 3, 5, 6, 7};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 5);
  std::vector<float> a_col_ind_vec = {0, 1, 3, 1, 2, 3, 3};
  Data<int> a_col_ind(a_col_ind_vec.data(), 7);

  std::vector<float> b_vec = {1, 2, 3, 4};
  Data<float> b_s(b_vec.data(), 4);
  Data<double> b_d(b_vec.data(), 4);
  Data<float2> b_c(b_vec.data(), 4);
  Data<double2> b_z(b_vec.data(), 4);

  Data<float> c_s(4);
  Data<double> c_d(4);
  Data<float2> c_c(4);
  Data<double2> c_z(4);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);
  hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_UPPER);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 7, (float *)alpha_s.d_data, descrA, (float *)a_s_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float *)b_s.d_data, (float *)beta_s.d_data, (float *)c_s.d_data);
  hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 7, (double *)alpha_d.d_data, descrA, (double *)a_d_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double *)b_d.d_data, (double *)beta_d.d_data, (double *)c_d.d_data);
  if (run_complex_datatype) {
    hipsparseCcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 7, (float2 *)alpha_c.d_data, descrA, (float2 *)a_c_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float2 *)b_c.d_data, (float2 *)beta_c.d_data, (float2 *)c_c.d_data);
    hipsparseZcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 4, 7, (double2 *)alpha_z.d_data, descrA, (double2 *)a_z_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double2 *)b_z.d_data, (double2 *)beta_z.d_data, (double2 *)c_z.d_data);
  }

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);

  float expect_c[4] = {130, 130, 280, 40};
  if (compare_result(expect_c, c_s.h_data, 4) &&
      compare_result(expect_c, c_d.h_data, 4) &&
      compare_result(expect_c, c_c.h_data, 4) &&
      compare_result(expect_c, c_z.h_data, 4))
    printf("Tcsrmv_tr pass\n");
  else {
    printf("Tcsrmv_tr fail\n");
    test_passed = false;
  }
}

void test_cusparseTcsrmm() {
  std::vector<float> a_val_vec = {1, 4, 2, 3, 5, 7, 8, 9, 6};
  Data<float> a_s_val(a_val_vec.data(), 9);
  Data<double> a_d_val(a_val_vec.data(), 9);
  Data<float2> a_c_val(a_val_vec.data(), 9);
  Data<double2> a_z_val(a_val_vec.data(), 9);
  std::vector<float> a_row_ptr_vec = {0, 2, 4, 7, 9};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 5);
  std::vector<float> a_col_ind_vec = {0, 1, 1, 2, 0, 3, 4, 2, 4};
  Data<int> a_col_ind(a_col_ind_vec.data(), 9);

  std::vector<float> b_vec = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  Data<float> b_s(b_vec.data(), 10);
  Data<double> b_d(b_vec.data(), 10);
  Data<float2> b_c(b_vec.data(), 10);
  Data<double2> b_z(b_vec.data(), 10);

  Data<float> c_s(8);
  Data<double> c_d(8);
  Data<float2> c_c(8);
  Data<double2> c_z(8);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 2, 5, 9, (float *)alpha_s.d_data, descrA, (float *)a_s_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float *)b_s.d_data, 5, (float *)beta_s.d_data, (float *)c_s.d_data, 4);
  hipsparseDcsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 2, 5, 9, (double *)alpha_d.d_data, descrA, (double *)a_d_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double *)b_d.d_data, 5, (double *)beta_d.d_data, (double *)c_d.d_data, 4);
  if (run_complex_datatype) {
    hipsparseCcsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 2, 5, 9, (float2 *)alpha_c.d_data, descrA, (float2 *)a_c_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float2 *)b_c.d_data, 5, (float2 *)beta_c.d_data, (float2 *)c_c.d_data, 4);
    hipsparseZcsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 2, 5, 9, (double2 *)alpha_z.d_data, descrA, (double2 *)a_z_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double2 *)b_z.d_data, 5, (double2 *)beta_z.d_data, (double2 *)c_z.d_data, 4);
  }

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);

  float expect_c[8] = {90, 130, 730, 570, 340, 380, 1730, 1320};
  if (compare_result(expect_c, c_s.h_data, 8) &&
      compare_result(expect_c, c_d.h_data, 8) &&
      compare_result(expect_c, c_c.h_data, 8) &&
      compare_result(expect_c, c_z.h_data, 8))
    printf("Tcsrmm pass\n");
  else {
    printf("Tcsrmm fail\n");
    test_passed = false;
  }
}

void test_cusparseTcsrmv_mp() {
  std::vector<float> a_val_vec = {1, 4, 2, 3, 5, 7, 8, 9, 6};
  Data<float> a_s_val(a_val_vec.data(), 9);
  Data<double> a_d_val(a_val_vec.data(), 9);
  Data<float2> a_c_val(a_val_vec.data(), 9);
  Data<double2> a_z_val(a_val_vec.data(), 9);
  std::vector<float> a_row_ptr_vec = {0, 2, 4, 7, 9};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 5);
  std::vector<float> a_col_ind_vec = {0, 1, 1, 2, 0, 3, 4, 2, 4};
  Data<int> a_col_ind(a_col_ind_vec.data(), 9);

  std::vector<float> b_vec = {1, 2, 3, 4, 5};
  Data<float> b_s(b_vec.data(), 5);
  Data<double> b_d(b_vec.data(), 5);
  Data<float2> b_c(b_vec.data(), 5);
  Data<double2> b_z(b_vec.data(), 5);

  Data<float> c_s(4);
  Data<double> c_d(4);
  Data<float2> c_c(4);
  Data<double2> c_z(4);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  cusparseScsrmv_mp(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, (float *)alpha_s.d_data, descrA, (float *)a_s_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float *)b_s.d_data, (float *)beta_s.d_data, (float *)c_s.d_data);
  cusparseDcsrmv_mp(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, (double *)alpha_d.d_data, descrA, (double *)a_d_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double *)b_d.d_data, (double *)beta_d.d_data, (double *)c_d.d_data);
  if (run_complex_datatype) {
    cusparseCcsrmv_mp(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, (float2 *)alpha_c.d_data, descrA, (float2 *)a_c_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (float2 *)b_c.d_data, (float2 *)beta_c.d_data, (float2 *)c_c.d_data);
    cusparseZcsrmv_mp(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, (double2 *)alpha_z.d_data, descrA, (double2 *)a_z_val.d_data, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, (double2 *)b_z.d_data, (double2 *)beta_z.d_data, (double2 *)c_z.d_data);
  }

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);

  float expect_c[4] = {90, 130, 730, 570};
  if (compare_result(expect_c, c_s.h_data, 4) &&
      compare_result(expect_c, c_d.h_data, 4) &&
      compare_result(expect_c, c_c.h_data, 4) &&
      compare_result(expect_c, c_z.h_data, 4))
    printf("Tcsrmv_mp pass\n");
  else {
    printf("Tcsrmv_mp fail\n");
    test_passed = false;
  }
}

void test_cusparseCsrmvEx() {
  std::vector<float> a_val_vec = {1, 4, 2, 3, 5, 7, 8, 9, 6};
  Data<float> a_s_val(a_val_vec.data(), 9);
  Data<double> a_d_val(a_val_vec.data(), 9);
  Data<float2> a_c_val(a_val_vec.data(), 9);
  Data<double2> a_z_val(a_val_vec.data(), 9);
  std::vector<float> a_row_ptr_vec = {0, 2, 4, 7, 9};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 5);
  std::vector<float> a_col_ind_vec = {0, 1, 1, 2, 0, 3, 4, 2, 4};
  Data<int> a_col_ind(a_col_ind_vec.data(), 9);

  std::vector<float> b_vec = {1, 2, 3, 4, 5};
  Data<float> b_s(b_vec.data(), 5);
  Data<double> b_d(b_vec.data(), 5);
  Data<float2> b_c(b_vec.data(), 5);
  Data<double2> b_z(b_vec.data(), 5);

  Data<float> c_s(4);
  Data<double> c_d(4);
  Data<float2> c_c(4);
  Data<double2> c_z(4);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  hipsparseMatDescr_t descrA;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  cusparseAlgMode_t alg;

  size_t ws_size_s;
  size_t ws_size_d;
  size_t ws_size_c;
  size_t ws_size_z;
  cusparseCsrmvEx_bufferSize(handle, alg, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, alpha_s.d_data, HIP_R_32F, descrA, a_s_val.d_data, HIP_R_32F, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, b_s.d_data, HIP_R_32F, beta_s.d_data, HIP_R_32F, c_s.d_data, HIP_R_32F, HIP_R_32F, &ws_size_s);
  cusparseCsrmvEx_bufferSize(handle, alg, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, alpha_d.d_data, HIP_R_64F, descrA, a_d_val.d_data, HIP_R_64F, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, b_d.d_data, HIP_R_64F, beta_d.d_data, HIP_R_64F, c_d.d_data, HIP_R_64F, HIP_R_64F, &ws_size_d);
  if (run_complex_datatype) {
    cusparseCsrmvEx_bufferSize(handle, alg, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, alpha_c.d_data, HIP_C_32F, descrA, a_c_val.d_data, HIP_C_32F, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, b_c.d_data, HIP_C_32F, beta_c.d_data, HIP_C_32F, c_c.d_data, HIP_C_32F, HIP_C_32F, &ws_size_c);
    cusparseCsrmvEx_bufferSize(handle, alg, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, alpha_z.d_data, HIP_C_64F, descrA, a_z_val.d_data, HIP_C_64F, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, b_z.d_data, HIP_C_64F, beta_z.d_data, HIP_C_64F, c_z.d_data, HIP_C_64F, HIP_C_64F, &ws_size_z);
  }

  void *ws_s;
  void *ws_d;
  void *ws_c;
  void *ws_z;
  hipMalloc(&ws_s, ws_size_s);
  hipMalloc(&ws_d, ws_size_d);
  hipMalloc(&ws_c, ws_size_c);
  hipMalloc(&ws_z, ws_size_z);

  cusparseCsrmvEx(handle, alg, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, alpha_s.d_data, HIP_R_32F, descrA, a_s_val.d_data, HIP_R_32F, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, b_s.d_data, HIP_R_32F, beta_s.d_data, HIP_R_32F, c_s.d_data, HIP_R_32F, HIP_R_32F, ws_s);
  cusparseCsrmvEx(handle, alg, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, alpha_d.d_data, HIP_R_64F, descrA, a_d_val.d_data, HIP_R_64F, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, b_d.d_data, HIP_R_64F, beta_d.d_data, HIP_R_64F, c_d.d_data, HIP_R_64F, HIP_R_64F, ws_d);
  if (run_complex_datatype) {
    cusparseCsrmvEx(handle, alg, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, alpha_c.d_data, HIP_C_32F, descrA, a_c_val.d_data, HIP_C_32F, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, b_c.d_data, HIP_C_32F, beta_c.d_data, HIP_C_32F, c_c.d_data, HIP_C_32F, HIP_C_32F, ws_c);
    cusparseCsrmvEx(handle, alg, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 5, 9, alpha_z.d_data, HIP_C_64F, descrA, a_z_val.d_data, HIP_C_64F, (int *)a_row_ptr.d_data, (int *)a_col_ind.d_data, b_z.d_data, HIP_C_64F, beta_z.d_data, HIP_C_64F, c_z.d_data, HIP_C_64F, HIP_C_64F, ws_z);
  }

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipFree(ws_s);
  hipFree(ws_d);
  hipFree(ws_c);
  hipFree(ws_z);
  hipStreamSynchronize(0);
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);

  float expect_c[4] = {90, 130, 730, 570};
  if (compare_result(expect_c, c_s.h_data, 4) &&
      compare_result(expect_c, c_d.h_data, 4) &&
      compare_result(expect_c, c_c.h_data, 4) &&
      compare_result(expect_c, c_z.h_data, 4))
    printf("CsrmvEx pass\n");
  else {
    printf("CsrmvEx fail\n");
    test_passed = false;
  }
}

int main() {
  test_cusparseSetGetStream();
  test_cusparseTcsrmv_ge();
  test_cusparseTcsrmv_sy();
  // test_cusparseTcsrmv_tr();
  // test_cusparseTcsrmm(); // Re-enable this test until MKL issue fixed
  test_cusparseTcsrmv_mp();
  test_cusparseCsrmvEx();

  if (test_passed)
    return 0;
  return -1;
}
