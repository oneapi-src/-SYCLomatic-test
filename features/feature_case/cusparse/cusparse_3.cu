// ===------- cusparse_3.cu -------------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include <cmath>
#include <complex>
#include <cstdio>
#include <vector>

template <class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float *input_data, int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num,
               hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num,
               hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }

private:
  inline void from_float_convert(float *in, d_data_t *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t *in, float *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float *in, float2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float *in, double2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float *expect, float *result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i] - expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float *expect, float *result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]] - expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

void test_cusparseSpMV() {
  std::vector<float> a_val_vec = {1, 4, 2, 3, 5, 7, 8, 9, 6};
  Data<float> a_s_val(a_val_vec.data(), 9);
  Data<double> a_d_val(a_val_vec.data(), 9);
  Data<float2> a_c_val(a_val_vec.data(), 9);
  Data<double2> a_z_val(a_val_vec.data(), 9);
  std::vector<float> a_row_ptr_vec = {0, 2, 4, 7, 9};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 5);
  std::vector<float> a_col_ind_vec = {0, 1, 1, 2, 0, 3, 4, 2, 4};
  Data<int> a_col_ind(a_col_ind_vec.data(), 9);

  std::vector<float> b_vec = {1, 2, 3, 4, 5};
  Data<float> b_s(b_vec.data(), 5);
  Data<double> b_d(b_vec.data(), 5);
  Data<float2> b_c(b_vec.data(), 5);
  Data<double2> b_z(b_vec.data(), 5);

  Data<float> c_s(4);
  Data<double> c_d(4);
  Data<float2> c_c(4);
  Data<double2> c_z(4);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  hipsparseSpMatDescr_t a_descr_s;
  hipsparseSpMatDescr_t a_descr_d;
  hipsparseSpMatDescr_t a_descr_c;
  hipsparseSpMatDescr_t a_descr_z;
  hipsparseCreateCsr(&a_descr_s, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_s_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
  hipsparseCreateCsr(&a_descr_d, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_d_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
  hipsparseCreateCsr(&a_descr_c, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_c_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
  hipsparseCreateCsr(&a_descr_z, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_z_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

  hipsparseDnVecDescr_t b_descr_s;
  hipsparseDnVecDescr_t b_descr_d;
  hipsparseDnVecDescr_t b_descr_c;
  hipsparseDnVecDescr_t b_descr_z;
  hipsparseCreateDnVec(&b_descr_s, 5, b_s.d_data, HIP_R_32F);
  hipsparseCreateDnVec(&b_descr_d, 5, b_d.d_data, HIP_R_64F);
  hipsparseCreateDnVec(&b_descr_c, 5, b_c.d_data, HIP_C_32F);
  hipsparseCreateDnVec(&b_descr_z, 5, b_z.d_data, HIP_C_64F);

  hipsparseDnVecDescr_t c_descr_s;
  hipsparseDnVecDescr_t c_descr_d;
  hipsparseDnVecDescr_t c_descr_c;
  hipsparseDnVecDescr_t c_descr_z;
  hipsparseCreateDnVec(&c_descr_s, 4, c_s.d_data, HIP_R_32F);
  hipsparseCreateDnVec(&c_descr_d, 4, c_d.d_data, HIP_R_64F);
  hipsparseCreateDnVec(&c_descr_c, 4, c_c.d_data, HIP_C_32F);
  hipsparseCreateDnVec(&c_descr_z, 4, c_z.d_data, HIP_C_64F);

  size_t ws_size_s;
  size_t ws_size_d;
  size_t ws_size_c;
  size_t ws_size_z;
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, beta_s.d_data, c_descr_s, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &ws_size_s);
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, beta_d.d_data, c_descr_d, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &ws_size_d);
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, beta_c.d_data, c_descr_c, HIP_C_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &ws_size_c);
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, beta_z.d_data, c_descr_z, HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &ws_size_z);

  void *ws_s = nullptr;
  void *ws_d = nullptr;
  void *ws_c = nullptr;
  void *ws_z = nullptr;
  hipMalloc(&ws_s, ws_size_s);
  hipMalloc(&ws_d, ws_size_d);
  hipMalloc(&ws_c, ws_size_c);
  hipMalloc(&ws_z, ws_size_z);

  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, beta_s.d_data, c_descr_s, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, ws_s);
  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, beta_d.d_data, c_descr_d, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, ws_d);
  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, beta_c.d_data, c_descr_c, HIP_C_32F, HIPSPARSE_SPMV_ALG_DEFAULT, ws_c);
  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, beta_z.d_data, c_descr_z, HIP_C_64F, HIPSPARSE_SPMV_ALG_DEFAULT, ws_z);

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);

  hipFree(ws_s);
  hipFree(ws_d);
  hipFree(ws_c);
  hipFree(ws_z);
  hipsparseDestroySpMat(a_descr_s);
  hipsparseDestroySpMat(a_descr_d);
  hipsparseDestroySpMat(a_descr_c);
  hipsparseDestroySpMat(a_descr_z);
  hipsparseDestroyDnVec(b_descr_s);
  hipsparseDestroyDnVec(b_descr_d);
  hipsparseDestroyDnVec(b_descr_c);
  hipsparseDestroyDnVec(b_descr_z);
  hipsparseDestroyDnVec(c_descr_s);
  hipsparseDestroyDnVec(c_descr_d);
  hipsparseDestroyDnVec(c_descr_c);
  hipsparseDestroyDnVec(c_descr_z);
  hipsparseDestroy(handle);

  float expect_c[4] = {90, 130, 730, 570};
  if (compare_result(expect_c, c_s.h_data, 4) &&
      compare_result(expect_c, c_d.h_data, 4) &&
      compare_result(expect_c, c_c.h_data, 4) &&
      compare_result(expect_c, c_z.h_data, 4))
    printf("SpMV pass\n");
  else {
    printf("SpMV fail\n");
    test_passed = false;
  }
}

void test_cusparseSpMM() {
  std::vector<float> a_val_vec = {1, 4, 2, 3, 5, 7, 8, 9, 6};
  Data<float> a_s_val(a_val_vec.data(), 9);
  Data<double> a_d_val(a_val_vec.data(), 9);
  Data<float2> a_c_val(a_val_vec.data(), 9);
  Data<double2> a_z_val(a_val_vec.data(), 9);
  std::vector<float> a_row_ptr_vec = {0, 2, 4, 7, 9};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 5);
  std::vector<float> a_col_ind_vec = {0, 1, 1, 2, 0, 3, 4, 2, 4};
  Data<int> a_col_ind(a_col_ind_vec.data(), 9);

  std::vector<float> b_vec = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  Data<float> b_s(b_vec.data(), 10);
  Data<double> b_d(b_vec.data(), 10);
  Data<float2> b_c(b_vec.data(), 10);
  Data<double2> b_z(b_vec.data(), 10);

  Data<float> c_s(8);
  Data<double> c_d(8);
  Data<float2> c_c(8);
  Data<double2> c_z(8);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  hipsparseSpMatDescr_t a_descr_s;
  hipsparseSpMatDescr_t a_descr_d;
  hipsparseSpMatDescr_t a_descr_c;
  hipsparseSpMatDescr_t a_descr_z;
  hipsparseCreateCsr(&a_descr_s, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_s_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
  hipsparseCreateCsr(&a_descr_d, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_d_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
  hipsparseCreateCsr(&a_descr_c, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_c_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
  hipsparseCreateCsr(&a_descr_z, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_z_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

  hipsparseDnMatDescr_t b_descr_s;
  hipsparseDnMatDescr_t b_descr_d;
  hipsparseDnMatDescr_t b_descr_c;
  hipsparseDnMatDescr_t b_descr_z;
  hipsparseCreateDnMat(&b_descr_s, 5, 2, 5, b_s.d_data, HIP_R_32F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&b_descr_d, 5, 2, 5, b_d.d_data, HIP_R_64F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&b_descr_c, 5, 2, 5, b_c.d_data, HIP_C_32F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&b_descr_z, 5, 2, 5, b_z.d_data, HIP_C_64F, HIPSPARSE_ORDER_COL);

  hipsparseDnMatDescr_t c_descr_s;
  hipsparseDnMatDescr_t c_descr_d;
  hipsparseDnMatDescr_t c_descr_c;
  hipsparseDnMatDescr_t c_descr_z;
  hipsparseCreateDnMat(&c_descr_s, 4, 2, 4, c_s.d_data, HIP_R_32F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&c_descr_d, 4, 2, 4, c_d.d_data, HIP_R_64F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&c_descr_c, 4, 2, 4, c_c.d_data, HIP_C_32F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&c_descr_z, 4, 2, 4, c_z.d_data, HIP_C_64F, HIPSPARSE_ORDER_COL);

  size_t ws_size_s;
  size_t ws_size_d;
  size_t ws_size_c;
  size_t ws_size_z;
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, beta_s.d_data, c_descr_s, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &ws_size_s);
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, beta_d.d_data, c_descr_d, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &ws_size_d);
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, beta_c.d_data, c_descr_c, HIP_C_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &ws_size_c);
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, beta_z.d_data, c_descr_z, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &ws_size_z);

  void *ws_s = nullptr;
  void *ws_d = nullptr;
  void *ws_c = nullptr;
  void *ws_z = nullptr;
  hipMalloc(&ws_s, ws_size_s);
  hipMalloc(&ws_d, ws_size_d);
  hipMalloc(&ws_c, ws_size_c);
  hipMalloc(&ws_z, ws_size_z);

  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, beta_s.d_data, c_descr_s, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_s);
  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, beta_d.d_data, c_descr_d, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_d);
  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, beta_c.d_data, c_descr_c, HIP_C_32F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_c);
  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, beta_z.d_data, c_descr_z, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_z);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, beta_s.d_data, c_descr_s, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_s);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, beta_d.d_data, c_descr_d, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_d);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, beta_c.d_data, c_descr_c, HIP_C_32F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_c);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, beta_z.d_data, c_descr_z, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_z);

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);

  hipFree(ws_s);
  hipFree(ws_d);
  hipFree(ws_c);
  hipFree(ws_z);
  hipsparseDestroySpMat(a_descr_s);
  hipsparseDestroySpMat(a_descr_d);
  hipsparseDestroySpMat(a_descr_c);
  hipsparseDestroySpMat(a_descr_z);
  hipsparseDestroyDnMat(b_descr_s);
  hipsparseDestroyDnMat(b_descr_d);
  hipsparseDestroyDnMat(b_descr_c);
  hipsparseDestroyDnMat(b_descr_z);
  hipsparseDestroyDnMat(c_descr_s);
  hipsparseDestroyDnMat(c_descr_d);
  hipsparseDestroyDnMat(c_descr_c);
  hipsparseDestroyDnMat(c_descr_z);
  hipsparseDestroy(handle);

  float expect_c[8] = {90, 130, 730, 570, 340, 380, 1730, 1320};
  if (compare_result(expect_c, c_s.h_data, 8) &&
      compare_result(expect_c, c_d.h_data, 8) &&
      compare_result(expect_c, c_c.h_data, 8) &&
      compare_result(expect_c, c_z.h_data, 8))
    printf("SpMM pass\n");
  else {
    printf("SpMM fail\n");
    test_passed = false;
  }
}

int main() {
  test_cusparseSpMV();
  test_cusparseSpMM();

  if (test_passed)
    return 0;
  return -1;
}
