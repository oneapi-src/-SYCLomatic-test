// ===------- cusparse_10.cu ------------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include <cmath>
#include <complex>
#include <cstdio>
#include <vector>

template <class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float *input_data, int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num,
               hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num,
               hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }

private:
  inline void from_float_convert(float *in, d_data_t *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t *in, float *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float *in, float2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float *in, double2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float *expect, float *result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i] - expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float *expect, float *result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]] - expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

// A
// 1 4 0 0 0
// 0 2 3 0 0
// 5 0 7 8 0
// 0 0 9 0 6
void test_cusparseSpMM_COO() {
  std::vector<float> a_val_vec = {1, 4, 2, 3, 5, 7, 8, 9, 6};
  Data<float> a_s_val(a_val_vec.data(), 9);
  Data<double> a_d_val(a_val_vec.data(), 9);
  Data<float2> a_c_val(a_val_vec.data(), 9);
  Data<double2> a_z_val(a_val_vec.data(), 9);
  std::vector<float> a_row_ptr_vec = {1, 1, 2, 2, 3, 3, 3, 4, 4};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 9);
  std::vector<float> a_col_ind_vec = {1, 2, 2, 3, 1, 4, 5, 3, 5};
  Data<int> a_col_ind(a_col_ind_vec.data(), 9);

  std::vector<float> b_vec = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  Data<float> b_s(b_vec.data(), 10);
  Data<double> b_d(b_vec.data(), 10);
  Data<float2> b_c(b_vec.data(), 10);
  Data<double2> b_z(b_vec.data(), 10);

  Data<float> c_s(8);
  Data<double> c_d(8);
  Data<float2> c_c(8);
  Data<double2> c_z(8);

  float alpha = 10;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  float beta = 0;
  Data<float> beta_s(&beta, 1);
  Data<double> beta_d(&beta, 1);
  Data<float2> beta_c(&beta, 1);
  Data<double2> beta_z(&beta, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();
  beta_s.H2D();
  beta_d.H2D();
  beta_c.H2D();
  beta_z.H2D();

  hipsparseSpMatDescr_t a_descr_s;
  hipsparseSpMatDescr_t a_descr_d;
  hipsparseSpMatDescr_t a_descr_c;
  hipsparseSpMatDescr_t a_descr_z;
  hipsparseCreateCoo(&a_descr_s, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_s_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F);
  hipsparseCreateCoo(&a_descr_d, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_d_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_64F);
  hipsparseCreateCoo(&a_descr_c, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_c_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_C_32F);
  hipsparseCreateCoo(&a_descr_z, 4, 5, 9, a_row_ptr.d_data, a_col_ind.d_data, a_z_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_C_64F);

  hipsparseDnMatDescr_t b_descr_s;
  hipsparseDnMatDescr_t b_descr_d;
  hipsparseDnMatDescr_t b_descr_c;
  hipsparseDnMatDescr_t b_descr_z;
  hipsparseCreateDnMat(&b_descr_s, 5, 2, 5, b_s.d_data, HIP_R_32F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&b_descr_d, 5, 2, 5, b_d.d_data, HIP_R_64F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&b_descr_c, 5, 2, 5, b_c.d_data, HIP_C_32F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&b_descr_z, 5, 2, 5, b_z.d_data, HIP_C_64F, HIPSPARSE_ORDER_COL);

  hipsparseDnMatDescr_t c_descr_s;
  hipsparseDnMatDescr_t c_descr_d;
  hipsparseDnMatDescr_t c_descr_c;
  hipsparseDnMatDescr_t c_descr_z;
  hipsparseCreateDnMat(&c_descr_s, 4, 2, 4, c_s.d_data, HIP_R_32F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&c_descr_d, 4, 2, 4, c_d.d_data, HIP_R_64F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&c_descr_c, 4, 2, 4, c_c.d_data, HIP_C_32F, HIPSPARSE_ORDER_COL);
  hipsparseCreateDnMat(&c_descr_z, 4, 2, 4, c_z.d_data, HIP_C_64F, HIPSPARSE_ORDER_COL);

  size_t ws_size_s;
  size_t ws_size_d;
  size_t ws_size_c;
  size_t ws_size_z;
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, beta_s.d_data, c_descr_s, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &ws_size_s);
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, beta_d.d_data, c_descr_d, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &ws_size_d);
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, beta_c.d_data, c_descr_c, HIP_C_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &ws_size_c);
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, beta_z.d_data, c_descr_z, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &ws_size_z);

  void *ws_s = nullptr;
  void *ws_d = nullptr;
  void *ws_c = nullptr;
  void *ws_z = nullptr;
  hipMalloc(&ws_s, ws_size_s);
  hipMalloc(&ws_d, ws_size_d);
  hipMalloc(&ws_c, ws_size_c);
  hipMalloc(&ws_z, ws_size_z);

  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, beta_s.d_data, c_descr_s, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_s);
  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, beta_d.d_data, c_descr_d, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_d);
  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, beta_c.d_data, c_descr_c, HIP_C_32F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_c);
  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, beta_z.d_data, c_descr_z, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_z);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, beta_s.d_data, c_descr_s, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_s);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, beta_d.d_data, c_descr_d, HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_d);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, beta_c.d_data, c_descr_c, HIP_C_32F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_c);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, beta_z.d_data, c_descr_z, HIP_C_64F, HIPSPARSE_SPMM_ALG_DEFAULT, ws_z);

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);

  hipFree(ws_s);
  hipFree(ws_d);
  hipFree(ws_c);
  hipFree(ws_z);
  hipsparseDestroySpMat(a_descr_s);
  hipsparseDestroySpMat(a_descr_d);
  hipsparseDestroySpMat(a_descr_c);
  hipsparseDestroySpMat(a_descr_z);
  hipsparseDestroyDnMat(b_descr_s);
  hipsparseDestroyDnMat(b_descr_d);
  hipsparseDestroyDnMat(b_descr_c);
  hipsparseDestroyDnMat(b_descr_z);
  hipsparseDestroyDnMat(c_descr_s);
  hipsparseDestroyDnMat(c_descr_d);
  hipsparseDestroyDnMat(c_descr_c);
  hipsparseDestroyDnMat(c_descr_z);
  hipsparseDestroy(handle);

  float expect_c[8] = {90, 130, 730, 570, 340, 380, 1730, 1320};
  if (compare_result(expect_c, c_s.h_data, 8) &&
      compare_result(expect_c, c_d.h_data, 8) &&
      compare_result(expect_c, c_c.h_data, 8) &&
      compare_result(expect_c, c_z.h_data, 8))
    printf("SpMM_COO pass\n");
  else {
    printf("SpMM_COO fail\n");
    test_passed = false;
  }
}

int main() {
  test_cusparseSpMM_COO();

  if (test_passed)
    return 0;
  return -1;
}
