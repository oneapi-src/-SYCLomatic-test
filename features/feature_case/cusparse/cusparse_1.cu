// ===------- cusparse_1.cu -------------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <cstdio>

bool test1() {
  hipsparseMatDescr_t descr;
  hipsparseMatrixType_t mt;
  hipsparseDiagType_t dt;
  hipsparseFillMode_t fm;
  hipsparseIndexBase_t ib;

  hipsparseCreateMatDescr(&descr);

  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  mt = hipsparseGetMatType(descr);
  if (mt != HIPSPARSE_MATRIX_TYPE_GENERAL)
    return false;

  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
  mt = hipsparseGetMatType(descr);
  if (mt != HIPSPARSE_MATRIX_TYPE_SYMMETRIC)
    return false;

  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_HERMITIAN);
  mt = hipsparseGetMatType(descr);
  if (mt != HIPSPARSE_MATRIX_TYPE_HERMITIAN)
    return false;

  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);
  mt = hipsparseGetMatType(descr);
  if (mt != HIPSPARSE_MATRIX_TYPE_TRIANGULAR)
    return false;

  hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_NON_UNIT);
  dt = hipsparseGetMatDiagType(descr);
  if (dt != HIPSPARSE_DIAG_TYPE_NON_UNIT)
    return false;

  hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_UNIT);
  dt = hipsparseGetMatDiagType(descr);
  if (dt != HIPSPARSE_DIAG_TYPE_UNIT)
    return false;

  hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_LOWER);
  fm = hipsparseGetMatFillMode(descr);
  if (fm != HIPSPARSE_FILL_MODE_LOWER)
    return false;

  hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_UPPER);
  fm = hipsparseGetMatFillMode(descr);
  if (fm != HIPSPARSE_FILL_MODE_UPPER)
    return false;

  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
  ib = hipsparseGetMatIndexBase(descr);
  if (ib != HIPSPARSE_INDEX_BASE_ZERO)
    return false;

  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ONE);
  ib = hipsparseGetMatIndexBase(descr);
  if (ib != HIPSPARSE_INDEX_BASE_ONE)
    return false;

  hipsparseDestroyMatDescr(descr);
  return true;
}

int main() {
  bool res = true;

  if ((res = test1())) {
    printf("test1 passed\n");
  } else {
    printf("test1 failed\n");
  }

  return 0;
}
