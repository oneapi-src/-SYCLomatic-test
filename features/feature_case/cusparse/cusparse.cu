// ====------ cusparse.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
float alpha;
const float* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const float* x;
float beta;
float* y;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int main(){
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;
  hipStream_t s;

  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_HOST;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  constexpr int zero = 0;
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)zero);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)zero);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)zero);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)zero);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  hipsparseCreate(&handle);
  hipsparseSetStream(handle,s);
  hipsparseGetStream(handle,&s);

  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseScsrmv(handle, (hipsparseOperation_t)zero, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipComplex alpha_C, beta_C, *csrValA_C, *x_C, *y_C;

  hipsparseCcsrmv(handle, transA, m, n, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);

  hipsparseScsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  hipsparseCcsrmm(handle, transA, m, n, k, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);

  hipsparseStatus_t status;

  if(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  for(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  switch(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseScsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDestroySolveAnalysisInfo(info);

  cusparseCcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_C, csrRowPtrA, csrColIndA, info);

  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

int foo(hipsparseMatDescr_t descrB){
  return hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

void foo2(hipsparseMatDescr_t descrB){
  hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}