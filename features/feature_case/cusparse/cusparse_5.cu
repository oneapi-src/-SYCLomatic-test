// ===------- cusparse_3.cu -------------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include <cmath>
#include <complex>
#include <cstdio>
#include <vector>

template <class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float *input_data, int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num,
               hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num,
               hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }

private:
  inline void from_float_convert(float *in, d_data_t *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t *in, float *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float *in, float2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float *in, double2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float *expect, float *result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i] - expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float *expect, float *result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]] - expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

const bool run_complex_datatype = true;

// A * C = B
//
// | 1 1 2 |   | 1 |   | 9  |  
// | 0 1 3 | * | 2 | = | 11 |
// | 0 0 1 |   | 3 |   | 3  |
void test_cusparseSpSV() {
  std::vector<float> a_val_vec = {1, 1, 2, 1, 3, 1};
  Data<float> a_s_val(a_val_vec.data(), 6);
  Data<double> a_d_val(a_val_vec.data(), 6);
  Data<float2> a_c_val(a_val_vec.data(), 6);
  Data<double2> a_z_val(a_val_vec.data(), 6);
  std::vector<float> a_row_ptr_vec = {0, 3, 5, 6};
  Data<int> a_row_ptr(a_row_ptr_vec.data(), 4);
  std::vector<float> a_col_ind_vec = {0, 1, 2, 1, 2, 2};
  Data<int> a_col_ind(a_col_ind_vec.data(), 6);

  std::vector<float> b_vec = {9, 11, 3};
  Data<float> b_s(b_vec.data(), 3);
  Data<double> b_d(b_vec.data(), 3);
  Data<float2> b_c(b_vec.data(), 3);
  Data<double2> b_z(b_vec.data(), 3);

  Data<float> c_s(3);
  Data<double> c_d(3);
  Data<float2> c_c(3);
  Data<double2> c_z(3);

  float alpha = 1;
  Data<float> alpha_s(&alpha, 1);
  Data<double> alpha_d(&alpha, 1);
  Data<float2> alpha_c(&alpha, 1);
  Data<double2> alpha_z(&alpha, 1);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_row_ptr.H2D();
  a_col_ind.H2D();
  b_s.H2D();
  b_d.H2D();
  b_c.H2D();
  b_z.H2D();
  alpha_s.H2D();
  alpha_d.H2D();
  alpha_c.H2D();
  alpha_z.H2D();

  hipsparseSpMatDescr_t a_descr_s;
  hipsparseSpMatDescr_t a_descr_d;
  hipsparseSpMatDescr_t a_descr_c;
  hipsparseSpMatDescr_t a_descr_z;
  hipsparseCreateCsr(&a_descr_s, 3, 3, 4, a_row_ptr.d_data, a_col_ind.d_data, a_s_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
  hipsparseCreateCsr(&a_descr_d, 3, 3, 4, a_row_ptr.d_data, a_col_ind.d_data, a_d_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
  hipsparseCreateCsr(&a_descr_c, 3, 3, 4, a_row_ptr.d_data, a_col_ind.d_data, a_c_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_32F);
  hipsparseCreateCsr(&a_descr_z, 3, 3, 4, a_row_ptr.d_data, a_col_ind.d_data, a_z_val.d_data, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F);

  hipsparseDnVecDescr_t b_descr_s;
  hipsparseDnVecDescr_t b_descr_d;
  hipsparseDnVecDescr_t b_descr_c;
  hipsparseDnVecDescr_t b_descr_z;
  hipsparseCreateDnVec(&b_descr_s, 3, b_s.d_data, HIP_R_32F);
  hipsparseCreateDnVec(&b_descr_d, 3, b_d.d_data, HIP_R_64F);
  hipsparseCreateDnVec(&b_descr_c, 3, b_c.d_data, HIP_C_32F);
  hipsparseCreateDnVec(&b_descr_z, 3, b_z.d_data, HIP_C_64F);

  hipsparseDnVecDescr_t c_descr_s;
  hipsparseDnVecDescr_t c_descr_d;
  hipsparseDnVecDescr_t c_descr_c;
  hipsparseDnVecDescr_t c_descr_z;
  hipsparseCreateDnVec(&c_descr_s, 3, c_s.d_data, HIP_R_32F);
  hipsparseCreateDnVec(&c_descr_d, 3, c_d.d_data, HIP_R_64F);
  hipsparseCreateDnVec(&c_descr_c, 3, c_c.d_data, HIP_C_32F);
  hipsparseCreateDnVec(&c_descr_z, 3, c_z.d_data, HIP_C_64F);

  hipsparseFillMode_t uplo = HIPSPARSE_FILL_MODE_UPPER;
  hipsparseSpMatSetAttribute(a_descr_s, HIPSPARSE_SPMAT_FILL_MODE, &uplo, sizeof(uplo));
  hipsparseSpMatSetAttribute(a_descr_d, HIPSPARSE_SPMAT_FILL_MODE, &uplo, sizeof(uplo));
  hipsparseSpMatSetAttribute(a_descr_c, HIPSPARSE_SPMAT_FILL_MODE, &uplo, sizeof(uplo));
  hipsparseSpMatSetAttribute(a_descr_z, HIPSPARSE_SPMAT_FILL_MODE, &uplo, sizeof(uplo));
  hipsparseDiagType_t diag = HIPSPARSE_DIAG_TYPE_UNIT;
  hipsparseSpMatSetAttribute(a_descr_s, HIPSPARSE_SPMAT_DIAG_TYPE, &diag, sizeof(diag));
  hipsparseSpMatSetAttribute(a_descr_d, HIPSPARSE_SPMAT_DIAG_TYPE, &diag, sizeof(diag));
  hipsparseSpMatSetAttribute(a_descr_c, HIPSPARSE_SPMAT_DIAG_TYPE, &diag, sizeof(diag));
  hipsparseSpMatSetAttribute(a_descr_z, HIPSPARSE_SPMAT_DIAG_TYPE, &diag, sizeof(diag));

  hipsparseSpSVDescr_t SpSVDescr_s;
  hipsparseSpSVDescr_t SpSVDescr_d;
  hipsparseSpSVDescr_t SpSVDescr_c;
  hipsparseSpSVDescr_t SpSVDescr_z;
  hipsparseSpSV_createDescr(&SpSVDescr_s);
  hipsparseSpSV_createDescr(&SpSVDescr_d);
  hipsparseSpSV_createDescr(&SpSVDescr_c);
  hipsparseSpSV_createDescr(&SpSVDescr_z);

  size_t ws_size_s = 0;
  size_t ws_size_d = 0;
  size_t ws_size_c = 0;
  size_t ws_size_z = 0;
  hipsparseSpSV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, c_descr_s, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_s, &ws_size_s);
  hipsparseSpSV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, c_descr_d, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_d, &ws_size_d);
  if (run_complex_datatype) {
    hipsparseSpSV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, c_descr_c, HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_c, &ws_size_c);
    hipsparseSpSV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, c_descr_z, HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_z, &ws_size_z);
  }

  void *ws_s;
  void *ws_d;
  void *ws_c;
  void *ws_z;
  hipMalloc(&ws_s, ws_size_s);
  hipMalloc(&ws_d, ws_size_d);
  hipMalloc(&ws_c, ws_size_c);
  hipMalloc(&ws_z, ws_size_z);

  hipsparseSpSV_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, c_descr_s, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_s, &ws_size_s);
  hipsparseSpSV_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, c_descr_d, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_d, &ws_size_d);
  if (run_complex_datatype) {
    hipsparseSpSV_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, c_descr_c, HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_c, &ws_size_c);
    hipsparseSpSV_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, c_descr_z, HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_z, &ws_size_z);
  }

  hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_s.d_data, a_descr_s, b_descr_s, c_descr_s, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_s);
  hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_d.d_data, a_descr_d, b_descr_d, c_descr_d, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_d);
  if (run_complex_datatype) {
    hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_c.d_data, a_descr_c, b_descr_c, c_descr_c, HIP_C_32F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_c);
    hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha_z.d_data, a_descr_z, b_descr_z, c_descr_z, HIP_C_64F, HIPSPARSE_SPSV_ALG_DEFAULT, SpSVDescr_z);
  }

  c_s.D2H();
  c_d.D2H();
  c_c.D2H();
  c_z.D2H();

  hipStreamSynchronize(0);

  hipFree(ws_s);
  hipFree(ws_d);
  hipFree(ws_c);
  hipFree(ws_z);
  hipsparseDestroySpMat(a_descr_s);
  hipsparseDestroySpMat(a_descr_d);
  hipsparseDestroySpMat(a_descr_c);
  hipsparseDestroySpMat(a_descr_z);
  hipsparseDestroyDnVec(b_descr_s);
  hipsparseDestroyDnVec(b_descr_d);
  hipsparseDestroyDnVec(b_descr_c);
  hipsparseDestroyDnVec(b_descr_z);
  hipsparseDestroyDnVec(c_descr_s);
  hipsparseDestroyDnVec(c_descr_d);
  hipsparseDestroyDnVec(c_descr_c);
  hipsparseDestroyDnVec(c_descr_z);
  hipsparseSpSV_destroyDescr(SpSVDescr_s);
  hipsparseSpSV_destroyDescr(SpSVDescr_d);
  hipsparseSpSV_destroyDescr(SpSVDescr_c);
  hipsparseSpSV_destroyDescr(SpSVDescr_z);
  hipsparseDestroy(handle);

  float expect_c[4] = {1, 2, 3};
  if (compare_result(expect_c, c_s.h_data, 3) &&
      compare_result(expect_c, c_d.h_data, 3) &&
      compare_result(expect_c, c_c.h_data, 3) &&
      compare_result(expect_c, c_z.h_data, 3))
    printf("SpSV pass\n");
  else {
    printf("SpSV fail\n");
    test_passed = false;
  }
}

int main() {
  test_cusparseSpSV();

  if (test_passed)
    return 0;
  return -1;
}
