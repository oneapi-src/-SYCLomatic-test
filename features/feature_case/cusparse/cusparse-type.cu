// ====------ cusparse-type.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int main(){
  hipsparseFillMode_t a1;
  a1 = HIPSPARSE_FILL_MODE_LOWER;
  a1 = HIPSPARSE_FILL_MODE_UPPER;

  hipsparseDiagType_t a2;
  a2 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  a2 = HIPSPARSE_DIAG_TYPE_UNIT;

  hipsparseIndexBase_t a3;
  a3 = HIPSPARSE_INDEX_BASE_ZERO;
  a3 = HIPSPARSE_INDEX_BASE_ONE;

  hipsparseMatrixType_t a4;
  a4 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  a4 = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  a4 = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  a4 = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  hipsparseOperation_t a5;
  a5 = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  a5 = HIPSPARSE_OPERATION_TRANSPOSE;
  a5 = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  hipsparseStatus_t a6;
  a6 = HIPSPARSE_STATUS_SUCCESS;
  a6 = HIPSPARSE_STATUS_NOT_INITIALIZED;
  a6 = HIPSPARSE_STATUS_ALLOC_FAILED;
  a6 = HIPSPARSE_STATUS_INVALID_VALUE;
  a6 = HIPSPARSE_STATUS_ARCH_MISMATCH;
  a6 = HIPSPARSE_STATUS_MAPPING_ERROR;
  a6 = HIPSPARSE_STATUS_EXECUTION_FAILED;
  a6 = HIPSPARSE_STATUS_INTERNAL_ERROR;
  a6 = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  a6 = HIPSPARSE_STATUS_ZERO_PIVOT;

  hipsparseMatDescr_t a7;

  hipsparseHandle_t a8;
}

void foo(hipsparseFillMode_t a1,
         hipsparseDiagType_t a2,
         hipsparseIndexBase_t a3,
         hipsparseMatrixType_t a4,
         hipsparseOperation_t a5,
         hipsparseStatus_t a6,
         hipsparseMatDescr_t a7,
         hipsparseHandle_t a8);

hipsparseFillMode_t foo1();
hipsparseDiagType_t foo2();
hipsparseIndexBase_t foo3();
hipsparseMatrixType_t foo4();
hipsparseOperation_t foo5();
hipsparseStatus_t foo6();
hipsparseMatDescr_t foo7();
hipsparseHandle_t foo8();

template<typename T>
void bar1(hipsparseFillMode_t a1,
         hipsparseDiagType_t a2,
         hipsparseIndexBase_t a3,
         hipsparseMatrixType_t a4,
         hipsparseOperation_t a5,
         hipsparseStatus_t a6,
         hipsparseMatDescr_t a7,
         hipsparseHandle_t a8){}

template<typename T>
void bar2(hipsparseFillMode_t a1,
         hipsparseDiagType_t a2,
         hipsparseIndexBase_t a3,
         hipsparseMatrixType_t a4,
         hipsparseOperation_t a5,
         hipsparseStatus_t a6,
         hipsparseMatDescr_t a7,
         hipsparseHandle_t a8){}

// specialization
template<>
void bar2<double>(hipsparseFillMode_t a1,
                  hipsparseDiagType_t a2,
                  hipsparseIndexBase_t a3,
                  hipsparseMatrixType_t a4,
                  hipsparseOperation_t a5,
                  hipsparseStatus_t a6,
                  hipsparseMatDescr_t a7,
                  hipsparseHandle_t a8){}

