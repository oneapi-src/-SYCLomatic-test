// ===------- cusparse_8.cu -------------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hipsparse.h"

#include <cmath>
#include <complex>
#include <cstdio>
#include <vector>

template <class d_data_t>
struct Data {
  float *h_data;
  d_data_t *d_data;
  int element_num;
  Data(int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    memset(h_data, 0, sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
  }
  Data(float *input_data, int element_num) : element_num(element_num) {
    h_data = (float *)malloc(sizeof(float) * element_num);
    hipMalloc(&d_data, sizeof(d_data_t) * element_num);
    hipMemset(d_data, 0, sizeof(d_data_t) * element_num);
    memcpy(h_data, input_data, sizeof(float) * element_num);
  }
  ~Data() {
    free(h_data);
    hipFree(d_data);
  }
  void H2D() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    from_float_convert(h_data, h_temp);
    hipMemcpy(d_data, h_temp, sizeof(d_data_t) * element_num,
               hipMemcpyHostToDevice);
    free(h_temp);
  }
  void D2H() {
    d_data_t *h_temp = (d_data_t *)malloc(sizeof(d_data_t) * element_num);
    memset(h_temp, 0, sizeof(d_data_t) * element_num);
    hipMemcpy(h_temp, d_data, sizeof(d_data_t) * element_num,
               hipMemcpyDeviceToHost);
    to_float_convert(h_temp, h_data);
    free(h_temp);
  }

private:
  inline void from_float_convert(float *in, d_data_t *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
  inline void to_float_convert(d_data_t *in, float *out) {
    for (int i = 0; i < element_num; i++)
      out[i] = in[i];
  }
};
template <>
inline void Data<float2>::from_float_convert(float *in, float2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}
template <>
inline void Data<double2>::from_float_convert(float *in, double2 *out) {
  for (int i = 0; i < element_num; i++)
    out[i].x = in[i];
}

template <>
inline void Data<float2>::to_float_convert(float2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}
template <>
inline void Data<double2>::to_float_convert(double2 *in, float *out) {
  for (int i = 0; i < element_num; i++)
    out[i] = in[i].x;
}

bool compare_result(float *expect, float *result, int element_num) {
  for (int i = 0; i < element_num; i++) {
    if (std::abs(result[i] - expect[i]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool compare_result(float *expect, float *result, std::vector<int> indices) {
  for (int i = 0; i < indices.size(); i++) {
    if (std::abs(result[indices[i]] - expect[indices[i]]) >= 0.05) {
      return false;
    }
  }
  return true;
}

bool test_passed = true;

// | 1 1 2 0 |
// | 0 1 3 0 |
// | 0 0 1 5 |
void test_cusparseCsr2csc_00() {
  std::vector<float> a_val_vec = {1, 1, 2, 1, 3, 1, 5};
  Data<float> a_s_val(a_val_vec.data(), 7);
  Data<double> a_d_val(a_val_vec.data(), 7);
  Data<float2> a_c_val(a_val_vec.data(), 7);
  Data<double2> a_z_val(a_val_vec.data(), 7);
  std::vector<float> a_row_ptr_vec = {0, 3, 5, 7};
  Data<int> a_s_row_ptr(a_row_ptr_vec.data(), 4);
  Data<int> a_d_row_ptr(a_row_ptr_vec.data(), 4);
  Data<int> a_c_row_ptr(a_row_ptr_vec.data(), 4);
  Data<int> a_z_row_ptr(a_row_ptr_vec.data(), 4);
  std::vector<float> a_col_ind_vec = {0, 1, 2, 1, 2, 2, 3};
  Data<int> a_s_col_ind(a_col_ind_vec.data(), 7);
  Data<int> a_d_col_ind(a_col_ind_vec.data(), 7);
  Data<int> a_c_col_ind(a_col_ind_vec.data(), 7);
  Data<int> a_z_col_ind(a_col_ind_vec.data(), 7);

  Data<float> b_s_val(a_val_vec.data(), 7);
  Data<double> b_d_val(a_val_vec.data(), 7);
  Data<float2> b_c_val(a_val_vec.data(), 7);
  Data<double2> b_z_val(a_val_vec.data(), 7);
  Data<int> b_s_col_ptr(a_row_ptr_vec.data(), 5);
  Data<int> b_d_col_ptr(a_row_ptr_vec.data(), 5);
  Data<int> b_c_col_ptr(a_row_ptr_vec.data(), 5);
  Data<int> b_z_col_ptr(a_row_ptr_vec.data(), 5);
  Data<int> b_s_row_ind(a_col_ind_vec.data(), 7);
  Data<int> b_d_row_ind(a_col_ind_vec.data(), 7);
  Data<int> b_c_row_ind(a_col_ind_vec.data(), 7);
  Data<int> b_z_row_ind(a_col_ind_vec.data(), 7);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_s_row_ptr.H2D();
  a_d_row_ptr.H2D();
  a_c_row_ptr.H2D();
  a_z_row_ptr.H2D();
  a_s_col_ind.H2D();
  a_d_col_ind.H2D();
  a_c_col_ind.H2D();
  a_z_col_ind.H2D();


  size_t ws_size_s = 0;
  size_t ws_size_d = 0;
  size_t ws_size_c = 0;
  size_t ws_size_z = 0;
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_s_val.d_data, a_s_row_ptr.d_data, a_s_col_ind.d_data, b_s_val.d_data, b_s_col_ptr.d_data, b_s_row_ind.d_data, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size_s);
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_d_val.d_data, a_d_row_ptr.d_data, a_d_col_ind.d_data, b_d_val.d_data, b_d_col_ptr.d_data, b_d_row_ind.d_data, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size_d);
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_c_val.d_data, a_c_row_ptr.d_data, a_c_col_ind.d_data, b_c_val.d_data, b_c_col_ptr.d_data, b_c_row_ind.d_data, HIP_C_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size_c);
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_z_val.d_data, a_z_row_ptr.d_data, a_z_col_ind.d_data, b_z_val.d_data, b_z_col_ptr.d_data, b_z_row_ind.d_data, HIP_C_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size_z);

  void *ws_s;
  void *ws_d;
  void *ws_c;
  void *ws_z;
  hipMalloc(&ws_s, ws_size_s);
  hipMalloc(&ws_d, ws_size_d);
  hipMalloc(&ws_c, ws_size_c);
  hipMalloc(&ws_z, ws_size_z);

  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_s_val.d_data, a_s_row_ptr.d_data, a_s_col_ind.d_data, b_s_val.d_data, b_s_col_ptr.d_data, b_s_row_ind.d_data, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws_s);
  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_d_val.d_data, a_d_row_ptr.d_data, a_d_col_ind.d_data, b_d_val.d_data, b_d_col_ptr.d_data, b_d_row_ind.d_data, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws_d);
  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_c_val.d_data, a_c_row_ptr.d_data, a_c_col_ind.d_data, b_c_val.d_data, b_c_col_ptr.d_data, b_c_row_ind.d_data, HIP_C_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws_c);
  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_z_val.d_data, a_z_row_ptr.d_data, a_z_col_ind.d_data, b_z_val.d_data, b_z_col_ptr.d_data, b_z_row_ind.d_data, HIP_C_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws_z);

  b_s_val.D2H();
  b_d_val.D2H();
  b_c_val.D2H();
  b_z_val.D2H();
  b_s_col_ptr.D2H();
  b_d_col_ptr.D2H();
  b_c_col_ptr.D2H();
  b_z_col_ptr.D2H();
  b_s_row_ind.D2H();
  b_d_row_ind.D2H();
  b_c_row_ind.D2H();
  b_z_row_ind.D2H();

  hipStreamSynchronize(0);

  hipFree(ws_s);
  hipFree(ws_d);
  hipFree(ws_c);
  hipFree(ws_z);
  hipsparseDestroy(handle);

  float expect_b_val[7] = {1, 1, 1, 2, 3, 1, 5};
  float expect_b_col_ptr[5] = {0, 1, 3, 6, 7};
  float expect_b_row_ind[7] = {0, 0, 1, 0, 1, 2, 2};
  if (compare_result(expect_b_val, b_s_val.h_data, 7) &&
      compare_result(expect_b_val, b_d_val.h_data, 7) &&
      compare_result(expect_b_val, b_c_val.h_data, 7) &&
      compare_result(expect_b_val, b_z_val.h_data, 7) &&
      compare_result(expect_b_col_ptr, b_s_col_ptr.h_data, 5) &&
      compare_result(expect_b_col_ptr, b_d_col_ptr.h_data, 5) &&
      compare_result(expect_b_col_ptr, b_c_col_ptr.h_data, 5) &&
      compare_result(expect_b_col_ptr, b_z_col_ptr.h_data, 5) &&
      compare_result(expect_b_row_ind, b_s_row_ind.h_data, 7) &&
      compare_result(expect_b_row_ind, b_d_row_ind.h_data, 7) &&
      compare_result(expect_b_row_ind, b_c_row_ind.h_data, 7) &&
      compare_result(expect_b_row_ind, b_z_row_ind.h_data, 7))
    printf("Csr2csc 00 pass\n");
  else {
    printf("Csr2csc 00 fail\n");
    test_passed = false;
  }
}

// | 1 1 2 0 |
// | 0 1 3 0 |
// | 0 0 1 5 |
void test_cusparseCsr2csc_01() {
  std::vector<float> a_val_vec = {1, 1, 2, 1, 3, 1, 5};
  Data<float> a_s_val(a_val_vec.data(), 7);
  Data<double> a_d_val(a_val_vec.data(), 7);
  Data<float2> a_c_val(a_val_vec.data(), 7);
  Data<double2> a_z_val(a_val_vec.data(), 7);
  std::vector<float> a_row_ptr_vec = {0, 3, 5, 7};
  Data<int> a_s_row_ptr(a_row_ptr_vec.data(), 4);
  Data<int> a_d_row_ptr(a_row_ptr_vec.data(), 4);
  Data<int> a_c_row_ptr(a_row_ptr_vec.data(), 4);
  Data<int> a_z_row_ptr(a_row_ptr_vec.data(), 4);
  std::vector<float> a_col_ind_vec = {0, 1, 2, 1, 2, 2, 3};
  Data<int> a_s_col_ind(a_col_ind_vec.data(), 7);
  Data<int> a_d_col_ind(a_col_ind_vec.data(), 7);
  Data<int> a_c_col_ind(a_col_ind_vec.data(), 7);
  Data<int> a_z_col_ind(a_col_ind_vec.data(), 7);

  Data<float> b_s_val(a_val_vec.data(), 7);
  Data<double> b_d_val(a_val_vec.data(), 7);
  Data<float2> b_c_val(a_val_vec.data(), 7);
  Data<double2> b_z_val(a_val_vec.data(), 7);
  Data<int> b_s_col_ptr(a_row_ptr_vec.data(), 5);
  Data<int> b_d_col_ptr(a_row_ptr_vec.data(), 5);
  Data<int> b_c_col_ptr(a_row_ptr_vec.data(), 5);
  Data<int> b_z_col_ptr(a_row_ptr_vec.data(), 5);
  Data<int> b_s_row_ind(a_col_ind_vec.data(), 7);
  Data<int> b_d_row_ind(a_col_ind_vec.data(), 7);
  Data<int> b_c_row_ind(a_col_ind_vec.data(), 7);
  Data<int> b_z_row_ind(a_col_ind_vec.data(), 7);

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  a_s_val.H2D();
  a_d_val.H2D();
  a_c_val.H2D();
  a_z_val.H2D();
  a_s_row_ptr.H2D();
  a_d_row_ptr.H2D();
  a_c_row_ptr.H2D();
  a_z_row_ptr.H2D();
  a_s_col_ind.H2D();
  a_d_col_ind.H2D();
  a_c_col_ind.H2D();
  a_z_col_ind.H2D();


  size_t ws_size_s = 0;
  size_t ws_size_d = 0;
  size_t ws_size_c = 0;
  size_t ws_size_z = 0;
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_s_val.d_data, a_s_row_ptr.d_data, a_s_col_ind.d_data, b_s_val.d_data, b_s_col_ptr.d_data, b_s_row_ind.d_data, HIP_R_32F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size_s);
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_d_val.d_data, a_d_row_ptr.d_data, a_d_col_ind.d_data, b_d_val.d_data, b_d_col_ptr.d_data, b_d_row_ind.d_data, HIP_R_64F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size_d);
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_c_val.d_data, a_c_row_ptr.d_data, a_c_col_ind.d_data, b_c_val.d_data, b_c_col_ptr.d_data, b_c_row_ind.d_data, HIP_C_32F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size_c);
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_z_val.d_data, a_z_row_ptr.d_data, a_z_col_ind.d_data, b_z_val.d_data, b_z_col_ptr.d_data, b_z_row_ind.d_data, HIP_C_64F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size_z);

  void *ws_s;
  void *ws_d;
  void *ws_c;
  void *ws_z;
  hipMalloc(&ws_s, ws_size_s);
  hipMalloc(&ws_d, ws_size_d);
  hipMalloc(&ws_c, ws_size_c);
  hipMalloc(&ws_z, ws_size_z);

  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_s_val.d_data, a_s_row_ptr.d_data, a_s_col_ind.d_data, b_s_val.d_data, b_s_col_ptr.d_data, b_s_row_ind.d_data, HIP_R_32F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws_s);
  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_d_val.d_data, a_d_row_ptr.d_data, a_d_col_ind.d_data, b_d_val.d_data, b_d_col_ptr.d_data, b_d_row_ind.d_data, HIP_R_64F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws_d);
  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_c_val.d_data, a_c_row_ptr.d_data, a_c_col_ind.d_data, b_c_val.d_data, b_c_col_ptr.d_data, b_c_row_ind.d_data, HIP_C_32F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws_c);
  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_z_val.d_data, a_z_row_ptr.d_data, a_z_col_ind.d_data, b_z_val.d_data, b_z_col_ptr.d_data, b_z_row_ind.d_data, HIP_C_64F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws_z);

  b_s_val.D2H();
  b_d_val.D2H();
  b_c_val.D2H();
  b_z_val.D2H();
  b_s_col_ptr.D2H();
  b_d_col_ptr.D2H();
  b_c_col_ptr.D2H();
  b_z_col_ptr.D2H();
  b_s_row_ind.D2H();
  b_d_row_ind.D2H();
  b_c_row_ind.D2H();
  b_z_row_ind.D2H();

  hipStreamSynchronize(0);

  hipFree(ws_s);
  hipFree(ws_d);
  hipFree(ws_c);
  hipFree(ws_z);
  hipsparseDestroy(handle);

  float expect_b_val[7] = {0, 0, 0, 0, 0, 0, 0};
  float expect_b_col_ptr[5] = {0, 1, 3, 6, 7};
  float expect_b_row_ind[7] = {0, 0, 1, 0, 1, 2, 2};
  if (compare_result(expect_b_val, b_s_val.h_data, 7) &&
      compare_result(expect_b_val, b_d_val.h_data, 7) &&
      compare_result(expect_b_val, b_c_val.h_data, 7) &&
      compare_result(expect_b_val, b_z_val.h_data, 7) &&
      compare_result(expect_b_col_ptr, b_s_col_ptr.h_data, 5) &&
      compare_result(expect_b_col_ptr, b_d_col_ptr.h_data, 5) &&
      compare_result(expect_b_col_ptr, b_c_col_ptr.h_data, 5) &&
      compare_result(expect_b_col_ptr, b_z_col_ptr.h_data, 5) &&
      compare_result(expect_b_row_ind, b_s_row_ind.h_data, 7) &&
      compare_result(expect_b_row_ind, b_d_row_ind.h_data, 7) &&
      compare_result(expect_b_row_ind, b_c_row_ind.h_data, 7) &&
      compare_result(expect_b_row_ind, b_z_row_ind.h_data, 7))
    printf("Csr2csc 01 pass\n");
  else {
    printf("Csr2csc 01 fail\n");
    test_passed = false;
  }
}

int main() {
  test_cusparseCsr2csc_00();
  test_cusparseCsr2csc_01();

  if (test_passed)
    return 0;
  return -1;
}
