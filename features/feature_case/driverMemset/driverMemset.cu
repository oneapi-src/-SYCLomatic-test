
#include <hip/hip_runtime.h>

int main(){
    int size = 32;
    hipDeviceptr_t f_D = 0;
    hipMalloc(&f_D, size);
    hipStream_t stream;
    unsigned int v32 = 50000;
    unsigned short v16 = 20000;
    unsigned char v8 = (unsigned char) 200;
    hipMemsetD32(f_D, v32, size);
    hipMemsetD16(f_D, v16, size * 2);
    hipMemsetD8(f_D, v8, size * 4);
    hipMemsetD32Async(f_D, v32, size, stream);
    hipMemsetD16Async(f_D, v16, size * 2, stream);
    hipMemsetD8Async(f_D, v8, size * 4, stream);
    hipMemsetD2D32(f_D, 1, v32, 4, 6);
    hipMemsetD2D16(f_D, 1, v16, 4 * 2, 6);
    hipMemsetD2D8(f_D, 1, v8, 4 * 4, 6);
    hipMemsetD2D32Async(f_D, 1, v32, 4, 6, stream);
    hipMemsetD2D16Async(f_D, 1, v16, 4 * 2, 6, stream);
    hipMemsetD2D8Async(f_D, 1, v8, 4 * 4, 6, stream);
}