//__constant__ array declared in header must be migrated to a static variable
#include "constant_array.cuh"

void init_h1(int *hvals) {
  hipMemcpyToSymbol(HIP_SYMBOL(dvals), hvals, 2 * sizeof(int), 0, hipMemcpyHostToDevice);
}

void get_h1(int *target) {
  hipMemcpyFromSymbol(target, HIP_SYMBOL(dvals), 2 * sizeof(int), 0, hipMemcpyDeviceToHost);
}
