
#include <hip/hip_runtime.h>
//__constant__ array must be migrated to a static variable
__constant__ int dvals[2];

void init_l1(int *hvals) {
  hipMemcpyToSymbol(HIP_SYMBOL(dvals), hvals, 2 * sizeof(int), 0, hipMemcpyHostToDevice);
}

void get_l1(int *target) {
  hipMemcpyFromSymbol(target, HIP_SYMBOL(dvals), 2 * sizeof(int), 0, hipMemcpyDeviceToHost);
}
