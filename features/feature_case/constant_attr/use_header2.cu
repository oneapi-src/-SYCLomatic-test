//__constant__ array declared in header must be migrated to a static variable
#include "constant_array.cuh"

void init_h2(int *hvals) {
  hipMemcpyToSymbol(HIP_SYMBOL(dvals), hvals, 2 * sizeof(int), 0, hipMemcpyHostToDevice);
}

void get_h2(int *target) {
  hipMemcpyFromSymbol(target, HIP_SYMBOL(dvals), 2 * sizeof(int), 0, hipMemcpyDeviceToHost);
}
