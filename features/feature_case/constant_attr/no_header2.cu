
#include <hip/hip_runtime.h>
//__constant__ array must be migrated to a static variable
__constant__ int dvals[2];

void init_l2(int *hvals) {
  hipMemcpyToSymbol(HIP_SYMBOL(dvals), hvals, 2 * sizeof(int), 0, hipMemcpyHostToDevice);
}

void get_l2(int *target) {
  hipMemcpyFromSymbol(target, HIP_SYMBOL(dvals), 2 * sizeof(int), 0, hipMemcpyDeviceToHost);
}
