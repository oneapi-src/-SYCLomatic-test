// ====------ curand.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

int main(){
  hiprandStatus_t s1;
  hiprandStatus s2;
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
  hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull);
  float *d_data;

  hiprandGenerateUniform(rng, d_data, 100*100);

  s1 = hiprandGenerateUniform(rng, d_data, 100*100);

  s1 = hiprandGenerateLogNormal(rng, d_data, 100*100, 123, 456);

  s1 = hiprandGenerateNormal(rng, d_data, 100*100, 123, 456);

  double* d_data_d;
  hiprandGenerateUniformDouble(rng, d_data_d, 100*100);

  hiprandGenerateLogNormalDouble(rng, d_data_d, 100*100, 123, 456);

  hiprandGenerateNormalDouble(rng, d_data_d, 100*100, 123, 456);

  unsigned int* d_data_ui;

  s1 = hiprandGenerate(rng, d_data_ui, 100*100);

  s1 = hiprandGeneratePoisson(rng, d_data_ui, 100*100, 123.456);

  unsigned long long* d_data_ull;
  hiprandGenerateLongLong(rng, d_data_ull, 100*100);

  if(s1 = hiprandGenerateLongLong(rng, d_data_ull, 100*100)){}

  if(hiprandGenerateLongLong(rng, d_data_ull, 100*100)){}

  hiprandGenerator_t rng2;
  hiprandCreateGenerator(&rng2, HIPRAND_RNG_QUASI_DEFAULT);
  hiprandSetQuasiRandomGeneratorDimensions(rng2, 1111);
  hiprandGenerateUniform(rng2, d_data, 100*100);

  hiprandSetGeneratorOffset(rng, 100);
  s1 = hiprandSetGeneratorOffset(rng2, 200);

  hipStream_t stream;
  hiprandSetStream(rng, stream);

  hiprandDestroyGenerator(rng);
  s1 = hiprandDestroyGenerator(rng);
}

hiprandStatus_t foo1();
hiprandStatus foo2();

class A{
public:
  A(){
    hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_DEFAULT);
    hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
  }
  ~A(){
    hiprandDestroyGenerator(rng);
  }
private:
  hiprandGenerator_t rng;
};



void bar1(){
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull);
}


void bar2(){
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64);
  hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
  if (stat != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
  }
}

void bar3(){
  hiprandGenerator_t rng;
  curandErrCheck(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull));
  float *d_data;
  curandErrCheck(hiprandGenerateUniform(rng, d_data, 100*100));
  curandErrCheck(hiprandDestroyGenerator(rng));
}

void bar4(){
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32);
  hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
}


int bar6(){
  float *d_data;
  hiprandGenerator_t rng2;
  hiprandCreateGenerator(&rng2, HIPRAND_RNG_QUASI_DEFAULT);
  hiprandSetQuasiRandomGeneratorDimensions(rng2, 1111);
  return hiprandGenerateUniform(rng2, d_data, 100*100);
}

void bar7() {
  hiprandGenerator_t rng;
  hiprandRngType_t rngT1 = HIPRAND_RNG_PSEUDO_DEFAULT;
  hiprandRngType_t rngT2 = HIPRAND_RNG_PSEUDO_XORWOW;
  hiprandRngType_t rngT3 = HIPRAND_RNG_PSEUDO_MRG32K3A;
  hiprandRngType_t rngT4 = HIPRAND_RNG_PSEUDO_MTGP32;
  hiprandRngType_t rngT5 = HIPRAND_RNG_PSEUDO_MT19937;
  hiprandRngType_t rngT6 = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
  hiprandRngType_t rngT7 = HIPRAND_RNG_QUASI_DEFAULT;
  hiprandRngType_t rngT8 = HIPRAND_RNG_QUASI_SOBOL32;
  hiprandRngType_t rngT9 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  hiprandRngType_t rngT10 = HIPRAND_RNG_QUASI_SOBOL64;
  hiprandRngType_t rngT11 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;
  hiprandCreateGeneratorHost(&rng, rngT1);
}
