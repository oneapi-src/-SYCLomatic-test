// ====------ curand-usm.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

int main(){
  hiprandStatus_t s1;
  hiprandStatus s2;
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
  hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull);
  float *d_data;

  hiprandGenerateUniform(rng, d_data, 100*100);


  s1 = hiprandGenerateUniform(rng, d_data, 100*100);

  s1 = hiprandGenerateLogNormal(rng, d_data, 100*100, 123, 456);

  s1 = hiprandGenerateNormal(rng, d_data, 100*100, 123, 456);

  double* d_data_d;
  hiprandGenerateUniformDouble(rng, d_data_d, 100*100);

  hiprandGenerateLogNormalDouble(rng, d_data_d, 100*100, 123, 456);

  hiprandGenerateNormalDouble(rng, d_data_d, 100*100, 123, 456);

  unsigned int* d_data_ui;
  s1 = hiprandGenerate(rng, d_data_ui, 100*100);

  s1 = hiprandGeneratePoisson(rng, d_data_ui, 100*100, 123.456);

  unsigned long long* d_data_ull;
  hiprandGenerateLongLong(rng, d_data_ull, 100*100);

  if(s1 = hiprandGenerateLongLong(rng, d_data_ull, 100*100)){}

  if(hiprandGenerateLongLong(rng, d_data_ull, 100*100)){}

  hiprandGenerator_t rng2;
  hiprandCreateGenerator(&rng2, HIPRAND_RNG_QUASI_DEFAULT);
  hiprandSetQuasiRandomGeneratorDimensions(rng2, 1111);
  hiprandGenerateUniform(rng2, d_data, 100*100);

  hiprandSetGeneratorOffset(rng, 100);
  s1 = hiprandSetGeneratorOffset(rng2, 200);

  hipStream_t stream;
  hiprandSetStream(rng, stream);

  hiprandDestroyGenerator(rng);
  s1 = hiprandDestroyGenerator(rng);
}

hiprandStatus_t foo1();
hiprandStatus foo2();

class A{
public:
  A(){
    hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_DEFAULT);
    hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
  }
  ~A(){
    hiprandDestroyGenerator(rng);
  }
private:
  hiprandGenerator_t rng;
};

class B{
public:
  B(){
    hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_DEFAULT);
    hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
    hipMalloc(&karg1, 32 * sizeof(int));
  }
  ~B(){
    hiprandDestroyGenerator(rng);
    hipFree(karg1);
  }
private:
  hiprandGenerator_t rng;
  int *karg1;
};

void bar1(){

  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull);
}


void bar2(){
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64);
  hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
  if (stat != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
  }
}

void bar3(){
  hiprandGenerator_t rng;
  curandErrCheck(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull));
  float *d_data;
  curandErrCheck(hiprandGenerateUniform(rng, d_data, 100*100));
  curandErrCheck(hiprandDestroyGenerator(rng));
}

void bar4(){
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32);
  hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
}

int bar5(){
  float *d_data;
  hiprandGenerator_t rng2;
  hiprandCreateGenerator(&rng2, HIPRAND_RNG_QUASI_DEFAULT);
  hiprandSetQuasiRandomGeneratorDimensions(rng2, 1111);
  return hiprandGenerateUniform(rng2, d_data, 100*100);
}

void bar6(float *x_gpu, size_t n) {
  static hiprandGenerator_t gen[16];
  static int init[16] = {0};
  int i = 0;
  if(!init[i]) {
    hiprandCreateGenerator(&gen[i], HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen[i], 1234);
    init[i] = 1;
  }
  hiprandGenerateUniform(gen[i], x_gpu, n);
}
