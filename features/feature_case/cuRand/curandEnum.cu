// ====------ curandEnum.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

hiprandStatus_t foo(
  hiprandStatus_t a1,
  hiprandStatus_t a2,
  hiprandStatus_t a3,
  hiprandStatus_t a4,
  hiprandStatus_t a5,
  hiprandStatus_t a6,
  hiprandStatus_t a7,
  hiprandStatus_t a8,
  hiprandStatus_t a9,
  hiprandStatus_t a10,
  hiprandStatus_t a11,
  hiprandStatus_t a12,
  hiprandStatus_t a13) {}

int main() {
  hiprandStatus_t a1 = HIPRAND_STATUS_SUCCESS;
  hiprandStatus_t a2 = HIPRAND_STATUS_VERSION_MISMATCH;
  hiprandStatus_t a3 = HIPRAND_STATUS_NOT_INITIALIZED;
  hiprandStatus_t a4 = HIPRAND_STATUS_ALLOCATION_FAILED;
  hiprandStatus_t a5 = HIPRAND_STATUS_TYPE_ERROR;
  hiprandStatus_t a6 = HIPRAND_STATUS_OUT_OF_RANGE;
  hiprandStatus_t a7 = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  hiprandStatus_t a8 = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  hiprandStatus_t a9 = HIPRAND_STATUS_LAUNCH_FAILURE;
  hiprandStatus_t a10 = HIPRAND_STATUS_PREEXISTING_FAILURE;
  hiprandStatus_t a11 = HIPRAND_STATUS_INITIALIZATION_FAILED;
  hiprandStatus_t a12 = HIPRAND_STATUS_ARCH_MISMATCH;
  hiprandStatus_t a13 = HIPRAND_STATUS_INTERNAL_ERROR;


  foo(
    HIPRAND_STATUS_SUCCESS,
    HIPRAND_STATUS_VERSION_MISMATCH,
    HIPRAND_STATUS_NOT_INITIALIZED,
    HIPRAND_STATUS_ALLOCATION_FAILED,
    HIPRAND_STATUS_TYPE_ERROR,
    HIPRAND_STATUS_OUT_OF_RANGE,
    HIPRAND_STATUS_LENGTH_NOT_MULTIPLE,
    HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED,
    HIPRAND_STATUS_LAUNCH_FAILURE,
    HIPRAND_STATUS_PREEXISTING_FAILURE,
    HIPRAND_STATUS_INITIALIZATION_FAILED,
    HIPRAND_STATUS_ARCH_MISMATCH,
    HIPRAND_STATUS_INTERNAL_ERROR);
}
