// ====------ curandEnum.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

hiprandStatus_t foo(
  hiprandStatus_t a1,
  hiprandStatus_t a2,
  hiprandStatus_t a3,
  hiprandStatus_t a4,
  hiprandStatus_t a5,
  hiprandStatus_t a6,
  hiprandStatus_t a7,
  hiprandStatus_t a8,
  hiprandStatus_t a9,
  hiprandStatus_t a10,
  hiprandStatus_t a11,
  hiprandStatus_t a12,
  hiprandStatus_t a13) {}

hiprandOrdering_t goo(hiprandOrdering_t b1, hiprandOrdering_t b2,
                     // curandOrdering_t b3,
                     hiprandOrdering_t b4, hiprandOrdering_t b5
                     // , curandOrdering_t b6
) {
  return b1;
}

int main() {
  hiprandStatus_t a1 = HIPRAND_STATUS_SUCCESS;
  hiprandStatus_t a2 = HIPRAND_STATUS_VERSION_MISMATCH;
  hiprandStatus_t a3 = HIPRAND_STATUS_NOT_INITIALIZED;
  hiprandStatus_t a4 = HIPRAND_STATUS_ALLOCATION_FAILED;
  hiprandStatus_t a5 = HIPRAND_STATUS_TYPE_ERROR;
  hiprandStatus_t a6 = HIPRAND_STATUS_OUT_OF_RANGE;
  hiprandStatus_t a7 = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  hiprandStatus_t a8 = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  hiprandStatus_t a9 = HIPRAND_STATUS_LAUNCH_FAILURE;
  hiprandStatus_t a10 = HIPRAND_STATUS_PREEXISTING_FAILURE;
  hiprandStatus_t a11 = HIPRAND_STATUS_INITIALIZATION_FAILED;
  hiprandStatus_t a12 = HIPRAND_STATUS_ARCH_MISMATCH;
  hiprandStatus_t a13 = HIPRAND_STATUS_INTERNAL_ERROR;


  foo(
    HIPRAND_STATUS_SUCCESS,
    HIPRAND_STATUS_VERSION_MISMATCH,
    HIPRAND_STATUS_NOT_INITIALIZED,
    HIPRAND_STATUS_ALLOCATION_FAILED,
    HIPRAND_STATUS_TYPE_ERROR,
    HIPRAND_STATUS_OUT_OF_RANGE,
    HIPRAND_STATUS_LENGTH_NOT_MULTIPLE,
    HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED,
    HIPRAND_STATUS_LAUNCH_FAILURE,
    HIPRAND_STATUS_PREEXISTING_FAILURE,
    HIPRAND_STATUS_INITIALIZATION_FAILED,
    HIPRAND_STATUS_ARCH_MISMATCH,
    HIPRAND_STATUS_INTERNAL_ERROR);

  hiprandOrdering_t b1 = HIPRAND_ORDERING_PSEUDO_BEST;
  hiprandOrdering_t b2 = HIPRAND_ORDERING_PSEUDO_DEFAULT;
  // curandOrdering_t b3 = CURAND_ORDERING_PSEUDO_SEEDED;
  hiprandOrdering_t b4 = HIPRAND_ORDERING_PSEUDO_LEGACY;
  hiprandOrdering_t b5 = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
  // curandOrdering_t b6 = CURAND_ORDERING_QUASI_DEFAULT;

  goo(HIPRAND_ORDERING_PSEUDO_BEST, HIPRAND_ORDERING_PSEUDO_DEFAULT,
      // CURAND_ORDERING_PSEUDO_SEEDED,
      HIPRAND_ORDERING_PSEUDO_LEGACY, HIPRAND_ORDERING_PSEUDO_DYNAMIC
      // , CURAND_ORDERING_QUASI_DEFAULT
  );
}
