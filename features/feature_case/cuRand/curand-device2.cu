// ====------ curand-device2.cu ---------------------------- *- CUDA -* ----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"


__global__ void kernel1() {
  unsigned int u;
  uint4 u4;
  float f;
  float2 f2;
  float4 f4;
  double d;
  double2 d2;
  double4 d4;

  hiprandStatePhilox4_32_10_t rng1;
  hiprand_init(1, 2, 3, &rng1);
  u = hiprand(&rng1);

  hiprandStatePhilox4_32_10_t rng2;
  hiprand_init(1, 2, 3, &rng2);
  u4 = curand4(&rng2);

  hiprandStatePhilox4_32_10_t rng3;
  hiprand_init(1, 2, 3, &rng3);
  f = hiprand_normal(&rng3);

  hiprandStatePhilox4_32_10_t rng4;
  hiprand_init(1, 2, 3, &rng4);
  f2 = hiprand_normal2(&rng4);

  hiprandStatePhilox4_32_10_t rng5;
  hiprand_init(1, 2, 3, &rng5);
  d2 = hiprand_normal2_double(&rng5);

  hiprandStatePhilox4_32_10_t rng6;
  hiprand_init(1, 2, 3, &rng6);
  f4 = hiprand_normal4(&rng6);

  hiprandStatePhilox4_32_10_t rng7;
  hiprand_init(1, 2, 3, &rng7);
  d = hiprand_normal_double(&rng7);

  hiprandStatePhilox4_32_10_t rng8;
  hiprand_init(1, 2, 3, &rng8);
  f = hiprand_log_normal(&rng8, 3, 7);

  hiprandStatePhilox4_32_10_t rng9;
  hiprand_init(1, 2, 3, &rng9);
  f2 = hiprand_log_normal2(&rng9, 3, 7);

  hiprandStatePhilox4_32_10_t rng10;
  hiprand_init(1, 2, 3, &rng10);
  d2 = hiprand_log_normal2_double(&rng10, 3, 7);

  hiprandStatePhilox4_32_10_t rng11;
  hiprand_init(1, 2, 3, &rng11);
  f4 = hiprand_log_normal4(&rng11, 3, 7);

  hiprandStatePhilox4_32_10_t rng12;
  hiprand_init(1, 2, 3, &rng12);
  d = hiprand_log_normal_double(&rng12, 3, 7);

  hiprandStatePhilox4_32_10_t rng13;
  hiprand_init(1, 2, 3, &rng13);
  f = hiprand_uniform(&rng13);

  hiprandStatePhilox4_32_10_t rng14;
  hiprand_init(1, 2, 3, &rng14);
  d2 = hiprand_uniform2_double(&rng14);

  hiprandStatePhilox4_32_10_t rng15;
  hiprand_init(1, 2, 3, &rng15);
  f4 = hiprand_uniform4(&rng15);

  hiprandStatePhilox4_32_10_t rng16;
  hiprand_init(1, 2, 3, &rng16);
  d = hiprand_uniform_double(&rng16);

  hiprandStatePhilox4_32_10_t rng17;
  hiprand_init(1, 2, 3, &rng17);
  u = hiprand_poisson(&rng17, 3);

  hiprandStatePhilox4_32_10_t rng18;
  hiprand_init(1, 2, 3, &rng18);
  u4 = hiprand_poisson4(&rng18, 3);

  hiprandStatePhilox4_32_10_t rng19;
  hiprand_init(1, 2, 3, &rng19);
  d4 = hiprand_uniform4_double(&rng19);

  hiprandStatePhilox4_32_10_t rng20;
  hiprand_init(1, 2, 3, &rng20);
  d4 = hiprand_normal4_double(&rng20);

  hiprandStatePhilox4_32_10_t rng21;
  hiprand_init(1, 2, 3, &rng21);
  d4 = hiprand_log_normal4_double(&rng21, 3, 7);
}

__global__ void kernel2() {
  hiprandStatePhilox4_32_10_t rng1;
  hiprandStatePhilox4_32_10_t rng2;
  hiprand_init(11, 1, 1234, &rng1);
  hiprand_init(22, 2, 4321, &rng2);
  float x = hiprand_uniform(&rng1);
  float2 y = hiprand_normal2(&rng2);
}

__global__ void kernel3() {
  hiprandStateMRG32k3a_t rng1;
  hiprandStatePhilox4_32_10_t rng2;
  hiprandStateXORWOW_t rng3;

  hiprand_init(1, 2, 3, &rng1);
  hiprand_init(1, 2, 3, &rng2);
  hiprand_init(1, 2, 3, &rng3);

  skipahead(1, &rng1);
  skipahead(2, &rng2);
  skipahead(3, &rng3);

  skipahead_sequence(1, &rng1);
  skipahead_sequence(2, &rng2);
  skipahead_sequence(3, &rng3);

  skipahead_subsequence(1, &rng1);

  hiprand_uniform(&rng1);
  hiprand_uniform(&rng2);
  hiprand_uniform(&rng3);
}

__global__ void type_test() {
  hiprandStateXORWOW_t rng1;
  hiprandStateXORWOW rng2;
  hiprandState_t rng3;
  hiprandState rng4;
  hiprandStatePhilox4_32_10_t rng5;
  hiprandStatePhilox4_32_10 rng6;
  hiprandStateMRG32k3a_t rng7;
  hiprandStateMRG32k3a rng8;
}

int main() {
  kernel1<<<1,1>>>();
  kernel2<<<1,1>>>();
  kernel3<<<1,1>>>();
  return 0;
}

__global__ void kernel4() {
  hiprandStateMRG32k3a_t rng;
  hiprand_init(1, 2 + 3, 4, &rng);
  skipahead_sequence(2 + 3, &rng);
  skipahead_subsequence(2 + 3, &rng);
}

