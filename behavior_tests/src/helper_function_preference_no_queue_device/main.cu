#include "common.cuh"
#include <stdio.h>

int main() {
  int *h_Data;
  int *d_Data;
  hipDeviceProp_t deviceProp;
  deviceProp.major = 0;
  hipGetDeviceProperties(&deviceProp, 0);
  h_Data = (int *)malloc(SIZE * sizeof(int));
  hipMalloc((void **)&d_Data, SIZE * sizeof(int));
  malloc1();
  kernelWrapper1(d_Data);
  hipDeviceSynchronize();
  hipMemcpy(h_Data, d_Data, SIZE * sizeof(int), hipMemcpyDeviceToHost);
  free1();
  malloc2();
  kernelWrapper2(d_Data);
  hipDeviceSynchronize();
  hipMemcpy(h_Data, d_Data, SIZE * sizeof(int), hipMemcpyDeviceToHost);
  free2();
  hipFree(d_Data);
  free(h_Data);
  printf("test pass!\n");
  return 0;
}
