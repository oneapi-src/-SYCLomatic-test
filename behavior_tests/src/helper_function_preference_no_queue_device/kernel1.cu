#include "hip/hip_runtime.h"
#include "common.cuh"

__global__ void kernel1(int *d_Data) {}

static uint *d_Data1;

void malloc1() { hipMalloc((void **)&d_Data1, SIZE * sizeof(int)); }

void free1() { hipFree(d_Data1); }

void kernelWrapper1(int *d_Data) {
  kernel1<<<1, 1>>>(d_Data);
  kernel1<<<1, 1>>>(d_Data);
}
