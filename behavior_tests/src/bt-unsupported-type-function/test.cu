// ====------ test.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//
#include "hip/hip_runtime.h"

#include "hipsparse.h"
#include "nvml.h"
#include <vector>

int main(int argc, char **argv) {

    HIP_ARRAY_DESCRIPTOR *pcad;

    hipMemcpy3DParms *p1c3d;

    const hipMemcpy3DParms *p2c3d;

    static hipMemcpy3DParms *p3c3d;

    static volatile hipMemcpy3DParms *p4c3d;

    std::vector<hipMemcpy3DParms *> vc3dp;

    hipExternalMemory_t cum;

    hipExternalSemaphore_t cus;

    hipGraph_t cug;

    hipGraphExec_t cuge;

    hipGraphNode_t cugn;

    hipGraphicsResource_t cugr;

    nvmlDevice_t nvmld;

    nvmlReturn_t nvmlr;

    nvmlMemory_t nvmlm;

    nvmlValueType_t nvmlvt;

    nvmlValue_t nvmlv;

    nvmlInit();

    nvmlInit_v2();

    char Ver[10];

    nvmlSystemGetDriverVersion(Ver, 10);

    unsigned int dc;

    nvmlDeviceGetCount_v2(&dc);

    nvmlShutdown();
    return 0;
}

